#include "hip/hip_runtime.h"
#include "NetworkCreator.h"

/////////////////////////////
//(C) Will Cunningham 2014 //
// Krioukov Research Group //
// Northeastern University //
/////////////////////////////

bool initVars(NetworkProperties * const network_properties, CaResources * const ca, CausetPerformance * const cp, Benchmark * const bm)
{
	#if DEBUG
	assert (network_properties != NULL);
	assert (ca != NULL);
	assert (cp != NULL);
	assert (bm != NULL);
	#endif

	//Benchmarking
	if (network_properties->flags.bench) {
		network_properties->graphID = 0;
		network_properties->flags.verbose = false;
		network_properties->flags.print_network = false;
	}

	int rank = network_properties->cmpi.rank;

	//Suppress queries if MPI is enabled
	#ifdef MPI_ENABLED
	if (network_properties->flags.verbose)
		network_properties->flags.yes = true;
	#endif

	//If a graph ID has been provided, warn user
	if (network_properties->graphID && network_properties->flags.verbose && !network_properties->flags.yes) {
		printf("You have chosen to load a graph from memory. Some parameters may be ignored as a result. Continue [y/N]? ");
		fflush(stdout);
		char response = getchar();
		getchar();
		if (response != 'y')
			return false;
	}

	//If the GPU is requested, optimize parameters
	#ifdef CUDA_ENABLED
	if (network_properties->flags.use_gpu && network_properties->N_tar % (BLOCK_SIZE << 1)) {
		printf_mpi(rank, "If you are using the GPU, set the target number of nodes (--nodes) to be a multiple of %d!\n", BLOCK_SIZE << 1);
		fflush(stdout);
		network_properties->cmpi.fail = 1;
	}

	if (checkMpiErrors(network_properties->cmpi))
		return false;

	//Adjacency matrix not implemented in certain GPU algorithms
	if (network_properties->flags.use_gpu && !LINK_NODES_GPU_V2)
		network_properties->core_edge_fraction = 0.0;
	#endif

	//Disable the default GSL Error Handler
	disableGSLErrHandler();

	try {
		if (network_properties->manifold == DE_SITTER || network_properties->manifold == DUST || network_properties->manifold == FLRW) {
			//Check for under-constrained system
			if (network_properties->N_tar == 0)
				throw CausetException("Flag '--nodes', number of nodes, must be specified!\n");
			if (network_properties->tau0 == 0.0)
				throw CausetException("Flag '--age', temporal cutoff, must be specified!\n");
			if (network_properties->manifold == DE_SITTER && !network_properties->flags.compact && !network_properties->r_max)
				throw CausetException("Flag '--slice', spatial scaling, must be specified!\n");
		}

		if (network_properties->manifold == DE_SITTER) {
			//Constrain the de Sitter system
			if (!network_properties->delta)
				network_properties->a = 1.0;

			if (network_properties->flags.compact) {
				network_properties->zeta = HALF_PI - network_properties->tau0;
				network_properties->tau0 = etaToTauCompact(HALF_PI - network_properties->zeta);
			} else {
				network_properties->zeta = HALF_PI + 1.0;
				network_properties->zeta1 = HALF_PI - tauToEtaFlat(network_properties->tau0);
			}

			#if DEBUG
			if (network_properties->flags.compact)
				assert (network_properties->zeta > 0.0 && network_properties->zeta < HALF_PI);
			else
				assert (network_properties->zeta > HALF_PI);
			assert (network_properties->tau0 > 0.0);
			#endif

			double eta0 = HALF_PI - network_properties->zeta;
			double eta1 = HALF_PI - network_properties->zeta1;
			if (network_properties->dim == 1) {
				#if DEBUG
				assert (network_properties->flags.compact);
				#endif

				if (network_properties->flags.symmetric) {
					network_properties->k_tar = network_properties->N_tar * (eta0 / (sin(eta0) * cos(eta0)) - 1.0) / (HALF_PI * tan(eta0));
					if (!!network_properties->delta)
						network_properties->a = SQRT(network_properties->N_tar / (4.0 * M_PI * network_properties->delta * TAN(eta0, STL)), STL);
					else
						network_properties->delta = network_properties->N_tar / (4.0 * M_PI * POW2(network_properties->a, EXACT) * TAN(eta0, STL));
				} else {
					network_properties->k_tar = network_properties->N_tar * (eta0 / TAN(eta0, STL) - LOG(COS(eta0, STL), STL) - 1.0) / (TAN(eta0, STL) * HALF_PI);
					if (!!network_properties->delta)
						network_properties->a = SQRT(network_properties->N_tar / (TWO_PI * network_properties->delta * TAN(eta0, STL)), STL);
					else
						network_properties->delta = network_properties->N_tar / (TWO_PI * POW2(network_properties->a, EXACT) * TAN(eta0, STL));
				}
			} else if (network_properties->dim == 3) {
				if (network_properties->flags.compact) {
					if (network_properties->flags.symmetric) {
						IntData idata;
						idata.limit = 50;
						idata.tol = 1e-5;
						idata.lower = -eta0;
						idata.upper = eta0;
						double t1 = (sin(eta0) + sin(5.0 * eta0)) / (3.0 * POW3(cos(eta0), EXACT));
						double t2 = 2.0 * integrate1D(&averageDegreeSym, NULL, &idata, QNG);
						double t3 = (2.0 * eta0 * eta0 - 1.0) * (3.0 * sin(eta0) + sin(3.0 * eta0)) / (3.0 * POW3(cos(eta0), EXACT));

						//DEBUG
						//double kappa = TWO_PI * (t1 + t2 + t3) / (tan(eta0) * (2.0 + 1.0 / POW2(cos(eta0), EXACT)));
						//printf("kappa: %f\n", kappa);

						network_properties->k_tar = 3.0 * network_properties->N_tar * POW3(cos(eta0), EXACT) * (t1 + t2 + t3) / (M_PI * tan(eta0) * (2.0 + 1.0 / POW2(cos(eta0), EXACT)) * (3.0 * sin(eta0) + sin(3.0 * eta0)));
						if (!!network_properties->delta)
							network_properties->a = POW(3.0 * network_properties->N_tar * POW3(cos(eta0), EXACT) / (TWO_PI * M_PI * network_properties->delta * (3.0 * sin(eta0) + sin(3.0 * eta0))), 0.25, STL);
						else
							network_properties->delta = 3.0 * network_properties->N_tar * POW3(cos(eta0), EXACT) / (TWO_PI * M_PI * POW2(POW2(network_properties->a, EXACT), EXACT) * (3.0 * sin(eta0) + sin(3.0 * eta0)));
					} else {
						network_properties->k_tar = network_properties->N_tar * (12.0 * (eta0 / TAN(eta0, STL) - LOG(COS(eta0, STL), STL)) - (6.0 * LOG(COS(eta0, STL), STL) + 5.0) / POW2(COS(eta0, STL), EXACT) - 7.0) / (POW2(2.0 + 1.0 / POW2(COS(eta0, STL), EXACT), EXACT) * TAN(eta0, STL) * 3.0 * HALF_PI);
						if (!!network_properties->delta)
							network_properties->a = POW(network_properties->N_tar * 3.0 / (2.0 * POW2(M_PI, EXACT) * network_properties->delta * (2.0 + 1.0 / POW2(COS(eta0, STL), EXACT)) * TAN(eta0, STL)), 0.25, STL);
						else
							network_properties->delta = network_properties->N_tar * 3.0 / (2.0 * POW2(M_PI * POW2(network_properties->a, EXACT), EXACT) * (2.0 + 1.0 / POW2(COS(eta0, STL), EXACT)) * TAN(eta0, STL));
					}
				} else {
					int seed = static_cast<int>(4000000000 * network_properties->mrng.rng());

					network_properties->k_tar = 9.0 * network_properties->N_tar * POW2(POW3(eta0 * eta1, EXACT), EXACT) * integrate2D(&rescaledDegreeDeSitterFlat, eta0, eta0, eta1, eta1, NULL, seed, 0) / (POW3(network_properties->r_max, EXACT) * POW2(POW3(eta1, EXACT) - POW3(eta0, EXACT), EXACT));
					if (!!network_properties->delta)
						network_properties->a = POW(9.0 * network_properties->N_tar * POW3(eta0 * eta1, EXACT) / (4.0 * M_PI * network_properties->delta * POW3(network_properties->r_max, EXACT) * (POW3(eta1, EXACT) - POW3(eta0, EXACT))), 0.25, STL);
					else
						network_properties->delta = 9.0 * network_properties->N_tar * POW3(eta0 * eta1, EXACT) / (4.0 * M_PI * POW2(POW2(network_properties->a, EXACT), EXACT) * POW3(network_properties->r_max, EXACT) * (POW3(eta1, EXACT) - POW3(eta0, EXACT)));
				}
			}

			#if DEBUG
			assert (network_properties->k_tar > 0.0);
			assert (network_properties->a > 0.0);
			assert (network_properties->delta > 0.0);
			#endif

			//Display Constraints
			printf_mpi(rank, "\n");
			printf_mpi(rank, "\tParameters Constraining %d+1 de Sitter Causal Set:\n", network_properties->dim);
			printf_mpi(rank, "\t--------------------------------------------------\n");
			if (!rank) printf_cyan();
			printf_mpi(rank, "\t > Number of Nodes:\t\t%d\n", network_properties->N_tar);
			printf_mpi(rank, "\t > Expected Degrees:\t\t%.6f\n", network_properties->k_tar);
			if (network_properties->flags.compact) {
				if (network_properties->flags.symmetric)
					printf_mpi(rank, "\t > Min. Conformal Time:\t\t%.6f\n", -eta0);
				else
					printf_mpi(rank, "\t > Min. Conformal Time:\t\t0.0\n");
				printf_mpi(rank, "\t > Max. Conformal Time:\t\t%.6f\n", eta0);
			} else {
				printf_mpi(rank, "\t > Min. Conformal Time:\t\t%.6f\n", eta0);
				printf_mpi(rank, "\t > Max. Conformal Time:\t\t%.6f\n", eta1);
			}
			printf_mpi(rank, "\t > Max. Rescaled  Time:\t\t%.6f\n", network_properties->tau0);
			if (!network_properties->flags.compact)
				printf_mpi(rank, "\t > Spatial Cutoff:\t\t%.6f\n", network_properties->r_max);
			printf_mpi(rank, "\t > Node Density: \t\t%.6f\n", network_properties->delta);
			printf_mpi(rank, "\t > Pseudoradius:\t\t%.6f\n", network_properties->a);
			printf_mpi(rank, "\t > Random Seed:\t\t\t%Ld\n", network_properties->seed);
			if (!rank) printf_std();
			fflush(stdout);

			//Miscellaneous Tasks
			if (!network_properties->cmpi.rank && network_properties->flags.gen_ds_table && !generateGeodesicLookupTable("geodesics_ds_table.cset.bin", 5.0, -5.0, 5.0, 0.01, 0.01, network_properties->manifold, network_properties->flags.verbose))
				network_properties->cmpi.fail = 1;

			if (checkMpiErrors(network_properties->cmpi))
				return false;
		} else if (network_properties->manifold == DUST) {
			//Check for under-constrained system
			if (!network_properties->alpha)
				throw CausetException("Flag '--alpha', spatial scale, must be specified!\n");

			if (network_properties->dim == 1)
				throw CausetException("Flag '--dim', spatial dimension, must be (3) in Dust spacetime!\n");

			//Constrain the dust system
			if (!network_properties->delta)
				network_properties->delta = 1000;

			//Let alpha characterize the boundary effects
			if (!network_properties->r_max)
				network_properties->r_max = 1.0;

			//Dust Constraints
			int method = 0;
			if (!solveExpAvgDegree(network_properties->k_tar, network_properties->N_tar, network_properties->dim, network_properties->manifold, network_properties->a, network_properties->r_max, network_properties->tau0, network_properties->alpha, network_properties->delta, network_properties->cmpi.rank, network_properties->mrng, ca, cp->sCalcDegrees, bm->bCalcDegrees, network_properties->flags.compact, network_properties->flags.verbose, network_properties->flags.bench, method))
				network_properties->cmpi.fail = 1;

			if (checkMpiErrors(network_properties->cmpi))
				return false;

			double q = network_properties->N_tar / (M_PI * POW3(network_properties->alpha * network_properties->r_max * network_properties->tau0, EXACT));
			network_properties->a = POW(q / network_properties->delta, 0.25, STL);
			network_properties->alpha *= network_properties->a;

			network_properties->zeta = HALF_PI - tauToEtaDust(network_properties->tau0, network_properties->a, network_properties->alpha);

			#if DEBUG
			assert (network_properties->a > 0.0);
			assert (network_properties->k_tar > 0.0);
			assert (network_properties->zeta < HALF_PI);
			#endif

			//Display Constraints
			printf_mpi(rank, "\n");
			printf_mpi(rank, "\tParameters Constraining the Dusty Causal Set:\n");
			printf_mpi(rank, "\t---------------------------------------------\n");
			if (!rank) printf_cyan();
			printf_mpi(rank, "\t > Number of Expected Nodes:\t%d\n", network_properties->N_tar);
			printf_mpi(rank, "\t > Expected Degrees:\t\t%.6f\n", network_properties->k_tar);
			printf_mpi(rank, "\t > Max. Rescaled Time:\t\t%.6f\n", network_properties->tau0);
			printf_mpi(rank, "\t > Max. Conformal Time:\t\t%.6f\n", HALF_PI - network_properties->zeta);
			printf_mpi(rank, "\t > Spatial Scaling:\t\t%.6f\n", network_properties->alpha);
			printf_mpi(rank, "\t > Temporal Scaling:\t\t%.6f\n", network_properties->a);
			printf_mpi(rank, "\t > Node Density:\t\t%.6f\n", network_properties->delta);
			printf_mpi(rank, "\t > Random Seed:\t\t\t%Ld\n", network_properties->seed);
			if (!rank) printf_std();
			fflush(stdout);
		} else if (network_properties->manifold == FLRW) {
			//Check for under-constrained system
			if (!network_properties->alpha)
				throw CausetException("Flag '--alpha', spatial scale, must be specified!\n");
			if (network_properties->dim == 1)
				throw CausetException("Flag '--dim', spatial dimension, must be (3) in FLRW spacetime!\n");

			//Constrain the FLRW system
			if (!network_properties->delta)
				network_properties->delta = 1000;

			if (network_properties->flags.compact) {
				double q = 3.0 * network_properties->N_tar / (POW2(M_PI, EXACT) * POW3(network_properties->alpha, EXACT) * (SINH(3.0 * network_properties->tau0, STL) - 3.0 * network_properties->tau0));
				network_properties->a = POW(q / network_properties->delta, 1.0 / 4.0, STL);
				//\tilde{\alpha} -> \alpha
				network_properties->alpha *= network_properties->a;
				//Use lookup table to solve for k_tar
				int method = 1;
				if (!solveExpAvgDegree(network_properties->k_tar, network_properties->N_tar, network_properties->dim, network_properties->manifold, network_properties->a, network_properties->r_max, network_properties->tau0, network_properties->alpha, network_properties->delta, network_properties->cmpi.rank, network_properties->mrng, ca, cp->sCalcDegrees, bm->bCalcDegrees, network_properties->flags.compact, network_properties->flags.verbose, network_properties->flags.bench, method))
					network_properties->cmpi.fail = 1;

				if (checkMpiErrors(network_properties->cmpi))
					return false;
			} else {
				//This makes alpha characterize the boundary effects
				if (!network_properties->r_max)
					network_properties->r_max = 1.0;

				//Non-Compact FLRW Constraints
				int method = 0;
				//int method = 1;
				if (!solveExpAvgDegree(network_properties->k_tar, network_properties->N_tar, network_properties->dim, network_properties->manifold, network_properties->a, network_properties->r_max, network_properties->tau0, network_properties->alpha, network_properties->delta, network_properties->cmpi.rank, network_properties->mrng, ca, cp->sCalcDegrees, bm->bCalcDegrees, network_properties->flags.compact, network_properties->flags.verbose, network_properties->flags.bench, method))
					network_properties->cmpi.fail = 1;

				if (checkMpiErrors(network_properties->cmpi))
					return false;
				
				double q = 9.0 * network_properties->N_tar / (TWO_PI * POW3(network_properties->alpha * network_properties->r_max, EXACT) * (SINH(3.0 * network_properties->tau0, STL) - 3.0 * network_properties->tau0));
				network_properties->a = POW(q / network_properties->delta, 0.25, STL);
				network_properties->alpha *= network_properties->a;
			}

			network_properties->zeta = HALF_PI - tauToEtaFLRWExact(network_properties->tau0, network_properties->a, network_properties->alpha);

			#if DEBUG
			assert (network_properties->a > 0.0);
			assert (network_properties->k_tar > 0.0);
			assert (network_properties->zeta < HALF_PI);
			#endif

			//Display Constraints
			printf_mpi(rank, "\n");
			printf_mpi(rank, "\tParameters Constraining the FLRW Causal Set:\n");
			printf_mpi(rank, "\t--------------------------------------------\n");
			if (!rank) printf_cyan();
			printf_mpi(rank, "\t > Number of Nodes:\t\t%d\n", network_properties->N_tar);
			printf_mpi(rank, "\t > Expected Degrees:\t\t%.6f\n", network_properties->k_tar);
			if (!rank) printf_red();
			printf_mpi(rank, "\t > Dark Energy Density:\t\t%.6f\n", network_properties->omegaL);
			if (!rank) printf_cyan();
			printf_mpi(rank, "\t > Max. Rescaled Time:\t\t%.6f\n", network_properties->tau0);
			printf_mpi(rank, "\t > Max. Conformal Time:\t\t%.6f\n", HALF_PI - network_properties->zeta);
			printf_mpi(rank, "\t > Spatial Scaling:\t\t%.6f\n", network_properties->alpha);
			printf_mpi(rank, "\t > Temporal Scaling:\t\t%.6f\n", network_properties->a);
			printf_mpi(rank, "\t > Node Density:\t\t%.6f\n", network_properties->delta);
			printf_mpi(rank, "\t > Random Seed:\t\t\t%Ld\n", network_properties->seed);
			if (!rank) printf_std();
			fflush(stdout);

			//Miscellaneous Tasks
			if (!network_properties->cmpi.rank && network_properties->flags.gen_flrw_table && !generateGeodesicLookupTable("geodesics_flrw_table.cset.bin", 2.0, -5.0, 5.0, 0.01, 0.01, network_properties->manifold, network_properties->flags.verbose))
				network_properties->cmpi.fail = 1;

			if (checkMpiErrors(network_properties->cmpi))
				return false;
		} else if (network_properties->manifold == HYPERBOLIC) {
			if (network_properties->dim != 1)
				throw CausetException("You must use --dim 1 for a hyperbolic manifold!\n");
			if (network_properties->zeta == 0.0)
				network_properties->zeta = 1.0;
		}

		//Miscellaneous Tasks
		if (network_properties->edge_buffer == 0.0)
			network_properties->edge_buffer = 0.2;

		#ifdef CUDA_ENABLED
		//Determine group size and decoding method
		if (network_properties->flags.use_gpu) {
			long glob_mem = 5000000000L;
			long mem = glob_mem + 1L;
			long d_edges_size = static_cast<long>(exp2(ceil(log2(network_properties->N_tar * network_properties->k_tar * (1.0 + network_properties->edge_buffer) / 2.0))));
			float gsize = 0.5f;
			bool dcpu = false;

			while (mem > glob_mem) {
				gsize *= 2.0f;
				//long mbsize = static_cast<long>(ceil(static_cast<float>(network_properties->N_tar) / (BLOCK_SIZE * gsize * 2)));
				long mbsize = static_cast<long>(ceil(static_cast<float>(network_properties->N_tar) / (BLOCK_SIZE * gsize)));
				long mtsize = mbsize * BLOCK_SIZE;
				long mesize = mtsize * mtsize;
				long gmbsize = static_cast<long>(network_properties->N_tar * network_properties->k_tar * (1.0 + network_properties->edge_buffer) / (BLOCK_SIZE * gsize * 2));
				long gmtsize = gmbsize * BLOCK_SIZE;

				long mem1 = (40L * mtsize + mesize) * NBUFFERS;
				long mem2 = 4L * (2L * d_edges_size + gmtsize);
				long mem3 = 8L * (network_properties->N_tar + 2L * BLOCK_SIZE);

				if (mem2 > glob_mem / 4L) {
					mem2 = 0L;
					dcpu = true;
				}

				long max = mem1;
				if (mem2 > max) max = mem2;
				if (mem3 > max) max = mem3;
				mem = max;
			}

			network_properties->group_size = gsize < NBUFFERS ? NBUFFERS : gsize;
			network_properties->flags.decode_cpu = dcpu;
		}
		#endif

		if (network_properties->flags.calc_deg_field && network_properties->tau_m >= network_properties->tau0)
			throw CausetException("You have chosen to measure the degree field at a time greater than the maximum time!\n");
		
		uint64_t pair_multiplier = static_cast<uint64_t>(network_properties->N_tar) * (network_properties->N_tar - 1) / 2;
		if (network_properties->flags.calc_success_ratio && network_properties->N_sr <= 1.0)
			network_properties->N_sr *= pair_multiplier;
		if (network_properties->flags.validate_embedding && network_properties->N_emb <= 1.0)
			network_properties->N_emb *= pair_multiplier;
		if (network_properties->flags.validate_distances && network_properties->N_dst <= 1.0)
			network_properties->N_dst *= pair_multiplier;

		if (network_properties->flags.calc_action) {
			#if DEBUG
			assert (network_properties->max_cardinality == -1 || network_properties->max_cardinality == 1);
			#endif
			if (network_properties->max_cardinality == -1)
				network_properties->max_cardinality = 5;
			else
				network_properties->max_cardinality = network_properties->N_tar - 1;
		}
	} catch (CausetException c) {
		fprintf(stderr, "CausetException in %s: %s on line %d\n", __FILE__, c.what(), __LINE__);
		network_properties->cmpi.fail = 1;
	} catch (std::bad_alloc) {
		fprintf(stderr, "Memory allocation failure in %s on line %d!\n", __FILE__, __LINE__);
		network_properties->cmpi.fail = 1;
	} catch (std::exception e) {
		fprintf(stderr, "Unknown exception in %s: %s on line %d\n", __FILE__, e.what(), __LINE__);
		network_properties->cmpi.fail = 1;
	}

	if (checkMpiErrors(network_properties->cmpi))
		return false;

	return true;
}

//Calculate Expected Average Degree in the Dust or FLRW Spacetime
//See Causal Set Notes for detailed explanation of methods
bool solveExpAvgDegree(float &k_tar, const int &N_tar, const int &dim, const Manifold &manifold, double &a, const double &r_max, double &tau0, const double &alpha, const double &delta, const int &rank, MersenneRNG &mrng, CaResources * const ca, Stopwatch &sCalcDegrees, double &bCalcDegrees, const bool &compact, const bool &verbose, const bool &bench, const int method)
{
	#if DEBUG
	assert (ca != NULL);
	assert (N_tar > 0);
	assert (dim == 3);
	assert (manifold == DUST || manifold == FLRW);
	assert (tau0 > 0.0);
	assert (alpha > 0.0);
	assert (delta > 0.0);
	assert (method == 0 || method == 1 || method == 2);
	if (manifold == DUST) {
		assert (!compact);
		assert (method == 0);
	}
	if (!compact) {
		assert (method == 0 || method == 1);
		assert (r_max > 0.0);
	} else
		assert (a > 0.0);
	#endif

	printf_mpi(rank, "\tEstimating Expected Average Degree...\n");
	fflush(stdout);

	int nb = static_cast<int>(bench) * NBENCH;
	int i;

	double *table;
	long size = 0L;
	int seed = static_cast<int>(4000000000 * mrng.rng());

	if (method == 0) {
		//Method 1 of 3: Use Monte Carlo integration
		double r0;
		if (tau0 > LOG(MTAU, STL) / 3.0)
			r0 = POW(0.5, 2.0 / 3.0, STL) * exp(tau0);
		else
			r0 = POW(SINH(1.5 * tau0, STL), 2.0 / 3.0, STL);

		for (i = 0; i <= nb; i++) {
			stopwatchStart(&sCalcDegrees);
			if (compact) {
				if (tau0 > LOG(MTAU, STL) / 3.0)
					k_tar = delta * POW2(POW2(a, EXACT), EXACT) * integrate2D(&rescaledDegreeFLRW, 0.0, 0.0, r0, r0, NULL, seed, 0) * 16.0 * M_PI * exp(-3.0 * tau0);
				else
					k_tar = delta * POW2(POW2(a, EXACT), EXACT) * integrate2D(&rescaledDegreeFLRW, 0.0, 0.0, r0, r0, NULL, seed, 0) * 8.0 * M_PI / (SINH(3.0 * tau0, STL) - 3.0 * tau0);
			} else {
				if (manifold == DUST) {
					double kappa = integrate2D(&rescaledDegreeDust, 0.0, 0.0, tau0, tau0, NULL, seed, 0);
					kappa *= 108 * M_PI / POW3(tau0, EXACT);
					printf("kappa: %.8e\n", kappa);
					k_tar = (N_tar * kappa) / (M_PI * POW3(alpha * r_max * tau0, EXACT));
				} else if (manifold == FLRW) {
					double kappa = integrate2D(&rescaledDegreeFLRW_NC, 0.0, 0.0, tau0, tau0, NULL, seed, 0);
					kappa *= 8.0 * M_PI;
					kappa /= SINH(3.0 * tau0, STL) - 3.0 * tau0;
					//printf("kappa: %.8e\n", kappa);
					k_tar = (9.0 * kappa * N_tar) / (TWO_PI * POW3(alpha * r_max, EXACT) * (SINH(3.0 * tau0, STL) - 3.0 * tau0));
				}
			}
			stopwatchStop(&sCalcDegrees);
		}	
	} else if (method == 1) {
		//Method 2 of 3: Lookup table to approximate method 1
		if (compact) {
			if (!getLookupTable("./etc/raduc_table.cset.bin", &table, &size))
				return false;
		} else {
			if (!getLookupTable("./etc/raducNC_table.cset.bin", &table, &size))
				return false;
		}
		ca->hostMemUsed += size;

		for (i = 0; i <= nb; i++) {
			stopwatchStart(&sCalcDegrees);
			if (compact)
				k_tar = lookupValue(table, size, &tau0, NULL, true) * delta * POW2(POW2(a, EXACT), EXACT);
			else
				k_tar = lookupValue(table, size, &tau0, NULL, true) * 9.0 * N_tar / (TWO_PI * POW3(alpha * r_max, EXACT) * (SINH(3.0 * tau0, STL) - 3.0 * tau0));
			stopwatchStop(&sCalcDegrees);
		}	

		//Check for NaN
		if (k_tar != k_tar)
			return false;

		free(table);
		table = NULL;
		ca->hostMemUsed -= size;
	} else if (method == 2) {
		//Method 3 of 3: Explicit Solution
		if (!getLookupTable("./etc/ctuc_table.cset.bin", &table, &size))
			return false;
		ca->hostMemUsed += size;

		double *params = (double*)malloc(size + sizeof(double) * 3);
		if (params == NULL)
			throw std::bad_alloc();
		ca->hostMemUsed += size + sizeof(double) * 3;

		double d_size = static_cast<double>(size);
		memcpy(params, &a, sizeof(double));
		memcpy(params + 1, &alpha, sizeof(double));
		memcpy(params + 2, &d_size, sizeof(double));
		memcpy(params + 3, table, size);

		IntData idata = IntData();
		idata.limit = 50;
		idata.tol = 1e-5;
		idata.workspace = gsl_integration_workspace_alloc(idata.nintervals);
		idata.upper = tau0;

		double max_time;
		for (i = 0; i <= nb; i++) {
			stopwatchStart(&sCalcDegrees);
			max_time = integrate1D(&tauToEtaFLRW, NULL, &idata, QAGS) * a / alpha;
			stopwatchStop(&sCalcDegrees);
		}

		gsl_integration_workspace_free(idata.workspace);

		k_tar = integrate2D(&averageDegreeFLRW, 0.0, 0.0, max_time, max_time, params, seed, 0);
		k_tar *= 4.0 * M_PI * delta * POW2(POW2(alpha, EXACT), EXACT);

		for (i = 0; i <= nb; i++) {
			stopwatchStart(&sCalcDegrees);
			integrate2D(&averageDegreeFLRW, 0.0, 0.0, max_time, max_time, params, seed, 0);
			stopwatchStop(&sCalcDegrees);
		}
	
		idata.workspace = gsl_integration_workspace_alloc(idata.nintervals);
		idata.upper = max_time;
		k_tar /= (3.0 * integrate1D(&psi, params, &idata, QAGS));

		for (i = 0; i <= nb; i++) {
			stopwatchStart(&sCalcDegrees);
			integrate1D(&psi, params, &idata, QAGS);
			stopwatchStop(&sCalcDegrees);
		}

		gsl_integration_workspace_free(idata.workspace);

		free(params);
		params = NULL;
		ca->hostMemUsed -= size + sizeof(double) * 3;

		free(table);
		table = NULL;
		ca->hostMemUsed -= size;
	}

	if (nb)
		bCalcDegrees = sCalcDegrees.elapsedTime / NBENCH;

	if (verbose) {
		printf_mpi(rank, "\t\tExecution Time: %5.6f sec\n", sCalcDegrees.elapsedTime);
		fflush(stdout);
	}

	if (!bench) {
		printf_mpi(rank, "\tExpected Average Degree Successfully Calculated.\n");
		printf_mpi(rank, "\t\t<k> = %f\n", k_tar);
		fflush(stdout);
	}

	return true;
}

//Allocates memory for network
//O(1) Efficiency
bool createNetwork(Node &nodes, Edge &edges, bool *& core_edge_exists, const int &N_tar, const float &k_tar, const int &dim, const Manifold &manifold, const float &core_edge_fraction, const float &edge_buffer, CausetMPI &cmpi, const int &group_size, CaResources * const ca, Stopwatch &sCreateNetwork, const bool &use_gpu, const bool &decode_cpu, const bool &link, const bool &relink, const bool &verbose, const bool &bench, const bool &yes)
{
	#if DEBUG
	assert (ca != NULL);
	assert (N_tar > 0);
	assert (k_tar > 0.0f);
	assert (dim == 1 || dim == 3);
	assert (manifold == DE_SITTER || manifold == DUST || manifold == FLRW || manifold == HYPERBOLIC);
	if (manifold == HYPERBOLIC)
		assert (dim == 1);
	assert (core_edge_fraction >= 0.0f && core_edge_fraction <= 1.0f);
	assert (edge_buffer >= 0.0f && edge_buffer <= 1.0f);
	#endif

	int rank = cmpi.rank;
	bool links_exist = link || relink;

	if (verbose && !yes) {
		//Estimate memory usage before allocating
		size_t mem = 0;
		if (dim == 3)
			mem += sizeof(float) * N_tar << 2;	//For Coordinate4D
		else if (dim == 1)
			mem += sizeof(float) * N_tar << 1;	//For Coordinate2D
		if (manifold == HYPERBOLIC)
			mem += sizeof(int) * N_tar;		//For AS
		else if (manifold == DE_SITTER || manifold == DUST || manifold == FLRW)
			mem += sizeof(float) * N_tar;		//For tau
		if (links_exist) {
			mem += sizeof(int) * (N_tar << 1);	//For k_in and k_out
			mem += sizeof(int) * static_cast<int>(N_tar * k_tar * (1.0 + edge_buffer));	//For edge lists
			mem += sizeof(int) * (N_tar << 1);	//For edge list pointers
			mem += sizeof(bool) * POW2(core_edge_fraction * N_tar, EXACT);	//For adjacency list
		}

		size_t dmem = 0;
		#ifdef CUDA_ENABLED
		size_t dmem1 = 0, dmem2 = 0, dmem3 = 0;
		if (use_gpu) {
			size_t d_edges_size = pow(2.0, ceil(log2(N_tar * k_tar * (1.0 + edge_buffer) / 2)));
			mem += sizeof(uint64_t) * d_edges_size;	//For encoded edge list
			mem += sizeof(int);			//For g_idx

			size_t mblock_size = static_cast<unsigned int>(ceil(static_cast<float>(N_tar) / (BLOCK_SIZE * group_size << 1)));
			size_t mthread_size = mblock_size * BLOCK_SIZE;
			size_t m_edges_size = mthread_size * mthread_size;
			size_t nbuf = GEN_ADJ_LISTS_GPU_V2 ? NBUFFERS : 1;
			mem += sizeof(int) * mthread_size * nbuf << 1;		//For k_in and k_out buffers (host)
			mem += sizeof(bool) * m_edges_size * nbuf;			//For adjacency matrix buffers (host)
			dmem1 += sizeof(float) * mthread_size * 4 * nbuf << 1;	//For 4-D coordinate buffers
			dmem1 += sizeof(int) * mthread_size * nbuf << 1;		//For k_in and k_out buffers (device)
			dmem1 += sizeof(bool) * m_edges_size * nbuf;			//For adjacency matrix buffers (device)

			size_t g_mblock_size = static_cast<unsigned int>(N_tar * k_tar * (1.0 + edge_buffer) / (BLOCK_SIZE * group_size << 1));
			size_t g_mthread_size = g_mblock_size * BLOCK_SIZE;
			dmem2 += sizeof(uint64_t) * d_edges_size;	//Encoded edge list used during parallel sorting
			dmem2 += sizeof(int) * (DECODE_LISTS_GPU_V2 ? g_mthread_size : d_edges_size);	//For edge lists
			if (decode_cpu)
				dmem2 = 0;

			dmem3 += sizeof(int) * N_tar << 1;	//Edge list pointers
			dmem3 += sizeof(int) * BLOCK_SIZE << 2;	//Buffers used for scanning

			dmem = dmem1 > dmem2 ? dmem1 : dmem2;
			dmem = dmem > dmem3 ? dmem : dmem3;
		}
		#endif

		printMemUsed("for Network (Estimation)", mem, dmem, rank);
		printf("\nContinue [y/N]?");
		fflush(stdout);
		char response = getchar();
		getchar();
		if (response != 'y')
			return false;
	}

	stopwatchStart(&sCreateNetwork);

	try {
		if (manifold == DE_SITTER || manifold == DUST || manifold == FLRW) {
			nodes.id.tau = (float*)malloc(sizeof(float) * N_tar);
			if (nodes.id.tau == NULL)
				throw std::bad_alloc();
			memset(nodes.id.tau, 0, sizeof(float) * N_tar);
			ca->hostMemUsed += sizeof(float) * N_tar;
		} else if (manifold == HYPERBOLIC) {
			nodes.id.AS = (int*)malloc(sizeof(int) * N_tar);
			if (nodes.id.AS == NULL)
				throw std::bad_alloc();
			memset(nodes.id.AS, 0, sizeof(int) * N_tar);
			ca->hostMemUsed += sizeof(int) * N_tar;
		}

		if (dim == 3) {
			nodes.crd = new Coordinates4D();

			nodes.crd->w() = (float*)malloc(sizeof(float) * N_tar);
			nodes.crd->x() = (float*)malloc(sizeof(float) * N_tar);
			nodes.crd->y() = (float*)malloc(sizeof(float) * N_tar);
			nodes.crd->z() = (float*)malloc(sizeof(float) * N_tar);

			if (nodes.crd->w() == NULL || nodes.crd->x() == NULL || nodes.crd->y() == NULL || nodes.crd->z() == NULL)
				throw std::bad_alloc();

			memset(nodes.crd->w(), 0, sizeof(float) * N_tar);
			memset(nodes.crd->x(), 0, sizeof(float) * N_tar);
			memset(nodes.crd->y(), 0, sizeof(float) * N_tar);
			memset(nodes.crd->z(), 0, sizeof(float) * N_tar);

			ca->hostMemUsed += sizeof(float) * N_tar * 4;
		} else if (dim == 1) {
			nodes.crd = new Coordinates2D();

			nodes.crd->x() = (float*)malloc(sizeof(float) * N_tar);
			nodes.crd->y() = (float*)malloc(sizeof(float) * N_tar);

			if (nodes.crd->x() == NULL || nodes.crd->y() == NULL)
				throw std::bad_alloc();

			memset(nodes.crd->x(), 0, sizeof(float) * N_tar);
			memset(nodes.crd->y(), 0, sizeof(float) * N_tar);

			ca->hostMemUsed += sizeof(float) * N_tar * 2;
		}

		if (links_exist) {
			nodes.k_in = (int*)malloc(sizeof(int) * N_tar);
			if (nodes.k_in == NULL)
				throw std::bad_alloc();
			memset(nodes.k_in, 0, sizeof(int) * N_tar);
			ca->hostMemUsed += sizeof(int) * N_tar;

			nodes.k_out = (int*)malloc(sizeof(int) * N_tar);
			if (nodes.k_out == NULL)
				throw std::bad_alloc();
			memset(nodes.k_out, 0, sizeof(int) * N_tar);
			ca->hostMemUsed += sizeof(int) * N_tar;
		}

		if (verbose)
			printMemUsed("for Nodes", ca->hostMemUsed, ca->devMemUsed, rank);

		if (links_exist) {
			edges.past_edges = (int*)malloc(sizeof(int) * static_cast<unsigned int>(N_tar * k_tar * (1.0 + edge_buffer) / 2));
			if (edges.past_edges == NULL)
				throw std::bad_alloc();
			memset(edges.past_edges, 0, sizeof(int) * static_cast<unsigned int>(N_tar * k_tar * (1.0 + edge_buffer) / 2));
			ca->hostMemUsed += sizeof(int) * static_cast<unsigned int>(N_tar * k_tar * (1.0 + edge_buffer) / 2);

			edges.future_edges = (int*)malloc(sizeof(int) * static_cast<unsigned int>(N_tar * k_tar * (1.0 + edge_buffer) / 2));
			if (edges.future_edges == NULL)
				throw std::bad_alloc();
			memset(edges.future_edges, 0, sizeof(int) * static_cast<unsigned int>(N_tar * k_tar * (1.0 + edge_buffer) / 2));
			ca->hostMemUsed += sizeof(int) * static_cast<unsigned int>(N_tar * k_tar * (1.0 + edge_buffer) / 2);

			edges.past_edge_row_start = (int*)malloc(sizeof(int) * N_tar);
			if (edges.past_edge_row_start == NULL)
				throw std::bad_alloc();
			memset(edges.past_edge_row_start, 0, sizeof(int) * N_tar);
			ca->hostMemUsed += sizeof(int) * N_tar;
	
			edges.future_edge_row_start = (int*)malloc(sizeof(int) * N_tar);
			if (edges.future_edge_row_start == NULL)
				throw std::bad_alloc();
			memset(edges.future_edge_row_start, 0, sizeof(int) * N_tar);
			ca->hostMemUsed += sizeof(int) * N_tar;

			core_edge_exists = (bool*)malloc(sizeof(bool) * static_cast<uint64_t>(POW2(core_edge_fraction * N_tar, EXACT)));
			if (core_edge_exists == NULL)
				throw std::bad_alloc();
			memset(core_edge_exists, 0, sizeof(bool) * static_cast<uint64_t>(POW2(core_edge_fraction * N_tar, EXACT)));
			ca->hostMemUsed += sizeof(bool) * static_cast<uint64_t>(POW2(core_edge_fraction * N_tar, EXACT));
		}

		memoryCheckpoint(ca->hostMemUsed, ca->maxHostMemUsed, ca->devMemUsed, ca->maxDevMemUsed);
		if (verbose)
			printMemUsed("for Network", ca->hostMemUsed, ca->devMemUsed, rank);
	} catch (std::bad_alloc) {
		fprintf(stderr, "Memory allocation failure in %s on line %d!\n", __FILE__, __LINE__);
		cmpi.fail = 1;
	}

	if (nodes.crd->isNull()) {
		printf("Null in thread %d\n", rank);
		cmpi.fail = 1;
	}

	if (checkMpiErrors(cmpi))
		return false;
	
	stopwatchStop(&sCreateNetwork);

	if (!bench) {
		printf_mpi(rank, "\tMemory Successfully Allocated.\n");
		fflush(stdout);
	}

	if (verbose) {
		printf_mpi(rank, "\t\tExecution Time: %5.6f sec\n", sCreateNetwork.elapsedTime);
		fflush(stdout);
	}

	return true;
}

//Poisson Sprinkling
//O(N) Efficiency
bool generateNodes(Node &nodes, const int &N_tar, const float &k_tar, const int &dim, const Manifold &manifold, const double &a, const double &zeta, const double &zeta1, const double &r_max, const double &tau0, const double &alpha, MersenneRNG &mrng, Stopwatch &sGenerateNodes, const bool &use_gpu, const bool &symmetric, const bool &compact, const bool &verbose, const bool &bench)
{
	#if DEBUG
	//Values are in correct ranges
	assert (!nodes.crd->isNull());
	assert (N_tar > 0);
	assert (k_tar > 0.0f);
	assert (dim == 1 || dim == 3);
	assert (manifold == DE_SITTER || manifold == DUST || manifold == FLRW);
	assert (a >= 0.0);
	assert (tau0 > 0.0);
	if (manifold == DUST || manifold == FLRW) {
		assert (nodes.crd->getDim() == 4);
		assert (nodes.crd->w() != NULL);
		assert (nodes.crd->x() != NULL);
		assert (nodes.crd->y() != NULL);
		assert (nodes.crd->z() != NULL);
		assert (dim == 3);
		assert (zeta < HALF_PI);
	} else if (manifold == DE_SITTER) {
		if (compact) {
			assert (zeta > 0.0);
			assert (zeta < HALF_PI);
		} else {
			assert (zeta > HALF_PI);
			assert (zeta1 > HALF_PI);
			assert (zeta > zeta1);
		}
	}
	if (!compact)
		assert (r_max > 0.0);
	#endif

	IntData idata = IntData();
	//Modify these two parameters to trade off between speed and accuracy
	idata.limit = 50;
	idata.tol = 1e-4;

	if (USE_GSL && manifold == FLRW)
		idata.workspace = gsl_integration_workspace_alloc(idata.nintervals);

	stopwatchStart(&sGenerateNodes);

	//Generate coordinates for each of N nodes
	double x, rval;
	int i;
	for (i = 0; i < N_tar; i++) {
		////////////////////////////////////////////////////////////
		//~~~~~~~~~~~~~~~~~~~~~~~~~Theta3~~~~~~~~~~~~~~~~~~~~~~~~~//
		//Sample Theta3 from (0, 2pi), as described on p. 2 of [1]//
		////////////////////////////////////////////////////////////

		x = TWO_PI * mrng.rng();
		#if DEBUG
		assert (x > 0.0 && x < TWO_PI);
		#endif
		//if (i % NPRINT == 0) printf("Theta3: %5.5f\n", x); fflush(stdout);

		if (dim == 1) {
			nodes.crd->y(i) = static_cast<float>(x);

			/////////////////////////////////////////////////
			//~~~~~~~~~~~~~~~~~~~~Eta~~~~~~~~~~~~~~~~~~~~~~//
			//CDF derived from PDF identified in (2) of [2]//
			/////////////////////////////////////////////////

			do nodes.crd->x(i) = static_cast<float>(ATAN(mrng.rng() / TAN(zeta, APPROX ? FAST : STL), APPROX ? INTEGRATION : STL, VERY_HIGH_PRECISION));
			while (nodes.crd->x(i) >= static_cast<float>(HALF_PI - zeta));

			#if DEBUG
			assert (nodes.crd->x(i) > 0.0f);
			assert (nodes.crd->x(i) < static_cast<float>(HALF_PI - zeta));
			#endif

			// Use the symmetric interval between (-eta0, eta0)
			if (symmetric) {
				if (!!(i % 2))
					nodes.crd->x(i) *= -1.0;
			}

			nodes.id.tau[i] = static_cast<float>(etaToTauCompact(static_cast<double>(nodes.crd->x(i))));
		} else if (dim == 3) {
			nodes.crd->z(i) = static_cast<float>(x);

			/////////////////////////////////////////////////////////
			//~~~~~~~~~~~~~~~~~~~~~~~~~~~~T~~~~~~~~~~~~~~~~~~~~~~~~//
			//CDF derived from PDF identified in (6) of [2] for 3+1//
			//and from PDF identified in (12) of [2] for FLRW      //
			/////////////////////////////////////////////////////////

			do {
				rval = mrng.rng();

				double p1[2];
				p1[1] = rval;

				if (manifold == FLRW) {
					x = 0.5;
					p1[0] = tau0;
					if (tau0 > 1.8) {	//Cutoff of 1.8 determined by trial and error
						if (!bisection(&solveTauUnivBisec, &x, 2000, 0.0, tau0, TOL, true, p1, NULL, NULL))
							return false;
					} else {
						if (!newton(&solveTauUniverse, &x, 1000, TOL, p1, NULL, NULL))
							return false;
					}
				} else if (manifold == DUST) {
					x = tau0 * POW(rval, 1.0 / 3.0, STL);
					nodes.crd->w(i) = tauToEtaDust(x, a, alpha);
				} else if (manifold == DE_SITTER) {
					if (compact) {
						x = 3.5;
						p1[0] = zeta;
						if (!newton(&solveTau, &x, 1000, TOL, p1, NULL, NULL))
							return false;

						if (symmetric) {
							if (!!(i % 2))
								x *= -1.0;
						}
					} else {
						//In this case the eta distribution is used
						double eta0 = HALF_PI - zeta;
						double eta1 = HALF_PI - zeta1;
						nodes.crd->w(i) = eta0 * POW(1.0 - rval * (1.0 - POW3(eta0 / eta1, EXACT)), -1.0 / 3.0, STL);
						x = etaToTauFlat(nodes.crd->w(i));
					}
				}

				nodes.id.tau[i] = static_cast<float>(x);
			} while (nodes.id.tau[i] >= static_cast<float>(tau0));

			#if DEBUG
			//assert (nodes.id.tau[i] >= 0.0f);
			assert (nodes.id.tau[i] < static_cast<float>(tau0));
			#endif

			//Save eta values as well
			if (manifold == FLRW) {
				if (USE_GSL) {
					//Numerical Integration
					idata.upper = static_cast<double>(nodes.id.tau[i]);
					nodes.crd->w(i) = static_cast<float>(integrate1D(&tauToEtaFLRW, NULL, &idata, QAGS) * a / alpha);
				} else
					//Exact Solution
					nodes.crd->w(i) = static_cast<float>(tauToEtaFLRWExact(nodes.id.tau[i], a, alpha));

				//#if DEBUG 
				//assert (nodes.crd->w(i) < tauToEtaFLRWExact(tau0, a, alpha));
				//#endif
			} else if (manifold == DE_SITTER && compact) {
				nodes.crd->w(i) = static_cast<float>(tauToEtaCompact(static_cast<double>(nodes.id.tau[i])));
				#if DEBUG
				assert (fabs(nodes.crd->w(i)) < tauToEtaCompact(tau0));
				#endif
			}
			#if DEBUG
			if (manifold == DE_SITTER && !compact)
				assert (nodes.crd->w(i) < 0.0);
			//else
			//	assert (nodes.crd->w(i) > 0.0);
			#endif
				
			///////////////////////////////////////////////////////
			//~~~~~~~~~~~~~~~~Theta1 and Theta2~~~~~~~~~~~~~~~~~~//	
			//CDFs derived from PDFs identified on p. 3 of [2]   //
			//Phi given by [3]				     //
			///////////////////////////////////////////////////////

			if (compact) {
				//Sample Theta1 from (0, pi)
				x = HALF_PI;
				rval = mrng.rng();
				if (!newton(&solveTheta1, &x, 250, TOL, &rval, NULL, NULL))
					return false;
				nodes.crd->x(i) = static_cast<float>(x);
				#if DEBUG
				assert (nodes.crd->x(i) > 0.0f && nodes.crd->x(i) < static_cast<float>(M_PI));
				#endif
			} else {
				nodes.crd->x(i) = static_cast<float>(POW(mrng.rng(), 1.0 / 3.0, APPROX ? FAST : STL) * r_max);
				#if DEBUG
				assert (nodes.crd->x(i) >= 0.0f && nodes.crd->x(i) <= static_cast<float>(r_max));
				#endif
			}
			//if (i % NPRINT == 0) printf("Theta1: %5.5f\n", nodes.crd->x(i)); fflush(stdout);

			//Sample Theta2 from (0, pi)
			nodes.crd->y(i) = static_cast<float>(ACOS(1.0 - 2.0 * mrng.rng(), APPROX ? INTEGRATION : STL, VERY_HIGH_PRECISION));
			#if DEBUG
			assert (nodes.crd->y(i) >= 0.0f && nodes.crd->y(i) < static_cast<float>(M_PI));
			#endif
			//if (i % NPRINT == 0) printf("Theta2: %5.5f\n", nodes.crd->y(i)); fflush(stdout);
		}
		//if (i % NPRINT == 0) printf("eta: %5.5f\n", nodes.crd->w(i));
		//if (i % NPRINT == 0) printf("tau: %E\n", nodes.id.tau[i]);
	}

	//Manually alter points
	/*nodes.id.tau[0] = tau0 * 0.95;
	nodes.crd->w(0) = tauToEtaFlat(nodes.id.tau[0]);
	nodes.crd->x(0) = r_max * 0.95;
	nodes.crd->y(0) = 0.01;
	nodes.crd->z(0) = 0.01;

	nodes.id.tau[1] = tau0 * 0.96;
	nodes.crd->w(1) = tauToEtaFlat(nodes.id.tau[1]);
	nodes.crd->x(1) = r_max * 0.95;
	nodes.crd->y(1) = HALF_PI / 12.0;
	nodes.crd->z(1) = 0.01;

	double omega12;
	nodesAreRelated(nodes.crd, N_tar, dim, manifold, a, zeta, zeta1, r_max, alpha, compact, 0, 1, &omega12);
	printf("omega12: %f\n", omega12);*/

	//Debugging statements used to check coordinate distributions
	/*if (!printValues(nodes, N_tar, "tau_dist.cset.dbg.dat", "tau")) return false;
	if (!printValues(nodes, N_tar, "eta_dist.cset.dbg.dat", "eta")) return false;
	if (!printValues(nodes, N_tar, "theta1_dist.cset.dbg.dat", "theta1")) return false;
	if (!printValues(nodes, N_tar, "theta2_dist.cset.dbg.dat", "theta2")) return false;
	if (!printValues(nodes, N_tar, "theta3_dist.cset.dbg.dat", "theta3")) return false;
	printf_red();
	printf("Check coordinate distributions now.\n");
	printf_std();
	fflush(stdout);
	printChk();*/

	stopwatchStop(&sGenerateNodes);

	if (USE_GSL && manifold == FLRW)
		gsl_integration_workspace_free(idata.workspace);

	if (!bench) {
		printf("\tNodes Successfully Generated.\n");
		fflush(stdout);
	}

	if (verbose) {
		printf("\t\tExecution Time: %5.6f sec\n", sGenerateNodes.elapsedTime);
		fflush(stdout);
	}

	return true;
}

//Identify Causal Sets
//O(k*N^2) Efficiency
bool linkNodes(Node &nodes, Edge &edges, bool * const &core_edge_exists, const int &N_tar, const float &k_tar, int &N_res, float &k_res, int &N_deg2, const int &dim, const Manifold &manifold, const double &a, const double &zeta, const double &zeta1, const double &r_max, const double &tau0, const double &alpha, const float &core_edge_fraction, const float &edge_buffer, Stopwatch &sLinkNodes, const bool &symmetric, const bool &compact, const bool &verbose, const bool &bench)
{
	#if DEBUG
	//No null pointers
	assert (!nodes.crd->isNull());
	assert (edges.past_edges != NULL);
	assert (edges.future_edges != NULL);
	assert (edges.past_edge_row_start != NULL);
	assert (edges.future_edge_row_start != NULL);
	assert (core_edge_exists != NULL);

	//Variables in correct ranges
	assert (N_tar > 0);
	assert (k_tar > 0.0f);
	assert (dim == 1 || dim == 3);
	assert (manifold == DE_SITTER || manifold == DUST || manifold == FLRW);
	assert (a > 0.0);
	assert (tau0 > 0.0);
	if (manifold == DE_SITTER) {
		if (compact) {
			assert (zeta > 0.0);
			assert (zeta < HALF_PI);
		} else {
			assert (zeta > HALF_PI);
			assert (zeta1 > HALF_PI);
			assert (zeta > zeta1);
		}
	} else if (manifold == DUST || manifold == FLRW) {
		assert (nodes.crd->getDim() == 4);
		assert (nodes.crd->w() != NULL);
		assert (nodes.crd->x() != NULL);
		assert (nodes.crd->y() != NULL);
		assert (nodes.crd->z() != NULL);
		assert (dim == 3);
		assert (zeta < HALF_PI);
		assert (alpha > 0.0);
	}
	if (!compact)
		assert (r_max > 0.0);
	assert (core_edge_fraction >= 0.0f && core_edge_fraction <= 1.0f);
	assert (edge_buffer >= 0.0f && edge_buffer <= 1.0f);
	#endif

	int core_limit = static_cast<int>((core_edge_fraction * N_tar));
	int future_idx = 0;
	int past_idx = 0;
	int i, j, k;

	bool related;

	stopwatchStart(&sLinkNodes);

	//Identify future connections
	for (i = 0; i < N_tar - 1; i++) {
		if (i < core_limit)
			core_edge_exists[(i*core_limit)+i] = false;
		edges.future_edge_row_start[i] = future_idx;

		for (j = i + 1; j < N_tar; j++) {
			//Apply Causal Condition (Light Cone)
			//Assume nodes are already temporally ordered
			related = nodesAreRelated(nodes.crd, N_tar, dim, manifold, a, zeta, zeta1, r_max, alpha, symmetric, compact, i, j, NULL);

			//Core Edge Adjacency Matrix
			if (i < core_limit && j < core_limit) {
				uint64_t idx1 = static_cast<uint64_t>(i) * core_limit + j;
				uint64_t idx2 = static_cast<uint64_t>(j) * core_limit + i;

				if (related) {
					core_edge_exists[idx1] = true;
					core_edge_exists[idx2] = true;
				} else {
					core_edge_exists[idx1] = false;
					core_edge_exists[idx2] = false;
				}
			}
						
			//Link timelike relations
			try {
				if (related) {
					//if (i % NPRINT == 0) printf("%d %d\n", i, j); fflush(stdout);
					edges.future_edges[future_idx++] = j;
	
					if (future_idx >= static_cast<int>(N_tar * k_tar * (1.0 + edge_buffer) / 2))
						throw CausetException("Not enough memory in edge adjacency list.  Increase edge buffer or decrease network size.\n");
	
					//Record number of degrees for each node
					nodes.k_in[j]++;
					nodes.k_out[i]++;
				}
			} catch (CausetException c) {
				fprintf(stderr, "CausetException in %s: %s on line %d\n", __FILE__, c.what(), __LINE__);
				return false;
			} catch (std::exception e) {
				fprintf(stderr, "Unknown Exception in %s: %s on line %d\n", __FILE__, e.what(), __LINE__);
				return false;
			}
		}

		//If there are no forward connections from node i, mark with -1
		if (edges.future_edge_row_start[i] == future_idx)
			edges.future_edge_row_start[i] = -1;
	}

	edges.future_edge_row_start[N_tar-1] = -1;

	//if (!printSpatialDistances(nodes, manifold, N_tar, dim)) return false;

	//Identify past connections
	edges.past_edge_row_start[0] = -1;
	for (i = 1; i < N_tar; i++) {
		edges.past_edge_row_start[i] = past_idx;
		for (j = 0; j < i; j++) {
			if (edges.future_edge_row_start[j] == -1)
				continue;

			for (k = 0; k < nodes.k_out[j]; k++) {
				if (i == edges.future_edges[edges.future_edge_row_start[j]+k]) {
					edges.past_edges[past_idx++] = j;
				}
			}
		}

		//If there are no backward connections from node i, mark with -1
		if (edges.past_edge_row_start[i] == past_idx)
			edges.past_edge_row_start[i] = -1;
	}

	//The quantities future_idx and past_idx should be equal
	#if DEBUG
	assert (future_idx == past_idx);
	#endif
	//printf("\t\tEdges (backward): %d\n", past_idx);
	//fflush(stdout);

	//Identify Resulting Network
	for (i = 0; i < N_tar; i++) {
		if (nodes.k_in[i] + nodes.k_out[i] > 0) {
			N_res++;
			k_res += nodes.k_in[i] + nodes.k_out[i];

			if (nodes.k_in[i] + nodes.k_out[i] > 1)
				N_deg2++;
		} 
	}

	#if DEBUG
	assert (N_res >= 0);
	assert (N_deg2 >= 0);
	assert (k_res >= 0.0);
	#endif

	if (N_res > 0)
		k_res /= N_res;

	//Debugging options used to visually inspect the adjacency lists and the adjacency pointer lists
	//compareAdjacencyLists(nodes, edges);
	//compareAdjacencyListIndices(nodes, edges);
	//if(!compareCoreEdgeExists(nodes.k_out, edges.future_edges, edges.future_edge_row_start, core_edge_exists, N_tar, core_edge_fraction))
	//	return false;

	//Print Results
	if (!printDegrees(nodes, N_tar, "in-degrees_CPU.cset.dbg.dat", "out-degrees_CPU.cset.dbg.dat")) return false;
	/*if (!printEdgeLists(edges, past_idx, "past-edges_CPU.cset.dbg.dat", "future-edges_CPU.cset.dbg.dat")) return false;
	if (!printEdgeListPointers(edges, N_tar, "past-edge-pointers_CPU.cset.dbg.dat", "future-edge-pointers_CPU.cset.dbg.dat")) return false;
	printf_red();
	printf("Check files now.\n");
	printf_std();
	fflush(stdout);*/
	printChk();

	stopwatchStop(&sLinkNodes);

	if (!bench) {
		printf("\tCausets Successfully Connected.\n");
		printf_cyan();
		printf("\t\tUndirected Links:         %d\n", future_idx);
		printf("\t\tResulting Network Size:   %d\n", N_res);
		printf("\t\tResulting Average Degree: %f\n", k_res);
		printf("\t\t    Incl. Isolated Nodes: %f\n", (k_res * N_res) / N_tar);
		printf_red();
		printf("\t\tResulting Error in <k>:   %f\n", fabs(k_tar - k_res) / k_tar);
		printf_std();
		fflush(stdout);
	}

	if (verbose) {
		printf("\t\tExecution Time: %5.6f sec\n", sLinkNodes.elapsedTime);
		fflush(stdout);
	}

	return true;
}
