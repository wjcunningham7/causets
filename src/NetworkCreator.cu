#include "hip/hip_runtime.h"
#include "NetworkCreator.h"

/////////////////////////////
//(C) Will Cunningham 2014 //
//         DK Lab          //
// Northeastern University //
/////////////////////////////

bool initVars(NetworkProperties * const network_properties, CaResources * const ca, CausetPerformance * const cp, Benchmark * const bm)
{
	#if DEBUG
	assert (network_properties != NULL);
	assert (ca != NULL);
	assert (cp != NULL);
	assert (bm != NULL);
	#endif

	unsigned int spacetime = network_properties->spacetime;
	int rank = network_properties->cmpi.rank;

	//Make sure the spacetime is fully defined
	if (!rank) printf_red();
	if (!get_stdim(spacetime)) {
		printf_mpi(rank, "The spacetime dimension has not been defined!  Use flag '--stdim' to continue.\n");
		network_properties->cmpi.fail = 1;
	}
	if (!get_manifold(spacetime)) {
		printf_mpi(rank, "The manifold has not been defined!  Use flag '--manifold' to continue.\n");
		network_properties->cmpi.fail = 1;
	}
	if (!get_region(spacetime)) {
		printf_mpi(rank, "The region has not been defined!  Use flag '--region' to continue.\n");
		network_properties->cmpi.fail = 1;
	}
	if (!get_curvature(spacetime)) {
		printf_mpi(rank, "The curvature has not been defined!  Use flag '--curvature' to continue.\n");
		network_properties->cmpi.fail = 1;
	}
	if (!get_symmetry(spacetime)) {
		printf_mpi(rank, "The symmetry has not been defined!  Use flag '--symmetry' to continue.\n");
		network_properties->cmpi.fail = 1;
	}
	if (!rank) printf_std();
	fflush(stdout);

	//Benchmarking
	if (network_properties->flags.bench) {
		network_properties->graphID = 0;
		network_properties->flags.verbose = false;
		network_properties->flags.print_network = false;
	}

	//Suppress queries if MPI is enabled
	#ifdef MPI_ENABLED
	if (network_properties->cmpi.num_mpi_threads > 1) {
		if (network_properties->flags.verbose)
			network_properties->flags.yes = true;
		network_properties->flags.use_bit = true;
		network_properties->core_edge_fraction = 1.0;
	}
	#endif

	//If a graph ID has been provided, warn user
	if (network_properties->graphID && network_properties->flags.verbose && !network_properties->flags.yes) {
		printf("You have chosen to load a graph from memory. Some parameters may be ignored as a result. Continue [y/N]? ");
		fflush(stdout);
		char response = getchar();
		getchar();
		if (response != 'y')
			return false;
	}

	//If no node positions used, require an edge list is being read
	if (!network_properties->graphID && network_properties->flags.no_pos) {
		printf_mpi(rank, "Flag 'nopos' cannot be used if a graph is not being read.\n");
		fflush(stdout);
		network_properties->cmpi.fail = 1;
	}

	if (network_properties->flags.relink && network_properties->flags.no_pos) {
		printf_mpi(rank, "Flag 'nopos' cannot be used together with 'relink'.\n");
		fflush(stdout);
		network_properties->cmpi.fail = 1;
	}

	#ifdef CUDA_ENABLED
	//If the GPU is requested, optimize parameters
	if (!LINK_NODES_GPU_V2 && network_properties->flags.use_gpu && network_properties->N_tar % (BLOCK_SIZE << 1)) {
		printf_mpi(rank, "If you are using the GPU, set the target number of nodes (--nodes) to be a multiple of %d!\n", BLOCK_SIZE << 1);
		printf_mpi(rank, "Alternatively, set LINK_NODES_GPU_V2=true in inc/Constants.h and recompile.\n");
		fflush(stdout);
		network_properties->cmpi.fail = 1;
	}

	if (network_properties->flags.use_gpu && network_properties->flags.no_pos) {
		printf_mpi(rank, "Conflicting parameters: no_pos and use_gpu.  GPU linking requires the use of node positions.\n");
		fflush(stdout);
		network_properties->cmpi.fail = 1;
	}
	#endif

	if (checkMpiErrors(network_properties->cmpi))
		return false;

	//Disable the default GSL Error Handler
	disableGSLErrHandler();

	try {
		double *table;
		double eta0 = 0.0, eta1 = 0.0;
		double q;
		long size = 0L;
		int method;

		//Check for an under-constrained system
		if (get_manifold(spacetime) & (MINKOWSKI | DE_SITTER | DUST | FLRW)) {
			if (!network_properties->N_tar)
				throw CausetException("Flag '--nodes', number of nodes, must be specified!\n");
			if (!network_properties->tau0)
				throw CausetException("Flag '--age', temporal cutoff, must be specified!\n");
			if (get_curvature(spacetime) & FLAT && get_region(spacetime) & SLAB) {
				if (get_manifold(spacetime) & DE_SITTER && !network_properties->r_max)
					throw CausetException("Flag '--slice', spatial scaling, must be specified!\n");
				else if (get_manifold(spacetime) & (DUST | FLRW) && !network_properties->alpha)
					throw CausetException("Flag '--alpha', spatial scale, must be specified!\n");
			}
			if (get_curvature(spacetime) & POSITIVE && get_manifold(spacetime) & FLRW && !network_properties->alpha)
				throw CausetException("Flag '--alpha', spatial scale, must be specified!\n");
		}

		//Default constraints
		if (get_manifold(spacetime) & MINKOWSKI) {
			#if SPECIAL_SAUCER
			if (get_region(spacetime) & SAUCER)
				network_properties->tau0 = network_properties->eta0 = 1.0;
			else
			#endif
				network_properties->eta0 = network_properties->tau0;
			eta0 = network_properties->eta0;
			network_properties->zeta = HALF_PI - eta0;
			network_properties->a = 1.0;

			#if DEBUG
			assert (network_properties->eta0 > 0.0);
			#endif
		} else if (get_manifold(spacetime) & DE_SITTER) {
			//The pseudoradius takes a default value of 1
			if (!network_properties->delta)
				network_properties->a = 1.0;

			if (get_curvature(spacetime) & FLAT) {
				//We take eta_min = -1 so that rescaled time
				//will begin at tau = 0
				//In this case, the '--age' flag reads tau0
				network_properties->zeta = HALF_PI + 1.0;
				network_properties->zeta1 = HALF_PI - tauToEtaFlat(network_properties->tau0);

				#if DEBUG
				assert (network_properties->zeta > HALF_PI);
				#endif
			} else if (get_curvature(spacetime) & POSITIVE) {
				//Re-write variables to their correct locations
				//This is because the '--age' flag has read eta0
				//into the tau0 variable
				network_properties->eta0 = network_properties->tau0;
				network_properties->zeta = HALF_PI - network_properties->tau0;
				network_properties->tau0 = etaToTauSph(HALF_PI - network_properties->zeta);

				#if DEBUG
				assert (network_properties->zeta > 0.0 && network_properties->zeta < HALF_PI);
				#endif
			}

			eta0 = HALF_PI - network_properties->zeta;
			eta1 = HALF_PI - network_properties->zeta1;
			network_properties->eta0 = eta0;
		} else if (get_manifold(spacetime) & (DUST | FLRW)) {
			//The pseudoradius takes a default value of 1
			if (!network_properties->delta)
				network_properties->a = 1.0;

			//The maximum radius takes a default value of 1
			//This allows alpha to characterize the spatial cutoff
			if (get_region(spacetime) & SLAB && !network_properties->r_max)
				network_properties->r_max = 1.0;
		} else if (get_manifold(spacetime) & HYPERBOLIC) {
			//The hyperbolic curvature takes a default value of 1
			if (!network_properties->zeta)
				network_properties->zeta = 1.0;
		}

		//Solve for the remaining constraints
		switch (spacetime) {
		case (2 | MINKOWSKI | DIAMOND | FLAT | ASYMMETRIC):
			network_properties->k_tar = network_properties->N_tar / 2.0;
			network_properties->delta = 2.0 * network_properties->N_tar / POW2(network_properties->eta0, EXACT);
			network_properties->r_max = network_properties->eta0 / 2.0;
			break;
		case (2 | MINKOWSKI | SAUCER | FLAT | SYMMETRIC):
		{
			//A guess here...
			network_properties->k_tar = network_properties->N_tar / 2.0;
			#if SPECIAL_SAUCER
			double volume = volume_77834_1(1.5) - volume_77834_1(-1.5);
			//printf_dbg("volume: %f\n", volume);
			network_properties->r_max = 1.5;
			#else
			double beta = 1.0 - eta0;
			double volume = 2.0 * (sqrt(1.0 - POW2(beta, EXACT)) - POW2(beta, EXACT) * log((1.0 + sqrt(1.0 - POW2(beta, EXACT))) / beta));
			network_properties->r_max = sqrt(1.0 - POW2(beta, EXACT));
			#endif
			network_properties->delta = static_cast<double>(network_properties->N_tar) / volume;
			//printf_dbg("delta: %f\n", network_properties->delta);
			network_properties->a = 1.0;
			break;
		}
		case (2 | DE_SITTER | SLAB | POSITIVE | ASYMMETRIC):
			network_properties->k_tar = network_properties->N_tar * (network_properties->eta0 / TAN(network_properties->eta0, STL) - LOG(COS(network_properties->eta0, STL), STL) - 1.0) / (TAN(network_properties->eta0, STL) * HALF_PI);
			if (!!network_properties->delta)
				network_properties->a = SQRT(network_properties->N_tar / (TWO_PI * network_properties->delta * TAN(network_properties->eta0, STL)), STL);
			else
				network_properties->delta = network_properties->N_tar / (TWO_PI * POW2(network_properties->a, EXACT) * TAN(network_properties->eta0, STL));
			break;
		case (2 | DE_SITTER | SLAB | POSITIVE | SYMMETRIC):
			network_properties->k_tar = (network_properties->N_tar / M_PI) * ((network_properties->eta0 / TAN(network_properties->eta0, STL) - 1.0) / TAN(network_properties->eta0, STL) + network_properties->eta0);
			if (!!network_properties->delta)
				network_properties->a = SQRT(network_properties->N_tar / (4.0 * M_PI * network_properties->delta * TAN(network_properties->eta0, STL)), STL);
			else
				network_properties->delta = network_properties->N_tar / (4.0 * M_PI * POW2(network_properties->a, EXACT) * TAN(network_properties->eta0, STL));
			break;
		case (2 | DE_SITTER | DIAMOND | FLAT | ASYMMETRIC):
			fprintf(stderr, "Not yet implemented on line %d in file %s\n", __LINE__, __FILE__);
			assert (false);
			break;
		case (2 | DE_SITTER | DIAMOND | POSITIVE | ASYMMETRIC):	
			fprintf(stderr, "Not yet implemented on line %d in file %s\n", __LINE__, __FILE__);
			assert (false);
			break;
		case (2 | DE_SITTER | DIAMOND | POSITIVE | SYMMETRIC):
			fprintf(stderr, "Not yet implemented on line %d in file %s\n", __LINE__, __FILE__);
			assert (false);
			break;
		case (2 | HYPERBOLIC | SLAB | FLAT | ASYMMETRIC):
			//Nothing else needs to be done
			//but we don't want to trigger 'default'
			break;
		case (4 | DE_SITTER | SLAB | FLAT | ASYMMETRIC):
		{
			int seed = static_cast<int>(4000000000 * network_properties->mrng.rng());
			network_properties->k_tar = 9.0 * network_properties->N_tar * POW2(POW3(eta0 * eta1, EXACT), EXACT) * integrate2D(&averageDegree_10788_0, eta0, eta0, eta1, eta1, NULL, seed, 0) / (POW3(network_properties->r_max, EXACT) * POW2(POW3(eta1, EXACT) - POW3(eta0, EXACT), EXACT));
			if (!!network_properties->delta)
				network_properties->a = POW(9.0 * network_properties->N_tar * POW3(eta0 * eta1, EXACT) / (4.0 * M_PI * network_properties->delta * POW3(network_properties->r_max, EXACT) * (POW3(eta1, EXACT) - POW3(eta0, EXACT))), 0.25, STL);
			else
				network_properties->delta = 9.0 * network_properties->N_tar * POW3(eta0 * eta1, EXACT) / (4.0 * M_PI * POW2(POW2(network_properties->a, EXACT), EXACT) * POW3(network_properties->r_max, EXACT) * (POW3(eta1, EXACT) - POW3(eta0, EXACT)));
			break;
		}
		case (4 | DE_SITTER | SLAB | POSITIVE | ASYMMETRIC):
			network_properties->k_tar = network_properties->N_tar * (12.0 * (eta0 / TAN(eta0, STL) - LOG(COS(eta0, STL), STL)) - (6.0 * LOG(COS(eta0, STL), STL) + 5.0) / POW2(COS(eta0, STL), EXACT) - 7.0) / (POW2(2.0 + 1.0 / POW2(COS(eta0, STL), EXACT), EXACT) * TAN(eta0, STL) * 3.0 * HALF_PI);
			if (!!network_properties->delta)
				network_properties->a = POW(network_properties->N_tar * 3.0 / (2.0 * POW2(M_PI, EXACT) * network_properties->delta * (2.0 + 1.0 / POW2(COS(eta0, STL), EXACT)) * TAN(eta0, STL)), 0.25, STL);
			else
				network_properties->delta = network_properties->N_tar * 3.0 / (2.0 * POW2(M_PI * POW2(network_properties->a, EXACT), EXACT) * (2.0 + 1.0 / POW2(COS(eta0, STL), EXACT)) * TAN(eta0, STL));
			break;
		case (4 | DE_SITTER | SLAB | POSITIVE | SYMMETRIC):
		{
			network_properties->k_tar = 2.0 * network_properties->N_tar * POW3(cos(eta0), EXACT) * (-51.0 * sin(eta0) + 7.0 * sin(3.0 * eta0) + 6.0 * (eta0 * (3.0 + 1.0 / POW2(cos(eta0), EXACT)) + tan(eta0)) / cos(eta0)) / (3.0 * M_PI * POW2(3.0 * sin(eta0) + sin(3.0 * eta0), EXACT));
			if (!!network_properties->delta)
				network_properties->a = POW(3.0 * network_properties->N_tar * POW3(cos(eta0), EXACT) / (2.0 * POW2(M_PI, EXACT) * network_properties->delta * (3.0 * sin(eta0) + sin(3.0 * eta0))), 0.25, STL);
			else
				network_properties->delta = 3.0 * network_properties->N_tar * POW3(cos(eta0), EXACT) / (2.0 * POW2(M_PI, EXACT) * POW2(POW2(network_properties->a, EXACT), EXACT) * (3.0 * sin(eta0) + sin(3.0 * eta0)));

			break;
		}
		case (4 | DE_SITTER | DIAMOND | FLAT | ASYMMETRIC):
		{
			double xi = eta0 / sqrt(2.0);
			double w = (eta1 - eta0) / sqrt(2.0);
			double mu = LOG(POW2(w + 2.0 * xi, EXACT) / (4.0 * xi * (w + xi)), STL) - POW2(w / (w + 2.0 * xi), EXACT);
			if (!!network_properties->delta)
				network_properties->a = POW(3.0 * network_properties->N_tar / (4.0 * M_PI * network_properties->delta * mu), 0.25, STL);
			else
				network_properties->delta = 3.0 * network_properties->N_tar / (4.0 * M_PI * POW2(POW2(network_properties->a, EXACT), EXACT) * mu);
			if (!getLookupTable("./etc/tables/average_degree_11300_0_table.cset.bin", &table, &size))
				throw CausetException("Average degree table not found!\n");
			network_properties->k_tar = network_properties->delta * POW2(POW2(network_properties->a, EXACT), EXACT) * lookupValue(table, size, &network_properties->tau0, NULL, true);
			if (network_properties->k_tar != network_properties->k_tar)
				throw CausetException("Value not found in average degree table!\n");
			//network_properties->k_tar = 5000;
			network_properties->r_max = w / sqrt(2.0);
			break;
		}
		case (4 | DE_SITTER | DIAMOND | POSITIVE | ASYMMETRIC):
		{
			double xi = eta0 / sqrt(2.0);
			double mu = log(0.5 * (1.0 / cos(sqrt(2.0) * xi) + 1.0)) - 1.0 / POW2(cos(xi / sqrt(2.0)), EXACT) + 1.0;
			if (!!network_properties->delta)
				network_properties->a = POW(3.0 * network_properties->N_tar / (4.0 * M_PI * network_properties->delta * mu), 0.25, STL);
			else
				network_properties->delta = 3.0 * network_properties->N_tar / (4.0 * M_PI * POW2(POW2(network_properties->a, EXACT), EXACT) * mu);
			if (!getLookupTable("./etc/tables/average_degree_13348_0_table.cset.bin", &table, &size))
				throw CausetException("Average degree table not found!\n");
			network_properties->k_tar = network_properties->delta * POW2(POW2(network_properties->a, EXACT), EXACT) * lookupValue(table, size, &eta0, NULL, true);
			if (network_properties->k_tar != network_properties->k_tar)
				throw CausetException("Value not found in average degree table!\n");
			break;
		}
		case (4 | DE_SITTER | DIAMOND | POSITIVE | SYMMETRIC):
			fprintf(stderr, "Not yet implemented on line %d in file %s\n", __LINE__, __FILE__);
			assert (false);
			break;
		case (4 | DUST | SLAB | FLAT | ASYMMETRIC):
		{
			if (!!network_properties->delta)
				network_properties->a = POW(network_properties->N_tar / (M_PI * network_properties->delta * POW3(network_properties->alpha * network_properties->tau0, EXACT)), 0.25, STL);
			else
				network_properties->delta = network_properties->N_tar / (M_PI * POW2(POW2(network_properties->a, EXACT), EXACT) * POW3(network_properties->alpha * network_properties->tau0, EXACT));
			
			int seed = static_cast<int>(4000000000 * network_properties->mrng.rng());
			network_properties->k_tar = (108.0 * M_PI / POW3(network_properties->tau0, EXACT)) * network_properties->delta * POW2(POW2(network_properties->a, EXACT), EXACT) * integrate2D(&averageDegree_10820_0, 0.0, 0.0, network_properties->tau0, network_properties->tau0, NULL, seed, 0);
			network_properties->alpha *= network_properties->a;
			eta0 = tauToEtaDust(network_properties->tau0, network_properties->a, network_properties->alpha);
			network_properties->zeta = HALF_PI - eta0;
			break;
		}
		case (4 | DUST | DIAMOND | FLAT | ASYMMETRIC):
		{
			double t = POW2(POW2(1.5 * network_properties->tau0, EXACT), EXACT);
			if (!!network_properties->delta)
				network_properties->a = POW(2970.0 * 64.0 * network_properties->N_tar / (1981.0 * M_PI * network_properties->delta * t), 0.25, STL);
			else
				network_properties->delta = 2970.0 * 64.0 * network_properties->N_tar / (1981.0 * M_PI * POW2(POW2(network_properties->a, EXACT), EXACT) * t);
			network_properties->alpha = 2.0 * network_properties->a; //This property should not affect results in the diamond
			if (!getLookupTable("./etc/tables/average_degree_11332_0_table.cset.bin", &table, &size))
				throw CausetException("Average degree table not found!\n");
			network_properties->k_tar = network_properties->delta * POW2(POW2(network_properties->a, EXACT), EXACT) * lookupValue(table, size, &network_properties->tau0, NULL, true);
			if (network_properties->k_tar != network_properties->k_tar)
				throw CausetException("Value not found in average degree table!\n");
			eta0 = tauToEtaDust(network_properties->tau0, network_properties->a, network_properties->alpha);
			network_properties->eta0 = eta0;
			network_properties->zeta = HALF_PI - eta0;
			network_properties->r_max = eta0 / 2.0;
			break;
		}
		case (4 | FLRW | SLAB | FLAT | ASYMMETRIC):
			method = 0;
			if (!solveExpAvgDegree(network_properties->k_tar, network_properties->spacetime, network_properties->N_tar, network_properties->a, network_properties->r_max, network_properties->tau0, network_properties->alpha, network_properties->delta, network_properties->cmpi.rank, network_properties->mrng, ca, cp->sCalcDegrees, bm->bCalcDegrees, network_properties->flags.verbose, network_properties->flags.bench, method))
				network_properties->cmpi.fail = 1;

			if (checkMpiErrors(network_properties->cmpi))
				return false;
				
			q = 9.0 * network_properties->N_tar / (TWO_PI * POW3(network_properties->alpha * network_properties->r_max, EXACT) * (SINH(3.0 * network_properties->tau0, STL) - 3.0 * network_properties->tau0));
			if (!!network_properties->delta)
				network_properties->a = POW(q / network_properties->delta, 0.25, STL);
			else
				network_properties->delta = q / POW2(POW2(network_properties->a, EXACT), EXACT);
			network_properties->alpha *= network_properties->a;
			eta0 = tauToEtaFLRWExact(network_properties->tau0, network_properties->a, network_properties->alpha);
			network_properties->zeta = HALF_PI - eta0;
			break;
		case (4 | FLRW | SLAB | POSITIVE | ASYMMETRIC):
			q = 3.0 * network_properties->N_tar / (POW2(M_PI, EXACT) * POW3(network_properties->alpha, EXACT) * (SINH(3.0 * network_properties->tau0, STL) - 3.0 * network_properties->tau0));
			if (!!network_properties->delta)
				network_properties->a = POW(q / network_properties->delta, 0.25, STL);
			else
				network_properties->delta = q / POW2(POW2(network_properties->a, EXACT), EXACT);
			network_properties->alpha *= network_properties->a;
			eta0 = tauToEtaFLRWExact(network_properties->tau0, network_properties->a, network_properties->alpha);
			network_properties->zeta = HALF_PI - eta0;

			method = 1;
			if (!solveExpAvgDegree(network_properties->k_tar, network_properties->spacetime, network_properties->N_tar, network_properties->a, network_properties->r_max, network_properties->tau0, network_properties->alpha, network_properties->delta, network_properties->cmpi.rank, network_properties->mrng, ca, cp->sCalcDegrees, bm->bCalcDegrees, network_properties->flags.verbose, network_properties->flags.bench, method))
				network_properties->cmpi.fail = 1;

			if (checkMpiErrors(network_properties->cmpi))
				return false;
			break;
		case (4 | FLRW | DIAMOND | FLAT | ASYMMETRIC):
		{
			//We REQUIRE a = alpha for this spacetime
			eta0 = tauToEtaFLRWExact(network_properties->tau0, 1.0, 1.0);
			network_properties->eta0 = eta0;
			network_properties->zeta = HALF_PI - eta0;
			network_properties->r_max = eta0 / 2.0;

			//Bisection Method
			double res = 1.0, tol = 1.0e-10;
			double lower = 0.0, upper = network_properties->tau0;
			int iter = 0, max_iter = 10000;

			double x0 = 0.0;
			while (upper - lower > tol && iter < max_iter) {
				x0 = (lower + upper) / 2.0;
				res = tauToEtaFLRWExact(x0, 1.0, 1.0);
				res -= eta0 / 2.0;
				if (res < 0.0)
					lower = x0;
				else
					upper = x0;
				iter++;
			}
			//Store the result in zeta1 variable
			network_properties->zeta1 = x0;

			IntData idata;
			idata.limit = 100;
			idata.tol = 1e-8;
			//idata.key = GSL_INTEG_GAUSS61;	//Was used for QAG instead of QAGS
			idata.workspace = gsl_integration_workspace_alloc(idata.nintervals);
			idata.upper = network_properties->zeta1;
			double params[3];
			params[0] = network_properties->tau0;
			params[1] = eta0;
			params[2] = network_properties->zeta1;
			double vol_lower = integrate1D(&volume_11396_0_lower, &params, &idata, QAGS);
			assert (vol_lower == vol_lower);
			idata.lower = idata.upper;
			idata.upper = network_properties->tau0;
			double vol_upper = integrate1D(&volume_11396_0_upper, &params, &idata, QAGS);
			assert (vol_upper == vol_upper);
			double mu = vol_lower + vol_upper;
			gsl_integration_workspace_free(idata.workspace);
			//printf("Reduced Volume: %.10f\n", mu);

			if (!!network_properties->delta)
				network_properties->a = POW(3.0 * network_properties->N_tar / (4.0 * M_PI * network_properties->delta * mu), 0.25, STL);
			else
				network_properties->delta = 3.0 * network_properties->N_tar / (4.0 * M_PI * POW2(POW2(network_properties->a, EXACT), EXACT) * mu);
			network_properties->alpha = network_properties->a;

			if (!getLookupTable("./etc/tables/average_degree_11396_0_table.cset.bin", &table, &size))
				throw CausetException("Average degree table not found!\n");
			network_properties->k_tar = network_properties->delta * POW2(POW2(network_properties->a, EXACT), EXACT) * lookupValue(table, size, &network_properties->tau0, NULL, true);
			if (network_properties->k_tar != network_properties->k_tar)
				throw CausetException("Value not found in average degree table!\n");
			break;
		}
		default:
			throw CausetException("Spacetime parameters not supported!\n");
		}

		if (get_manifold(spacetime) & (MINKOWSKI | DE_SITTER | DUST | FLRW)) {
			#if DEBUG
			assert (network_properties->k_tar > 0.0);
			assert (network_properties->a > 0.0);
			assert (network_properties->delta > 0.0);
			if (!((get_manifold(spacetime) & DE_SITTER) && (get_curvature(spacetime) & FLAT)))
				assert (network_properties->zeta < HALF_PI);
			#endif

			//Display Constraints
			printf_mpi(rank, "\n");
			printf_mpi(rank, "\tParameters Constraining the %d+1 %s Causal Set:\n", get_stdim(spacetime) - 1, manifoldNames[(unsigned int)(log2((float)get_manifold(spacetime) / ManifoldFirst))].c_str());
			printf_mpi(rank, "\t--------------------------------------------\n");
			if (!rank) printf_cyan();
			printf_mpi(rank, "\t > Manifold:\t\t\t%s\n", manifoldNames[(unsigned int)(log2((float)get_manifold(spacetime) / ManifoldFirst))].c_str());
			printf_mpi(rank, "\t > Spacetime Dimension:\t\t%d+1\n", get_stdim(spacetime) - 1);
			printf_mpi(rank, "\t > Region:\t\t\t%s", regionNames[(unsigned int)(log2((float)get_region(spacetime) / RegionFirst))].c_str());
			#if SPECIAL_SAUCER
			if (get_manifold(spacetime) & MINKOWSKI && get_region(spacetime) & SAUCER)
				printf_mpi(rank, " (Special)\n");
			else
			#endif
				printf_mpi(rank, "\n");
			printf_mpi(rank, "\t > Curvature:\t\t\t%s\n", curvatureNames[(unsigned int)(log2((float)get_curvature(spacetime) / CurvatureFirst))].c_str());
			printf_mpi(rank, "\t > Temporal Symmetry:\t\t%s\n", symmetryNames[(unsigned int)(log2((float)get_symmetry(spacetime) / SymmetryFirst))].c_str());
			printf_mpi(rank, "\t > Spacetime ID:\t\t%u\n", network_properties->spacetime);
			if (!rank) printf_std();
			printf_mpi(rank, "\t--------------------------------------------\n");
			if (!rank) printf_cyan();
			printf_mpi(rank, "\t > Number of Nodes:\t\t%d\n", network_properties->N_tar);
			printf_mpi(rank, "\t > Node Density:\t\t%.6f\n", network_properties->delta);
			printf_mpi(rank, "\t > Expected Degrees:\t\t%.6f\n", network_properties->k_tar);
			if (get_symmetry(spacetime) & SYMMETRIC) {
				printf_mpi(rank, "\t > Min. Conformal Time:\t\t%.6f\n", -eta0);
				printf_mpi(rank, "\t > Max. Conformal Time:\t\t%.6f\n", eta0);
			} else if ((get_manifold(spacetime) & DE_SITTER) && (get_curvature(spacetime) & FLAT)) {
				printf_mpi(rank, "\t > Min. Conformal Time:\t\t%.6f\n", eta0);
				printf_mpi(rank, "\t > Max. Conformal Time:\t\t%.6f\n", eta1);
			} else {
				printf_mpi(rank, "\t > Min. Conformal Time:\t\t0.0\n");
				printf_mpi(rank, "\t > Max. Conformal Time:\t\t%.6f\n", eta0);
			}
			if (!(get_manifold(spacetime) & MINKOWSKI))
				printf_mpi(rank, "\t > Max. Rescaled Time:\t\t%.6f\n", network_properties->tau0);
			if (get_manifold(spacetime) & (DE_SITTER | FLRW))
				printf_mpi(rank, "\t > Dark Energy Density:\t\t%.6f\n", network_properties->omegaL);
			if (get_manifold(spacetime) & (DUST | FLRW))
				printf_mpi(rank, "\t > Spatial Scaling:\t\t%.6f\n", network_properties->alpha);
			if (get_curvature(spacetime) & FLAT && get_region(spacetime) & SLAB)
				printf_mpi(rank, "\t > Spatial Cutoff:\t\t%.6f\n", network_properties->r_max);
			if (get_manifold(spacetime) & (DE_SITTER | DUST | FLRW))
				printf_mpi(rank, "\t > Temporal Scaling:\t\t%.6f\n", network_properties->a);
			printf_mpi(rank, "\t > Random Seed:\t\t\t%Ld\n", network_properties->seed);
			if (!rank) { printf_std(); printf("\n"); }
			fflush(stdout);

			//Miscellaneous Tasks
			if (get_manifold(spacetime) & DE_SITTER) {
				if (!network_properties->cmpi.rank && network_properties->flags.gen_ds_table && !generateGeodesicLookupTable("etc/tables/geodesics_ds_table.cset.bin", 5.0, -5.0, 5.0, 0.01, 0.01, network_properties->spacetime, network_properties->flags.verbose))
					network_properties->cmpi.fail = 1;

				if (checkMpiErrors(network_properties->cmpi))
					return false;
			} else if (get_manifold(spacetime) & FLRW) {
				if (!network_properties->cmpi.rank && network_properties->flags.gen_flrw_table && !generateGeodesicLookupTable("etc/tables/geodesics_flrw_table.cset.bin", 2.0, -5.0, 5.0, 0.01, 0.01, network_properties->spacetime, network_properties->flags.verbose))
					network_properties->cmpi.fail = 1;

				if (checkMpiErrors(network_properties->cmpi))
					return false;
			}
			
		}

		//Miscellaneous Tasks
		if (!network_properties->edge_buffer)
			network_properties->edge_buffer = 0.2;

		if (network_properties->k_tar >= network_properties->N_tar / 32 - 1) {
			//This is when a bit array is smaller than the adjacency lists
			//network_properties->flags.use_bit = true;
			//network_properties->core_edge_fraction = 1.0;
			//printf_dbg("USE_BIT = true\n");
		}

		#ifdef CUDA_ENABLED
		//Adjacency matrix not implemented in certain GPU algorithms
		if (network_properties->flags.use_gpu && !LINK_NODES_GPU_V2) {
			network_properties->flags.use_bit = false;
			network_properties->core_edge_fraction = 0.0;
		}

		//Determine group size and decoding method
		if (network_properties->flags.use_gpu) {
			long mem = GLOB_MEM + 1L;
			long d_edges_size = network_properties->flags.use_bit ? 1L : static_cast<long>(exp2(ceil(log2(network_properties->N_tar * network_properties->k_tar * (1.0 + network_properties->edge_buffer) / 2.0))));
			float gsize = 0.5f;
			bool dcpu = false;

			while (mem > GLOB_MEM) {
				//Used in generateLists_v2
				//The group size - the number of groups, along one index, the full matrix is broken up into
				gsize *= 2.0f;
				//The 'mega-block' size - the number of thread blocks along index 'i' within a group	
				long mbsize = static_cast<long>(ceil(static_cast<float>(network_properties->N_tar) / (BLOCK_SIZE * gsize)));
				//The 'mega-thread' size - the number of threads along a dimension of a group
				long mtsize = mbsize * BLOCK_SIZE;
				//The 'mega-edges' size - the number of edges represented by the sub-matrix passed to the GPU
				long mesize = mtsize * mtsize;

				//Used in decodeLists_v2
				long gmbsize = static_cast<long>(network_properties->N_tar * network_properties->k_tar * (1.0 + network_properties->edge_buffer) / (BLOCK_SIZE * gsize * 2));
				long gmtsize = gmbsize * BLOCK_SIZE;

				long mem1 = (40L * mtsize + mesize) * NBUFFERS;							//For generating
				long mem2 = network_properties->flags.use_bit ? 0L : 4L * (2L * d_edges_size + gmtsize);	//For decoding

				if (mem2 > GLOB_MEM / 4L) {
					mem2 = 0L;
					dcpu = true;
				}

				mem = mem1 > mem2 ? mem1 : mem2;
			}

			network_properties->group_size = gsize < NBUFFERS ? NBUFFERS : gsize;
			network_properties->flags.decode_cpu = dcpu;

			/*printf("MPI Threads: %d\n", network_properties->cmpi.num_mpi_threads);
			printf("gsize:       %d\n", static_cast<int>(gsize));
			printf("Group Size:  %d\n", network_properties->group_size);*/
		}
		#endif

		if (network_properties->flags.calc_deg_field && network_properties->tau_m >= network_properties->tau0)
			throw CausetException("You have chosen to measure the degree field at a time greater than the maximum time!\n");
		
		uint64_t pair_multiplier = static_cast<uint64_t>(network_properties->N_tar) * (network_properties->N_tar - 1) / 2;
		if (network_properties->flags.calc_success_ratio && network_properties->N_sr <= 1.0)
			network_properties->N_sr *= pair_multiplier;
		if (network_properties->flags.validate_embedding && network_properties->N_emb <= 1.0)
			network_properties->N_emb *= pair_multiplier;
		if (network_properties->flags.validate_distances && network_properties->N_dst <= 1.0)
			network_properties->N_dst *= pair_multiplier;

		if (network_properties->flags.calc_action) {
			#if DEBUG
			assert (network_properties->max_cardinality == -1 || network_properties->max_cardinality == 1);
			#endif
			if (network_properties->max_cardinality == -1)
				network_properties->max_cardinality = 5;
			else
				network_properties->max_cardinality = network_properties->N_tar - 1;
		}
	} catch (CausetException c) {
		fprintf(stderr, "CausetException in %s: %s on line %d\n", __FILE__, c.what(), __LINE__);
		network_properties->cmpi.fail = 1;
	} catch (std::bad_alloc) {
		fprintf(stderr, "Memory allocation failure in %s on line %d!\n", __FILE__, __LINE__);
		network_properties->cmpi.fail = 1;
	} catch (std::exception e) {
		fprintf(stderr, "Unknown exception in %s: %s on line %d\n", __FILE__, e.what(), __LINE__);
		network_properties->cmpi.fail = 1;
	}

	if (checkMpiErrors(network_properties->cmpi))
		return false;

	return true;
}

//Calculate Expected Average Degree in the FLRW Spacetime
//See Causal Set Notes for detailed explanation of methods
//NOTE: This method is largely historical - only a small portion is used
//  in practice, but it offers several methods to achieve the same outcome
bool solveExpAvgDegree(float &k_tar, const unsigned int &spacetime, const int &N_tar, double &a, const double &r_max, double &tau0, const double &alpha, const double &delta, const int &rank, MersenneRNG &mrng, CaResources * const ca, Stopwatch &sCalcDegrees, double &bCalcDegrees, const bool &verbose, const bool &bench, const int method)
{
	#if DEBUG
	assert (ca != NULL);
	assert (get_stdim(spacetime) & 4);
	assert (get_manifold(spacetime) & FLRW);
	assert (N_tar > 0);
	assert (tau0 > 0.0);
	assert (alpha > 0.0);
	assert (method == 0 || method == 1 || method == 2);
	if (get_curvature(spacetime) & FLAT) {
		assert (method == 0 || method == 1);
		assert (r_max > 0.0);
	} else {
		assert (delta > 0.0);
		assert (a > 0.0);
	}
	#endif

	printf_mpi(rank, "\tEstimating Expected Average Degree...\n");
	fflush(stdout);

	int nb = static_cast<int>(bench) * NBENCH;
	int i;

	double *table;
	double kappa;
	long size = 0L;
	int seed = static_cast<int>(4000000000 * mrng.rng());

	switch (method) {
	case 0:
	{
		//Method 1 of 3: Use Monte Carlo integration
		double r0;
		if (tau0 > LOG(MTAU, STL) / 3.0)
			r0 = POW(0.5, 2.0 / 3.0, STL) * exp(tau0);
		else
			r0 = POW(SINH(1.5 * tau0, STL), 2.0 / 3.0, STL);

		for (i = 0; i <= nb; i++) {
			stopwatchStart(&sCalcDegrees);
			switch (spacetime) {
			case (4 | FLRW | SLAB | FLAT | ASYMMETRIC):
				kappa = integrate2D(&averageDegree_10884_0, 0.0, 0.0, tau0, tau0, NULL, seed, 0);
				kappa *= 8.0 * M_PI;
				kappa /= SINH(3.0 * tau0, STL) - 3.0 * tau0;
				k_tar = (9.0 * kappa * N_tar) / (TWO_PI * POW3(alpha * r_max, EXACT) * (SINH(3.0 * tau0, STL) - 3.0 * tau0));
				break;
			case (4 | FLRW | SLAB | POSITIVE | ASYMMETRIC):
				if (tau0 > LOG(MTAU, STL) / 3.0)
					k_tar = delta * POW2(POW2(a, EXACT), EXACT) * integrate2D(&averageDegree_12932_0, 0.0, 0.0, r0, r0, NULL, seed, 0) * 16.0 * M_PI * exp(-3.0 * tau0);
				else
					k_tar = delta * POW2(POW2(a, EXACT), EXACT) * integrate2D(&averageDegree_12932_0, 0.0, 0.0, r0, r0, NULL, seed, 0) * 8.0 * M_PI / (SINH(3.0 * tau0, STL) - 3.0 * tau0);
				break;
			default:
				fprintf(stderr, "Spacetime parameters not supported!\n");
				return false;
			}
			stopwatchStop(&sCalcDegrees);
		}
		break;
	}
	case 1:
	{
		//Method 2 of 3: Lookup table to approximate method 1
		if (get_curvature(spacetime) & POSITIVE) {
			if (!getLookupTable("./etc/tables/raduc_table.cset.bin", &table, &size))
				return false;
		} else if (get_curvature(spacetime) & FLAT) {
			if (!getLookupTable("./etc/tables/raducNC_table.cset.bin", &table, &size))
				return false;
		} else
			return false;
		
		ca->hostMemUsed += size;

		for (i = 0; i <= nb; i++) {
			stopwatchStart(&sCalcDegrees);
			if (get_curvature(spacetime) & POSITIVE)
				k_tar = lookupValue(table, size, &tau0, NULL, true) * delta * POW2(POW2(a, EXACT), EXACT);
			else if (get_curvature(spacetime) & FLAT)
				k_tar = lookupValue(table, size, &tau0, NULL, true) * 9.0 * N_tar / (TWO_PI * POW3(alpha * r_max, EXACT) * (SINH(3.0 * tau0, STL) - 3.0 * tau0));
			else
				return false;
			stopwatchStop(&sCalcDegrees);
		}	

		//Check for NaN
		if (k_tar != k_tar)
			return false;

		free(table);
		table = NULL;
		ca->hostMemUsed -= size;
		break;
	}
	case 2:
	{
		//Method 3 of 3: Explicit Solution
		if (!getLookupTable("./etc/tables/ctuc_table.cset.bin", &table, &size))
			return false;
		ca->hostMemUsed += size;

		double *params = (double*)malloc(size + sizeof(double) * 3);
		if (params == NULL)
			throw std::bad_alloc();
		ca->hostMemUsed += size + sizeof(double) * 3;

		double d_size = static_cast<double>(size);
		memcpy(params, &a, sizeof(double));
		memcpy(params + 1, &alpha, sizeof(double));
		memcpy(params + 2, &d_size, sizeof(double));
		memcpy(params + 3, table, size);

		IntData idata = IntData();
		idata.limit = 50;
		idata.tol = 1e-5;
		idata.workspace = gsl_integration_workspace_alloc(idata.nintervals);
		idata.upper = tau0;

		double max_time;
		for (i = 0; i <= nb; i++) {
			stopwatchStart(&sCalcDegrees);
			max_time = integrate1D(&tauToEtaFLRW, NULL, &idata, QAGS) * a / alpha;
			stopwatchStop(&sCalcDegrees);
		}

		gsl_integration_workspace_free(idata.workspace);

		k_tar = integrate2D(&averageDegree_12932_0_a, 0.0, 0.0, max_time, max_time, params, seed, 0);
		k_tar *= 4.0 * M_PI * delta * POW2(POW2(alpha, EXACT), EXACT);

		for (i = 0; i <= nb; i++) {
			stopwatchStart(&sCalcDegrees);
			integrate2D(&averageDegree_12932_0_a, 0.0, 0.0, max_time, max_time, params, seed, 0);
			stopwatchStop(&sCalcDegrees);
		}
	
		idata.workspace = gsl_integration_workspace_alloc(idata.nintervals);
		idata.upper = max_time;
		k_tar /= (3.0 * integrate1D(&averageDegree_12932_0_b, params, &idata, QAGS));

		for (i = 0; i <= nb; i++) {
			stopwatchStart(&sCalcDegrees);
			integrate1D(&averageDegree_12932_0_b, params, &idata, QAGS);
			stopwatchStop(&sCalcDegrees);
		}

		gsl_integration_workspace_free(idata.workspace);

		free(params);
		params = NULL;
		ca->hostMemUsed -= size + sizeof(double) * 3;

		free(table);
		table = NULL;
		ca->hostMemUsed -= size;
		break;
	}
	default:
		return false;
	}

	if (nb)
		bCalcDegrees = sCalcDegrees.elapsedTime / NBENCH;

	if (verbose) {
		printf_mpi(rank, "\t\tExecution Time: %5.6f sec\n", sCalcDegrees.elapsedTime);
		fflush(stdout);
	}

	if (!bench) {
		printf_mpi(rank, "\tExpected Average Degree Successfully Calculated.\n");
		printf_mpi(rank, "\t\t<k> = %f\n", k_tar);
		fflush(stdout);
	}

	return true;
}

//Allocates memory for network
//O(1) Efficiency
bool createNetwork(Node &nodes, Edge &edges, Bitvector &adj, const unsigned int &spacetime, const int &N_tar, const float &k_tar, const float &core_edge_fraction, const float &edge_buffer, CausetMPI &cmpi, const int &group_size, CaResources * const ca, Stopwatch &sCreateNetwork, const bool &use_gpu, const bool &decode_cpu, const bool &link, const bool &relink, const bool &no_pos, const bool &use_bit, const bool &verbose, const bool &bench, const bool &yes)
{
	#if DEBUG
	assert (ca != NULL);
	assert (N_tar > 0);
	assert (k_tar > 0.0f);
	assert (get_stdim(spacetime) & (2 | 4));
	assert (get_manifold(spacetime) & (MINKOWSKI | DE_SITTER | DUST | FLRW | HYPERBOLIC));
	if (get_manifold(spacetime) & HYPERBOLIC)
		assert (get_stdim(spacetime) == 2);
	assert (core_edge_fraction >= 0.0f && core_edge_fraction <= 1.0f);
	assert (edge_buffer >= 0.0f && edge_buffer <= 1.0f);
	#endif

	int rank = cmpi.rank;
	bool links_exist = link || relink;

	if (verbose && !yes) {
		//Estimate memory usage before allocating
		size_t mem = 0;
		if (!no_pos) {
			#if EMBED_NODES
			if (get_stdim(spacetime) == 4)
				mem += sizeof(float) * N_tar * 5;	//For Coordinate5D
			else if (get_stdim(spacetime) == 2)
				mem += sizeof(float) * N_tar * 3;	//For Coordinate3D
			#else
			if (get_stdim(spacetime) == 4)
				mem += sizeof(float) * N_tar << 2;	//For Coordinate4D
			else if (get_stdim(spacetime) == 2)
				mem += sizeof(float) * N_tar << 1;	//For Coordinate2D
			#endif
			if (get_manifold(spacetime) & HYPERBOLIC)
				mem += sizeof(int) * N_tar;		//For AS
			else if (get_manifold(spacetime) & (DE_SITTER | DUST | FLRW))
				mem += sizeof(float) * N_tar;		//For tau
		}
		if (links_exist) {
			mem += sizeof(int) * (N_tar << 1);	//For k_in and k_out
			if (!use_bit) {
				mem += sizeof(int) * static_cast<int64_t>(N_tar) * k_tar * (1.0 + edge_buffer);	//For edge lists
				mem += sizeof(int64_t) * (N_tar << 1);	//For edge list pointers
			}
			mem += static_cast<uint64_t>(POW2(core_edge_fraction * N_tar, EXACT)) / (8 * cmpi.num_mpi_threads);	//Adjacency matrix
			#ifdef MPI_ENABLED
			mem += static_cast<uint64_t>(core_edge_fraction * N_tar) * ceil(static_cast<int>(N_tar * core_edge_fraction) / (2.0 * POW2(cmpi.num_mpi_threads, EXACT))) / 8;
			#endif
		}

		size_t dmem = 0;
		#ifdef CUDA_ENABLED
		size_t dmem1 = 0, dmem2 = 0;
		if (use_gpu) {
			size_t d_edges_size = pow(2.0, ceil(log2(N_tar * k_tar * (1.0 + edge_buffer) / 2)));
			if (!use_bit)
				mem += sizeof(uint64_t) * d_edges_size;	//For encoded edge list
			mem += sizeof(int64_t);				//For g_idx

			size_t mblock_size = static_cast<unsigned int>(ceil(static_cast<float>(N_tar) / (BLOCK_SIZE * group_size)));
			size_t mthread_size = mblock_size * BLOCK_SIZE;
			size_t m_edges_size = mthread_size * mthread_size;
			size_t nbuf = GEN_ADJ_LISTS_GPU_V2 ? NBUFFERS : 1;
			mem += sizeof(int) * mthread_size * nbuf << 1;		//For k_in and k_out buffers (host)
			mem += sizeof(bool) * m_edges_size * nbuf;		//For adjacency matrix buffers (host)
			#if EMBED_NODES
			fprintf(stderr, "Not yet implemented on line %d in file %s\n", __LINE__, __FILE__);
			assert (false);
			#else
			dmem1 += sizeof(float) * mthread_size * get_stdim(spacetime) * nbuf << 1;	//For 4-D coordinate buffers
			#endif
			dmem1 += sizeof(int) * mthread_size * nbuf << 1;		//For k_in and k_out buffers (device)
			dmem1 += sizeof(bool) * m_edges_size * nbuf;			//For adjacency matrix buffers (device)

			if (!use_bit) {
				size_t g_mblock_size = static_cast<uint64_t>(N_tar) * k_tar * (1.0 + edge_buffer) / (BLOCK_SIZE * group_size << 1);
				size_t g_mthread_size = g_mblock_size * BLOCK_SIZE;
				dmem2 += sizeof(uint64_t) * d_edges_size;	//Encoded edge list used during parallel sorting
				dmem2 += sizeof(int) * (DECODE_LISTS_GPU_V2 ? g_mthread_size : d_edges_size);	//For edge lists
				if (decode_cpu)
					dmem2 = 0;
			}

			dmem = dmem1 > dmem2 ? dmem1 : dmem2;
		}
		#endif

		printMemUsed("for Network (Estimation)", mem, dmem, rank);
		printf("\nContinue [y/N]?");
		fflush(stdout);
		char response = getchar();
		getchar();
		if (response != 'y')
			return false;
	}

	stopwatchStart(&sCreateNetwork);

	try {
		if (!no_pos) {
			if (get_manifold(spacetime) & (DE_SITTER | DUST | FLRW)) {
				nodes.id.tau = (float*)malloc(sizeof(float) * N_tar);
				if (nodes.id.tau == NULL)
					throw std::bad_alloc();
				memset(nodes.id.tau, 0, sizeof(float) * N_tar);
				ca->hostMemUsed += sizeof(float) * N_tar;
			} else if (get_manifold(spacetime) & HYPERBOLIC) {
				nodes.id.AS = (int*)malloc(sizeof(int) * N_tar);
				if (nodes.id.AS == NULL)
					throw std::bad_alloc();
				memset(nodes.id.AS, 0, sizeof(int) * N_tar);
				ca->hostMemUsed += sizeof(int) * N_tar;
			}

			#if EMBED_NODES
			if (get_stdim(spacetime) == 4) {
				nodes.crd = new Coordinates5D();

				nodes.crd->v() = (float*)malloc(sizeof(float) * N_tar);
				nodes.crd->w() = (float*)malloc(sizeof(float) * N_tar);
				nodes.crd->x() = (float*)malloc(sizeof(float) * N_tar);
				nodes.crd->y() = (float*)malloc(sizeof(float) * N_tar);
				nodes.crd->z() = (float*)malloc(sizeof(float) * N_tar);

				if (nodes.crd->v() == NULL || nodes.crd->w() == NULL || nodes.crd->x() == NULL || nodes.crd->y() == NULL || nodes.crd->z() == NULL)
					throw std::bad_alloc();

				memset(nodes.crd->v(), 0, sizeof(float) * N_tar);
				memset(nodes.crd->w(), 0, sizeof(float) * N_tar);
				memset(nodes.crd->x(), 0, sizeof(float) * N_tar);
				memset(nodes.crd->y(), 0, sizeof(float) * N_tar);
				memset(nodes.crd->z(), 0, sizeof(float) * N_tar);

				ca->hostMemUsed += sizeof(float) * N_tar * 5;
			} else if (get_stdim(spacetime) == 2) {
				nodes.crd = new Coordinates3D();

				nodes.crd->x() = (float*)malloc(sizeof(float) * N_tar);
				nodes.crd->y() = (float*)malloc(sizeof(float) * N_tar);
				nodes.crd->z() = (float*)malloc(sizeof(float) * N_tar);

				if (nodes.crd->x() == NULL || nodes.crd->y() == NULL || nodes.crd->z() == NULL)
					throw std::bad_alloc();

				memset(nodes.crd->x(), 0, sizeof(float) * N_tar);
				memset(nodes.crd->y(), 0, sizeof(float) * N_tar);
				memset(nodes.crd->z(), 0, sizeof(float) * N_tar);

				ca->hostMemUsed += sizeof(float) * N_tar * 3;
			}
			#else
			if (get_stdim(spacetime) == 4) {
				nodes.crd = new Coordinates4D();
	
				nodes.crd->w() = (float*)malloc(sizeof(float) * N_tar);
				nodes.crd->x() = (float*)malloc(sizeof(float) * N_tar);
				nodes.crd->y() = (float*)malloc(sizeof(float) * N_tar);
				nodes.crd->z() = (float*)malloc(sizeof(float) * N_tar);

				if (nodes.crd->w() == NULL || nodes.crd->x() == NULL || nodes.crd->y() == NULL || nodes.crd->z() == NULL)
					throw std::bad_alloc();

				memset(nodes.crd->w(), 0, sizeof(float) * N_tar);
				memset(nodes.crd->x(), 0, sizeof(float) * N_tar);
				memset(nodes.crd->y(), 0, sizeof(float) * N_tar);
				memset(nodes.crd->z(), 0, sizeof(float) * N_tar);

				ca->hostMemUsed += sizeof(float) * N_tar * 4;
			} else if (get_stdim(spacetime) == 2) {
				nodes.crd = new Coordinates2D();

				nodes.crd->x() = (float*)malloc(sizeof(float) * N_tar);
				nodes.crd->y() = (float*)malloc(sizeof(float) * N_tar);

				if (nodes.crd->x() == NULL || nodes.crd->y() == NULL)
					throw std::bad_alloc();

				memset(nodes.crd->x(), 0, sizeof(float) * N_tar);
				memset(nodes.crd->y(), 0, sizeof(float) * N_tar);

				ca->hostMemUsed += sizeof(float) * N_tar * 2;
			}
			#endif
		}

		if (links_exist) {
			nodes.k_in = (int*)malloc(sizeof(int) * N_tar);
			if (nodes.k_in == NULL)
				throw std::bad_alloc();
			memset(nodes.k_in, 0, sizeof(int) * N_tar);
			ca->hostMemUsed += sizeof(int) * N_tar;

			nodes.k_out = (int*)malloc(sizeof(int) * N_tar);
			if (nodes.k_out == NULL)
				throw std::bad_alloc();
			memset(nodes.k_out, 0, sizeof(int) * N_tar);
			ca->hostMemUsed += sizeof(int) * N_tar;
		}

		if (verbose)
			printMemUsed("for Nodes", ca->hostMemUsed, ca->devMemUsed, rank);

		if (links_exist) {
			if (!use_bit) {
				edges.past_edges = (int*)malloc(sizeof(int) * static_cast<uint64_t>(N_tar * k_tar * (1.0 + edge_buffer) / 2));
				if (edges.past_edges == NULL)
					throw std::bad_alloc();
				memset(edges.past_edges, 0, sizeof(int) * static_cast<uint64_t>(N_tar * k_tar * (1.0 + edge_buffer) / 2));
				ca->hostMemUsed += sizeof(int) * static_cast<uint64_t>(N_tar * k_tar * (1.0 + edge_buffer) / 2);

				edges.future_edges = (int*)malloc(sizeof(int) * static_cast<uint64_t>(N_tar * k_tar * (1.0 + edge_buffer) / 2));
				if (edges.future_edges == NULL)
					throw std::bad_alloc();
				memset(edges.future_edges, 0, sizeof(int) * static_cast<uint64_t>(N_tar * k_tar * (1.0 + edge_buffer) / 2));
				ca->hostMemUsed += sizeof(int) * static_cast<uint64_t>(N_tar * k_tar * (1.0 + edge_buffer) / 2);

				edges.past_edge_row_start = (int64_t*)malloc(sizeof(int64_t) * N_tar);
				if (edges.past_edge_row_start == NULL)
					throw std::bad_alloc();
				memset(edges.past_edge_row_start, 0, sizeof(int64_t) * N_tar);
				ca->hostMemUsed += sizeof(int64_t) * N_tar;
	
				edges.future_edge_row_start = (int64_t*)malloc(sizeof(int64_t) * N_tar);
				if (edges.future_edge_row_start == NULL)
					throw std::bad_alloc();
				memset(edges.future_edge_row_start, 0, sizeof(int64_t) * N_tar);
				ca->hostMemUsed += sizeof(int64_t) * N_tar;
			}

			int length = static_cast<int>(ceil(static_cast<float>(static_cast<int>(N_tar * core_edge_fraction)) / cmpi.num_mpi_threads));
			int n = static_cast<unsigned int>(POW2(cmpi.num_mpi_threads, EXACT)) << 1;
			if (length % n)
				length += n - (length % n);
			//printf("adj length: %d\n", length);
			adj.reserve(length);
			for (int i = 0; i < length; i++) {
				FastBitset fb(static_cast<uint64_t>(core_edge_fraction * N_tar));
				adj.push_back(fb);
				ca->hostMemUsed += sizeof(BlockType) * fb.getNumBlocks();
			}

			#ifdef MPI_ENABLED
			if (cmpi.num_mpi_threads > 1) {
				int buflen = length / (cmpi.num_mpi_threads << 1);
				cmpi.adj_buf.reserve(buflen);
				for (int i = 0; i < buflen; i++) {
					FastBitset fb(static_cast<uint64_t>(core_edge_fraction * N_tar));
					cmpi.adj_buf.push_back(fb);
					ca->hostMemUsed += sizeof(BlockType) * fb.getNumBlocks();
					//printf_mpi(rank, "buffer row [%d] has %" PRIu64 " blocks\n", i, fb.getNumBlocks());
					//if (!rank) fflush(stdout);
					//MPI_Barrier(MPI_COMM_WORLD);
				}
			}
			#endif
		}

		memoryCheckpoint(ca->hostMemUsed, ca->maxHostMemUsed, ca->devMemUsed, ca->maxDevMemUsed);
		if (verbose)
			printMemUsed("for Network", ca->hostMemUsed, ca->devMemUsed, rank);
	} catch (std::bad_alloc) {
		fprintf(stderr, "Memory allocation failure in %s on line %d!\n", __FILE__, __LINE__);
		cmpi.fail = 1;
	}

	if (!no_pos && nodes.crd->isNull()) {
		printf("Null in thread %d\n", rank);
		cmpi.fail = 1;
	}

	if (checkMpiErrors(cmpi))
		return false;
	
	stopwatchStop(&sCreateNetwork);

	if (!bench) {
		printf_mpi(rank, "\tMemory Successfully Allocated.\n");
		fflush(stdout);
	}

	if (verbose) {
		printf_mpi(rank, "\t\tExecution Time: %5.6f sec\n", sCreateNetwork.elapsedTime);
		fflush(stdout);
	}

	return true;
}

//Poisson Sprinkling
//O(N) Efficiency
bool generateNodes(Node &nodes, const unsigned int &spacetime, const int &N_tar, const float &k_tar, const double &a, const double &eta0, const double &zeta, const double &zeta1, const double &r_max, const double &tau0, const double &alpha, CausetMPI &cmpi, MersenneRNG &mrng, Stopwatch &sGenerateNodes, const bool &verbose, const bool &bench)
{
	#if DEBUG
	//Values are in correct ranges
	assert (!nodes.crd->isNull());
	assert (N_tar > 0);
	assert (k_tar > 0.0f);
	assert (get_stdim(spacetime) & (2 | 4));
	assert (get_manifold(spacetime) & (MINKOWSKI | DE_SITTER | DUST | FLRW));
	assert (a >= 0.0);
	assert (tau0 > 0.0);
	if (get_manifold(spacetime) & (DUST | FLRW)) {
		#if EMBED_NODES
		assert (nodes.crd->getDim() == 5);
		assert (nodes.crd->v() != NULL);
		#else
		assert (nodes.crd->getDim() == 4);
		#endif
		assert (nodes.crd->w() != NULL);
		assert (nodes.crd->x() != NULL);
		assert (nodes.crd->y() != NULL);
		assert (nodes.crd->z() != NULL);
		assert (get_stdim(spacetime) == 4);
		assert (zeta < HALF_PI);
	} else if (get_manifold(spacetime) & DE_SITTER) {
		if (get_curvature(spacetime) & POSITIVE) {
			assert (zeta > 0.0);
			assert (zeta < HALF_PI);
		} else if (get_curvature(spacetime) & FLAT) {
			assert (zeta > HALF_PI);
			assert (zeta1 > HALF_PI);
			assert (zeta > zeta1);
		}
	}
	if (get_curvature(spacetime) & FLAT)
		assert (r_max > 0.0);
	#endif

	//Enable this to validate the nodes are being generated with the correct
	//distributions - it will use rejection sampling from the slab's distributions
	bool DEBUG_DIAMOND = false;

	stopwatchStart(&sGenerateNodes);

	IntData *idata = NULL;
	double params[3];
	double xi = eta0 / sqrt(2.0);
	double w = (zeta - zeta1) / sqrt(2.0);
	double mu = 0.0, mu1 = 0.0, mu2 = 0.0;
	double p1 = 0.0;

	//Rejection sampling vs exact CDF inversion
	bool use_rejection = false;
	if (DEBUG_DIAMOND && get_region(spacetime) & DIAMOND)
		use_rejection = true;
	if (get_region(spacetime) & SAUCER)
		use_rejection = true;

	//Initialize GSL integration structure
	//There is one 'workspace' per OpenMP thread to avoid
	//write conflicts in the for loop
	size_t i_size = (use_rejection ? 1 : omp_get_max_threads()) * sizeof(IntData);
	if ((USE_GSL || get_region(spacetime) & DIAMOND) && get_manifold(spacetime) & FLRW) {
		try {
			idata = (IntData*)malloc(i_size);
			if (idata == NULL)
				throw std::bad_alloc();
			memset(idata, 0, i_size);
		} catch (std::bad_alloc) {
			fprintf(stderr, "Memory allocation failure in %s on line %d!\n", __FILE__, __LINE__);
			return false;
		}

		for (int i = 0; i < (int)(i_size / sizeof(IntData)); i++) {
			idata[i] = IntData();
			//Modify these two parameters to trade off between speed and accuracy
			idata[i].limit = 50;
			idata[i].tol = 1e-4;
			idata[i].workspace = gsl_integration_workspace_alloc(idata[i].nintervals);
		}
	}

	//Initialize constants
	switch (spacetime) {
	case (2 | MINKOWSKI | SAUCER | FLAT | SYMMETRIC):
		mu1 = volume_77834_1(1.5);
		mu2 = volume_77834_1(-1.5);
		mu = mu1 - mu2;
		break;
	case (4 | DE_SITTER | DIAMOND | FLAT | ASYMMETRIC):
		mu = LOG(POW2(w + 2.0 * xi, EXACT) / (4.0 * xi * (w + xi)), STL) - POW2(w / (w + 2.0 * xi), EXACT);
		break;
	case (4 | DE_SITTER | DIAMOND | POSITIVE | ASYMMETRIC):
		mu = LOG(0.5 * (1.0 / COS(sqrt(2.0) * xi, APPROX ? FAST : STL) + 1.0), STL) - 1.0 / POW2(COS(xi / sqrt(2.0), APPROX ? FAST : STL), EXACT) + 1.0;
		break;
	case (4 | FLRW | DIAMOND | FLAT | ASYMMETRIC):
		params[0] = tau0;
		params[1] = HALF_PI - zeta;
		params[2] = zeta1;
		(*idata).limit = 100;
		(*idata).tol = 1e-8;
		(*idata).upper = zeta1;
		mu1 = integrate1D(&volume_11396_0_lower, params, idata, QAGS);
		(*idata).lower = (*idata).upper;
		(*idata).upper = tau0;
		mu2 = integrate1D(&volume_11396_0_upper, params, idata, QAGS);
		mu = mu1 + mu2;
		p1 = mu1 / mu;
		(*idata).limit = 50;
		(*idata).tol = 1e-4;
		break;
	default:
		mu = 1.0;
	}

	#ifndef _OPENMP
	UGenerator &urng = mrng.rng;
	NDistribution ndist(0.0, 1.0);
	NGenerator nrng(mrng.eng, ndist);
	#endif

	//Generate coordinates for each of N nodes
	int mpi_chunk = N_tar / cmpi.num_mpi_threads;
	int mpi_offset = mpi_chunk * cmpi.rank;
	if (use_rejection) {
		#ifdef _OPENMP
		UGenerator &urng = mrng.rng;
		NDistribution ndist(0.0, 1.0);
		NGenerator nrng(mrng.eng, ndist);
		#endif
		#if EMBED_NODES
		float2 emb2;
		float3 emb3;
		float4 emb4;
		#endif
		double eta;

		//Use the rejection method
		int i = 0;
		while (i < N_tar) {
			switch (spacetime) {
			case (2 | MINKOWSKI | SAUCER | FLAT | SYMMETRIC):
				#if SPECIAL_SAUCER
				nodes.crd->x(i) = 2.0 * urng() - 1.0;
				nodes.crd->y(i) = 3.0 * urng() - 1.5;
				if (fabs(nodes.crd->x(i)) > eta_77834_1(nodes.crd->y(i), eta0))
					continue;
				#else
				nodes.crd->x(i) = (2.0 * urng() - 1.0) * eta0;
				nodes.crd->y(i) = (2.0 * urng() - 1.0) * r_max;
				if (fabs(nodes.crd->x(i)) > eta_77834_1(nodes.crd->y(i), eta0))
					continue;
				#endif
				break;
			case (4 | DE_SITTER | DIAMOND | FLAT | ASYMMETRIC):
				#if EMBED_NODES
				nodes.crd->v(i) = get_4d_asym_flat_deSitter_slab_eta(urng, HALF_PI - zeta, HALF_PI - zeta1);
				emb3 = get_4d_asym_flat_deSitter_slab_cartesian(urng, nrng, r_max);
				r = sqrt(POW2(emb3.x, EXACT) + POW2(emb3.y, EXACT) + POW2(emb3.z, EXACT));
				if (!iad(nodes.crd->v(i), r, eta0, HALF_PI  - zeta, HALF_PI - zeta1))
					continue;
				nodes.id.tau[i] = etaToTauFlat(nodes.crd->v(i));
				nodes.crd->x(i) = emb3.x;
				nodes.crd->y(i) = emb3.y;
				nodes.crd->z(i) = emb3.z;
				#else
				nodes.crd->w(i) = get_4d_asym_flat_deSitter_slab_eta(urng, HALF_PI - zeta, HALF_PI - zeta1);
				nodes.crd->x(i) = get_4d_asym_flat_deSitter_slab_radius(urng, r_max);
				if (!iad(nodes.crd->w(i), nodes.crd->x(i), HALF_PI - zeta, HALF_PI - zeta1))
					continue;
				nodes.id.tau[i] = etaToTauFlat(nodes.crd->w(i));
				nodes.crd->y(i) = get_4d_asym_flat_deSitter_diamond_theta2(urng);
				nodes.crd->z(i) = get_4d_asym_flat_deSitter_diamond_theta3(urng);
				#endif
				break;
			case (4 | DE_SITTER | DIAMOND | POSITIVE | ASYMMETRIC):
				#if EMBED_NODES
				nodes.crd->v(i) = get_4d_asym_sph_deSitter_slab_eta(urng, zeta);
				emb4 = get_4d_asym_sph_deSitter_slab_emb(nrng);
				r = acosf(emb4.w);
				if (!iad(nodes.crd->v(i), r, 0.0, eta0))
					continue;
				nodes.id.tau[i] = etaToTauSph(nodes.crd->v(i));
				nodes.crd->w(i) = emb4.w;
				nodes.crd->x(i) = emb4.x;
				nodes.crd->y(i) = emb4.y;
				nodes.crd->z(i) = emb4.z;
				#else
				nodes.crd->w(i) = get_4d_asym_sph_deSitter_slab_eta(urng, zeta);
				nodes.crd->x(i) = get_4d_asym_sph_deSitter_slab_theta1(urng);
				if (!iad(nodes.crd->w(i), nodes.crd->x(i), 0.0, HALF_PI - zeta))
					continue;
				nodes.id.tau[i] = etaToTauSph(nodes.crd->w(i));
				nodes.crd->y(i) = get_4d_asym_sph_deSitter_diamond_theta2(urng);
				nodes.crd->z(i) = get_4d_asym_sph_deSitter_diamond_theta3(urng);
				#endif
				break;
			case (4 | DUST | DIAMOND | FLAT | ASYMMETRIC):
			{
				nodes.id.tau[i] = get_4d_asym_flat_dust_slab_tau(urng, tau0);
				#if EMBED_NODES
				nodes.crd->v(i) = tauToEtaDust(nodes.id.tau[i], a, alpha);
				emb3 = get_4d_asym_flat_dust_slab_cartesian(urng, nrng, r_max);
				r = sqrt(POW2(emb3.x, EXACT) + POW2(emb3.y, EXACT) + POW2(emb3.z, EXACT));
				if (!iad(nodes.crd->v(i), r, 0.0, HALF_PI - zeta))
					continue;
				nodes.crd->x(i) = emb3.x;
				nodes.crd->y(i) = emb3.y;
				nodes.crd->z(i) = emb3.z;
				#else
				nodes.crd->w(i) = tauToEtaDust(nodes.id.tau[i], a, alpha);
				nodes.crd->x(i) = get_4d_asym_flat_dust_slab_radius(urng, r_max);
				if (!iad(nodes.crd->w(i), nodes.crd->x(i), 0.0, HALF_PI - zeta))
					continue;
				nodes.crd->y(i) = get_4d_asym_flat_dust_diamond_theta2(urng);
				nodes.crd->z(i) = get_4d_asym_flat_dust_diamond_theta3(urng);
				#endif
				break;
			}
			case (4 | FLRW | DIAMOND | FLAT | ASYMMETRIC):
			{
				nodes.id.tau[i] = get_4d_asym_flat_flrw_slab_tau(urng, tau0);
				if (USE_GSL) {
					(*idata).lower = 0.0;
					(*idata).upper = nodes.id.tau[i];
					eta = integrate1D(&tauToEtaFLRW, NULL, idata, QAGS) * a / alpha;
				} else
					eta = tauToEtaFLRWExact(nodes.id.tau[i], a, alpha);
				#if EMBED_NODES
				nodes.crd->v(i) = eta;
				emb3 = get_4d_asym_flat_flrw_slab_cartesian(urng, nrng, r_max);
				r = sqrt(POW2(emb3.x, EXACT) + POW2(emb3.y, EXACT) + POW2(emb3.z, EXACT));
				if (!iad(nodes.crd->v(i), r, 0.0, HALF_PI - zeta))
					continue;
				nodes.crd->x(i) = emb3.x;
				nodes.crd->y(i) = emb3.y;
				nodes.crd->z(i) = emb3.z;
				#else
				nodes.crd->w(i) = eta;
				nodes.crd->x(i) = get_4d_asym_flat_flrw_slab_radius(urng, r_max);
				if (!iad(nodes.crd->w(i), nodes.crd->x(i), 0.0, HALF_PI - zeta))
					continue;
				nodes.crd->y(i) = get_4d_asym_flat_flrw_slab_theta2(urng);
				nodes.crd->z(i) = get_4d_asym_flat_flrw_slab_theta3(urng);
				#endif 
				break;
			}
			default:
				fprintf(stderr, "Spacetime parameters not supported!\n");
				assert (false);
			}

			#if DEBUG
			if (!validateCoordinates(nodes, spacetime, eta0, zeta, zeta1, r_max, tau0, i))
				i--;
			#endif

			i++;
		}
	} else {
		//Use exact CDF inversion formulae (see notes)
		int start = mpi_offset;
		int finish = start + mpi_chunk;

		#ifdef _OPENMP
		unsigned int seed = static_cast<unsigned int>(mrng.rng() * 400000000);
		#pragma omp parallel if (N_tar < 1000)
		{
		//Initialize one RNG per thread
		Engine eng(seed ^ omp_get_thread_num());
		UDistribution udist(0.0, 1.0);
		UGenerator urng(eng, udist);
		NDistribution ndist(0.0, 1.0);
		NGenerator nrng(eng, ndist);
		#pragma omp for schedule (dynamic, 8)
		#endif
		for (int i = start; i < finish; i++) {
			#if EMBED_NODES
			float2 emb2;
			float3 emb3;
			float4 emb4;
			#endif
			double eta, r;
			double u, v;
			int tid = omp_get_thread_num();

			do {
				switch (spacetime) {
				case (2 | MINKOWSKI | DIAMOND | FLAT | ASYMMETRIC):
					u = get_2d_asym_flat_minkowski_diamond_u(urng, xi);
					v = get_2d_asym_flat_minkowski_diamond_v(urng, xi);
					nodes.crd->x(i) = (u + v) / sqrt(2.0);
					nodes.crd->y(i) = (u - v) / sqrt(2.0);
					break;
				case (2 | MINKOWSKI | SAUCER | FLAT | SYMMETRIC):
					fprintf(stderr, "Not yet implemented on line %d in file %s\n", __LINE__, __FILE__);
					assert (false);
					//nodes.crd->y(i) = get_2d_sym_flat_minkowski_saucer_x(urng, mu, mu2);
					//nodes.crd->x(i) = get_2d_sym_flat_minkowski_saucer_eta(urng, nodes.crd->y(i));
					break;
				case (2 | DE_SITTER | SLAB | POSITIVE | ASYMMETRIC):
					nodes.crd->x(i) = get_2d_asym_sph_deSitter_slab_eta(urng, eta0);
					nodes.id.tau[i] = etaToTauSph(nodes.crd->x(i));
					#if EMBED_NODES
					emb2 = get_2d_asym_sph_deSitter_slab_emb(urng);
					nodes.crd->y(i) = emb2.x;
					nodes.crd->z(i) = emb2.y;
					#else
					nodes.crd->y(i) = get_2d_asym_sph_deSitter_slab_theta(urng);
					#endif
					break;
				case (2 | DE_SITTER | SLAB | POSITIVE | SYMMETRIC):
					nodes.crd->x(i) = get_2d_sym_sph_deSitter_slab_eta(urng, eta0);
					nodes.id.tau[i] = etaToTauSph(nodes.crd->x(i));
					#if EMBED_NODES
					emb2 = get_2d_sym_sph_deSitter_slab_emb(urng);
					nodes.crd->y(i) = emb2.x;
					nodes.crd->z(i) = emb2.y;
					#else
					nodes.crd->y(i) = get_2d_sym_sph_deSitter_slab_theta(urng);
					#endif
					break;
				case (2 | DE_SITTER | DIAMOND | FLAT | ASYMMETRIC):
					fprintf(stderr, "Not yet implemented on line %d in file %s\n", __LINE__, __FILE__);
					assert (false);
					break;
				case (2 | DE_SITTER | DIAMOND | POSITIVE | ASYMMETRIC):
					nodes.crd->x(i) = get_2d_asym_sph_deSitter_diamond_eta(urng);
					nodes.id.tau[i] = etaToTauSph(nodes.crd->x(i));
					#if EMBED_NODES
					emb2 = get_2d_asym_sph_deSitter_diamond_emb(mrng.rng, nodes.crd->x(i));
					nodes.crd->y(i) = emb2.x;
					nodes.crd->z(i) = emb2.y;
					#else
					nodes.crd->y(i) = get_2d_asym_sph_deSitter_diamond_theta(urng, nodes.crd->x(i));
					#endif
					break;
				case (2 | DE_SITTER | DIAMOND | POSITIVE | SYMMETRIC):
					fprintf(stderr, "Not yet implemented on line %d in file %s\n", __LINE__, __FILE__);
					assert (false);
					break;
				case (4 | DE_SITTER | SLAB | FLAT | ASYMMETRIC):
					#if EMBED_NODES
					nodes.crd->v(i) = get_4d_asym_flat_deSitter_slab_eta(urng, HALF_PI - zeta, HALF_PI - zeta1);
					nodes.id.tau[i] = etaToTauFlat(nodes.crd->v(i));
					emb3 = get_4d_asym_flat_deSitter_slab_cartesian(urng, nrng, r_max);
					nodes.crd->x(i) = emb3.x;
					nodes.crd->y(i) = emb3.y;
					nodes.crd->z(i) = emb3.z;
					#else
					nodes.crd->w(i) = get_4d_asym_flat_deSitter_slab_eta(urng, HALF_PI - zeta, HALF_PI - zeta1);
					nodes.id.tau[i] = etaToTauFlat(nodes.crd->w(i));
					nodes.crd->x(i) = get_4d_asym_flat_deSitter_slab_radius(mrng.rng, r_max);
					nodes.crd->y(i) = get_4d_asym_flat_deSitter_slab_theta2(mrng.rng);
					nodes.crd->z(i) = get_4d_asym_flat_deSitter_slab_theta3(mrng.rng);
					#endif
					break;
				case (4 | DE_SITTER | SLAB | POSITIVE | ASYMMETRIC):
					#if EMBED_NODES
					nodes.crd->v(i) = get_4d_asym_sph_deSitter_slab_eta(urng, zeta);
					nodes.id.tau[i] = etaToTauSph(nodes.crd->v(i));
					emb4 = get_4d_asym_sph_deSitter_slab_emb(nrng);
					nodes.crd->w(i) = emb4.w;
					nodes.crd->x(i) = emb4.x;
					nodes.crd->y(i) = emb4.y;
					nodes.crd->z(i) = emb4.z;
					#else
					nodes.crd->w(i) = get_4d_asym_sph_deSitter_slab_eta(urng, zeta);
					nodes.id.tau[i] = etaToTauSph(nodes.crd->w(i));
					nodes.crd->x(i) = get_4d_asym_sph_deSitter_slab_theta1(urng);
					nodes.crd->y(i) = get_4d_asym_sph_deSitter_slab_theta2(urng);
					nodes.crd->z(i) = get_4d_asym_sph_deSitter_slab_theta3(urng);
					#endif
					break;
				case (4 | DE_SITTER | SLAB | POSITIVE | SYMMETRIC):
					#if EMBED_NODES
					nodes.crd->v(i) = get_4d_sym_sph_deSitter_slab_eta(urng, zeta);
					nodes.id.tau[i] = etaToTauSph(nodes.crd->v(i));
					emb4 = get_4d_sym_sph_deSitter_slab_emb(nrng);
					nodes.crd->w(i) = emb4.w;
					nodes.crd->x(i) = emb4.x;
					nodes.crd->y(i) = emb4.y;
					nodes.crd->z(i) = emb4.z;
					#else
					nodes.crd->w(i) = get_4d_sym_sph_deSitter_slab_eta(urng, zeta);
					//This statement makes the slab partially asymmetric on the upper half
					//if (nodes.crd->w(i) > eta0 / 2.0)
					//	nodes.crd->w(i) = M_PI;
					nodes.id.tau[i] = etaToTauSph(nodes.crd->w(i));
					nodes.crd->x(i) = get_4d_sym_sph_deSitter_slab_theta1(urng);
					nodes.crd->y(i) = get_4d_sym_sph_deSitter_slab_theta2(urng);
					nodes.crd->z(i) = get_4d_sym_sph_deSitter_slab_theta3(urng);
					#endif
					break;
				case (4 | DE_SITTER | DIAMOND | FLAT | ASYMMETRIC):
					u = get_4d_asym_flat_deSitter_diamond_u(urng, xi, mu);
					v = get_4d_asym_flat_deSitter_diamond_v(urng, u, xi);
					#if EMBED_NODES
					nodes.crd->v(i) = (u + v) / sqrt(2.0);
					nodes.id.tau[i] = etaToTauFlat(nodes.crd->v(i));
					emb3 = get_4d_asym_flat_deSitter_diamond_cartesian(urng, nrng);
					nodes.crd->x(i) = emb3.x;
					nodes.crd->y(i) = emb3.y;
					nodes.crd->z(i) = emb3.z;
					#else
					nodes.crd->w(i) = (u + v) / sqrt(2.0);
					nodes.crd->x(i) = (u - v) / sqrt(2.0);
					nodes.id.tau[i] = etaToTauFlat(nodes.crd->w(i));
					nodes.crd->y(i) = get_4d_asym_flat_deSitter_diamond_theta2(urng);
					nodes.crd->z(i) = get_4d_asym_flat_deSitter_diamond_theta3(urng);
					#endif
					break;
				case (4 | DE_SITTER | DIAMOND | POSITIVE | ASYMMETRIC):
					u = get_4d_asym_sph_deSitter_diamond_u(urng, xi, mu);
					v = get_4d_asym_sph_deSitter_diamond_v(urng, u);
					#if EMBED_NODES
					nodes.crd->v(i) = (u + v) / sqrt(2.0);
					nodes.id.tau[i] = etaToTauSph(nodes.crd->v(i));
					emb4 = get_4d_asym_sph_deSitter_diamond_emb(urng, nrng, u, v);
					nodes.crd->w(i) = emb4.w;
					nodes.crd->x(i) = emb4.x;
					nodes.crd->y(i) = emb4.y;
					nodes.crd->z(i) = emb4.z;
					#else
					nodes.crd->w(i) = (u + v) / sqrt(2.0);
					nodes.crd->x(i) = (u - v) / sqrt(2.0);
					nodes.id.tau[i] = etaToTauSph(nodes.crd->w(i));
					nodes.crd->y(i) = get_4d_asym_sph_deSitter_diamond_theta2(urng);
					nodes.crd->z(i) = get_4d_asym_sph_deSitter_diamond_theta3(urng);
					#endif
					break;
				case (4 | DE_SITTER | DIAMOND | POSITIVE | SYMMETRIC):
					fprintf(stderr, "Not yet implemented on line %d in file %s\n", __LINE__, __FILE__);
					assert (false);
					break;
				case (4 | DUST | SLAB | FLAT | ASYMMETRIC):
					nodes.id.tau[i] = get_4d_asym_flat_dust_slab_tau(urng, tau0);
					#if EMBED_NODES
					nodes.crd->v(i) = tauToEtaDust(nodes.id.tau[i], a, alpha);
					emb3 = get_4d_asym_flat_dust_slab_cartesian(urng, nrng, r_max);
					nodes.crd->x(i) = emb3.x;
					nodes.crd->y(i) = emb3.y;
					nodes.crd->z(i) = emb3.z;
					#else
					nodes.crd->w(i) = tauToEtaDust(nodes.id.tau[i], a, alpha);
					nodes.crd->x(i) = get_4d_asym_flat_dust_slab_radius(urng, r_max);
					nodes.crd->y(i) = get_4d_asym_flat_dust_slab_theta2(urng);
					nodes.crd->z(i) = get_4d_asym_flat_dust_slab_theta3(urng);
					#endif
					break;
				case (4 | DUST | DIAMOND | FLAT | ASYMMETRIC):
					u = get_4d_asym_flat_dust_diamond_u(urng, xi);
					v = get_4d_asym_flat_dust_diamond_v(urng, u);
					#if EMBED_NODES
					nodes.crd->v(i) = (u + v) / sqrt(2.0);
					nodes.id.tau[i] = etaToTauDust(nodes.crd->v(i), a, alpha);
					emb3 = get_4d_asym_flat_dust_diamond_cartesian(urng, nrng, u, v);
					nodes.crd->x(i) = emb3.x;
					nodes.crd->y(i) = emb3.y;
					nodes.crd->z(i) = emb3.z;
					#else
					nodes.crd->w(i) = (u + v) / sqrt(2.0);
					nodes.id.tau[i] = etaToTauDust(nodes.crd->w(i), a, alpha);
					nodes.crd->x(i) = (u - v) / sqrt(2.0);
					nodes.crd->y(i) = get_4d_asym_flat_dust_diamond_theta2(urng);
					nodes.crd->z(i) = get_4d_asym_flat_dust_diamond_theta3(urng);
					#endif
					break;
				case (4 | FLRW | SLAB | FLAT | ASYMMETRIC):
					nodes.id.tau[i] = get_4d_asym_flat_flrw_slab_tau(urng, tau0);
					if (USE_GSL) {
						idata[tid].lower = 0.0;
						idata[tid].upper = nodes.id.tau[i];
						eta = integrate1D(&tauToEtaFLRW, NULL, &idata[tid], QAGS) * a / alpha;
					} else
						eta = tauToEtaFLRWExact(nodes.id.tau[i], a, alpha);
					#if EMBED_NODES
					nodes.crd->v(i) = eta;
					emb3 = get_4d_asym_flat_flrw_slab_cartesian(urng, nrng, r_max);
					nodes.crd->x(i) = emb3.x;
					nodes.crd->y(i) = emb3.y;
					nodes.crd->z(i) = emb3.z;
					#else
					nodes.crd->w(i) = eta;
					nodes.crd->x(i) = get_4d_asym_flat_flrw_slab_radius(urng, r_max);
					nodes.crd->y(i) = get_4d_asym_flat_flrw_slab_theta2(urng);
					nodes.crd->z(i) = get_4d_asym_flat_flrw_slab_theta3(urng);
					#endif
					break;
				case (4 | FLRW | SLAB | POSITIVE | ASYMMETRIC):
					nodes.id.tau[i] = get_4d_asym_sph_flrw_slab_tau(urng, tau0);
					if (USE_GSL) {
						idata[tid].lower = 0.0;
						idata[tid].upper = nodes.id.tau[i];
						eta = integrate1D(&tauToEtaFLRW, NULL, &idata[tid], QAGS) * a / alpha;
					} else
						eta = tauToEtaFLRWExact(nodes.id.tau[i], a, alpha);
					#if EMBED_NODES
					nodes.crd->v(i) = eta;
					emb4 = get_4d_asym_sph_flrw_slab_cartesian(nrng);
					nodes.crd->w(i) = emb4.w;
					nodes.crd->x(i) = emb4.x;
					nodes.crd->y(i) = emb4.y;
					nodes.crd->z(i) = emb4.z;
					#else
					nodes.crd->w(i) = eta;
					nodes.crd->x(i) = get_4d_asym_sph_flrw_slab_theta1(urng);
					nodes.crd->y(i) = get_4d_asym_sph_flrw_slab_theta2(urng);
					nodes.crd->z(i) = get_4d_asym_sph_flrw_slab_theta3(urng);
					#endif
					break;
				case (4 | FLRW | DIAMOND | FLAT | ASYMMETRIC):
				{
					nodes.id.tau[i] = get_4d_asym_flat_flrw_diamond_tau(urng, &idata[tid], params, tau0, zeta1, p1, mu, mu1);
					if (USE_GSL) {
						idata[tid].lower = 0.0;
						idata[tid].upper = nodes.id.tau[i];
						eta = integrate1D(&tauToEtaFLRW, NULL, &idata[tid], QAGS) * a / alpha;
					} else
						eta = tauToEtaFLRWExact(nodes.id.tau[i], a, alpha);
					r = get_4d_asym_flat_flrw_diamond_radius(urng, eta, zeta);
					#if EMBED_NODES
					nodes.crd->v(i) = eta;
					emb3 = get_sph_d3(nrng);
					nodes.crd->x(i) = r * emb3.x;
					nodes.crd->y(i) = r * emb3.y;
					nodes.crd->z(i) = r * emb3.z;
					#else
					nodes.crd->w(i) = eta;
					nodes.crd->x(i) = r;
					nodes.crd->y(i) = get_4d_asym_flat_flrw_diamond_theta2(urng);
					nodes.crd->z(i) = get_4d_asym_flat_flrw_diamond_theta3(urng);
					#endif
					break;
				}
				default:
					fprintf(stderr, "Spacetime parameters not supported!\n");
					assert (false);
				}
			} while (!validateCoordinates(nodes, spacetime, eta0, zeta, zeta1, r_max, tau0, i));
		}
		#ifdef _OPENMP
		}
		#endif
	}

	//Free GSL workspace memory
	if ((USE_GSL || get_region(spacetime) & DIAMOND) && get_manifold(spacetime) & FLRW) {
		for (int i = 0; i < (int)(i_size / sizeof(IntData)); i++)
			gsl_integration_workspace_free(idata[i].workspace);
		free(idata);
		idata = NULL;
	}

	#ifdef MPI_ENABLED
	if (nodes.id.tau != NULL)
		MPI_Allgather(MPI_IN_PLACE, 0, MPI_DATATYPE_NULL, nodes.id.tau, mpi_chunk, MPI_FLOAT, MPI_COMM_WORLD);
	if (nodes.crd->v() != NULL)
		MPI_Allgather(MPI_IN_PLACE, 0, MPI_DATATYPE_NULL, nodes.crd->v(), mpi_chunk, MPI_FLOAT, MPI_COMM_WORLD);
	if (nodes.crd->w() != NULL)
		MPI_Allgather(MPI_IN_PLACE, 0, MPI_DATATYPE_NULL, nodes.crd->w(), mpi_chunk, MPI_FLOAT, MPI_COMM_WORLD);
	if (nodes.crd->x() != NULL)
		MPI_Allgather(MPI_IN_PLACE, 0, MPI_DATATYPE_NULL, nodes.crd->x(), mpi_chunk, MPI_FLOAT, MPI_COMM_WORLD);
	if (nodes.crd->y() != NULL)
		MPI_Allgather(MPI_IN_PLACE, 0, MPI_DATATYPE_NULL, nodes.crd->y(), mpi_chunk, MPI_FLOAT, MPI_COMM_WORLD);
	if (nodes.crd->z() != NULL)
		MPI_Allgather(MPI_IN_PLACE, 0, MPI_DATATYPE_NULL, nodes.crd->z(), mpi_chunk, MPI_FLOAT, MPI_COMM_WORLD);
	#endif

	//Debugging statements used to check coordinate distributions
	//if (cmpi.rank == 0 && !printValues(nodes, spacetime, N_tar, "tau_dist_rank0.cset.dbg.dat", "tau")) return false;
	//if (cmpi.rank == 1 && !printValues(nodes, spacetime, N_tar, "tau_dist_rank1.cset.dbg.dat", "tau")) return false;
	//if (!printValues(nodes, spacetime, N_tar, "eta_dist.cset.dbg.dat", "eta")) return false;
	//if (!printValues(nodes, spacetime, N_tar, "u_dist.cset.dbg.dat", "u")) return false;
	//if (!printValues(nodes, spacetime, N_tar, "v_dist.cset.dbg.dat", "v")) return false;
	//if (!printValues(nodes, spacetime, N_tar, "theta1_dist.cset.dbg.dat", "theta1")) return false;
	//if (!printValues(nodes, spacetime, N_tar, "theta2_dist.cset.dbg.dat", "theta2")) return false;
	//if (!printValues(nodes, spacetime, N_tar, "theta3_dist.cset.dbg.dat", "theta3")) return false;
	/*printf_red();
	printf("Check coordinate distributions now.\n");
	printf_std();
	fflush(stdout);*/
	//printChk();

	stopwatchStop(&sGenerateNodes);

	if (!bench) {
		printf_mpi(cmpi.rank, "\tNodes Successfully Generated.\n");
		fflush(stdout);
	}

	if (verbose) {
		printf_mpi(cmpi.rank, "\t\tExecution Time: %5.6f sec\n", sGenerateNodes.elapsedTime);
		fflush(stdout);
	}

	return true;
}

bool linkNodes_v2(Node &nodes, Bitvector &adj, const unsigned int &spacetime, const int &N_tar, const float &k_tar, int &N_res, float &k_res, int &N_deg2, const double &a, const double &zeta, const double &zeta1, const double &r_max, const double &tau0, const double &alpha, CausetMPI &cmpi, Stopwatch &sLinkNodes, const bool &use_bit, const bool &verbose, const bool &bench)
{
	#if DEBUG
	//No null pointers
	assert (!nodes.crd->isNull());

	//Variables in correct ranges
	assert (N_tar > 0);
	assert (k_tar > 0.0f);
	assert (get_stdim(spacetime) & (2 | 4));
	assert (get_manifold(spacetime) & (MINKOWSKI | DE_SITTER | DUST | FLRW));
	assert (a > 0.0);
	assert (tau0 > 0.0);
	if (get_manifold(spacetime) & DE_SITTER) {
		if (get_curvature(spacetime) & POSITIVE) {
			assert (zeta > 0.0);
			assert (zeta < HALF_PI);
		} else if (get_curvature(spacetime) & FLAT) {
			assert (zeta > HALF_PI);
			assert (zeta1 > HALF_PI);
			assert (zeta > zeta1);
		}
	} else if (get_manifold(spacetime) & (DUST | FLRW)) {
		#if EMBED_NODES
		assert (nodes.crd->getDim() == 5);
		assert (nodes.crd->v() != NULL);
		#else
		assert (nodes.crd->getDim() == 4);
		#endif
		assert (nodes.crd->w() != NULL);
		assert (nodes.crd->x() != NULL);
		assert (nodes.crd->y() != NULL);
		assert (nodes.crd->z() != NULL);
		assert (get_stdim(spacetime) == 4);
		assert (zeta < HALF_PI);
		assert (alpha > 0.0);
	}
	if (get_curvature(spacetime) & FLAT)
		assert (r_max > 0.0);
	assert (use_bit);
	#endif

	//#ifdef MPI_ENABLED
	if (!cmpi.rank) printf_mag();
	printf_mpi(cmpi.rank, "Using Version 2 (linkNodes).\n");
	if (!cmpi.rank) printf_std();
	//#endif

	int64_t idx = 0;
	int rank = cmpi.rank;
	int mpi_chunk = N_tar / cmpi.num_mpi_threads;
	int mpi_offset = rank * mpi_chunk;

	uint64_t npairs = static_cast<uint64_t>(N_tar) * mpi_chunk;
	uint64_t start = rank * npairs;
	uint64_t finish = start + npairs;
	stopwatchStart(&sLinkNodes);

	//printf_mpi(rank - 1, "mpi_offset: %d\n", mpi_offset);
	//printf_mpi(rank, "start: %d\tfinish: %d\n", start, finish);

	#ifdef _OPENMP
	#pragma omp parallel for schedule (dynamic, 1) reduction (+ : idx) if (finish - start > 1024)
	#endif
	for (uint64_t k = start; k < finish; k++) {
		int i = static_cast<int>(k / N_tar);
		int j = static_cast<int>(k % N_tar);

		if (i == j) continue;
		//printf_mpi(rank - 1, "i: %d\tj: %d\t", i, j);
		bool related = nodesAreRelated(nodes.crd, spacetime, N_tar, a, zeta, zeta1, r_max, alpha, i, j, NULL);
		//printf_mpi(rank - 1, "related: %d\n", (int)related);

		if (related) {
			#ifdef _OPENMP
			#pragma omp critical
			#endif
			{
				//printf_mpi(rank - 1, "Setting row [%d] column [%d]\n", i-mpi_offset, j);
				adj[i-mpi_offset].set(j);
				//printf_mpi(rank, "Setting row [%d] column [%d]\n", j, i);
				//adj[j].set(i);
				//printf_mpi(rank, "Successfully set both elements.\n");
			}

			if (i < j) {
				#ifdef _OPENMP
				#pragma omp atomic
				#endif
				nodes.k_in[j]++;
				#ifdef _OPENMP
				#pragma omp atomic
				#endif
				nodes.k_out[i]++;

				idx++;
			}
		}
	}

	#ifdef MPI_ENABLED
	MPI_Barrier(MPI_COMM_WORLD);
	MPI_Allreduce(MPI_IN_PLACE, nodes.k_in, N_tar, MPI_INT, MPI_SUM, MPI_COMM_WORLD);
	MPI_Allreduce(MPI_IN_PLACE, nodes.k_out, N_tar, MPI_INT, MPI_SUM, MPI_COMM_WORLD);
	MPI_Allreduce(MPI_IN_PLACE, &idx, 1, MPI_INT64_T, MPI_SUM, MPI_COMM_WORLD);
	#endif

	uint64_t kr = 0;
	for (int i = 0; i < N_tar; i++) {
		if (nodes.k_in[i] + nodes.k_out[i] > 0) {
			N_res++;
			kr += nodes.k_in[i] + nodes.k_out[i];

			if (nodes.k_in[i] + nodes.k_out[i] > 1)
				N_deg2++;
		} 
	}

	#if DEBUG
	assert (N_res >= 0);
	assert (N_deg2 >= 0);
	#endif

	if (N_res > 0)
		k_res = static_cast<long double>(kr) / N_res;

	//if (!rank && !printDegrees(nodes, N_tar, "in-degrees_MPI_CPU.cset.dbg.dat", "out-degrees_MPI_CPU.cset.dbg.dat")) return false;
	//if (!printAdjMatrix(adj, N_tar, "adj_matrix_MPI_CPU.cset.dbg.dat", cmpi.num_mpi_threads, cmpi.rank)) return false;

	stopwatchStop(&sLinkNodes);

	if (!bench) {
		printf_mpi(rank, "\tCausets Successfully Connected.\n");
		if (!rank) printf_cyan();
		printf_mpi(rank, "\t\tUndirected Links:         %" PRIu64 "\n", idx);
		printf_mpi(rank, "\t\tResulting Network Size:   %d\n", N_res);
		printf_mpi(rank, "\t\tResulting Average Degree: %f\n", k_res);
		printf_mpi(rank, "\t\t    Incl. Isolated Nodes: %f\n", k_res * ((float)N_res / N_tar));
		if (!rank) printf_red();
		printf_mpi(rank, "\t\tResulting Error in <k>:   %f\n", fabs(k_tar - k_res) / k_tar);
		if (!rank) printf_std();
		if (!rank) fflush(stdout);
	}

	if (verbose) {
		printf_mpi(rank, "\t\tExecution Time: %5.6f sec\n", sLinkNodes.elapsedTime);
		fflush(stdout);
	}
	
	#ifdef MPI_ENABLED
	MPI_Barrier(MPI_COMM_WORLD);
	#endif

	return true;
}

//Identify Causal Sets
//O(k*N^2) Efficiency
bool linkNodes(Node &nodes, Edge &edges, Bitvector &adj, const unsigned int &spacetime, const int &N_tar, const float &k_tar, int &N_res, float &k_res, int &N_deg2, const double &a, const double &zeta, const double &zeta1, const double &r_max, const double &tau0, const double &alpha, const float &core_edge_fraction, const float &edge_buffer, Stopwatch &sLinkNodes, const bool &use_bit, const bool &verbose, const bool &bench)
{
	#if DEBUG
	//No null pointers
	assert (!nodes.crd->isNull());
	if (!use_bit) {
		assert (edges.past_edges != NULL);
		assert (edges.future_edges != NULL);
		assert (edges.past_edge_row_start != NULL);
		assert (edges.future_edge_row_start != NULL);
	} else
		assert (core_edge_fraction == 1.0f);

	//Variables in correct ranges
	assert (N_tar > 0);
	assert (k_tar > 0.0f);
	assert (get_stdim(spacetime) & (2 | 4));
	assert (get_manifold(spacetime) & (MINKOWSKI | DE_SITTER | DUST | FLRW));
	assert (a > 0.0);
	assert (tau0 > 0.0);
	if (get_manifold(spacetime) & DE_SITTER) {
		if (get_curvature(spacetime) & POSITIVE) {
			assert (zeta > 0.0);
			assert (zeta < HALF_PI);
		} else if (get_curvature(spacetime) & FLAT) {
			assert (zeta > HALF_PI);
			assert (zeta1 > HALF_PI);
			assert (zeta > zeta1);
		}
	} else if (get_manifold(spacetime) & (DUST | FLRW)) {
		#if EMBED_NODES
		assert (nodes.crd->getDim() == 5);
		assert (nodes.crd->v() != NULL);
		#else
		assert (nodes.crd->getDim() == 4);
		#endif
		assert (nodes.crd->w() != NULL);
		assert (nodes.crd->x() != NULL);
		assert (nodes.crd->y() != NULL);
		assert (nodes.crd->z() != NULL);
		assert (get_stdim(spacetime) == 4);
		assert (zeta < HALF_PI);
		assert (alpha > 0.0);
	}
	if (get_curvature(spacetime) & FLAT)
		assert (r_max > 0.0);
	assert (core_edge_fraction >= 0.0f && core_edge_fraction <= 1.0f);
	assert (edge_buffer >= 0.0f && edge_buffer <= 1.0f);
	#endif

	//#ifdef MPI_ENABLED
	printf_dbg("Using Version 1 (linkNodes).\n");
	//#endif

	uint64_t future_idx = 0;
	uint64_t past_idx = 0;
	int core_limit = static_cast<int>(core_edge_fraction * N_tar);
	int i, j, k;

	bool related;

	stopwatchStart(&sLinkNodes);

	//Identify future connections
	for (i = 0; i < N_tar - 1; i++) {
		if (!use_bit)
			edges.future_edge_row_start[i] = future_idx;

		for (j = i + 1; j < N_tar; j++) {
			//Apply Causal Condition (Light Cone)
			//Assume nodes are already temporally ordered
			related = nodesAreRelated(nodes.crd, spacetime, N_tar, a, zeta, zeta1, r_max, alpha, i, j, NULL);

			//Core Edge Adjacency Matrix
			if (i < core_limit && j < core_limit) {
				if (related) {
					//printf("[%d] and [%d] are related\n", i, j);
					adj[i].set(j);
					adj[j].set(i);
				}
			}
						
			//Link timelike relations
			try {
				if (related) {
					if (!use_bit) {
						//if (i % NPRINT == 0) printf("%d %d\n", i, j); fflush(stdout);
						edges.future_edges[future_idx++] = j;
	
						if (future_idx >= static_cast<int64_t>(N_tar) * k_tar * (1.0 + edge_buffer) / 2)
							throw CausetException("Not enough memory in edge adjacency list.  Increase edge buffer or decrease network size.\n");
					} else
						future_idx++;
	
					//Record number of degrees for each node
					nodes.k_in[j]++;
					nodes.k_out[i]++;
				}
			} catch (CausetException c) {
				fprintf(stderr, "CausetException in %s: %s on line %d\n", __FILE__, c.what(), __LINE__);
				return false;
			} catch (std::exception e) {
				fprintf(stderr, "Unknown Exception in %s: %s on line %d\n", __FILE__, e.what(), __LINE__);
				return false;
			}
		}

		if (!use_bit) {
			//If there are no forward connections from node i, mark with -1
			if (static_cast<uint64_t>(edges.future_edge_row_start[i]) == future_idx)
				edges.future_edge_row_start[i] = -1;
		}
	}

	if (!use_bit) {
		edges.future_edge_row_start[N_tar-1] = -1;

		//Identify past connections
		edges.past_edge_row_start[0] = -1;
		for (i = 1; i < N_tar; i++) {
			edges.past_edge_row_start[i] = past_idx;
			for (j = 0; j < i; j++) {
				if (edges.future_edge_row_start[j] == -1)
					continue;

				for (k = 0; k < nodes.k_out[j]; k++) {
					if (i == edges.future_edges[edges.future_edge_row_start[j]+k]) {
						edges.past_edges[past_idx++] = j;
					}
				}
			}

			//If there are no backward connections from node i, mark with -1
			if (static_cast<uint64_t>(edges.past_edge_row_start[i]) == past_idx)
				edges.past_edge_row_start[i] = -1;
		}

		//The quantities future_idx and past_idx should be equal
		#if DEBUG
		assert (future_idx == past_idx);
		#endif
		//printf("\t\tEdges (backward): %" PRId64 "\n", past_idx);
		//fflush(stdout);
	}

	//Identify Resulting Network
	uint64_t kr = 0;
	for (i = 0; i < N_tar; i++) {
		if (nodes.k_in[i] + nodes.k_out[i] > 0) {
			N_res++;
			kr += nodes.k_in[i] + nodes.k_out[i];

			if (nodes.k_in[i] + nodes.k_out[i] > 1)
				N_deg2++;
		} 
	}

	#if DEBUG
	assert (N_res >= 0);
	assert (N_deg2 >= 0);
	#endif

	if (N_res > 0)
		k_res = static_cast<long double>(kr) / N_res;

	//Debugging options used to visually inspect the adjacency lists and the adjacency pointer lists
	//compareAdjacencyLists(nodes, edges);
	//compareAdjacencyListIndices(nodes, edges);
	//if(!compareCoreEdgeExists(nodes.k_out, edges.future_edges, edges.future_edge_row_start, adj, N_tar, core_edge_fraction))
	//	return false;

	//Print Results
	/*if (!printDegrees(nodes, N_tar, "in-degrees_CPU.cset.dbg.dat", "out-degrees_CPU.cset.dbg.dat")) return false;
	if (!printAdjMatrix(adj, N_tar, "adj_matrix_CPU.cset.dbg.dat", 1, 0)) return false;
	if (!printEdgeLists(edges, past_idx, "past-edges_CPU.cset.dbg.dat", "future-edges_CPU.cset.dbg.dat")) return false;
	if (!printEdgeListPointers(edges, N_tar, "past-edge-pointers_CPU.cset.dbg.dat", "future-edge-pointers_CPU.cset.dbg.dat")) return false;
	printf_red();
	printf("Check files now.\n");
	printf_std();
	fflush(stdout);
	printChk();*/

	stopwatchStop(&sLinkNodes);

	if (!bench) {
		printf("\tCausets Successfully Connected.\n");
		printf_cyan();
		printf("\t\tUndirected Links:         %" PRIu64 "\n", future_idx);
		printf("\t\tResulting Network Size:   %d\n", N_res);
		printf("\t\tResulting Average Degree: %f\n", k_res);
		printf("\t\t    Incl. Isolated Nodes: %f\n", k_res * ((float)N_res / N_tar));
		printf_red();
		printf("\t\tResulting Error in <k>:   %f\n", fabs(k_tar - k_res) / k_tar);
		printf_std();
		fflush(stdout);
	}

	if (verbose) {
		printf("\t\tExecution Time: %5.6f sec\n", sLinkNodes.elapsedTime);
		fflush(stdout);
	}

	return true;
}
