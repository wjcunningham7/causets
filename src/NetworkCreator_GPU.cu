#include "hip/hip_runtime.h"
#include "NetworkCreator_GPU.h"

/////////////////////////////
//(C) Will Cunningham 2014 //
//         DK Lab          //
// Northeastern University //
/////////////////////////////

__global__ void GenerateAdjacencyLists_v2(float *w0, float *x0, float *y0, float *z0, float *w1, float *x1, float *y1, float *z1, int *k_in, int *k_out, bool *edges, size_t size0, size_t size1, bool diag, bool compact)
{
	__shared__ float shr_w1[THREAD_SIZE];
	__shared__ float shr_x1[THREAD_SIZE];
	__shared__ float shr_y1[THREAD_SIZE];
	__shared__ float shr_z1[THREAD_SIZE];
	__shared__ int n[BLOCK_SIZE][THREAD_SIZE];

	float4 n0, n1;

	unsigned int tid = threadIdx.y;
	//Index 'i' marks the row and 'j' marks the column
	unsigned int i = blockDim.y * blockIdx.y + threadIdx.y;
	unsigned int j = blockIdx.x;
	unsigned int k;

	//Each thread compares 1 node in 'nodes0' to 'THREAD_SIZE' nodes in 'nodes1'
	if (!tid) {
		for (k = 0; k < THREAD_SIZE; k++) {
			if (j * THREAD_SIZE + k < size1) {
				shr_w1[k] = w1[j*THREAD_SIZE+k];
				shr_x1[k] = x1[j*THREAD_SIZE+k];
				shr_y1[k] = y1[j*THREAD_SIZE+k];
				shr_z1[k] = z1[j*THREAD_SIZE+k];
			}
		}
	}
	__syncthreads();

	float dt[THREAD_SIZE];
	float dx[THREAD_SIZE];

	for (k = 0; k < THREAD_SIZE; k++) {
		if ((!diag || i < j * THREAD_SIZE + k) && (i < size0 && j * THREAD_SIZE + k < size1)) {
			//Identify nodes to compare
			n0.w = w0[i];
			n0.x = x0[i];
			n0.y = y0[i];
			n0.z = z0[i];

			n1.w = shr_w1[k];
			n1.x = shr_x1[k];
			n1.y = shr_y1[k];
			n1.z = shr_z1[k];

			//Identify spacetime interval
			dt[k] = n1.w - n0.w;

			if (compact) {
				if (DIST_V2)
					dx[k] = acosf(sphProduct_GPU_v2(n0, n1));
				else
					dx[k] = acosf(sphProduct_GPU_v1(n0, n1));
			} else {
				if (DIST_V2)
					dx[k] = sqrtf(flatProduct_GPU_v2(n0, n1));
				else
					dx[k] = sqrtf(flatProduct_GPU_v1(n0, n1));
			}
		}
	}

	bool edge[THREAD_SIZE];
	int out = 0;
	for (k = 0; k < THREAD_SIZE; k++) {
		//Mark if edge is present (register memory)
		edge[k] = (!diag || i < j * THREAD_SIZE + k) && (i < size0 && j * THREAD_SIZE + k < size1) && dx[k] < dt[k];
		//Copy to shared memory to prepare for reduction
		n[tid][k] = (int)edge[k];
		//Identify number of out-degrees found by a single thread
		out += (int)edge[k];
	}
	__syncthreads();

	//Reduction algorithm (used to optimize atomic operations below)
	int stride;
	for (stride = 1; stride < BLOCK_SIZE; stride <<= 1) {
		if (!(tid % (stride << 1)))
			for (k = 0; k < THREAD_SIZE; k++)
				n[tid][k] += n[tid+stride][k];
		__syncthreads();
	}

	//Global Memory Operations

	//Write edges to global memory
	for (k = 0; k < THREAD_SIZE; k++)
		if ((!diag || i < j * THREAD_SIZE + k) && (i < size0 && j * THREAD_SIZE + k < size1))
			edges[(i*THREAD_SIZE*gridDim.x)+(j*THREAD_SIZE)+k] = edge[k];

	//Write out-degrees
	atomicAdd(&k_out[i], out);

	//Wrtie in-degrees
	if (!tid)
		for (k = 0; k < THREAD_SIZE; k++)
			if ((!diag || i < j * THREAD_SIZE + k) && (i < size0 && j * THREAD_SIZE + k < size1))
				atomicAdd(&k_in[j*THREAD_SIZE+k], n[0][k]);
}

__global__ void DecodeFutureEdges(uint64_t *edges, int *future_edges, int elements, int offset)
{
	unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;

	if (idx < elements) {
		//Decode Future Edges
		uint64_t key = edges[idx + offset];
		unsigned int i = key >> 32;
		unsigned int j = key & 0x00000000FFFFFFFF;

		//Write Future Edges
		future_edges[idx] = j;

		//Encode Past Edges
		edges[idx+offset] = ((uint64_t)j) << 32 | ((uint64_t)i);
	}
}

__global__ void DecodePastEdges(uint64_t *edges, int *past_edges, int elements, int offset)
{
	unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;

	if (idx < elements) {
		//Decode Past Edges
		uint64_t key = edges[idx + offset];

		//Write Past Edges
		past_edges[idx] = key & 0x00000000FFFFFFFF;
	}
}

__global__ void ResultingProps(int *k_in, int *k_out, int *N_res, int *N_deg2, int elements)
{
	unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;

	if (idx < elements) {
		int k = k_in[idx] + k_out[idx];
		if (k <= 1) {
			atomicAdd(N_deg2, 1);
			if (!k)
				atomicAdd(N_res, 1);
		}
	}
}

bool linkNodesGPU_v2(Node &nodes, const Edge &edges, std::vector<bool> &core_edge_exists, const int &N_tar, const float &k_tar, int &N_res, float &k_res, int &N_deg2, const float &core_edge_fraction, const float &edge_buffer, const int &group_size, CaResources * const ca, Stopwatch &sLinkNodesGPU, const hipCtx_t &ctx, const bool &decode_cpu, const bool &use_bit, const bool &compact, const bool &verbose, const bool &bench)
{
	#if DEBUG
	assert (nodes.crd->getDim() == 4);
	assert (!nodes.crd->isNull());
	assert (nodes.crd->w() != NULL);
	assert (nodes.crd->x() != NULL);
	assert (nodes.crd->y() != NULL);
	assert (nodes.crd->z() != NULL);
	if (!use_bit) {
		assert (edges.past_edges != NULL);
		assert (edges.future_edges != NULL);
		assert (edges.past_edge_row_start != NULL);
		assert (edges.future_edge_row_start != NULL);
	} else
		assert (core_edge_fraction == 1.0f);
	assert (ca != NULL);
	assert (N_tar > 0);
	assert (k_tar > 0.0f);
	assert (core_edge_fraction >= 0.0f && core_edge_fraction <= 1.0f);
	assert (edge_buffer >= 0.0f && edge_buffer <= 1.0f);
	#endif

	Stopwatch sGenAdjList = Stopwatch();
	Stopwatch sDecodeLists = Stopwatch();
	Stopwatch sScanLists = Stopwatch();
	Stopwatch sProps = Stopwatch();

	hipDeviceptr_t d_k_in, d_k_out;
	uint64_t *h_edges;
	int *g_idx;

	size_t d_edges_size = use_bit ? 1 : pow(2.0, ceil(log2(N_tar * k_tar * (1.0 + edge_buffer) / 2)));

	stopwatchStart(&sLinkNodesGPU);

	//Allocate Overhead on Host
	try {
		h_edges = (uint64_t*)malloc(sizeof(uint64_t) * d_edges_size);
		if (h_edges == NULL)
			throw std::bad_alloc();
		memset(h_edges, 0, sizeof(uint64_t) * d_edges_size);
		ca->hostMemUsed += sizeof(uint64_t) * d_edges_size;

		g_idx = (int*)malloc(sizeof(int));
		if (g_idx == NULL)
			throw std::bad_alloc();
		memset(g_idx, 0, sizeof(int));
		ca->hostMemUsed += sizeof(int);
	} catch (std::bad_alloc) {
		fprintf(stderr, "Memory allocation failure in %s on line %d!\n", __FILE__, __LINE__);
		return false;
	}

	stopwatchStart(&sGenAdjList);
	#if GEN_ADJ_LISTS_GPU_V2
	if (!generateLists_v2(nodes, h_edges, core_edge_exists, g_idx, N_tar, core_edge_fraction, d_edges_size, group_size, ca, ctx, use_bit, compact, verbose))
		return false;
	#else
	if (!generateLists_v1(nodes, h_edges, core_edge_exists, g_idx, N_tar, core_edge_fraction, d_edges_size, group_size, ca, compact, verbose))
		return false;
	#endif
	stopwatchStop(&sGenAdjList);

	if (!use_bit) {
		try {
			if (*g_idx + 1 >= static_cast<int>(N_tar * k_tar * (1.0 + edge_buffer) / 2))
				throw CausetException("Not enough memory in edge adjacency list.  Increase edge buffer or decrease network size.\n");
		} catch (CausetException c) {
			fprintf(stderr, "CausetException in %s: %s on line %d\n", __FILE__, c.what(), __LINE__);
			return false;
		} catch (std::exception e) {
			fprintf(stderr, "Unknown Exception in %s: %s on line %d\n", __FILE__, e.what(), __LINE__);
			return false;
		}

		/*if (!printDegrees(nodes, N_tar, "in-degrees_GPU_v2.cset.dbg.dat", "out-degrees_GPU_v2.cset.dbg.dat")) return false;
		printf_red();
		printf("Check files now.\n");
		printf_std();
		fflush(stdout);
		printChk();*/

		//Decode Adjacency Lists
		stopwatchStart(&sDecodeLists);
		if (decode_cpu) {
			if (!decodeListsCPU(edges, h_edges, g_idx))
				return false;
		} else {
			#if DECODE_LISTS_GPU_V2
			if (!decodeLists_v2(edges, h_edges, g_idx, d_edges_size, group_size, ca, verbose))
				return false;
			#else
			if (!decodeLists_v1(edges, h_edges, g_idx, d_edges_size, ca, verbose))
				return false;
			#endif
		}
		stopwatchStop(&sDecodeLists);
	}

	//Free Host Memory
	free(h_edges);
	h_edges = NULL;
	ca->hostMemUsed -= sizeof(uint64_t) * d_edges_size;
	
	//Allocate Device Memory
	checkCudaErrors(hipMalloc(&d_k_in, sizeof(int) * N_tar));
	ca->devMemUsed += sizeof(int) * N_tar;

	checkCudaErrors(hipMalloc(&d_k_out, sizeof(int) * N_tar));
	ca->devMemUsed += sizeof(int) * N_tar;

	//Copy Memory from Host to Device
	checkCudaErrors(hipMemcpyHtoD(d_k_in, nodes.k_in, sizeof(int) * N_tar));
	checkCudaErrors(hipMemcpyHtoD(d_k_out, nodes.k_out, sizeof(int) * N_tar));

	//Identify Resulting Network Properties
	stopwatchStart(&sProps);
	if (!identifyListProperties(nodes, d_k_in, d_k_out, g_idx, N_tar, N_res, N_deg2, k_res, ca, verbose))
		return false;
	stopwatchStop(&sProps);	

	if (!use_bit) {
		//Prefix Scan of Degrees
		stopwatchStart(&sScanLists);
		scan(nodes.k_in, nodes.k_out, edges.past_edge_row_start, edges.future_edge_row_start, N_tar);
		stopwatchStop(&sScanLists);
	}

	//Free Device Memory
	hipFree(d_k_in);
	d_k_in = 0;
	ca->devMemUsed -= sizeof(int) * N_tar;

	hipFree(d_k_out);
	d_k_out = 0;
	ca->devMemUsed -= sizeof(int) * N_tar;

	stopwatchStop(&sLinkNodesGPU);

	if (!bench) {
		printf("\tCausets Successfully Connected.\n");
		printf_cyan();
		printf("\t\tUndirected Links:         %d\n", *g_idx);
		printf("\t\tResulting Network Size:   %d\n", N_res);
		printf("\t\tResulting Average Degree: %f\n", k_res);
		printf("\t\t    Incl. Isolated Nodes: %f\n", (k_res * N_res) / N_tar);
		printf_red();
		printf("\t\tResulting Error in <k>:   %f\n", fabs(k_tar - k_res) / k_tar);
		printf_std();
		fflush(stdout);
	}

	//if(!compareCoreEdgeExists(nodes.k_out, edges.future_edges, edges.future_edge_row_start, core_edge_exists, N_tar, core_edge_fraction))
	//	return false;

	//Print Results
	/*if (!printDegrees(nodes, N_tar, "in-degrees_GPU_v2.cset.dbg.dat", "out-degrees_GPU_v2.cset.dbg.dat")) return false;
	if (!printEdgeLists(edges, *g_idx, "past-edges_GPU_v2.cset.dbg.dat", "future-edges_GPU_v2.cset.dbg.dat")) return false;
	if (!printEdgeListPointers(edges, N_tar, "past-edge-pointers_GPU_v2.cset.dbg.dat", "future-edge-pointers_GPU_v2.cset.dbg.dat")) return false;
	printf_red();
	printf("Check files now.\n");
	printf_std();
	fflush(stdout);
	printChk();*/

	//Free Host Memory
	free(g_idx);
	g_idx = NULL;
	ca->hostMemUsed -= sizeof(int);

	if (verbose) {
		printf("\t\tExecution Time: %5.6f sec\n", sLinkNodesGPU.elapsedTime);
		printf("\t\t\tAdjacency List Function Time: %5.6f sec\n", sGenAdjList.elapsedTime);
		printf("\t\t\tDecode Lists Function Time: %5.6f sec\n", sDecodeLists.elapsedTime);
		printf("\t\t\tScan Lists Function Time: %5.6f sec\n", sScanLists.elapsedTime);
		fflush(stdout);
	}

	return true;
}

//Uses multiple buffers and asynchronous operations
bool generateLists_v2(Node &nodes, uint64_t * const &edges, std::vector<bool> &core_edge_exists, int * const &g_idx, const int &N_tar, const float &core_edge_fraction, const size_t &d_edges_size, const int &group_size, CaResources * const ca, const hipCtx_t &ctx, const bool &use_bit, const bool &compact, const bool &verbose)
{
	#if DEBUG
	assert (nodes.crd->getDim() == 4);
	assert (!nodes.crd->isNull());
	assert (nodes.crd->w() != NULL);
	assert (nodes.crd->x() != NULL);
	assert (nodes.crd->y() != NULL);
	assert (nodes.crd->z() != NULL);
	assert (nodes.k_in != NULL);
	assert (nodes.k_out != NULL);
	assert (g_idx != NULL);
	assert (ca != NULL);
	assert (N_tar > 0);
	assert (core_edge_fraction >= 0.0f && core_edge_fraction <= 1.0f);
	if (use_bit)
		assert (core_edge_fraction == 1.0f);
	#endif

	//CUDA Streams
	hipStream_t stream[NBUFFERS];

	//Arrays of Buffers
	int *h_k_in[NBUFFERS];
	int *h_k_out[NBUFFERS];
	bool *h_edges[NBUFFERS];

	hipDeviceptr_t d_w0[NBUFFERS];
	hipDeviceptr_t d_x0[NBUFFERS];
	hipDeviceptr_t d_y0[NBUFFERS];
	hipDeviceptr_t d_z0[NBUFFERS];

	hipDeviceptr_t d_w1[NBUFFERS];
	hipDeviceptr_t d_x1[NBUFFERS];
	hipDeviceptr_t d_y1[NBUFFERS];
	hipDeviceptr_t d_z1[NBUFFERS];

	hipDeviceptr_t d_k_in[NBUFFERS];
	hipDeviceptr_t d_k_out[NBUFFERS];
	hipDeviceptr_t d_edges[NBUFFERS];

	unsigned int core_limit = static_cast<unsigned int>(core_edge_fraction * N_tar);
	unsigned int i, j, m;
	bool diag;

	//Thread blocks are grouped into "mega" blocks
	size_t mblock_size = static_cast<unsigned int>(ceil(static_cast<float>(N_tar) / (BLOCK_SIZE * group_size)));
	size_t mthread_size = mblock_size * BLOCK_SIZE;
	size_t m_edges_size = mthread_size * mthread_size;

	//Create Streams
	for (i = 0; i < NBUFFERS; i++)
		checkCudaErrors(hipStreamCreateWithFlags(&stream[i], hipStreamNonBlocking));

	//Allocate Memory
	for (i = 0; i < NBUFFERS; i++) {
		checkCudaErrors(hipHostAlloc((void**)&h_k_in[i], sizeof(int) * mthread_size, hipHostMallocPortable));
		ca->hostMemUsed += sizeof(int) * mthread_size;

		checkCudaErrors(hipHostAlloc((void**)&h_k_out[i], sizeof(int) * mthread_size, hipHostMallocPortable));
		ca->hostMemUsed += sizeof(int) * mthread_size;

		checkCudaErrors(hipHostAlloc((void**)&h_edges[i], sizeof(bool) * m_edges_size, hipHostMallocPortable));
		ca->hostMemUsed += sizeof(bool) * m_edges_size;

		checkCudaErrors(hipMalloc(&d_w0[i], sizeof(float) * mthread_size));
		checkCudaErrors(hipMalloc(&d_x0[i], sizeof(float) * mthread_size));
		checkCudaErrors(hipMalloc(&d_y0[i], sizeof(float) * mthread_size));
		checkCudaErrors(hipMalloc(&d_z0[i], sizeof(float) * mthread_size));
		ca->devMemUsed += sizeof(float) * mthread_size * 4;

		checkCudaErrors(hipMalloc(&d_w1[i], sizeof(float) * mthread_size));
		checkCudaErrors(hipMalloc(&d_x1[i], sizeof(float) * mthread_size));
		checkCudaErrors(hipMalloc(&d_y1[i], sizeof(float) * mthread_size));
		checkCudaErrors(hipMalloc(&d_z1[i], sizeof(float) * mthread_size));
		ca->devMemUsed += sizeof(float) * mthread_size * 4;

		checkCudaErrors(hipMalloc(&d_k_in[i], sizeof(int) * mthread_size));
		ca->devMemUsed += sizeof(int) * mthread_size;

		checkCudaErrors(hipMalloc(&d_k_out[i], sizeof(int) * mthread_size));
		ca->devMemUsed += sizeof(int) * mthread_size;

		checkCudaErrors(hipMalloc(&d_edges[i], sizeof(bool) * m_edges_size));
		ca->devMemUsed += sizeof(bool) * m_edges_size;
	}

	memoryCheckpoint(ca->hostMemUsed, ca->maxHostMemUsed, ca->devMemUsed, ca->maxDevMemUsed);
	if (verbose)
		printMemUsed("for Generating Lists on GPU", ca->hostMemUsed, ca->devMemUsed, 0);

	//CUDA Grid Specifications
	unsigned int gridx = static_cast<unsigned int>(ceil(static_cast<float>(mthread_size) / THREAD_SIZE));
	unsigned int gridy = mblock_size;
	dim3 threads_per_block(1, BLOCK_SIZE, 1);
	dim3 blocks_per_grid(gridx, gridy, 1);

	size_t final_size = N_tar - mthread_size * (group_size - 1);
	size_t size0, size1;

	//Index 'i' marks the row and 'j' marks the column
	for (i = 0; i < group_size; i++) {
		for (j = 0; j < group_size / NBUFFERS; j++) {
			for (m = 0; m < NBUFFERS; m++) {
				if (i > j * NBUFFERS + m)
					continue;

				diag = (i == j * NBUFFERS + m);

				size0 = (i < group_size - 1) ? mthread_size : final_size;
				size1 = (j * NBUFFERS + m < group_size - 1) ? mthread_size : final_size;

				//Clear Device Buffers
				checkCudaErrors(hipMemsetD32Async(d_k_in[m], 0, mthread_size, stream[m]));
				checkCudaErrors(hipMemsetD32Async(d_k_out[m], 0, mthread_size, stream[m]));
				checkCudaErrors(hipMemsetD8Async(d_edges[m], 0, m_edges_size, stream[m]));					
			
				//Transfer Nodes to Device Buffers
				checkCudaErrors(hipMemcpyHtoDAsync(d_w0[m], nodes.crd->w() + i * mthread_size, sizeof(float) * size0, stream[m]));
				checkCudaErrors(hipMemcpyHtoDAsync(d_x0[m], nodes.crd->x() + i * mthread_size, sizeof(float) * size0, stream[m]));
				checkCudaErrors(hipMemcpyHtoDAsync(d_y0[m], nodes.crd->y() + i * mthread_size, sizeof(float) * size0, stream[m]));
				checkCudaErrors(hipMemcpyHtoDAsync(d_z0[m], nodes.crd->z() + i * mthread_size, sizeof(float) * size0, stream[m]));

				checkCudaErrors(hipMemcpyHtoDAsync(d_w1[m], nodes.crd->w() + (j * NBUFFERS + m) * mthread_size, sizeof(float) * size1, stream[m]));
				checkCudaErrors(hipMemcpyHtoDAsync(d_x1[m], nodes.crd->x() + (j * NBUFFERS + m) * mthread_size, sizeof(float) * size1, stream[m]));
				checkCudaErrors(hipMemcpyHtoDAsync(d_y1[m], nodes.crd->y() + (j * NBUFFERS + m) * mthread_size, sizeof(float) * size1, stream[m]));
				checkCudaErrors(hipMemcpyHtoDAsync(d_z1[m], nodes.crd->z() + (j * NBUFFERS + m) * mthread_size, sizeof(float) * size1, stream[m]));

				//Execute Kernel
				GenerateAdjacencyLists_v2<<<blocks_per_grid, threads_per_block, 0, stream[m]>>>((float*)d_w0[m], (float*)d_x0[m], (float*)d_y0[m], (float*)d_z0[m], (float*)d_w1[m], (float*)d_x1[m], (float*)d_y1[m], (float*)d_z1[m], (int*)d_k_in[m], (int*)d_k_out[m], (bool*)d_edges[m], size0, size1, diag, compact);
				getLastCudaError("Kernel 'NetworkCreator_GPU.GenerateAdjacencyLists_v2' Failed to Execute!\n");

				//Copy Memory to Host Buffers
				checkCudaErrors(hipMemcpyDtoHAsync(h_k_in[m], d_k_in[m], sizeof(int) * size1, stream[m]));
				checkCudaErrors(hipMemcpyDtoHAsync(h_k_out[m], d_k_out[m], sizeof(int) * size0, stream[m]));
				checkCudaErrors(hipMemcpyDtoHAsync(h_edges[m], d_edges[m], sizeof(bool) * m_edges_size, stream[m]));

				//Synchronize
				checkCudaErrors(hipStreamSynchronize(stream[m]));

				//Read Data from Buffers
				readDegrees(nodes.k_in, h_k_in[m], (j * NBUFFERS + m) * mthread_size, size1);
				readDegrees(nodes.k_out, h_k_out[m], i * mthread_size, size0);
				readEdges(edges, h_edges[m], core_edge_exists, g_idx, core_limit, d_edges_size, mthread_size, size0, size1, i, j*NBUFFERS+m, use_bit);
			}				
		}
	}

	//Free Buffers
	for (i = 0; i < NBUFFERS; i++) {
		hipHostFree(h_k_in[i]);
		h_k_in[i] = NULL;
		ca->hostMemUsed -= sizeof(int) * mthread_size;

		hipHostFree(h_k_out[i]);
		h_k_out[i] = NULL;
		ca->hostMemUsed -= sizeof(int) * mthread_size;

		hipHostFree(h_edges[i]);
		h_edges[i] = NULL;
		ca->hostMemUsed -= sizeof(bool) * m_edges_size;

		hipFree(d_w0[i]);
		d_w0[i] = 0;

		hipFree(d_x0[i]);
		d_x0[i] = 0;

		hipFree(d_y0[i]);
		d_y0[i] = 0;

		hipFree(d_z0[i]);
		d_z0[i] = 0;

		ca->devMemUsed -= sizeof(float) * mthread_size * 4;

		hipFree(d_w1[i]);
		d_w1[i] = 0;

		hipFree(d_x1[i]);
		d_x1[i] = 0;

		hipFree(d_y1[i]);
		d_y1[i] = 0;

		hipFree(d_z1[i]);
		d_z1[i] = 0;

		ca->devMemUsed -= sizeof(float) * mthread_size * 4;

		hipFree(d_k_in[i]);
		d_k_in[i] = 0;
		ca->devMemUsed -= sizeof(int) * mthread_size;

		hipFree(d_k_out[i]);
		d_k_out[i] = 0;
		ca->devMemUsed -= sizeof(int) * mthread_size;

		hipFree(d_edges[i]);
		d_edges[i] = 0;
		ca->devMemUsed -= sizeof(bool) * m_edges_size;
	}

	//Destroy Streams
	for (i = 0; i < NBUFFERS; i++)
		checkCudaErrors(hipStreamDestroy(stream[i]));

	//Final Synchronization
	checkCudaErrors(hipCtxSynchronize());

	return true;
}

//Decode past and future edge lists using Bitonic Sort
bool decodeLists_v2(const Edge &edges, const uint64_t * const h_edges, const int * const g_idx, const size_t &d_edges_size, const int &group_size, CaResources * const ca, const bool &verbose)
{
	#if DEBUG
	assert (edges.past_edges != NULL);
	assert (edges.future_edges != NULL);
	assert (h_edges != NULL);
	assert (g_idx != NULL);
	assert (ca != NULL);
	assert (*g_idx > 0);
	assert (d_edges_size > 0);
	#endif

	hipDeviceptr_t d_edges;
	hipDeviceptr_t d_past_edges, d_future_edges;
	int cpy_size;
	int i, j, k;

	size_t g_mblock_size = static_cast<unsigned int>(ceil(static_cast<float>(*g_idx) / (BLOCK_SIZE * group_size)));
	size_t g_mthread_size = g_mblock_size * BLOCK_SIZE;

	//DEBUG
	/*printf_red();
	printf("G_IDX:          %d\n", *g_idx);
	printf("BLOCK_SIZE:     %d\n", BLOCK_SIZE);
	printf("GROUP_SIZE:     %d\n", group_size);
	printf("G_MBLOCK_SIZE:  %zu\n", g_mblock_size);
	printf("G_MTHREAD_SIZE: %zu\n", g_mthread_size);
	printf_std();
	fflush(stdout);*/

	//Allocate Global Device Memory
	checkCudaErrors(hipMalloc(&d_edges, sizeof(uint64_t) * d_edges_size));
	ca->devMemUsed += sizeof(uint64_t) * d_edges_size;

	//Copy Memory from Host to Device
	checkCudaErrors(hipMemcpyHtoD(d_edges, h_edges, sizeof(uint64_t) * d_edges_size));

	//Synchronize
	checkCudaErrors(hipCtxSynchronize());

	//CUDA Grid Specifications
	unsigned int gridx_bitonic = d_edges_size / BLOCK_SIZE;
	dim3 threads_per_block(BLOCK_SIZE, 1, 1);
	dim3 blocks_per_grid_bitonic(gridx_bitonic, 1, 1);

	//Execute Kernel
	for (k = 2; k <= d_edges_size; k <<= 1) {
		for (j = k >> 1; j > 0; j >>= 1) {
			BitonicSort<<<blocks_per_grid_bitonic, threads_per_block>>>((uint64_t*)d_edges, j, k);
			getLastCudaError("Kernel 'Subroutines_GPU.BitonicSort' Failed to Execute!\n");
			checkCudaErrors(hipCtxSynchronize());
		}
	}

	//Allocate Device Memory
	checkCudaErrors(hipMalloc(&d_future_edges, sizeof(int) * g_mthread_size));
	ca->devMemUsed += sizeof(int) * g_mthread_size;

	memoryCheckpoint(ca->hostMemUsed, ca->maxHostMemUsed, ca->devMemUsed, ca->maxDevMemUsed);
	if (verbose)
		printMemUsed("for Bitonic Sorting", ca->hostMemUsed, ca->devMemUsed, 0);

	//CUDA Grid Specifications
	unsigned int gridx_decode = static_cast<unsigned int>(ceil(static_cast<float>(g_mthread_size) / BLOCK_SIZE));
	dim3 blocks_per_grid_decode(gridx_decode, 1, 1);

	for (i = 0; i < group_size; i++) {
		//Clear Device Buffers
		checkCudaErrors(hipMemsetD32(d_future_edges, 0, g_mthread_size));

		//Execute Kernel
		DecodeFutureEdges<<<blocks_per_grid_decode, threads_per_block>>>((uint64_t*)d_edges, (int*)d_future_edges, *g_idx - i * g_mthread_size, d_edges_size - (*g_idx - i * g_mthread_size));
		getLastCudaError("Kernel 'NetworkCreator_GPU.DecodeFutureEdges' Failed to Execute!\n");

		//Synchronize
		checkCudaErrors(hipCtxSynchronize());

		//Copy Memory from Device to Host
		if (*g_idx > g_mthread_size)
			cpy_size = *g_idx - static_cast<int>(i * g_mthread_size) >= 0 ? g_mthread_size : static_cast<int>(i * g_mthread_size) - *g_idx;
		else
			cpy_size = *g_idx;
		checkCudaErrors(hipMemcpyDtoH(edges.future_edges + i * g_mthread_size, d_future_edges, sizeof(int) * cpy_size));

		if (cpy_size < g_mthread_size)
			break;
	}

	//Free Device Memory
	hipFree(d_future_edges);
	d_future_edges = 0;
	ca->devMemUsed -= sizeof(int) * g_mthread_size;

	//Resort Edges with New Encoding
	for (k = 2; k <= d_edges_size; k <<= 1) {
		for (j = k >> 1; j > 0; j >>= 1) {
			BitonicSort<<<blocks_per_grid_bitonic, threads_per_block>>>((uint64_t*)d_edges, j, k);
			getLastCudaError("Kernel 'Subroutines_GPU.BitonicSort' Failed to Execute!\n");
			checkCudaErrors(hipCtxSynchronize());
		}
	}

	//Allocate Device Memory
	checkCudaErrors(hipMalloc(&d_past_edges, sizeof(int) * g_mthread_size));
	ca->devMemUsed += sizeof(int) * g_mthread_size;

	for (i = 0; i < group_size; i++) {
		//Clear Device Buffers
		checkCudaErrors(hipMemsetD32(d_past_edges, 0, g_mthread_size));

		//Execute Kernel
		DecodePastEdges<<<blocks_per_grid_decode, threads_per_block>>>((uint64_t*)d_edges, (int*)d_past_edges, *g_idx - i * g_mthread_size, d_edges_size - (*g_idx - i * g_mthread_size));
		getLastCudaError("Kernel 'NetworkCreator_GPU.DecodePastEdges' Failed to Execute!\n");

		//Synchronize
		checkCudaErrors(hipCtxSynchronize());

		//Copy Memory from Device to Host
		if (*g_idx > g_mthread_size)
			cpy_size = *g_idx - static_cast<int>(i * g_mthread_size) >= 0 ? g_mthread_size : static_cast<int>(i * g_mthread_size) - *g_idx;
		else
			cpy_size = *g_idx;
		checkCudaErrors(hipMemcpyDtoH(edges.past_edges + i * g_mthread_size, d_past_edges, sizeof(int) * cpy_size));

		if (cpy_size < g_mthread_size)
			break;
	}

	//Free Device Memory
	hipFree(d_past_edges);
	d_past_edges = 0;
	ca->devMemUsed -= sizeof(int) * g_mthread_size;

	hipFree(d_edges);
	d_edges = 0;
	ca->devMemUsed -= sizeof(uint64_t) * d_edges_size;

	return true;
}

bool decodeListsCPU(const Edge &edges, uint64_t *h_edges, const int * const g_idx)
{
	#if DEBUG
	assert (edges.past_edges != NULL);
	assert (edges.future_edges != NULL);
	assert (h_edges != NULL);
	assert (g_idx != NULL);
	assert (*g_idx > 0);
	#endif

	uint64_t key;
	unsigned int idx0, idx1;
	int i;

	quicksort(h_edges, 0, *g_idx - 1);

	for (i = 0; i < *g_idx; i++) {
		key = h_edges[i];
		idx0 = key >> 32;
		idx1 = key & 0x00000000FFFFFFFF;
		edges.future_edges[i] = idx1;
		h_edges[i] = ((uint64_t)idx1) << 32 | ((uint64_t)idx0);
	}

	quicksort(h_edges, 0, *g_idx - 1);

	for (i = 0; i < *g_idx; i++) {
		key = h_edges[i];
		idx0 = key >> 32;
		idx1 = key & 0x00000000FFFFFFFF;
		edges.past_edges[i] = idx1;
	}

	return true;
}

//Parallel Prefix Sum of 'k_in' and 'k_out' and Write to Edge Pointers
//This function works, but has been deprecated since it doesn't provide much speedup
bool scanLists(const Edge &edges, const hipDeviceptr_t &d_k_in, const hipDeviceptr_t d_k_out, const int &N_tar, CaResources * const ca, const bool &verbose)
{
	#if DEBUG
	assert (edges.past_edge_row_start != NULL);
	assert (edges.future_edge_row_start != NULL);
	assert (ca != NULL);
	assert (N_tar > 0);
	#endif

	hipDeviceptr_t d_past_edge_row_start, d_future_edge_row_start;
	hipDeviceptr_t d_buf, d_buf_scanned;
	int i;

	//Allocate Device Memory
	checkCudaErrors(hipMalloc(&d_past_edge_row_start, sizeof(int) * N_tar));
	ca->devMemUsed += sizeof(int) * N_tar;

	checkCudaErrors(hipMalloc(&d_future_edge_row_start, sizeof(int) * N_tar));
	ca->devMemUsed += sizeof(int) * N_tar;

	checkCudaErrors(hipMalloc(&d_buf, sizeof(int) * (BLOCK_SIZE << 1)));
	ca->devMemUsed += sizeof(int) * (BLOCK_SIZE << 1);

	checkCudaErrors(hipMalloc(&d_buf_scanned, sizeof(int) * (BLOCK_SIZE << 1)));
	ca->devMemUsed += sizeof(int) * (BLOCK_SIZE << 1);
	
	memoryCheckpoint(ca->hostMemUsed, ca->maxHostMemUsed, ca->devMemUsed, ca->maxDevMemUsed);
	if (verbose)
		printMemUsed("for Parallel Prefix Sum", ca->hostMemUsed, ca->devMemUsed, 0);

	//Initialize Memory on Device
	checkCudaErrors(hipMemsetD32(d_past_edge_row_start, 0, N_tar));
	checkCudaErrors(hipMemsetD32(d_future_edge_row_start, 0, N_tar));

	//CUDA Grid Specifications
	unsigned int gridx_scan = static_cast<unsigned int>(ceil(static_cast<float>(N_tar) / (BLOCK_SIZE << 1)));
	dim3 threads_per_block(BLOCK_SIZE, 1, 1);
	dim3 blocks_per_grid_scan(gridx_scan, 1, 1);

	//Execute Kernels
	Scan<<<blocks_per_grid_scan, threads_per_block>>>((int*)d_k_in, (int*)d_past_edge_row_start, (int*)d_buf, N_tar);
	getLastCudaError("Kernel 'Subroutines_GPU.Scan' Failed to Execute!\n");
	checkCudaErrors(hipCtxSynchronize());

	Scan<<<dim3(1,1,1), threads_per_block>>>((int*)d_buf, (int*)d_buf_scanned, NULL, BLOCK_SIZE << 1);
	getLastCudaError("Kernel 'Subroutines_GPU.Scan' Failed to Execute!\n");
	checkCudaErrors(hipCtxSynchronize());

	PostScan<<<blocks_per_grid_scan, threads_per_block>>>((int*)d_past_edge_row_start, (int*)d_buf_scanned, N_tar);
	getLastCudaError("Kernel 'Subroutines_GPU.PostScan' Failed to Execute!\n");
	checkCudaErrors(hipCtxSynchronize());

	Scan<<<blocks_per_grid_scan, threads_per_block>>>((int*)d_k_out, (int*)d_future_edge_row_start, (int*)d_buf, N_tar);
	getLastCudaError("Kernel 'Subroutines_GPU.Scan' Failed to Execute!\n");
	checkCudaErrors(hipCtxSynchronize());

	Scan<<<dim3(1,1,1), threads_per_block>>>((int*)d_buf, (int*)d_buf_scanned, NULL, BLOCK_SIZE << 1);
	getLastCudaError("Kernel 'Subroutines_GPU.Scan' Failed to Execute!\n");
	checkCudaErrors(hipCtxSynchronize());

	PostScan<<<blocks_per_grid_scan, threads_per_block>>>((int*)d_future_edge_row_start, (int*)d_buf_scanned, N_tar);
	getLastCudaError("Kernel 'Subroutines_GPU.PostScan' Failed to Execute!\n");
	checkCudaErrors(hipCtxSynchronize());

	//Copy Memory from Device to Host
	checkCudaErrors(hipMemcpyDtoH(edges.past_edge_row_start, d_past_edge_row_start, sizeof(int) * N_tar));
	checkCudaErrors(hipMemcpyDtoH(edges.future_edge_row_start, d_future_edge_row_start, sizeof(int) * N_tar));

	//Synchronize
	checkCudaErrors(hipCtxSynchronize());

	//Formatting
	for (i = N_tar - 1; i > 0; i--) {
		edges.past_edge_row_start[i] = edges.past_edge_row_start[i-1];
		edges.future_edge_row_start[i] = edges.future_edge_row_start[i-1];
	}

	edges.past_edge_row_start[0] = -1;
	edges.future_edge_row_start[0] = 0;

	int pv = edges.past_edge_row_start[N_tar-1];
	int fv = edges.future_edge_row_start[N_tar-1];

	for (i = N_tar-2; i >= 0; i--) {
		if (pv == edges.past_edge_row_start[i])
			edges.past_edge_row_start[i] = -1;
		else
			pv = edges.past_edge_row_start[i];

		if (fv == edges.future_edge_row_start[i])
			edges.future_edge_row_start[i] = -1;
		else
			fv = edges.future_edge_row_start[i];
	}

	edges.future_edge_row_start[N_tar-1] = -1;
	
	//Free Device Memory
	hipFree(d_past_edge_row_start);
	d_past_edge_row_start = 0;
	ca->devMemUsed -= sizeof(int) * N_tar;

	hipFree(d_future_edge_row_start);
	d_future_edge_row_start = 0;
	ca->devMemUsed -= sizeof(int) * N_tar;

	hipFree(d_buf);
	d_buf = 0;
	ca->devMemUsed -= sizeof(int) * (BLOCK_SIZE << 1);

	hipFree(d_buf_scanned);
	d_buf_scanned = 0;
	ca->devMemUsed -= sizeof(int) * (BLOCK_SIZE << 1);

	return true;
}

bool identifyListProperties(const Node &nodes, const hipDeviceptr_t &d_k_in, const hipDeviceptr_t &d_k_out, const int *g_idx, const int &N_tar, int &N_res, int &N_deg2, float &k_res, CaResources * const ca, const bool &verbose)
{
	#if DEBUG
	assert (nodes.k_in != NULL);
	assert (nodes.k_out != NULL);
	assert (g_idx != NULL);
	assert (N_tar > 0);
	#endif

	hipDeviceptr_t d_N_res, d_N_deg2;

	//Allocate Device Memory
	checkCudaErrors(hipMalloc(&d_N_res, sizeof(int)));
	ca->devMemUsed += sizeof(int);

	checkCudaErrors(hipMalloc(&d_N_deg2, sizeof(int)));
	ca->devMemUsed += sizeof(int);
	
	memoryCheckpoint(ca->hostMemUsed, ca->maxHostMemUsed, ca->devMemUsed, ca->maxDevMemUsed);
	if (verbose)
		printMemUsed("for Identifying List Properties", ca->hostMemUsed, ca->devMemUsed, 0);
	
	//Initialize Memory on Device
	checkCudaErrors(hipMemsetD32(d_N_res, 0, 1));
	checkCudaErrors(hipMemsetD32(d_N_deg2, 0, 1));

	//CUDA Grid Specifications
	unsigned int gridx_res_prop = static_cast<unsigned int>(ceil(static_cast<float>(N_tar) / BLOCK_SIZE));
	dim3 threads_per_block(BLOCK_SIZE, 1, 1);
	dim3 blocks_per_grid_res_prop(gridx_res_prop, 1, 1);

	//Execute Kernel
	ResultingProps<<<gridx_res_prop, threads_per_block>>>((int*)d_k_in, (int*)d_k_out, (int*)d_N_res, (int*)d_N_deg2, N_tar);
	getLastCudaError("Kernel 'NetworkCreator_GPU.ResultingProps' Failed to Execute!\n");
	checkCudaErrors(hipCtxSynchronize());

	//Copy Memory from Device to Host
	checkCudaErrors(hipMemcpyDtoH(nodes.k_in, d_k_in, sizeof(int) * N_tar));
	checkCudaErrors(hipMemcpyDtoH(nodes.k_out, d_k_out, sizeof(int) * N_tar));
	checkCudaErrors(hipMemcpyDtoH(&N_res, d_N_res, sizeof(int)));
	checkCudaErrors(hipMemcpyDtoH(&N_deg2, d_N_deg2, sizeof(int)));

	//Synchronize
	checkCudaErrors(hipCtxSynchronize());

	N_res = N_tar - N_res;
	N_deg2 = N_tar - N_deg2;
	k_res = static_cast<float>(*g_idx * 2) / N_res;

	#if DEBUG
	assert (N_res >= 0);
	assert (N_deg2 >= 0);
	assert (k_res >= 0.0);
	#endif

	//Free Device Memory
	hipFree(d_N_res);
	d_N_res = 0;
	ca->devMemUsed -= sizeof(int);

	hipFree(d_N_deg2);
	d_N_deg2 = 0;
	ca->devMemUsed -= sizeof(int);

	return true;
}
