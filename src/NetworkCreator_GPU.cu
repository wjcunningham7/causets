#include "hip/hip_runtime.h"
#include "NetworkCreator_GPU.h"

/////////////////////////////
//(C) Will Cunningham 2014 //
// Northeastern University //
// Krioukov Research Group //
/////////////////////////////

/*__global__ void Generate(Node *nodes, int N_tar, long seed)
{
	//int i = blockDim.x * blockIdx.x + threadIdx.x;
	//int j = blockDim.y * blockIdx.y + threadIdx.y;
	//if ((j * width) + i > N_tar)
	//	return;

	//Implement CURAND package here for random number generation
}*/

__global__ void GenerateAdjacencyLists(float4 *nodes, int *past_edges, int *future_edges, int *g_idx, int N_tar, int max)
{
	///////////////////////////////////////
	// Identify Node Pair with Thread ID //
	///////////////////////////////////////

	//Thread ID (unique within each block)
	int tid = blockDim.x * threadIdx.y + threadIdx.x;

	//Global Thread ID (unique among all threads)
	int _i = blockDim.x * blockIdx.x + threadIdx.x;
	int _j = blockDim.y * blockIdx.y + threadIdx.y;
	int gid = _j * blockDim.x * gridDim.x + _i;

	if (gid >= max)
		return;

	//Identify Node Pair
	int i, j;
	int k = vec2MatIdx(N_tar, gid);
	i = k % N_tar;
	j = k / N_tar;
	//__syncthreads(); (Use this for unit testing)

	//Read Coordinates from Global Memory
	float4 node0 = nodes[i];
	float4 node1 = nodes[j];

	//////////////////////////////////
	// Identify Causal Relationship //
	//////////////////////////////////

	//Calculate dt
	float dt = fabsf(node0.w - node1.w);

	//Calculate dx
	float dx = acosf(X1_GPU(node0.y) * X1_GPU(node1.y) +
			 X2_GPU(node0.y, node0.z) * X2_GPU(node1.y, node1.z) +
			 X3_GPU(node0.y, node0.z, node0.x) * X3_GPU(node1.y, node1.z, node1.x) +
			 X4_GPU(node0.y, node0.z, node0.x) * X4_GPU(node1.y, node1.z, node1.x));

	//Calculate Pair Key
	int key = (dx < dt) ? i * N_tar + j : 0;

	/////////////////////////////////
	// Operations in Shared Memory //
	/////////////////////////////////

	__shared__ int l_key[BLOCK_SIZE * BLOCK_SIZE];
	__shared__ int d;
	__shared__ int l_idx;

	//Prefix Sum
	l_key[tid] = (int)(!key);
	__syncthreads();

	int diff = 0;
	for (k = 0; k < tid; k++)
		diff += l_key[i];
	__syncthreads();

	//Reduction
	int lstride;
	for (lstride = 1; lstride < BLOCK_SIZE * BLOCK_SIZE; lstride <<= 1) {
		if (!(tid % (lstride << 1)))
			l_key[tid] += l_key[tid + lstride];
		__syncthreads();
	}

	if (!tid)
		d = l_key[0];
	__syncthreads();

	//Compaction
	if (key)
		l_key[tid - diff] = key;

	////////////////////////////
	// Write to Global Memory //
	////////////////////////////

	if (!tid)
		l_idx = atomicAdd(g_idx, d);
	__syncthreads();

	//Check if Memory is Full
	if (l_idx >= max)
		return;

	if (tid < BLOCK_SIZE * BLOCK_SIZE - d) {
		//Read Keys from Local to Register Memory
		int f_r_key = l_key[tid];
		int p_r_key = (f_r_key % N_tar) * N_tar + (f_r_key / N_tar);

		//Write Keys from Register to Global Memory
		future_edges[l_idx + tid] = f_r_key;
		past_edges[l_idx + tid] = p_r_key;
	}
}

__global__ void DecodeAdjacencyLists(int *past_edges, int *future_edges, int *past_edge_row_start, int *future_edge_row_start)
{
	//
}

__global__ void FindNodeDegrees(int *past_edge_row_start, int *future_edge_row_start, int *k_in, int *k_out)
{
	//
}

/*bool generateNodesGPU(Node * const &nodes, const int &N_tar, const float &k_tar, const int &dim, const Manifold &manifold, const double &a, const double &zeta, const double &tau0, const double &alpha, long &seed, Stopwatch &sGenerateNodesGPU, const bool &universe, const bool &verbose, const bool &bench)
{
	//CURAND
	hiprandGenerator_t prng;
	
	try {
		if (HIPRAND_STATUS_SUCCESS != hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_DEFAULT))
			throw CausetException("Failed to create hiprand generator.\n");
		if (HIPRAND_STATUS_SUCCESS != hiprandSetPseudoRandomGeneratorSeed(prng, (int)network->network_properties.seed))
			throw CausetException("Failed to set hiprand seed.\n");

		//Need to redesign Node for GPU so memory for points is contiguous
		//Lots of thought should go into this...
		//if (HIPRAND_STATUS_SUCCESS != hiprandGenerateUniform(prng, (float*)d_points, network->network_properties.N_tar))
		//	throw CausetException("Failed to generate hiprand uniform number distribution.\n");

		if (HIPRAND_STATUS_SUCCESS != hiprandDestroyGenerator(prng))
			throw CausetException("Failed to destroy hiprand generator.\n");
	} catch (CausetException c) {
		fprintf(stderr, "CausetException in %s: %s on line %d\n", __FILE__, c.what(), __LINE__);
		return false;
	}

	//Invoke Kernel
	Generate<<<network->network_properties.network_exec.blocks_per_grid, network->network_properties.network_exec.threads_per_block>>>((Node*)network->d_nodes, network->network_properties.N_tar, network->network_properties.seed);
	getLastCudaError("Kernel 'Generate' Failed to Execute!");
	checkCudaErrors(hipCtxSynchronize());

	//Copy Values to Host
	checkCudaErrors(hipMemcpyDtoH(network->nodes, network->d_nodes, sizeof(Node) * network->network_properties.N_tar));

	return true;
}*/

bool linkNodesGPU(Node * const &nodes, hipDeviceptr_t &d_nodes, int * const &past_edges, hipDeviceptr_t &d_past_edges, int * const &future_edges, hipDeviceptr_t &d_future_edges, int * const &past_edge_row_start, hipDeviceptr_t &d_past_edge_row_start, int * const &future_edge_row_start, hipDeviceptr_t &d_future_edge_row_start, bool * const &core_edge_exists, hipDeviceptr_t &d_k_in, hipDeviceptr_t &d_k_out, const int &N_tar, const float &k_tar, int &N_res, float &k_res, int &N_deg2, const int &dim, const Manifold &manifold, const double &a, const double &zeta, const double &tau0, const double &alpha, const float &core_edge_fraction, const int &edge_buffer, Stopwatch &sLinkNodesGPU, size_t &hostMemUsed, size_t &maxHostMemUsed, size_t &devMemUsed, size_t &maxDevMemUsed, const bool &universe, const bool &verbose, const bool &bench)
{
	//Add assert statements

	float4 *coord;
	int *k_in;
	int *k_out;
	int *g_idx;
	int i;

	stopwatchStart(&sLinkNodesGPU);

	//Allocate memory for contiguous coordinates and degree counters on Host
	try {
		coord = (float4*)malloc(sizeof(float4) * N_tar);
		if (coord == NULL)
			throw std::bad_alloc();
		hostMemUsed += sizeof(float4) * N_tar;

		k_in = (int*)malloc(sizeof(int) * N_tar);
		if (k_in == NULL)
			throw std::bad_alloc();
		hostMemUsed += sizeof(int) * N_tar;

		k_out = (int*)malloc(sizeof(int) * N_tar);
		if (k_out == NULL)
			throw std::bad_alloc();
		hostMemUsed += sizeof(int) * N_tar;

		g_idx = (int*)malloc(sizeof(int));
		if (g_idx == NULL)
			throw std::bad_alloc();
		hostMemUsed += sizeof(int);
	} catch (std::bad_alloc) {
		fprintf(stderr, "Memory allocation failure in %s on line %d!\n", __FILE__, __LINE__);
		return false;
	}

	//Copy node coordinates to contiguous memory on Host
	for (i = 0; i < N_tar; i++) {
		coord[i].w = nodes[i].eta;
		coord[i].x = nodes[i].theta;
		coord[i].y = nodes[i].phi;
		coord[i].z = nodes[i].chi;
	}

	//Allocate memory for global index on Device
	hipDeviceptr_t d_g_idx;
	checkCudaErrors(hipMalloc(&d_g_idx, sizeof(int)));
	devMemUsed += sizeof(int);

	memoryCheckpoint(hostMemUsed, maxHostMemUsed, devMemUsed, maxDevMemUsed);
	if (verbose)
		printMemUsed("for Parallel Node Linking", hostMemUsed, devMemUsed);

	//Initialize global index on Device
	checkCudaErrors(hipMemsetD32(d_g_idx, 0, 1));

	//Copy memory from Host to Device
	checkCudaErrors(hipMemcpyHtoD(d_nodes, coord, sizeof(float4) * N_tar));

	//Synchronize
	checkCudaErrors(hipCtxSynchronize());

	free(coord);
	coord = NULL;
	hostMemUsed -= sizeof(float4) * N_tar;

	//Set up kernel arguments
	dim3 threads_per_block(BLOCK_SIZE, BLOCK_SIZE, 1);
	dim3 blocks_per_grid(256, 256, 1);
	int max = N_tar * k_tar / 2 + edge_buffer;

	//Execute kernel to generate forward/reverse adjacency list
	GenerateAdjacencyLists<<<blocks_per_grid, threads_per_block>>>((float4*)d_nodes, (int*)d_past_edges, (int*)d_future_edges, (int*)d_g_idx, N_tar, max);

	//Synchronize
	checkCudaErrors(hipCtxSynchronize());

	//DEBUG

	//Check Number of Connections
	checkCudaErrors(hipMemcpyDtoH(g_idx, d_g_idx, sizeof(int)));
	printf("Number of edges: %d\n", g_idx);
	checkCudaErrors(hipCtxSynchronize());

	free(g_idx);
	g_idx = NULL;
	hostMemUsed -= sizeof(int);

	hipFree(d_g_idx);
	g_idx = NULL;
	devMemUsed -= sizeof(int);

	//Execute kernel to sort and decode adjacency lists and their pointers
	/*DecodeAdjacencyLists<<<blocks_per_grid, threads_per_block>>>(d_past_edges, d_future_edges, d_past_edge_row_start, d_future_edge_row_start);

	//Synchronize
	checkCudaErrors(hipCtxSynchronize());

	//Copy adjacency lists from Device to Host
	checkCudaErrors(hipMemcpyDtoH(past_edges, d_past_edges, sizeof(int) * (N_tar * k_tar / 2 + edge_buffer)));
	checkCudaErrors(hipMemcpyDtoH(future_edges, d_future_edges, sizeof(int) * (N_tar * k_tar / 2 + edge_buffer)));

	//Execute kernel to increment in-degrees and out-degrees from adjacency list pointers
	FindNodeDegrees<<<blocks_per_grid, threads_per_block>>>(d_past_edge_row_start, d_future_edge_row_start, d_k_in, d_k_out);

	//Synchronize
	checkCudaErrors(hipCtxSynchronize());

	//Copy adjacency list pointers from Device to Host
	checkCudaErrors(hipMemcpyDtoH(past_edge_row_start, d_past_edge_row_start, sizeof(int) * N_tar));
	checkCudaErrors(hipMemcpyDtoH(future_edge_row_start, d_future_edge_row_start, sizeof(int) * N_tar));

	//Copy in-degree and out-degree counters from Device to Host
	checkCudaErrors(hipMemcpyDtoH(k_in, d_k_in, sizeof(int) * N_tar));
	checkCudaErrors(hipMemcpyDtoH(k_out, d_k_out, sizeof(int) * N_tar));

	//Synchronize
	checkCudaErrors(hipCtxSynchronize());

	//Write contiguous degree counters back to 'nodes'
	for (i = 0; i < N_tar; i++) {
		nodes[i].k_in = k_in[i];
		nodes[i].k_out = k_out[i];
	}*/

	stopwatchStop(&sLinkNodesGPU);

	if (!bench) {
		printf("\tCausets Successfully Connected.\n");
		fflush(stdout);
	}

	if (verbose) {
		printf("\t\tExecution Time: %5.6f sec\n", sLinkNodesGPU.elapsedTime);
		fflush(stdout);
	}

	return true;
}
