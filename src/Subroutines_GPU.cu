#include "hip/hip_runtime.h"
#include "Subroutines_GPU.h"

/////////////////////////////
//(C) Will Cunningham 2014 //
// Krioukov Research Group //
// Northeastern University //
/////////////////////////////

//Input:  Index from vectorized upper diagonal matrix
//Output: i*N+j where i,j are matrix indices
//Efficiency: O(N)
/*__host__ __device__ uint64_t vec2MatIdx(const int &N, const uint64_t &vecIdx)
{
	int i = 0, j = 0;
	int delta = 1;
	int k;
	
	for (k = 0; k < N - 1; k++) {
		if (vecIdx < (k + 1) * static_cast<uint64_t>(N) - delta) {
			i = k;
			j = static_cast<int>(vecIdx - (k * static_cast<uint64_t>(N)) + delta);
			break;
		}
		delta += k + 2;
	}

	return i * static_cast<uint64_t>(N) + j;
}*/

//Bitonic Sort
//Borrowed from https://gist.github.com/mre/1392067

__global__ void BitonicSort(uint64_t *edges, int j, int k)
{
        //Sorting Parameters
        unsigned i = blockDim.x * blockIdx.x + threadIdx.x;
        unsigned ixj = i ^ j;

        //Threads with the lowest IDs sort the list
        if (i < ixj) {
                //Sort Ascending
                if (!(i & k) && edges[i] > edges[ixj])
                        swap(edges, i, ixj);

                //Sort Descending
                if ((i & k) && edges[i] < edges[ixj])
                        swap(edges, i, ixj);
        }   
}

__device__ void swap(uint64_t * const &edges, const unsigned int &i, const unsigned int &j)
{
	uint64_t temp = edges[i];
	edges[i] = edges[j];
	edges[j] = temp;
}

//Parallel Prefix Sum
//Borrowed from https://gist.github.com/wh5a/4500706

__global__ void Scan(int *input, int *output, int *buf, int elements)
{
	__shared__ int s_vals[BLOCK_SIZE << 1];
	unsigned int tid = threadIdx.x;
	unsigned int start = (blockDim.x * blockIdx.x) << 1;

	//Read 'input' to shared memory
	if (start + tid < elements)
		s_vals[tid] = input[start + tid];
	else
		s_vals[tid] = 0;

	if (start + blockDim.x + tid < elements)
		s_vals[blockDim.x + tid] = input[start + blockDim.x + tid];
	else
		s_vals[blockDim.x + tid] = 0;
	__syncthreads();

	//Primary Reduction
	int stride, index;
	for (stride = 1; stride <= blockDim.x; stride <<= 1) {
		index = (stride * (tid + 1) << 1) - 1;
		if (index < blockDim.x << 1)
			s_vals[index] += s_vals[index - stride];
		__syncthreads();
	}

	//Secondary Reduction
	for (stride = blockDim.x >> 1; stride; stride >>= 1) {
		index = (stride * (tid + 1) << 1) - 1;
		if (index + stride < blockDim.x << 1)
			s_vals[index + stride] += s_vals[index];
		__syncthreads();
	}

	if (start + tid < elements)
		output[start + tid] = s_vals[tid];

	if (start + blockDim.x + tid < elements)
		output[start + blockDim.x + tid] = s_vals[blockDim.x + tid];

	if (buf && tid == 0)
		buf[blockIdx.x] = s_vals[(blockDim.x << 1) - 1];
}

__global__ void PostScan(int *input, int *buf, int elements)
{
	unsigned int tid = threadIdx.x;
	unsigned int start = blockDim.x * blockIdx.x << 1;

	if (blockIdx.x) {
		if (start + tid < elements)
			input[start + tid] += buf[blockIdx.x - 1];

		if (start + blockDim.x + tid < elements)
			input[start + blockDim.x + tid] += buf[blockIdx.x - 1];
	}
}


















