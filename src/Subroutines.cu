#include "hip/hip_runtime.h"
#include "Subroutines.h"

/////////////////////////////
//(C) Will Cunningham 2014 //
//         DK Lab          //
// Northeastern University //
/////////////////////////////

//Linear Interpolation using Lookup Table
bool getLookupTable(const char *filename, double **lt, long *size)
{
	#if DEBUG
	assert (filename != NULL);
	assert (lt != NULL);
	assert (size != NULL);
	#endif

	double *table;
	std::ifstream ltable(filename, std::ios::in | std::ios::binary | std::ios::ate);

	try {
		if (ltable.is_open()) {
			//Find size of file
			*size = ltable.tellg();

			if (*size == 0)
				throw CausetException("Lookup table file is empty!\n");

			//Allocate Memory for Buffer
			char *memblock = (char*)malloc(*size);
			if (memblock == NULL)
				throw std::bad_alloc();

			//Allocate Memory for Lookup Table
			table = (double*)malloc(*size);
			if (table == NULL)
				throw std::bad_alloc();

			//Read File
			ltable.seekg(0, std::ios::beg);
			ltable.read(memblock, *size);
			memcpy(table, memblock, *size);

			//Free Memory
			free(memblock);
			memblock = NULL;

			//Close Stream
			ltable.close();
		} else
			throw CausetException("Failed to open lookup table file!\n");

		//Return Table
		*lt = table;
	} catch (CausetException c) {
		fprintf(stderr, "CausetException in %s: %s on line %d\n", __FILE__, c.what(), __LINE__);
		return false;
	} catch (std::bad_alloc) {
		fprintf(stderr, "Memory allocation failure in %s on line %d!\n", __FILE__, __LINE__);
		return false;
	} catch (std::exception e) {
		fprintf(stderr, "Unknown Exception in %s: %s on line %d\n", __FILE__, e.what(), __LINE__);
		return false;
	}

	return true;
}

//Lookup value in table of (x, y) coordinates -> 2D parameter space
double lookupValue(const double *table, const long &size, double *x, double *y, bool increasing)
{
	#if DEBUG
	assert (table != NULL);
	assert (size > 0);
	assert ((x == NULL) ^ (y == NULL));
	#endif
	
	//Identify which is being calculated
	bool first = (x == NULL);
	//Identify input value
	double input = first ? *y : *x;
	double output = 0.0;
	int t_idx = (int)(!first);
	int i;

	try {
		//Identify Value in Table
		//Assumes values are written (y, x)
		for (i = (int)(!first); i < size / (int)sizeof(double); i += 2) {
			if ((increasing && table[i] >= input) || (!increasing && table[i] <= input)) {
				t_idx = i;
				break;
			}
		}

		//Check if Table is Insufficient
		if (t_idx == (int)(!first) && input != table[i]) {
			//printf("%f\n", input);
			throw CausetException("Values from lookup table do not include requested input.  Recreate table or change input.\n");
		}

		//Linear Interpolation
		if (first)
			output = table[t_idx-1] + (table[t_idx+1] - table[t_idx-1]) * (input - table[t_idx-2]) / (table[t_idx] - table[t_idx-2]);
		else
			output = table[t_idx-3] + (table[t_idx-1] - table[t_idx-3]) * (input - table[t_idx-2]) / (table[t_idx] - table[t_idx-2]);
	} catch (CausetException c) {
		fprintf(stderr, "CausetException in %s: %s on line %d\n", __FILE__, c.what(), __LINE__);
		output = std::numeric_limits<double>::quiet_NaN();
	} catch (std::exception e) {
		fprintf(stderr, "Unknown Exception in %s: %s on line %d\n", __FILE__, e.what(), __LINE__);
		output = std::numeric_limits<double>::quiet_NaN();
	}

	return output;
}

//Lookup value in table of (t1, t2, omega12, lambda) coordinates -> 4D parameter space
//Used for geodesic distance calculations
//Returns the transcendental integration parameter 'lambda'
double lookupValue4D(const double *table, const long &size, const double &omega12, double t1, double t2)
{
	#if DEBUG
	assert (table != NULL);
	assert (size > 0);
	assert (omega12 >= 0.0);
	assert (t1 >= 0.0);
	assert (t2 >= 0.0);
	#endif

	if (t2 < t1) {
		double temp = t1;
		t1 = t2;
		t2 = temp;
	}

	double lambda = 0.0;
	double tol = 1e-2;
	int tau_step, lambda_step, step;
	int counter;
	int i;

	try {
		//The first two table elements should be zero
		if (table[0] != 0.0 || table[1] != 0.0)
			throw CausetException("Corrupted lookup table!\n");

		tau_step = table[2];
		lambda_step = table[3];
		step = 4 * tau_step * lambda_step;
		counter = 0;

		//Identify Value in Table
		//Assumes values are written (tau1, tau2, omega12, lambda)
		for (i = 4; i < size / (int)sizeof(double); i += step) {
			counter++;

			if (step == 4 * tau_step * lambda_step && table[i] > t1) {
				i -= (step - 1);
				step = 4 * lambda_step;
				i -= step;
				counter = 0;
			} else if (step == 4 * lambda_step && table[i] > t2) {
				i -= (step - 1);
				step = 4;
				i -= step;
				counter = 0;
			} else if (step == 4 && ABS(table[i] - omega12, STL) / omega12 < tol && table[i] != 0.0) {
				i -= step;
				step = 1;
			} else if (step == 1) {
				lambda = table[i];
				break;
			}

			if ((step == 4 * tau_step * lambda_step && counter == tau_step) ||
			    (step == 4 * lambda_step && counter == tau_step) ||
			    (step == 4 && counter == lambda_step))
				break;
		}

		//Perhaps do some linear interpolation here?

		//If no value found
		if (lambda == 0.0) {
			if (step == 4 * tau_step * lambda_step)
				throw CausetException("tau1 value not found in geodesic lookup table.\n");
			else if (step == 4 * lambda_step)
				throw CausetException("tau2 value not found in geodesic lookup table.\n");
			else if (step == 4)
				throw CausetException("omega12 value not found in geodesic lookup table.\n");
			else if (step == 1)
				throw std::exception();
		}
	} catch (CausetException c) {
		fprintf(stderr, "CausetException in %s: %s on line %d\n", __FILE__, c.what(), __LINE__);
		lambda = std::numeric_limits<double>::quiet_NaN();
	} catch (std::exception e) {
		fprintf(stderr, "Unknown Exception in %s: %s on line %d\n", __FILE__, e.what(), __LINE__);
		lambda = std::numeric_limits<double>::quiet_NaN();
	}

	return lambda;
}

//Sort nodes temporally
//O(N*log(N)) Efficiency
void quicksort(Node &nodes, const unsigned int &spacetime, int low, int high)
{
	#if DEBUG
	assert (!nodes.crd->isNull());
	assert (get_stdim(spacetime) & (2 | 4));
	assert (get_manifold(spacetime) & (MINKOWSKI | DE_SITTER | DUST | FLRW | HYPERBOLIC));
	if (get_manifold(spacetime) & HYPERBOLIC)
		assert (get_stdim(spacetime) == 2);
	#endif

	int i, j, k;
	float key = 0.0;
	#if EMBED_NODES
	float *& time = get_stdim(spacetime) == 2 ? nodes.crd->x() : nodes.crd->v();
	#else
	float *& time = get_stdim(spacetime) == 2 ? nodes.crd->x() : nodes.crd->w();
	#endif

	if (low < high) {
		k = (low + high) >> 1;
		swap(nodes, spacetime, low, k);
		key = time[low];
		i = low + 1;
		j = high;

		while (i <= j) {
			while ((i <= high) && (time[i] <= key))
				i++;
			while ((j >= low) && (time[j] > key))
				j--;
			if (i < j)
				swap(nodes, spacetime, i, j);
		}

		swap(nodes, spacetime, low, j);
		quicksort(nodes, spacetime, low, j - 1);
		quicksort(nodes, spacetime, j + 1, high);
	}
}

//Sort edge list
void quicksort(uint64_t *edges, int64_t low, int64_t high)
{
	#if DEBUG
	assert (edges != NULL);
	#endif

	int64_t i, j, k;
	uint64_t key;

	if (low < high) {
		k = (low + high) >> 1;
		swap(edges, low, k);
		key = edges[low];
		i = low + 1;
		j = high;

		while (i <= j) {
			while ((i <= high) && (edges[i] <= key))
				i++;
			while ((j >= low) && (edges[j] > key))
				j--;
			if (i < j)
				swap(edges, i, j);
		}

		swap(edges, low, j);
		quicksort(edges, low, j - 1);
		quicksort(edges, j + 1, high);
	}
}

//Exchange two nodes
void swap(Node &nodes, const unsigned int &spacetime, const int i, const int j)
{
	#if DEBUG
	assert (!nodes.crd->isNull());
	assert (get_stdim(spacetime) & (2 | 4));
	assert (get_manifold(spacetime) & (MINKOWSKI | DE_SITTER | DUST | FLRW | HYPERBOLIC));
	if (get_manifold(spacetime) & HYPERBOLIC)
		assert (get_stdim(spacetime) == 2);
	#endif

	#if EMBED_NODES
	if (get_stdim(spacetime) == 2) {
		float3 hc = nodes.crd->getFloat3(i);
		nodes.crd->setFloat3(nodes.crd->getFloat3(j), i);
		nodes.crd->setFloat3(hc, j);
	} else if (get_stdim(spacetime) == 4) {
		float5 sc = nodes.crd->getFloat5(i);
		nodes.crd->setFloat5(nodes.crd->getFloat5(j), i);
		nodes.crd->setFloat5(sc, j);
	}
	#else
	if (get_stdim(spacetime) == 2) {
		float2 hc = nodes.crd->getFloat2(i);
		nodes.crd->setFloat2(nodes.crd->getFloat2(j), i);
		nodes.crd->setFloat2(hc, j);
	} else if (get_stdim(spacetime) == 4) {
		float4 sc = nodes.crd->getFloat4(i);
		nodes.crd->setFloat4(nodes.crd->getFloat4(j), i);
		nodes.crd->setFloat4(sc, j);
	}
	#endif

	if (get_manifold(spacetime) & (DE_SITTER | DUST | FLRW)) {
		float tau = nodes.id.tau[i];
		nodes.id.tau[i] = nodes.id.tau[j];
		nodes.id.tau[j] = tau;
	} else if (get_manifold(spacetime) & HYPERBOLIC) {
		int AS = nodes.id.AS[i];
		nodes.id.AS[i] = nodes.id.AS[j];
		nodes.id.AS[j] = AS;
	}
}

//Exchange two edges
void swap(uint64_t *edges, const int64_t i, const int64_t j)
{
	#if DEBUG
	assert (edges != NULL);
	assert (i >= 0);
	assert (j >= 0);
	#endif

	uint64_t tmp = edges[i];
	edges[i] = edges[j];
	edges[j] = tmp;
}

//Exchange references to two lists
//as well as related indices (used in causet_intersection)
void swap(const int * const *& list0, const int * const *& list1, int64_t &idx0, int64_t &idx1, int64_t &max0, int64_t &max1)
{
	#if DEBUG
	assert (idx0 >= 0);
	assert (idx1 >= 0);
	assert (max0 >= 0);
	assert (max1 >= 0);
	#endif

	const int * const * tmp_list = list0;
	list0 = list1;
	list1 = tmp_list;

	//Bitwise swaps
	idx0 ^= idx1;
	idx1 ^= idx0;
	idx0 ^= idx1;

	max0 ^= max1;
	max1 ^= max0;
	max0 ^= max1;
}

//Bisection Method
//Use when Newton-Raphson fails
bool bisection(double (*solve)(const double &x, const double * const p1, const float * const p2, const int * const p3), double *x, const int max_iter, const double lower, const double upper, const double tol, const bool increasing, const double * const p1, const float * const p2, const int * const p3)
{
	#if DEBUG
	assert (solve != NULL);
	#endif

	double res = 1.0;
	double a = lower;
	double b = upper;
	int iter = 0;

	try {
		if (b <= a)
			throw CausetException("Invalid Bounds in Bisection!\n");

		//Initial test point
		*x = (b + a) / 2;
		while (ABS(res, STL) > tol && iter < max_iter) {
			//Residual Value
			res = (*solve)(*x, p1, p2, p3);
			//printf("res:   %.16e\n\n", res);
			//printf("x: %.16e\n\n", *x);

			//Check for NaN
			if (res != res)
				throw CausetException("NaN Error in Bisection!\n");

			//Change bounds
			if (increasing) {
				if (res > 0)
					b = *x;
				else
					a = *x;
			} else {
				if (res > 0)
					a = *x;
				else
					b = *x;
			}

			//New test point
			*x = (b + a) / 2;
			iter++;
		}
	} catch (CausetException c) {
		fprintf(stderr, "CausetException in %s: %s on line %d\n", __FILE__, c.what(), __LINE__);
		return false;
	} catch (std::exception e) {
		fprintf(stderr, "Unknown Exception in %s: %s on line %d\n", __FILE__, e.what(), __LINE__);
		return false;
	}
	
	//printf("Bisection Results:\n");
	//printf("Tolerance: %E\n", tol);
	//printf("%d of %d iterations performed.\n", iter, max_iter);
	//printf("Residual: %E\n", y - res);
	//printf("Solution: %E\n", *x);
	//fflush(stdout);

	return true;
}

//Newton-Raphson Method
//Solves Transcendental Equations
bool newton(double (*solve)(const double &x, const double * const p1, const float * const p2, const int * const p3), double *x, const int max_iter, const double tol, const double * const p1, const float * const p2, const int * const p3)
{
	#if DEBUG
	assert (solve != NULL);
	#endif

	double res = 1.0;
	double x1;
	int iter = 0;

	try {
		while (ABS(res, STL) > tol && iter < max_iter) {
			//Residual Value
			res = (*solve)(*x, p1, p2, p3);
			//printf("res: %E\n", res);

			//Check for NaN
			if (res != res)
				throw CausetException("NaN Error in Newton-Raphson\n");
	
			//New test value
			x1 = *x + res;
			//printf("x1: %E\n", x1);
	
			*x = x1;
			iter++;

			fflush(stdout);
		}
	} catch (CausetException c) {
		fprintf(stderr, "CausetException in %s: %s on line %d\n", __FILE__, c.what(), __LINE__);
		return false;
	} catch (std::exception e) {
		fprintf(stderr, "Unknown Exception in %s: %s on line %d\n", __FILE__, e.what(), __LINE__);
		return false;
	}

	//printf("Newton-Raphson Results:\n");
	//printf("Tolerance: %E\n", tol);
	//printf("%d of %d iterations performed.\n", iter, max_iter);
	//printf("Residual: %E\n", res);
	//printf("Solution: %E\n", *x);
	//fflush(stdout);

	return true;
}

//Returns true if two nodes are causally connected
//Note: past_idx must be less than future_idx
//O(1) Efficiency for Adjacency Matrix
//O(k) Efficiency for Adjacency List
bool nodesAreConnected(const Node &nodes, const int * const future_edges, const int64_t * const future_edge_row_start, const Bitvector &adj, const int &N_tar, const float &core_edge_fraction, int past_idx, int future_idx)
{
	#if DEBUG
	//No null pointers
	assert (future_edges != NULL);
	assert (future_edge_row_start != NULL);

	//Parameters in correct ranges
	assert (core_edge_fraction >= 0.0f && core_edge_fraction <= 1.0f);
	assert (past_idx >= 0 && past_idx < N_tar);
	assert (future_idx >= 0 && future_idx < N_tar);
	assert (past_idx != future_idx);

	assert (!(future_edge_row_start[past_idx] == -1 && nodes.k_out[past_idx] > 0));
	assert (!(future_edge_row_start[past_idx] != -1 && nodes.k_out[past_idx] == 0));
	#endif

	int core_limit = static_cast<int>(core_edge_fraction * N_tar);
	int i;

	//Make sure past_idx < future_idx
	if (past_idx > future_idx) {
		past_idx ^= future_idx;
		future_idx ^= past_idx;
		past_idx ^= future_idx;
	}

	//Check if the adjacency matrix can be used
	if (past_idx < core_limit && future_idx < core_limit)
		return (bool)adj[past_idx].read(future_idx);
	//Check if past node is not connected to anything
	else if (future_edge_row_start[past_idx] == -1)
		return false;
	//Check adjacency list
	else
		for (i = 0; i < nodes.k_out[past_idx]; i++)
			if (future_edges[future_edge_row_start[past_idx] + i] == future_idx)
				return true;

	return false;
}

//Returns true if two nodes are causally connected
//Note: past_idx must be less than future_idx
//O(1) Efficiency for Adjacency Matrix
bool nodesAreConnected_v2(const Bitvector &adj, const int &N_tar, int past_idx, int future_idx)
{
	#if DEBUG
	assert (past_idx >= 0 && past_idx < N_tar);
	assert (future_idx >= 0 && future_idx < N_tar);
	//assert (past_idx != future_idx);
	#endif

	return (bool)adj[past_idx].read(future_idx);
}

//Breadth First Search
void bfsearch(const Node &nodes, const Edge &edges, const int index, const int id, int &elements)
{
	#if DEBUG
	assert (nodes.k_in != NULL);
	assert (nodes.k_out != NULL);
	assert (nodes.cc_id != NULL);
	assert (edges.past_edges != NULL);
	assert (edges.future_edges != NULL);
	assert (edges.past_edge_row_start != NULL);
	assert (edges.future_edge_row_start != NULL);
	assert (index >= 0);
	assert (id >= 0);
	assert (elements >= 0);
	#endif

	int64_t ps = edges.past_edge_row_start[index];
	int64_t fs = edges.future_edge_row_start[index];
	int i;

	nodes.cc_id[index] = id;
	elements++;

	//Move to past nodes
	for (i = 0; i < nodes.k_in[index]; i++)
		if (!nodes.cc_id[edges.past_edges[ps+i]])
			bfsearch(nodes, edges, edges.past_edges[ps+i], id, elements);

	//Move to future nodes
	for (i = 0; i < nodes.k_out[index]; i++)
		if (!nodes.cc_id[edges.future_edges[fs+i]])
			bfsearch(nodes, edges, edges.future_edges[fs+i], id, elements);
}

//Breadth First Search
//Uses adjacency matrix only
void bfsearch_v2(const Node &nodes, const Bitvector &adj, const int &N_tar, const int index, const int id, int &elements)
{
	#if DEBUG
	assert (nodes.cc_id != NULL);
	assert (N_tar >= 0);
	assert (index >= 0 && index < N_tar);
	assert (id >= 0 && id <= N_tar / 2);
	assert (elements >= 0 && elements < N_tar);
	#endif

	nodes.cc_id[index] = id;
	elements++;

	int i;
	for (i = 0; i < N_tar; i++)
		if (adj[index].read(i) && !nodes.cc_id[i])
			bfsearch_v2(nodes, adj, N_tar, i, id, elements);
}

void causet_intersection_v2(int &elements, const int * const past_edges, const int * const future_edges, const int &k_i, const int &k_o, const int &max_cardinality, const int64_t &pstart, const int64_t &fstart, bool &too_many)
{
	#if DEBUG
	assert (past_edges != NULL);
	assert (future_edges != NULL);
	assert (k_i >= 0);
	assert (k_o >= 0);
	assert (!(k_i == 0 && k_o == 0));
	assert (max_cardinality > 1);
	assert (pstart >= 0);
	assert (fstart >= 0);
	#endif

	if (k_i == 1 || k_o == 1) {
		elements = 0;
		return;
	}

	//int larger = k_i > k_o ? k_i : k_o;
	//int smaller = k_i <= k_o ? k_i : k_o;

	//if (larger + smaller > smaller * LOG(larger, APPROX ? FAST : STL)) {
		//Binary search
	//} else {
		int64_t idx0 = pstart;	//Index of past neighbors of 'future element j'
		int64_t idx1 = fstart;	//Index of future neighbors of 'past element i'
		int64_t max0 = idx0 + k_i;
		int64_t max1 = idx1 + k_o;

		while (idx0 < max0 && idx1 < max1 && !too_many) {
			if (past_edges[idx0] > future_edges[idx1])
				idx1++;
			else if (past_edges[idx0] < future_edges[idx1])
				idx0++;
			else {
				elements++;

				if (elements >= max_cardinality - 1) {
					too_many = true;
					//printChk();
					break;
				}

				idx0++;
				idx1++;
			}
		}
	//}

	//printf("(%d - %d):\t%d\n", past_edges[pstart], future_edges[fstart], elements);
}

//Intersection of Sorted Lists
//Used to find the cardinality of an interval
//Complexity: O(k*log(k))
void causet_intersection(int &elements, const int * const past_edges, const int * const future_edges, const int &k_i, const int &k_o, const int &max_cardinality, const int64_t &pstart, const int64_t &fstart, bool &too_many)
{
	#if DEBUG
	assert (past_edges != NULL);
	assert (future_edges != NULL);
	assert (k_i >= 0);
	assert (k_o >= 0);
	assert (!(k_i == 0 && k_o == 0));
	assert (max_cardinality > 1);
	assert (pstart >= 0);
	assert (fstart >= 0);
	#endif

	int64_t idx0 = pstart;
	int64_t idx1 = fstart;
	int64_t max0 = idx0 + k_i;
	int64_t max1 = idx1 + k_o;

	if (k_i == 1 || k_o == 1) {
		elements = 0;
		return;
	}

	//Pointers are used here so that 'primary' and 'secondary'
	//can be switched as needed.  References are static, so they
	//cannot be used.  The 'const' specifiers are kept since the
	//edge list values and their locations in memory should
	//not be modified in this algorithm.

	const int * const * primary = &past_edges;
	const int * const * secondary = &future_edges;

	/*printf("Future Edge List:\n");
	for (int i = 0; i < k_o; i++)
		printf("\t%d\n", (*secondary)[i+fstart]);
	printf("Past Edge List:\n");
	for (int i = 0; i < k_i; i++)
		printf("\t%d\n", (*primary)[i+pstart]);*/

	//printf("idx0: %d\tidx1: %d\n", idx0, idx1);

	while (idx0 < max0 && idx1 < max1) {
		if ((*secondary)[idx1] > (*primary)[idx0])
			swap(primary, secondary, idx0, idx1, max0, max1);

		/*if (*primary == past_edges)
			printf("Primary: PAST\n");
		else if (*primary == future_edges)
			printf("Primary: FUTURE\n");
		if (*secondary == past_edges)
			printf("Secondary: PAST\n");
		else if (*secondary == future_edges)
			printf("Secondary: FUTURE\n");*/

		while (idx1 < max1 && (*secondary)[idx1] < (*primary)[idx0])
			idx1++;

		if (idx1 == max1)
			//continue;
			break;

		//printf("idx0: %d\tidx1: %d\n", idx0, idx1);

		if ((*primary)[idx0] == (*secondary)[idx1]) {
			//printf_red();
			//printf("Element Found!\n");
			//printf_std();
			elements++;
			if (elements >= max_cardinality - 1) {
				too_many = true;
				//printf("TOO MANY!\n");
				return;
			}
			idx0++;
			idx1++;
		}
	}

	/*printf_red();
	printf("Found %d Elements.\n", elements);
	printf_std();*/
}

//Data formatting used when reading the degree
//sequences found on the GPU
void readDegrees(int * const &degrees, const int * const h_k, const size_t &offset, const size_t &size)
{
	#if DEBUG
	assert (degrees != NULL);
	assert (h_k != NULL);
	#endif

	unsigned int i;
	for (i = 0; i < size; i++)
		degrees[offset+i] += h_k[i];
}

//Data formatting used when reading output of
//the adjacency list created by the GPU
void readEdges(uint64_t * const &edges, const bool * const h_edges, Bitvector &adj, int64_t * const &g_idx, const unsigned int &core_limit_row, const unsigned int &core_limit_col, const size_t &d_edges_size, const size_t &mthread_size, const size_t &size0, const size_t &size1, const int x, const int y, const bool &use_bit, const bool &use_mpi)
{
	#if DEBUG
	if (!use_bit)
		assert (edges != NULL);
	assert (h_edges != NULL);
	assert (g_idx != NULL);
	assert (*g_idx >= 0);
	assert (x >= 0);
	assert (y >= 0);
	//assert (x <= y);
	#endif

	//printf("x: %d\tsize0: %zd\n", x, size0);
	//printf("I have a bitvector of length %zd\n", adj.size());

	unsigned int i, j;
	for (i = 0; i < size0; i++) {
		for (j = 0; j < size1; j++) {
			if (h_edges[i*mthread_size+j] && (use_bit || g_idx[0] < (int64_t)d_edges_size)) {
				if (!use_bit)
					edges[g_idx[0]++] = (static_cast<uint64_t>(x*mthread_size+i)) << 32 | (static_cast<uint64_t>(y*mthread_size+j));
				else
					g_idx[0]++;
				if (x*mthread_size+i < core_limit_row && y*mthread_size+j < core_limit_col) {
					adj[x*mthread_size+i].set(y*mthread_size+j);
					if (!use_mpi)
						adj[y*mthread_size+j].set(x*mthread_size+i);
				}
			}
		}
	}
}

//Remake adjacency sub-matrix using 'l' rows, beginning at row 'i'
void remakeAdjMatrix(bool * const adj0, bool * const adj1, const int * const k_in, const int * const k_out, const int * const past_edges, const int * const future_edges, const int64_t * const past_edge_row_start, const int64_t * const future_edge_row_start, int * const idx_buf0, int * const idx_buf1, const int &N_tar, const int &i, const int &j, const int64_t &l)
{
	#if DEBUG
	assert (adj0 != NULL);
	assert (adj1 != NULL);
	assert (k_in != NULL);
	assert (k_out != NULL);
	assert (past_edges != NULL);
	assert (future_edges != NULL);
	assert (past_edge_row_start != NULL);
	assert (future_edge_row_start != NULL);
	assert (idx_buf0 != NULL);
	assert (idx_buf1 != NULL);
	assert (N_tar > 0);
	assert (i >= 0);
	assert (j >= 0);
	assert (l > 0);
	#endif

	//Map tile indices to global indices
	for (int m = 0; m < l; m++) {
		int M = m + i * l;
		for (int n = 0; n < l; n++) {
			int N = n + j * l;
			if (!N)
				continue;

			//Use triangular mapping
			int do_map = M >= N;
			if (N < N_tar >> 1) {
				M = M + do_map * ((((N_tar >> 1) - M) << 1) - 1);
				N = N + do_map * (((N_tar >> 1) - N) << 1);
			}

			idx_buf0[m] = M;
			idx_buf1[n] = N;
		}
	}

	//Fill Adjacency Submatrix 0
	#ifdef _OPENMP
	#pragma omp parallel for schedule (dynamic, 1)
	#endif
	for (int m = 0; m < l; m++) {
		int M = idx_buf0[m];
		int element;

		//Past Neighbors
		int64_t start = past_edge_row_start[M];
		for (int p = 0; p < k_in[M]; p++) {
			element = past_edges[start+p];
			adj0[m*N_tar+element] = true;
		}

		//Future Neighbors
		start = future_edge_row_start[M];
		for (int p = 0; p < k_out[M]; p++) {
			element = future_edges[start+p];
			adj0[m*N_tar+element] = true;
		}
	}

	//Fill Adjacency Submatrix 1
	#ifdef _OPENMP
	#pragma omp parallel for schedule (dynamic, 1)
	#endif
	for (int n = 0; n < l; n++) {
		int N = idx_buf1[n];
		int element;

		if (!N)
			continue;

		//Past Neighbors
		int64_t start = past_edge_row_start[N];
		for (int p = 0; p < k_in[N]; p++) {
			element = past_edges[start+p];
			adj1[n*N_tar+element] = true;
		}

		//Future Neighbors
		start = future_edge_row_start[N];
		for (int p = 0; p < k_out[N]; p++) {
			element = future_edges[start+p];
			adj1[n*N_tar+element] = true;
		}
	}
}

//Data formatting used when reading output of
//the interval matrix created by the GPU
void readIntervals(int * const cardinalities, const unsigned int * const N_ij, const int &l)
{
	#if DEBUG
	assert (cardinalities != NULL);
	assert (N_ij != NULL);
	assert (l > 0);
	#endif

	int i, j;
	for (i = 0; i < l; i++)
		for (j = 0; j < l; j++)
			cardinalities[N_ij[j*l+i]+1]++;
}

//Scanning algorithm used when decoding
//lists found using GPU algorithms
void scan(const int * const k_in, const int * const k_out, int64_t * const &past_edge_pointers, int64_t * const &future_edge_pointers, const int &N_tar)
{
	int64_t past_idx = 0, future_idx = 0;
	int i;

	for (i = 0; i < N_tar; i++) {
		if (k_in[i] != 0) {
			past_edge_pointers[i] = past_idx;
			past_idx += k_in[i];
		} else
			past_edge_pointers[i] = -1;

		if (k_out[i] != 0) {
			future_edge_pointers[i] = future_idx;
			future_idx += k_out[i];
		} else
			future_edge_pointers[i] = -1;
	}
}

//Debug Print Variadic Function
int printf_dbg(const char * format, ...)
{
	printf_mag();
	va_list argp;
	va_start(argp, format);
	vprintf(format, argp);
	va_end(argp);
	printf_std();
	fflush(stdout);

	return 0;
}

//Allows only the master process to print to stdout
//If MPI is not enabled, rank == 0
int printf_mpi(int rank, const char * format, ...)
{
	int retval = 0;

	if (rank == 0) {
		va_list argp;
		va_start(argp, format);
		vprintf(format, argp);
		va_end(argp);
	}

	return retval;
}

void printCardinalities(const uint64_t * const cardinalities, unsigned int Nc, unsigned int nthreads, unsigned int idx0, unsigned int idx1, unsigned int version)
{
	#if DEBUG
	assert (cardinalities != NULL);
	assert (Nc > 0);
	assert (nthreads >= 1);
	assert (idx0 != idx1);
	#endif

	if (idx0 > idx1) {
		idx0 ^= idx1;
		idx1 ^= idx0;
		idx0 ^= idx1;
	}

	std::ofstream os;
	char filename[80];
	strcpy(filename, "cards_");
	strcat(filename, std::to_string(idx0).c_str());
	strcat(filename, std::to_string(idx1).c_str());
	strcat(filename, "-v");
	strcat(filename, std::to_string(version).c_str());
	strcat(filename, ".cset.dbg.dat");

	os.open(filename);
	for (int i = 0; i < Nc; i++) {
		int sum = 0;
		for (int j = 0; j < nthreads; j++)
			sum += cardinalities[j*Nc+i];
		os << sum << std::endl;
	}

	os.flush();
	os.close();
}

MPI_Request* sendSignal(const int signal, const int rank, const int num_mpi_threads)
{
	MPI_Request *req = (MPI_Request*)malloc(sizeof(MPI_Request) * num_mpi_threads);;
	for (int i = 0; i < num_mpi_threads; i++) {
		if (signal != 1 && i == rank) continue;
		MPI_Isend((void*)&signal, 1, MPI_INT, i, MPI_ANY_TAG, MPI_COMM_WORLD, &req[i]);
	}

	return req;
}

/*MPI_Request* requestLock(CausetSpinlock * const lock, const int rank, const int num_mpi_threads)
{
	MPI_Request *req;
	//int success;
	//lock[rank] = LOCKED;
	printf("Rank [%d] is sending signal 0 to other ranks.\n", rank);
	req = sendSignal(0, rank, num_mpi_threads);
	//for (unsigned int i = 0; i < num_mpi_threads; i++) {
	//	if (i == rank) continue;
	//	MPI_Isend(lock, num_mpi_threads, MPI_INT, i, MPI_ANY_TAG, MPI_COMM_WORLD, &req[1]);
	//	MPI_Irecv(&success, 1, MPI_INT, i, MPI_ANY_TAG, MPI_COMM_WORLD, &req[2]);
	//}
	//MPI_Request r;
	//req[1] = r;
	//req[2] = r;
	//lock[rank] = UNLOCKED;

	return req;
}

void requestUnlock(CausetSpinlock * const lock, const int rank, const int num_mpi_threads)
{
	CausetSpinlock req_lock = UNLOCKED;
	sendSignal(1, rank, num_mpi_threads);
	MPI_Bcast(&req_lock, 1, MPI_INT, rank, MPI_COMM_WORLD);
}*/

//MPI Print Variadic Function
bool checkMpiErrors(CausetMPI &cmpi)
{
	#ifdef MPI_ENABLED
	MPI_Barrier(MPI_COMM_WORLD);
	if (!cmpi.rank)
		MPI_Reduce(MPI_IN_PLACE, &cmpi.fail, 1, MPI_INT, MPI_SUM, 0, MPI_COMM_WORLD);
	else
		MPI_Reduce(&cmpi.fail, NULL, 1, MPI_INT, MPI_SUM, 0, MPI_COMM_WORLD);
	MPI_Bcast(&cmpi.fail, cmpi.num_mpi_threads, MPI_INT, 0, MPI_COMM_WORLD);
	#endif

	return !!cmpi.fail;
}

void perm_to_binary(FastBitset &fb, std::vector<unsigned int> perm)
{
	#if DEBUG
	assert (!(fb.size() % 2));
	assert (!(perm.size() % 2));
	#endif

	for (uint64_t k = 0; k < perm.size(); k += 2) {
		unsigned int i = perm[k];
		unsigned int j = perm[k+1];

		if (i > j) {
			i ^= j;
			j ^= i;
			i ^= j;
		}

		//printf("Adding element at index (%d, %d)\n", i, j);
		unsigned int do_map = i >= perm.size() >> 1;
		i -= do_map * (((i - (perm.size() >> 1)) << 1) + 1);
		j -= do_map * ((j - (perm.size() >> 1)) << 1);

		//printf("Index mapped to (%d, %d)\n", i, j);
		unsigned int m = i * (perm.size() - 1) + j - 1;
		//printf("Linear Index: %d\n\n", m);
		fb.set(m);
	}
}

void binary_to_perm(std::vector<unsigned int> &perm, const FastBitset &fb, const unsigned int len)
{
	#if DEBUG
	assert (!(perm.size() % 2));
	assert (!(fb.size() % 2));
	#endif

	for (uint64_t k = 0; k < fb.size(); k++) {
		if (!fb.read(k)) continue;

		unsigned int i = k / (len - 1);
		unsigned int j = k % (len - 1) + 1;

		unsigned int do_map = i >= j;
		i += do_map * ((((len >> 1) - i) << 1) - 1);
		j += do_map * (((len >> 1) - j) << 1);

		perm.push_back(i);
		perm.push_back(j);
	}
}

void init_mpi_permutations_v2(std::unordered_set<FastBitset> &permutations, std::vector<unsigned int> perm)
{
	uint64_t len = static_cast<uint64_t>(perm.size()) * (perm.size() - 1) >> 1;
	while (std::next_permutation(perm.begin(), perm.end())) {
		FastBitset fb(len);
		perm_to_binary(fb, perm);

		bool ins = true;
		for (std::unordered_set<FastBitset>::iterator fb0 = permutations.begin(); fb0 != permutations.end(); fb0++)
			for (uint64_t i = 0; i < len; i++)
				if (fb.read(i) & fb0->read(i))
					ins = false;

		if (ins) {
			//
		}		
	}
}

void init_mpi_permutations(std::unordered_set<FastBitset> &permutations, std::vector<unsigned int> perm)
{
	uint64_t len = static_cast<uint64_t>(perm.size()) * (perm.size() - 1) >> 1;
	while (std::next_permutation(perm.begin(), perm.end())) {
		FastBitset fb(len);
		perm_to_binary(fb, perm);
		bool ins = true;
		for (std::unordered_set<FastBitset>::iterator fb0 = permutations.begin(); fb0 != permutations.end(); fb0++)
			for (uint64_t i = 0; i < len; i++)
				if (fb.read(i) & fb0->read(i))
					ins = false;

		if (ins) permutations.insert(fb);
		/*std::pair<std::unordered_set<FastBitset>::iterator, bool> p = permutations.insert(fb);
		if (p.second) {
			printf("Successfully added element:\n");
			for (size_t i = 0; i < perm.size(); i += 2)
				printf("(%d, %d) ", perm[i], perm[i+1]);
			printf("\n");
			fb.printBitset();
		} else
			printf("Did not add element.\n");*/
	}

	//The first element will be the ordered one, so remove it

	/*for (std::unordered_set<FastBitset>::iterator fb = permutations.begin(); fb != permutations.end(); fb++) {
		std::vector<unsigned int> p;
		binary_to_perm(p, *fb, 8);
		for (size_t j = 0; j < p.size(); j += 2)
			printf("(%d, %d) ", p[j], p[j+1]);
		printf("\n");
	}*/
}

void init_mpi_pairs(std::unordered_set<std::pair<int,int> > &pairs, std::vector<unsigned int> current, int nbuf)
{
	#if DEBUG
	assert (nbuf > 0 && !(nbuf % 2));
	#endif

	for (size_t k = 0; k < current.size(); k += 2) {
		unsigned int i = current[k];
		unsigned int j = current[k+1];

		for (unsigned int m = 0; m < nbuf; m++) {
			if (m == i || m == j) continue;
			pairs.insert(std::make_pair(std::min(i, m), std::max(i, m)));
			pairs.insert(std::make_pair(std::min(j, m), std::max(j, m)));
		}
	}

	for (size_t k = 0; k < current.size(); k += 2)
		pairs.erase(std::make_pair(k, k+1));
}

void fill_mpi_similar(std::vector<std::vector<unsigned int> > &similar, std::vector<unsigned int> perm)
{
	uint64_t len = static_cast<uint64_t>(perm.size()) * (perm.size() - 1) >> 1;
	FastBitset fb(len);
	perm_to_binary(fb, perm);
	similar.push_back(perm);

	while (std::next_permutation(perm.begin(), perm.end())) {
		FastBitset sim(len);
		perm_to_binary(sim, perm);
		if (fb == sim)
			similar.push_back(perm);
	}
}

void print_pairs(std::vector<unsigned int> vec)
{
	for (size_t i = 0; i < vec.size(); i += 2)
		printf("(%d, %d) ", vec[i], vec[i+1]);
	printf("\n");
}

void cyclesort(unsigned int &writes, std::vector<unsigned int> c, std::vector<std::pair<int,int> > *swaps)
{
	unsigned int it, p;
	size_t len = c.size();
	writes = 0;
	for (size_t j = 0; j < len - 1; j++) {
		it = c[j];
		p = j;

		for (unsigned int k = j + 1; k < len; k++)
			if (c[k] < it)
				p++;

		if (j == p) continue;

		//Swap
		while (c[p] == it)
			p++;
		//printf("Copying spot [%d] to buffer\n", p);
		//printf("Copying spot [%zd] to spot [%d]\n", j, p);
		it ^= c[p];
		c[p] ^= it;
		it ^= c[p];
		writes += 2;
		if (swaps != NULL) {
			swaps->push_back(std::make_pair(p, -1));
			swaps->push_back(std::make_pair(j, p));
		}
		//print_pairs(c);

		while (j != p) {
			p = j;
			for (size_t k = j + 1; k < len; k++)
				if (c[k] < it)
					p++;
			
			while (c[p] == it)
				p++;
			//printf("Copying buffer to spot [%d]\n", p);
			it ^= c[p];
			c[p] ^= it;
			it ^= c[p];
			//print_pairs(c);
			writes++;
			if (swaps != NULL)
				swaps->push_back(std::make_pair(-1, p));
		}
	}
}

void relabel_vector(std::vector<unsigned int> &output, std::vector<unsigned int> input)
{
	#if DEBUG
	assert (input.size() == output.size());
	#endif

	size_t len = input.size();
	std::vector<unsigned int> out(len);

	for (size_t i = 0; i < len; i++)
		for (size_t j = 0; j < len; j++)
			if (output[j] == input[i])
				out[j] = i;

	output = out;
}

void get_most_similar(std::vector<unsigned int> &sim, unsigned int &nsteps, std::vector<std::vector<unsigned int> > candidates, std::vector<unsigned int> current)
{
	unsigned int idx = candidates.size();
	nsteps = 100000000;

	for (size_t i = 0; i < candidates.size(); i++) {
		size_t len = candidates[i].size();
		std::vector<unsigned int> c(len);

		/*printf("Current: ");
		for (size_t j = 0; j < len; j += 2)
			printf("(%d, %d) ", current[j], current[j+1]);
		printf("\n");
		
		printf("First candidate: ");
		for (size_t j = 0; j < len; j += 2)
			printf("(%d, %d) ", candidates[i][j], candidates[i][j+1]);
		printf("\n");*/

		//Relabel
		/*for (size_t j = 0; j < len; j++)
			for (size_t k = 0; k < len; k++)
				if (candidates[i][k] == current[j])
					c[k] = j;*/
		c = candidates[i];
		relabel_vector(c, current);
		
		//printf("Relabeled permutation: ");
		//print_pairs(candidates[i]);
		/*for (size_t j = 0; j < len; j += 2)
			printf("(%d, %d) ", c[j], c[j+1]);
		printf("\n");*/

		//Cycle Sort
		//I/O: writes
		//I: c

		unsigned int writes;
		//cyclesort(writes, candidates[i], NULL);
		cyclesort(writes, c, NULL);
		//printf("Configuration [%zd] requires %d memory transfers.\n", i, writes);
		if (writes < nsteps) {
			nsteps = writes;
			idx = i;
		}
	}

	sim = candidates[idx];
}

#ifdef MPI_ENABLED
//NOTE: Add macro for MPI_UINT64_T to fastbitset
void mpi_swaps(std::vector<std::pair<int,int> > swaps, Bitvector &adj, Bitvector &adj_buf, const int N_tar, const int num_mpi_threads, const int rank)
{
	int mpi_offset = N_tar / (num_mpi_threads << 1);
	int cpy_offset = mpi_offset / num_mpi_threads;
	int start = rank * cpy_offset;
	int finish = start + cpy_offset;
	MPI_Status status;

	/*MPI_Barrier(MPI_COMM_WORLD);
	printf("Rank [%d] beginning mpi_swaps.\n", rank);
	fflush(stdout); sleep(1);
	MPI_Barrier(MPI_COMM_WORLD);*/

	for (size_t i = 0; i < swaps.size(); i++) {
		int idx0 = std::get<0>(swaps[i]);
		int idx1 = std::get<1>(swaps[i]);

		//printf("Rank [%d] identified swap [%d->%d].\n", rank, idx0, idx1);
		//fflush(stdout); sleep(1);
		MPI_Barrier(MPI_COMM_WORLD);
		if (idx0 != -1 && idx1 != -1) {
			//if (rank << 1 != idx0 && (rank << 1) + 1 != idx0) break;
			//if (rank << 1 != idx1 && (rank << 1) + 1 != idx1) break;
			if (idx0 >> 1 == idx1 >> 1 && (rank<<1)+(idx0%2) != idx0) continue;
		}

		//printf_mpi(rank, "swap number: %d\n", i);
		//if (!rank) fflush(stdout);
		//MPI_Barrier(MPI_COMM_WORLD);
		//printf_mpi(rank, "rank: 0\tidx0: %d\tidx1: %d\n", idx0, idx1);
		//fflush(stdout);
		//MPI_Barrier(MPI_COMM_WORLD);
		//printf_mpi(rank - 1, "rank: 1\tidx0: %d\tidx1: %d\n", idx0, idx1);
		//fflush(stdout);

		if (idx0 == -1) {	//Copy buffer to idx1
			int buf_offset = mpi_offset * (idx1 % 2);
			start = rank * cpy_offset;
			finish = start + cpy_offset;
			//printf("rank [%d] start: %d\n", rank, start);
			//printf("rank [%d] finish: %d\n", rank, finish);
			//fflush(stdout);
			//MPI_Barrier(MPI_COMM_WORLD);
			for (int j = start; j < finish; j++) {
				int loc_idx = j % cpy_offset;
				for (int k = 0; k < num_mpi_threads; k++) {
					int adj_idx = buf_offset + k * cpy_offset + loc_idx;
					if ((rank<<1) + (idx1%2) == idx1) {
						if (k == rank) {
							//printf("Rank [%d] copying to adj row [%d] to from local buffer row [%d]\n", rank, adj_idx, loc_idx);
							memcpy(adj[adj_idx].getAddress(), adj_buf[loc_idx].getAddress(), sizeof(BlockType) * adj_buf[loc_idx].getNumBlocks());
						}
						if (k == rank) continue;
						//printf("Rank [%d] receiving adj row [%d] from rank [%d]\n", rank, adj_idx, k);
						MPI_Recv(adj[adj_idx].getAddress(), adj[adj_idx].getNumBlocks(), MPI_UINT64_T, k, 0, MPI_COMM_WORLD, &status);
					} else {
						if ((k<<1)+(idx1%2)!=idx1) continue;
						//printf("Rank [%d] sending [%d] to rank [%d]\n", rank, loc_idx, k);
						MPI_Send(adj_buf[loc_idx].getAddress(), adj_buf[loc_idx].getNumBlocks(), MPI_UINT64_T, k, 0, MPI_COMM_WORLD);
					}
				}
			}
		} else if (idx1 == -1) {	//Copy idx0 to buffer
			//int buf_offset = (mpi_offset >> 1) * (idx0 % 2);
			int buf_offset = mpi_offset * (idx0 % 2);
			start = rank * cpy_offset;
			finish = start + cpy_offset;
			//printf_mpi(rank, "buf_offset: %d\n", buf_offset);
			//printf_mpi(rank, "cpy_offset: %d\n", cpy_offset);
			//fflush(stdout);
			//MPI_Barrier(MPI_COMM_WORLD);
			//printf("rank [%d] start: %d\n", rank, start);
			//printf("rank [%d] finish: %d\n", rank, finish);
			//fflush(stdout);
			//MPI_Barrier(MPI_COMM_WORLD);
			for (int j = start; j < finish; j++) {
				int loc_idx = j % cpy_offset;
				for (int k = 0; k < num_mpi_threads; k++) {
					int adj_idx = buf_offset + k * cpy_offset + loc_idx;
					if ((rank<<1) + (idx0%2) == idx0) {
						if (k == rank) {
							//printf("Rank [%d] copying adj row [%d] to local buffer row [%d]\n", rank, adj_idx, loc_idx);
							memcpy(adj_buf[loc_idx].getAddress(), adj[adj_idx].getAddress(), sizeof(BlockType) * adj[adj_idx].getNumBlocks());
						} else {
							//printf("Rank [%d] sending adj row [%d] to rank [%d]\n", rank, adj_idx, k);
							MPI_Send(adj[adj_idx].getAddress(), adj[adj_idx].getNumBlocks(), MPI_UINT64_T, k, 0, MPI_COMM_WORLD);
						}
					//} else if ((rank<<1) + (idx0%2) != idx0) {
					} else {
						if ((k<<1)+(idx0%2)!=idx0) continue;
						//printf("Rank [%d] receiving to local buffer [%d] from rank [%d]\n", rank, j % cpy_offset, k);
						MPI_Recv(adj_buf[loc_idx].getAddress(), adj_buf[loc_idx].getNumBlocks(), MPI_UINT64_T, k, 0, MPI_COMM_WORLD, &status);
					}
				}
			}

			//fflush(stdout);
			//sleep(1);
			//MPI_Barrier(MPI_COMM_WORLD);

			//Print buffer
			/*printf_mpi(rank, "\nPrinting Buffers:\n");
			if (!rank) fflush(stdout);
			for (int j = 0; j < num_mpi_threads; j++) {
				MPI_Barrier(MPI_COMM_WORLD);
				if (j == rank) {
					for (int k = 0; k < cpy_offset; k++) {
						for (int l = 0; l < N_tar; l++)
							printf("%" PRIu64 " ", adj_buf[k].read(l));
						printf(" [%d]\n", rank);
						fflush(stdout);
					}
					if (j != num_mpi_threads - 1) {
						for (int k = 0; k < N_tar; k++)
							printf("==");
						printf("\n");
						fflush(stdout);
					}
					sleep(1);
				}
			}
			sleep(1);
			printf_mpi(rank, "\n");
			if (!rank) fflush(stdout);
			MPI_Barrier(MPI_COMM_WORLD);*/
		} else {
			int buf_offset0 = mpi_offset * (idx0%2);
			int buf_offset1 = mpi_offset * (idx1%2);
			start = rank * mpi_offset;
			finish = start + mpi_offset;
			//printf("rank [%d] start: %d\n", rank, start);
			//printf("rank [%d] finish: %d\n", rank, finish);
			//fflush(stdout);
			//MPI_Barrier(MPI_COMM_WORLD);
			for (int j = start; j < finish; j++) {
				int loc_idx = j % mpi_offset;
				if (idx0>>1==idx1>>1) {
					//printf("Rank [%d] copying adj row [%d] to adj row [%d]\n", rank, buf_offset0+loc_idx, buf_offset1+loc_idx);
					memcpy(adj[buf_offset1+loc_idx].getAddress(), adj[buf_offset0+loc_idx].getAddress(), sizeof(BlockType) * adj[buf_offset0+loc_idx].getNumBlocks());
				} else if ((rank<<1)+(idx0%2)==idx0) {
					//printf("Rank [%d] sending adj row [%d] to rank [%d]\n", rank, buf_offset0+j%mpi_offset, idx1/2);
					MPI_Send(adj[buf_offset0+loc_idx].getAddress(), adj[buf_offset0+loc_idx].getNumBlocks(), MPI_UINT64_T, idx1/2, 0, MPI_COMM_WORLD);
				} else if ((rank<<1)+(idx1%2)==idx1) {
					//printf("Rank [%d] receiving to adj row [%d] from rank [%d]\n", rank, buf_offset1+j%mpi_offset, idx0/2);
					MPI_Recv(adj[buf_offset1+loc_idx].getAddress(), adj[buf_offset1+loc_idx].getNumBlocks(), MPI_UINT64_T, idx0/2, 0, MPI_COMM_WORLD, &status);
				}
			}
		}

		//MPI_Barrier(MPI_COMM_WORLD);
		//printf_mpi(rank, "\n");
		//fflush(stdout);
		//sleep(1);
		//MPI_Barrier(MPI_COMM_WORLD);
		//break;
	}

	/*printf("Rank [%d] finished mpi_swaps.\n", rank);
	fflush(stdout); sleep(1);
	MPI_Barrier(MPI_COMM_WORLD);*/
}
#endif

unsigned int loc_to_glob_idx(std::vector<unsigned int> config, unsigned int idx, const int N_tar, const int num_mpi_threads, const int rank)
{
	int mpi_offset = N_tar / (num_mpi_threads << 1);
	int loc_idx = idx - (idx / mpi_offset) * mpi_offset;
	int seq_idx = config[(rank << 1) + (idx / mpi_offset)];
	int glob_idx = seq_idx * mpi_offset + loc_idx;

	return glob_idx;
}
