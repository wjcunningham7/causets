#include "hip/hip_runtime.h"
#include "Subroutines.h"

/////////////////////////////
//(C) Will Cunningham 2014 //
//         DK Lab          //
// Northeastern University //
/////////////////////////////

//Linear Interpolation using Lookup Table
bool getLookupTable(const char *filename, double **lt, long *size)
{
	#if DEBUG
	assert (filename != NULL);
	assert (lt != NULL);
	assert (size != NULL);
	#endif

	double *table;
	std::ifstream ltable(filename, std::ios::in | std::ios::binary | std::ios::ate);

	try {
		if (ltable.is_open()) {
			//Find size of file
			*size = ltable.tellg();

			if (*size == 0)
				throw CausetException("Lookup table file is empty!\n");

			//Allocate Memory for Buffer
			char *memblock = (char*)malloc(*size);
			if (memblock == NULL)
				throw std::bad_alloc();

			//Allocate Memory for Lookup Table
			table = (double*)malloc(*size);
			if (table == NULL)
				throw std::bad_alloc();

			//Read File
			ltable.seekg(0, std::ios::beg);
			ltable.read(memblock, *size);
			memcpy(table, memblock, *size);

			//Free Memory
			free(memblock);
			memblock = NULL;

			//Close Stream
			ltable.close();
		} else
			throw CausetException("Failed to open lookup table file!\n");

		//Return Table
		*lt = table;
	} catch (CausetException c) {
		fprintf(stderr, "CausetException in %s: %s on line %d\n", __FILE__, c.what(), __LINE__);
		return false;
	} catch (std::bad_alloc) {
		fprintf(stderr, "Memory allocation failure in %s on line %d!\n", __FILE__, __LINE__);
		return false;
	} catch (std::exception e) {
		fprintf(stderr, "Unknown Exception in %s: %s on line %d\n", __FILE__, e.what(), __LINE__);
		return false;
	}

	return true;
}

//Lookup value in table of (x, y) coordinates -> 2D parameter space
double lookupValue(const double *table, const long &size, double *x, double *y, bool increasing)
{
	#if DEBUG
	assert (table != NULL);
	assert (size > 0);
	assert ((x == NULL) ^ (y == NULL));
	#endif
	
	//Identify which is being calculated
	bool first = (x == NULL);
	//Identify input value
	double input = first ? *y : *x;
	double output = 0.0;
	int t_idx = (int)(!first);
	int i;

	try {
		//Identify Value in Table
		//Assumes values are written (y, x)
		for (i = (int)(!first); i < size / (int)sizeof(double); i += 2) {
			if ((increasing && table[i] >= input) || (!increasing && table[i] <= input)) {
				t_idx = i;
				break;
			}
		}

		//Check if Table is Insufficient
		if (t_idx == (int)(!first) && input != table[i]) {
			//printf("%f\n", input);
			throw CausetException("Values from lookup table do not include requested input.  Recreate table or change input.\n");
		}

		//Linear Interpolation
		if (first)
			output = table[t_idx-1] + (table[t_idx+1] - table[t_idx-1]) * (input - table[t_idx-2]) / (table[t_idx] - table[t_idx-2]);
		else
			output = table[t_idx-3] + (table[t_idx-1] - table[t_idx-3]) * (input - table[t_idx-2]) / (table[t_idx] - table[t_idx-2]);
	} catch (CausetException c) {
		fprintf(stderr, "CausetException in %s: %s on line %d\n", __FILE__, c.what(), __LINE__);
		output = std::numeric_limits<double>::quiet_NaN();
	} catch (std::exception e) {
		fprintf(stderr, "Unknown Exception in %s: %s on line %d\n", __FILE__, e.what(), __LINE__);
		output = std::numeric_limits<double>::quiet_NaN();
	}

	return output;
}

//Lookup value in table of (t1, t2, omega12, lambda) coordinates -> 4D parameter space
//Used for geodesic distance calculations
//Returns the transcendental integration parameter 'lambda'
double lookupValue4D(const double *table, const long &size, const double &omega12, double t1, double t2)
{
	#if DEBUG
	assert (table != NULL);
	assert (size > 0);
	assert (omega12 >= 0.0);
	assert (t1 >= 0.0);
	assert (t2 >= 0.0);
	#endif

	if (t2 < t1) {
		double temp = t1;
		t1 = t2;
		t2 = temp;
	}

	double lambda = 0.0;
	double tol = 1e-2;
	int tau_step, lambda_step, step;
	int counter;
	int i;

	try {
		//The first two table elements should be zero
		if (table[0] != 0.0 || table[1] != 0.0)
			throw CausetException("Corrupted lookup table!\n");

		tau_step = table[2];
		lambda_step = table[3];
		step = 4 * tau_step * lambda_step;
		counter = 0;

		//Identify Value in Table
		//Assumes values are written (tau1, tau2, omega12, lambda)
		for (i = 4; i < size / (int)sizeof(double); i += step) {
			counter++;

			if (step == 4 * tau_step * lambda_step && table[i] > t1) {
				i -= (step - 1);
				step = 4 * lambda_step;
				i -= step;
				counter = 0;
			} else if (step == 4 * lambda_step && table[i] > t2) {
				i -= (step - 1);
				step = 4;
				i -= step;
				counter = 0;
			} else if (step == 4 && ABS(table[i] - omega12, STL) / omega12 < tol && table[i] != 0.0) {
				i -= step;
				step = 1;
			} else if (step == 1) {
				lambda = table[i];
				break;
			}

			if ((step == 4 * tau_step * lambda_step && counter == tau_step) ||
			    (step == 4 * lambda_step && counter == tau_step) ||
			    (step == 4 && counter == lambda_step))
				break;
		}

		//Perhaps do some linear interpolation here?

		//If no value found
		if (lambda == 0.0) {
			if (step == 4 * tau_step * lambda_step)
				throw CausetException("tau1 value not found in geodesic lookup table.\n");
			else if (step == 4 * lambda_step)
				throw CausetException("tau2 value not found in geodesic lookup table.\n");
			else if (step == 4)
				throw CausetException("omega12 value not found in geodesic lookup table.\n");
			else if (step == 1)
				throw std::exception();
		}
	} catch (CausetException c) {
		fprintf(stderr, "CausetException in %s: %s on line %d\n", __FILE__, c.what(), __LINE__);
		lambda = std::numeric_limits<double>::quiet_NaN();
	} catch (std::exception e) {
		fprintf(stderr, "Unknown Exception in %s: %s on line %d\n", __FILE__, e.what(), __LINE__);
		lambda = std::numeric_limits<double>::quiet_NaN();
	}

	return lambda;
}

//Sort nodes temporally
//O(N*log(N)) Efficiency
void quicksort(Node &nodes, const unsigned int &spacetime, int low, int high)
{
	#if DEBUG
	assert (!nodes.crd->isNull());
	assert (get_stdim(spacetime) & (2 | 4));
	assert (get_manifold(spacetime) & (MINKOWSKI | DE_SITTER | DUST | FLRW | HYPERBOLIC));
	if (get_manifold(spacetime) & HYPERBOLIC)
		assert (get_stdim(spacetime) == 2);
	#endif

	int i, j, k;
	float key = 0.0;
	#if EMBED_NODES
	float *& time = get_stdim(spacetime) == 2 ? nodes.crd->x() : nodes.crd->v();
	#else
	float *& time = get_stdim(spacetime) == 2 ? nodes.crd->x() : nodes.crd->w();
	#endif

	if (low < high) {
		k = (low + high) >> 1;
		swap(nodes, spacetime, low, k);
		key = time[low];
		i = low + 1;
		j = high;

		while (i <= j) {
			while ((i <= high) && (time[i] <= key))
				i++;
			while ((j >= low) && (time[j] > key))
				j--;
			if (i < j)
				swap(nodes, spacetime, i, j);
		}

		swap(nodes, spacetime, low, j);
		quicksort(nodes, spacetime, low, j - 1);
		quicksort(nodes, spacetime, j + 1, high);
	}
}

//Sort edge list
//O(N*log(N)) Efficiency
void quicksort(uint64_t *edges, int64_t low, int64_t high)
{
	#if DEBUG
	assert (edges != NULL);
	#endif

	int64_t i, j, k;
	uint64_t key;

	if (low < high) {
		k = (low + high) >> 1;
		swap(edges, low, k);
		key = edges[low];
		i = low + 1;
		j = high;

		while (i <= j) {
			while ((i <= high) && (edges[i] <= key))
				i++;
			while ((j >= low) && (edges[j] > key))
				j--;
			if (i < j)
				swap(edges, i, j);
		}

		swap(edges, low, j);
		quicksort(edges, low, j - 1);
		quicksort(edges, j + 1, high);
	}
}

//Exchange two nodes
void swap(Node &nodes, const unsigned int &spacetime, const int i, const int j)
{
	#if DEBUG
	assert (!nodes.crd->isNull());
	assert (get_stdim(spacetime) & (2 | 4));
	assert (get_manifold(spacetime) & (MINKOWSKI | DE_SITTER | DUST | FLRW | HYPERBOLIC));
	if (get_manifold(spacetime) & HYPERBOLIC)
		assert (get_stdim(spacetime) == 2);
	#endif

	#if EMBED_NODES
	if (get_stdim(spacetime) == 2) {
		float3 hc = nodes.crd->getFloat3(i);
		nodes.crd->setFloat3(nodes.crd->getFloat3(j), i);
		nodes.crd->setFloat3(hc, j);
	} else if (get_stdim(spacetime) == 4) {
		float5 sc = nodes.crd->getFloat5(i);
		nodes.crd->setFloat5(nodes.crd->getFloat5(j), i);
		nodes.crd->setFloat5(sc, j);
	}
	#else
	if (get_stdim(spacetime) == 2) {
		float2 hc = nodes.crd->getFloat2(i);
		nodes.crd->setFloat2(nodes.crd->getFloat2(j), i);
		nodes.crd->setFloat2(hc, j);
	} else if (get_stdim(spacetime) == 4) {
		float4 sc = nodes.crd->getFloat4(i);
		nodes.crd->setFloat4(nodes.crd->getFloat4(j), i);
		nodes.crd->setFloat4(sc, j);
	}
	#endif

	if (get_manifold(spacetime) & (DE_SITTER | DUST | FLRW)) {
		float tau = nodes.id.tau[i];
		nodes.id.tau[i] = nodes.id.tau[j];
		nodes.id.tau[j] = tau;
	} else if (get_manifold(spacetime) & HYPERBOLIC) {
		int AS = nodes.id.AS[i];
		nodes.id.AS[i] = nodes.id.AS[j];
		nodes.id.AS[j] = AS;
	}
}

//Exchange two edges
void swap(uint64_t *edges, const int64_t i, const int64_t j)
{
	#if DEBUG
	assert (edges != NULL);
	assert (i >= 0);
	assert (j >= 0);
	#endif

	uint64_t tmp = edges[i];
	edges[i] = edges[j];
	edges[j] = tmp;
}

//Exchange references to two lists
//as well as related indices (used in causet_intersection)
void swap(const int * const *& list0, const int * const *& list1, int64_t &idx0, int64_t &idx1, int64_t &max0, int64_t &max1)
{
	#if DEBUG
	assert (idx0 >= 0);
	assert (idx1 >= 0);
	assert (max0 >= 0);
	assert (max1 >= 0);
	#endif

	const int * const * tmp_list = list0;
	list0 = list1;
	list1 = tmp_list;

	//Bitwise swaps
	idx0 ^= idx1;
	idx1 ^= idx0;
	idx0 ^= idx1;

	max0 ^= max1;
	max1 ^= max0;
	max0 ^= max1;
}

//Cyclesort
//Comparison sort which performs minimum swaps
//This algorithm returns the number of writes, and the sequence
//of swaps if a non-null vector is passed as the third argument
//NOTE: It is important the integers in 'elements' are unique for this to work
//O(N^2) Efficiency
void cyclesort(unsigned int &writes, std::vector<unsigned int> elements, std::vector<std::pair<int,int> > *swaps)
{
	unsigned int it, p;
	size_t len = elements.size();
	writes = 0;
	for (size_t j = 0; j < len - 1; j++) {
		it = elements[j];
		p = j;

		for (unsigned int k = j + 1; k < len; k++)
			if (elements[k] < it)
				p++;

		if (j == p) continue;

		//Swap
		while (elements[p] == it)
			p++;
		it ^= elements[p];
		elements[p] ^= it;
		it ^= elements[p];
		writes += 2;
		if (swaps != NULL) {
			swaps->push_back(std::make_pair(p, -1));
			swaps->push_back(std::make_pair(j, p));
		}

		while (j != p) {
			p = j;
			for (size_t k = j + 1; k < len; k++)
				if (elements[k] < it)
					p++;
			
			while (elements[p] == it)
				p++;
			it ^= elements[p];
			elements[p] ^= it;
			it ^= elements[p];
			writes++;
			if (swaps != NULL)
				swaps->push_back(std::make_pair(-1, p));
		}
	}
}

//Bisection Method
//Use when Newton-Raphson fails
bool bisection(double (*solve)(const double &x, const double * const p1, const float * const p2, const int * const p3), double *x, const int max_iter, const double lower, const double upper, const double tol, const bool increasing, const double * const p1, const float * const p2, const int * const p3)
{
	#if DEBUG
	assert (solve != NULL);
	#endif

	double res = 1.0;
	double a = lower;
	double b = upper;
	int iter = 0;

	try {
		if (b <= a)
			throw CausetException("Invalid Bounds in Bisection!\n");

		//Initial test point
		*x = (b + a) / 2;
		while (ABS(res, STL) > tol && iter < max_iter) {
			//Residual Value
			res = (*solve)(*x, p1, p2, p3);
			//printf("res:   %.16e\n\n", res);
			//printf("x: %.16e\n\n", *x);

			//Check for NaN
			if (res != res)
				throw CausetException("NaN Error in Bisection!\n");

			//Change bounds
			if (increasing) {
				if (res > 0)
					b = *x;
				else
					a = *x;
			} else {
				if (res > 0)
					a = *x;
				else
					b = *x;
			}

			//New test point
			*x = (b + a) / 2;
			iter++;
		}
	} catch (CausetException c) {
		fprintf(stderr, "CausetException in %s: %s on line %d\n", __FILE__, c.what(), __LINE__);
		return false;
	} catch (std::exception e) {
		fprintf(stderr, "Unknown Exception in %s: %s on line %d\n", __FILE__, e.what(), __LINE__);
		return false;
	}
	
	//printf("Bisection Results:\n");
	//printf("Tolerance: %E\n", tol);
	//printf("%d of %d iterations performed.\n", iter, max_iter);
	//printf("Residual: %E\n", y - res);
	//printf("Solution: %E\n", *x);
	//fflush(stdout);

	return true;
}

//Newton-Raphson Method
//Solves Transcendental Equations
bool newton(double (*solve)(const double &x, const double * const p1, const float * const p2, const int * const p3), double *x, const int max_iter, const double tol, const double * const p1, const float * const p2, const int * const p3)
{
	#if DEBUG
	assert (solve != NULL);
	#endif

	double res = 1.0;
	double x1;
	int iter = 0;

	try {
		while (ABS(res, STL) > tol && iter < max_iter) {
			//Residual Value
			res = (*solve)(*x, p1, p2, p3);
			//printf("res: %E\n", res);

			//Check for NaN
			if (res != res)
				throw CausetException("NaN Error in Newton-Raphson\n");
	
			//New test value
			x1 = *x + res;
			//printf("x1: %E\n", x1);
	
			*x = x1;
			iter++;

			fflush(stdout);
		}
	} catch (CausetException c) {
		fprintf(stderr, "CausetException in %s: %s on line %d\n", __FILE__, c.what(), __LINE__);
		return false;
	} catch (std::exception e) {
		fprintf(stderr, "Unknown Exception in %s: %s on line %d\n", __FILE__, e.what(), __LINE__);
		return false;
	}

	//printf("Newton-Raphson Results:\n");
	//printf("Tolerance: %E\n", tol);
	//printf("%d of %d iterations performed.\n", iter, max_iter);
	//printf("Residual: %E\n", res);
	//printf("Solution: %E\n", *x);
	//fflush(stdout);

	return true;
}

//Returns true if two nodes are causally connected
//Note: past_idx must be less than future_idx
//O(1) Efficiency for Adjacency Matrix
//O(<k>) Efficiency for Adjacency List
bool nodesAreConnected(const Node &nodes, const int * const future_edges, const int64_t * const future_edge_row_start, const Bitvector &adj, const int &N_tar, const float &core_edge_fraction, int past_idx, int future_idx)
{
	#if DEBUG
	//No null pointers
	assert (future_edges != NULL);
	assert (future_edge_row_start != NULL);

	//Parameters in correct ranges
	assert (core_edge_fraction >= 0.0f && core_edge_fraction <= 1.0f);
	assert (past_idx >= 0 && past_idx < N_tar);
	assert (future_idx >= 0 && future_idx < N_tar);
	assert (past_idx != future_idx);

	assert (!(future_edge_row_start[past_idx] == -1 && nodes.k_out[past_idx] > 0));
	assert (!(future_edge_row_start[past_idx] != -1 && nodes.k_out[past_idx] == 0));
	#endif

	int core_limit = static_cast<int>(core_edge_fraction * N_tar);
	int i;

	//Make sure past_idx < future_idx
	if (past_idx > future_idx) {
		past_idx ^= future_idx;
		future_idx ^= past_idx;
		past_idx ^= future_idx;
	}

	//Check if the adjacency matrix can be used
	if (past_idx < core_limit && future_idx < core_limit)
		return (bool)adj[past_idx].read(future_idx);
	//Check if past node is not connected to anything
	else if (future_edge_row_start[past_idx] == -1)
		return false;
	//Check adjacency list
	else
		for (i = 0; i < nodes.k_out[past_idx]; i++)
			if (future_edges[future_edge_row_start[past_idx] + i] == future_idx)
				return true;

	return false;
}

//Returns true if two nodes are causally connected
//Note: past_idx must be less than future_idx
//O(1) Efficiency
bool nodesAreConnected_v2(const Bitvector &adj, const int &N_tar, int past_idx, int future_idx)
{
	#if DEBUG
	assert (past_idx >= 0 && past_idx < N_tar);
	assert (future_idx >= 0 && future_idx < N_tar);
	//assert (past_idx != future_idx);
	#endif

	return (bool)adj[past_idx].read(future_idx);
}

//Breadth First Search
//O(N+E) Efficiency
void bfsearch(const Node &nodes, const Edge &edges, const int index, const int id, int &elements)
{
	#if DEBUG
	assert (nodes.k_in != NULL);
	assert (nodes.k_out != NULL);
	assert (nodes.cc_id != NULL);
	assert (edges.past_edges != NULL);
	assert (edges.future_edges != NULL);
	assert (edges.past_edge_row_start != NULL);
	assert (edges.future_edge_row_start != NULL);
	assert (index >= 0);
	assert (id >= 0);
	assert (elements >= 0);
	#endif

	int64_t ps = edges.past_edge_row_start[index];
	int64_t fs = edges.future_edge_row_start[index];
	int i;

	nodes.cc_id[index] = id;
	elements++;

	//Move to past nodes
	for (i = 0; i < nodes.k_in[index]; i++)
		if (!nodes.cc_id[edges.past_edges[ps+i]])
			bfsearch(nodes, edges, edges.past_edges[ps+i], id, elements);

	//Move to future nodes
	for (i = 0; i < nodes.k_out[index]; i++)
		if (!nodes.cc_id[edges.future_edges[fs+i]])
			bfsearch(nodes, edges, edges.future_edges[fs+i], id, elements);
}

//Breadth First Search
//Uses adjacency matrix only
//O(N+E) Efficiency
void bfsearch_v2(const Node &nodes, const Bitvector &adj, const int &N_tar, const int index, const int id, int &elements)
{
	#if DEBUG
	assert (nodes.cc_id != NULL);
	assert (N_tar >= 0);
	assert (index >= 0 && index < N_tar);
	assert (id >= 0 && id <= N_tar / 2);
	assert (elements >= 0 && elements < N_tar);
	#endif

	nodes.cc_id[index] = id;
	elements++;

	int i;
	for (i = 0; i < N_tar; i++)
		if (adj[index].read(i) && !nodes.cc_id[i])
			bfsearch_v2(nodes, adj, N_tar, i, id, elements);
}

//Modification of v1 which eliminates swaps
//O(k*log(k)) Efficiency
void causet_intersection_v2(int &elements, const int * const past_edges, const int * const future_edges, const int &k_i, const int &k_o, const int &max_cardinality, const int64_t &pstart, const int64_t &fstart, bool &too_many)
{
	#if DEBUG
	assert (past_edges != NULL);
	assert (future_edges != NULL);
	assert (k_i >= 0);
	assert (k_o >= 0);
	assert (!(k_i == 0 && k_o == 0));
	assert (max_cardinality > 1);
	assert (pstart >= 0);
	assert (fstart >= 0);
	#endif

	if (k_i == 1 || k_o == 1) {
		elements = 0;
		return;
	}

	int64_t idx0 = pstart;	//Index of past neighbors of 'future element j'
	int64_t idx1 = fstart;	//Index of future neighbors of 'past element i'
	int64_t max0 = idx0 + k_i;
	int64_t max1 = idx1 + k_o;

	while (idx0 < max0 && idx1 < max1 && !too_many) {
		if (past_edges[idx0] > future_edges[idx1])
			idx1++;
		else if (past_edges[idx0] < future_edges[idx1])
			idx0++;
		else {
			elements++;

			if (elements >= max_cardinality - 1) {
				too_many = true;
				break;
			}

			idx0++;
			idx1++;
		}
	}
}

//Intersection of Sorted Lists
//Used to find the cardinality of an interval
//O(k*log(k)) Efficiency
void causet_intersection(int &elements, const int * const past_edges, const int * const future_edges, const int &k_i, const int &k_o, const int &max_cardinality, const int64_t &pstart, const int64_t &fstart, bool &too_many)
{
	#if DEBUG
	assert (past_edges != NULL);
	assert (future_edges != NULL);
	assert (k_i >= 0);
	assert (k_o >= 0);
	assert (!(k_i == 0 && k_o == 0));
	assert (max_cardinality > 1);
	assert (pstart >= 0);
	assert (fstart >= 0);
	#endif

	int64_t idx0 = pstart;
	int64_t idx1 = fstart;
	int64_t max0 = idx0 + k_i;
	int64_t max1 = idx1 + k_o;

	if (k_i == 1 || k_o == 1) {
		elements = 0;
		return;
	}

	//Pointers are used here so that 'primary' and 'secondary'
	//can be switched as needed.  References are static, so they
	//cannot be used.  The 'const' specifiers are kept since the
	//edge list values and their locations in memory should
	//not be modified in this algorithm.

	const int * const * primary = &past_edges;
	const int * const * secondary = &future_edges;

	while (idx0 < max0 && idx1 < max1) {
		if ((*secondary)[idx1] > (*primary)[idx0])
			swap(primary, secondary, idx0, idx1, max0, max1);

		while (idx1 < max1 && (*secondary)[idx1] < (*primary)[idx0])
			idx1++;

		if (idx1 == max1)
			break;

		if ((*primary)[idx0] == (*secondary)[idx1]) {
			elements++;
			if (elements >= max_cardinality - 1) {
				too_many = true;
				return;
			}
			idx0++;
			idx1++;
		}
	}
}

//Data formatting used when reading the degree
//sequences found on the GPU
void readDegrees(int * const &degrees, const int * const h_k, const size_t &offset, const size_t &size)
{
	#if DEBUG
	assert (degrees != NULL);
	assert (h_k != NULL);
	#endif

	unsigned int i;
	for (i = 0; i < size; i++)
		degrees[offset+i] += h_k[i];
}

//Data formatting used when reading output of
//the adjacency list created by the GPU
void readEdges(uint64_t * const &edges, const bool * const h_edges, Bitvector &adj, int64_t * const &g_idx, const unsigned int &core_limit_row, const unsigned int &core_limit_col, const size_t &d_edges_size, const size_t &mthread_size, const size_t &size0, const size_t &size1, const int x, const int y, const bool &use_bit, const bool &use_mpi)
{
	#if DEBUG
	if (!use_bit)
		assert (edges != NULL);
	assert (h_edges != NULL);
	assert (g_idx != NULL);
	assert (*g_idx >= 0);
	assert (x >= 0);
	assert (y >= 0);
	//assert (x <= y);
	#endif

	//printf("x: %d\tsize0: %zd\n", x, size0);
	//printf("I have a bitvector of length %zd\n", adj.size());

	unsigned int i, j;
	for (i = 0; i < size0; i++) {
		for (j = 0; j < size1; j++) {
			if (h_edges[i*mthread_size+j] && (use_bit || g_idx[0] < (int64_t)d_edges_size)) {
				if (!use_bit)
					edges[g_idx[0]++] = (static_cast<uint64_t>(x*mthread_size+i)) << 32 | (static_cast<uint64_t>(y*mthread_size+j));
				else
					g_idx[0]++;
				if (x*mthread_size+i < core_limit_row && y*mthread_size+j < core_limit_col) {
					adj[x*mthread_size+i].set(y*mthread_size+j);
					if (!use_mpi)
						adj[y*mthread_size+j].set(x*mthread_size+i);
				}
			}
		}
	}
}

//Remake adjacency sub-matrix using 'l' rows, beginning at row 'i'
void remakeAdjMatrix(bool * const adj0, bool * const adj1, const int * const k_in, const int * const k_out, const int * const past_edges, const int * const future_edges, const int64_t * const past_edge_row_start, const int64_t * const future_edge_row_start, int * const idx_buf0, int * const idx_buf1, const int &N_tar, const int &i, const int &j, const int64_t &l)
{
	#if DEBUG
	assert (adj0 != NULL);
	assert (adj1 != NULL);
	assert (k_in != NULL);
	assert (k_out != NULL);
	assert (past_edges != NULL);
	assert (future_edges != NULL);
	assert (past_edge_row_start != NULL);
	assert (future_edge_row_start != NULL);
	assert (idx_buf0 != NULL);
	assert (idx_buf1 != NULL);
	assert (N_tar > 0);
	assert (i >= 0);
	assert (j >= 0);
	assert (l > 0);
	#endif

	//Map tile indices to global indices
	for (int m = 0; m < l; m++) {
		int M = m + i * l;
		for (int n = 0; n < l; n++) {
			int N = n + j * l;
			if (!N)
				continue;

			//Use triangular mapping
			int do_map = M >= N;
			if (N < N_tar >> 1) {
				M = M + do_map * ((((N_tar >> 1) - M) << 1) - 1);
				N = N + do_map * (((N_tar >> 1) - N) << 1);
			}

			idx_buf0[m] = M;
			idx_buf1[n] = N;
		}
	}

	//Fill Adjacency Submatrix 0
	#ifdef _OPENMP
	#pragma omp parallel for schedule (dynamic, 1)
	#endif
	for (int m = 0; m < l; m++) {
		int M = idx_buf0[m];
		int element;

		//Past Neighbors
		int64_t start = past_edge_row_start[M];
		for (int p = 0; p < k_in[M]; p++) {
			element = past_edges[start+p];
			adj0[m*N_tar+element] = true;
		}

		//Future Neighbors
		start = future_edge_row_start[M];
		for (int p = 0; p < k_out[M]; p++) {
			element = future_edges[start+p];
			adj0[m*N_tar+element] = true;
		}
	}

	//Fill Adjacency Submatrix 1
	#ifdef _OPENMP
	#pragma omp parallel for schedule (dynamic, 1)
	#endif
	for (int n = 0; n < l; n++) {
		int N = idx_buf1[n];
		int element;

		if (!N)
			continue;

		//Past Neighbors
		int64_t start = past_edge_row_start[N];
		for (int p = 0; p < k_in[N]; p++) {
			element = past_edges[start+p];
			adj1[n*N_tar+element] = true;
		}

		//Future Neighbors
		start = future_edge_row_start[N];
		for (int p = 0; p < k_out[N]; p++) {
			element = future_edges[start+p];
			adj1[n*N_tar+element] = true;
		}
	}
}

//Data formatting used when reading output of
//the interval matrix created by the GPU
void readIntervals(int * const cardinalities, const unsigned int * const N_ij, const int &l)
{
	#if DEBUG
	assert (cardinalities != NULL);
	assert (N_ij != NULL);
	assert (l > 0);
	#endif

	int i, j;
	for (i = 0; i < l; i++)
		for (j = 0; j < l; j++)
			cardinalities[N_ij[j*l+i]+1]++;
}

//Scanning algorithm used when decoding
//lists found using GPU algorithms
void scan(const int * const k_in, const int * const k_out, int64_t * const &past_edge_pointers, int64_t * const &future_edge_pointers, const int &N_tar)
{
	int64_t past_idx = 0, future_idx = 0;
	int i;

	for (i = 0; i < N_tar; i++) {
		if (k_in[i] != 0) {
			past_edge_pointers[i] = past_idx;
			past_idx += k_in[i];
		} else
			past_edge_pointers[i] = -1;

		if (k_out[i] != 0) {
			future_edge_pointers[i] = future_idx;
			future_idx += k_out[i];
		} else
			future_edge_pointers[i] = -1;
	}
}

//Debug Print Variadic Function
int printf_dbg(const char * format, ...)
{
	printf_mag();
	va_list argp;
	va_start(argp, format);
	vprintf(format, argp);
	va_end(argp);
	printf_std();
	fflush(stdout);

	return 0;
}

//MPI Print Variadic Function
//Allows only the master process to print to stdout
//If MPI is not enabled, rank == 0
int printf_mpi(int rank, const char * format, ...)
{
	int retval = 0;

	if (rank == 0) {
		va_list argp;
		va_start(argp, format);
		vprintf(format, argp);
		va_end(argp);
	}

	return retval;
}

//Update all processes regarding failure status
bool checkMpiErrors(CausetMPI &cmpi)
{
	#ifdef MPI_ENABLED
	MPI_Barrier(MPI_COMM_WORLD);
	if (!cmpi.rank)
		MPI_Reduce(MPI_IN_PLACE, &cmpi.fail, 1, MPI_INT, MPI_SUM, 0, MPI_COMM_WORLD);
	else
		MPI_Reduce(&cmpi.fail, NULL, 1, MPI_INT, MPI_SUM, 0, MPI_COMM_WORLD);
	MPI_Bcast(&cmpi.fail, cmpi.num_mpi_threads, MPI_INT, 0, MPI_COMM_WORLD);
	#endif

	return !!cmpi.fail;
}

//Enumerate permutations of the unique integers stored in 'elements'
//Permutations are saved as binary strings
//The goal is to find all permutations of 2 unlabeled elements in 'elements.size()/2' unlabeled bins
// > swapping two elements in the same bin is not a unique permutation
// > swapping two bins is not a unique permutation
// > swapping two elements in different bins is a unique permutation
void init_mpi_permutations(std::unordered_set<FastBitset> &permutations, std::vector<unsigned int> elements)
{
	#if DEBUG
	assert (elements.size() > 0);
	#endif

	//Ensure output is empty to begin
	permutations.clear();
	permutations.swap(permutations);

	//This is the length of the binary string
	uint64_t len = static_cast<uint64_t>(elements.size()) * (elements.size() - 1) >> 1;
	while (std::next_permutation(elements.begin(), elements.end())) {
		FastBitset fb(len);
		perm_to_binary(fb, elements);
		bool insert = true;

		//Check if a similar element has already been added
		//Unique binary strings will be completely orthogonal
		for (std::unordered_set<FastBitset>::iterator fb0 = permutations.begin(); fb0 != permutations.end(); fb0++)
			for (uint64_t i = 0; i < len; i++)
				if (fb.read(i) & fb0->read(i))
					insert = false;

		if (insert) permutations.insert(fb);
	}
}

//Enumerate all unique pairs, assuming the first element is smaller
//The ordered pairs are not included (e.g. (0,1) (2,3), etc.) 
//The variable 'nbuf' should be twice the number of computers used
//'Elements' is given as a sequence of natural numbers beginning at zero
void init_mpi_pairs(std::unordered_set<std::pair<int,int> > &pairs, const std::vector<unsigned int> elements)
{
	#if DEBUG
	assert (elements.size() > 0 && !(elements.size() % 2));
	#endif

	//Ensure output is empty
	pairs.clear();
	pairs.swap(pairs);

	for (size_t k = 0; k < elements.size(); k += 2) {
		//Elements are understood to be stored in pairs
		unsigned int i = elements[k];
		unsigned int j = elements[k+1];

		for (unsigned int m = 0; m < elements.size(); m++) {
			if (m == i || m == j) continue;
			pairs.insert(std::make_pair(std::min(i, m), std::max(i, m)));
			pairs.insert(std::make_pair(std::min(j, m), std::max(j, m)));
		}
	}

	//Remove the ordered pairs
	for (size_t k = 0; k < elements.size(); k += 2)
		pairs.erase(std::make_pair(k, k+1));
}

//Saves all permutations which are recognized as non-unique by
//the subroutine 'init_mpi_permutations'
void fill_mpi_similar(std::vector<std::vector<unsigned int> > &similar, std::vector<unsigned int> elements)
{
	#if DEBUG
	assert (elements.size() > 0);
	#endif

	//Ensure output is empty
	similar.clear();
	similar.swap(similar);

	//Translate initial elements to a binary string
	uint64_t len = static_cast<uint64_t>(elements.size()) * (elements.size() - 1) >> 1;
	FastBitset fb(len);
	perm_to_binary(fb, elements);
	similar.push_back(elements);

	//Compare each permutation's binary string to the original permutation's string
	while (std::next_permutation(elements.begin(), elements.end())) {
		FastBitset sim(len);
		perm_to_binary(sim, elements);
		//If their binary string is equal, they are the same permutation
		if (fb == sim)
			similar.push_back(elements);
	}
}

//Use the cyclesort algorithm to determine which permutation is "most similar" to
//the original, thereby minimizing the number of swaps done using MPI
void get_most_similar(std::vector<unsigned int> &sim, unsigned int &nsteps, const std::vector<std::vector<unsigned int> > candidates, const std::vector<unsigned int> elements)
{
	#if DEBUG
	assert (candidates.size() > 0);
	assert (elements.size() > 0);
	#endif

	unsigned int idx = 0;
	nsteps = 100000000;	//Taken to be integer-infinity

	//Check each candidate
	for (size_t i = 0; i < candidates.size(); i++) {
		size_t len = candidates[i].size();
		std::vector<unsigned int> c(len);

		//Create a local copy of the candidate, and relabel
		//it so when sorted, it will equal 'elements'
		c = candidates[i];
		relabel_vector(c, elements);
		
		unsigned int writes;
		//Perform cyclesort, record number of writes
		cyclesort(writes, c, NULL);
		if (writes < nsteps) {
			//Save minimum
			nsteps = writes;
			idx = i;
		}
	}

	sim = candidates[idx];
}

//Relabel the output vector so when sorted the elements will equal the input
//This allows a set of numbers to be 'sorted' to a permutation other than the truly sorted one
void relabel_vector(std::vector<unsigned int> &output, const std::vector<unsigned int> input)
{
	#if DEBUG
	assert (input.size() == output.size());
	#endif

	size_t len = input.size();
	std::vector<unsigned int> out(len);

	for (size_t i = 0; i < len; i++)
		for (size_t j = 0; j < len; j++)
			if (output[j] == input[i])
				//Save the index of the element rather than the element itself
				out[j] = i;

	output = out;
}

//Generate a binary string from a set of unique integers
//This is used to identify which permutations are equal
//Recall, we wish to identify all unique permutations of 2 unlabeled elements in '2N'
//unlabeled bins, where 'N' is the number of computers we have
void perm_to_binary(FastBitset &fb, const std::vector<unsigned int> perm)
{
	#if DEBUG
	assert (!(fb.size() % 2));
	assert (!(perm.size() % 2));
	#endif

	for (uint64_t k = 0; k < perm.size(); k += 2) {
		//Group numbers into pairs (2 unlabeled elements)
		unsigned int i = perm[k];
		unsigned int j = perm[k+1];

		//Swap them if the first is larger
		if (i > j) {
			i ^= j;
			j ^= i;
			i ^= j;
		}

		//This (i,j) is a row/column of a matrix
		//We wish to transform this into a 1-D index
		//identified with elements in an upper-triangular matrix
		unsigned int do_map = i >= perm.size() >> 1;
		i -= do_map * (((i - (perm.size() >> 1)) << 1) + 1);
		j -= do_map * ((j - (perm.size() >> 1)) << 1);

		//This transformed linear index is stored in 'm'
		unsigned int m = i * (perm.size() - 1) + j - 1;
		fb.set(m);
	}
}

//The inverse operation of perm_to_binary
void binary_to_perm(std::vector<unsigned int> &perm, const FastBitset &fb, const unsigned int len)
{
	#if DEBUG
	assert (!(perm.size() % 2));
	assert (!(fb.size() % 2));
	#endif

	//Transform each set bit to an (i,j) pair
	for (uint64_t k = 0; k < fb.size(); k++) {
		if (!fb.read(k)) continue;

		unsigned int i = k / (len - 1);
		unsigned int j = k % (len - 1) + 1;

		unsigned int do_map = i >= j;
		i += do_map * ((((len >> 1) - i) << 1) - 1);
		j += do_map * (((len >> 1) - j) << 1);

		//These (i,j) pairs are the pair associated with the permutation
		perm.push_back(i);
		perm.push_back(j);
	}
}

//When the adjacency matrix is broken across computers, local indices refer to those within
//the sub-matrix stored in a particular buffer. When these buffers are shuffled, this subroutine will
//return the global index, with respect to the whole unshuffled matrix, provided the current permutation
unsigned int loc_to_glob_idx(std::vector<unsigned int> perm, const unsigned int idx, const int N_tar, const int num_mpi_threads, const int rank)
{
	#if DEBUG
	assert (idx < (unsigned int)N_tar);
	assert (N_tar > 0);
	assert (num_mpi_threads > 1);
	assert (rank >= 0);
	#endif

	//The number of rows in a single buffer (two buffers per computer)
	int mpi_offset = N_tar / (num_mpi_threads << 1);
	//Index local to a single buffer, whereas idx spans two buffers
	int loc_idx = idx - (idx / mpi_offset) * mpi_offset;
	//The sequence index - i.e. which buffer loc_idx belongs to
	int seq_idx = perm[(rank << 1) + (idx / mpi_offset)];
	//The original global index, spanning [0, N_tar)
	int glob_idx = seq_idx * mpi_offset + loc_idx;

	return glob_idx;
}

#ifdef MPI_ENABLED
//Perform MPI trades across multiple computers
//When a swap is performed, the memory in one buffer is scattered to all other computers
//because the temporary storage is split across all computers. Then, the second buffer is moved to
//the first buffer. Finally, the temporary storage is moved back to the second buffer. The index
//used for "storage" is '-1'. The list of swaps needed is stored in 'swaps', and this variable
//is populated using the 'cyclesort' algorithm.
void mpi_swaps(const std::vector<std::pair<int,int> > swaps, Bitvector &adj, Bitvector &adj_buf, const int N_tar, const int num_mpi_threads, const int rank)
{
	#if DEBUG
	assert (swaps.size() > 0);
	assert (adj.size() > 0);
	assert (adj_buf.size() > 0);
	assert (N_tar > 0);
	assert (num_mpi_threads > 1);
	assert (rank >= 0);
	#endif

	//Number of rows per buffer
	int mpi_offset = N_tar / (num_mpi_threads << 1);
	//Number of rows per temporary buffer
	int cpy_offset = mpi_offset / num_mpi_threads;

	int loc_idx;
	int start, finish;
	MPI_Status status;

	//Perform all swaps requested
	for (size_t i = 0; i < swaps.size(); i++) {
		//The two swap indices
		int idx0 = std::get<0>(swaps[i]);
		int idx1 = std::get<1>(swaps[i]);

		MPI_Barrier(MPI_COMM_WORLD);
		//If this is a simple trade from one computer to a second, computers not involved
		//can continue to the next iteration and wait at the barrier
		if (idx0 != -1 && idx1 != -1 && idx0 >> 1 == idx1 >> 1 && (rank << 1) + (idx0 % 2) != idx0)
			continue;

		if (idx0 == -1) {	//Copy buffer to idx1
			//Distinguish between two local buffers
			int buf_offset = mpi_offset * (idx1 % 2);
			//Range of rows which will be copied
			start = rank * cpy_offset;
			finish = start + cpy_offset;
			//Iterate over all rows
			for (int j = start; j < finish; j++) {
				//Index internal to a temporary storage buffer
				loc_idx = j % cpy_offset;
				//All MPI processes have work to do
				for (int k = 0; k < num_mpi_threads; k++) {
					//Index of the row being addressed
					int adj_idx = buf_offset + k * cpy_offset + loc_idx;
					if ((rank << 1) + (idx1 % 2) == idx1) {	//Receiving data to buffer idx1
						if (k == rank)
							//Copy local buffer row 'loc_idx' to adj row 'adj_idx'
							memcpy(adj[adj_idx].getAddress(), adj_buf[loc_idx].getAddress(), sizeof(BlockType) * adj_buf[loc_idx].getNumBlocks());
						else
							//Receive foreign buffer in rank 'k' to adj row 'adj_idx'
							MPI_Recv(adj[adj_idx].getAddress(), adj[adj_idx].getNumBlocks(), BlockTypeMPI, k, 0, MPI_COMM_WORLD, &status);
					} else if ((k << 1) + (idx1 % 2) == idx1)	//Sending data to buffer idx1
						//Send local buffer row 'loc_idx' to foreign buffer in rank 'k'
						MPI_Send(adj_buf[loc_idx].getAddress(), adj_buf[loc_idx].getNumBlocks(), BlockTypeMPI, k, 0, MPI_COMM_WORLD);
				}
			}
		} else if (idx1 == -1) {	//Copy idx0 to buffer
			//Distinguish between two local buffers
			int buf_offset = mpi_offset * (idx0 % 2);
			//Range of rows which will be copied
			start = rank * cpy_offset;
			finish = start + cpy_offset;
			//Iterate over all rows
			for (int j = start; j < finish; j++) {
				//Index internal to a temporary storage buffer
				loc_idx = j % cpy_offset;
				//All MPI processes have work to do
				for (int k = 0; k < num_mpi_threads; k++) {
					//Index of the row being addressed
					int adj_idx = buf_offset + k * cpy_offset + loc_idx;
					if ((rank << 1) + (idx0 % 2) == idx0) {	//Sending data to buffer idx0
						if (k == rank)
							//Copy adj row 'adj_idx' to local buffer row 'loc_idx'
							memcpy(adj_buf[loc_idx].getAddress(), adj[adj_idx].getAddress(), sizeof(BlockType) * adj[adj_idx].getNumBlocks());
						else
							//Send adj row 'adj_idx' to foreign buffer in rank 'k'
							MPI_Send(adj[adj_idx].getAddress(), adj[adj_idx].getNumBlocks(), BlockTypeMPI, k, 0, MPI_COMM_WORLD);
					} else if ((k << 1) + (idx0 % 2) == idx0)	//Receiving data to buffer idx0
						//Receive foreign buffer in rank 'k' to local buffer row 'loc_idx'
						MPI_Recv(adj_buf[loc_idx].getAddress(), adj_buf[loc_idx].getNumBlocks(), BlockTypeMPI, k, 0, MPI_COMM_WORLD, &status);
				}
			}
		} else {	//Copy from idx0 to idx1
			//Distinguish between two local buffers
			int buf_offset0 = mpi_offset * (idx0 % 2);
			int buf_offset1 = mpi_offset * (idx1 % 2);
			//Range of rows which will be copied
			start = rank * mpi_offset;
			finish = start + mpi_offset;
			//Iterate over all rows
			for (int j = start; j < finish; j++) {
				//Index internal to a temporary storage buffer
				loc_idx = j % mpi_offset;
				if (idx0 >> 1 == idx1 >> 1)	//Sending data to buffer idx1, both on same computer
					//Copy adj row 'buf_offset0 + loc_idx' to adj row 'buf_offset1 + loc_idx'
					memcpy(adj[buf_offset1+loc_idx].getAddress(), adj[buf_offset0+loc_idx].getAddress(), sizeof(BlockType) * adj[buf_offset0+loc_idx].getNumBlocks());
				else if ((rank << 1) + (idx0 % 2) == idx0)	//Send data to buffer idx1
					//Send adj row 'buf_offset0 + loc_idx' to foreign rank 'idx1 / 2'
					MPI_Send(adj[buf_offset0+loc_idx].getAddress(), adj[buf_offset0+loc_idx].getNumBlocks(), BlockTypeMPI, idx1 / 2, 0, MPI_COMM_WORLD);
				else if ((rank << 1) + (idx1 % 2) == idx1)	//Receive data to buffer idx1
					//Receive adj row 'buf_offset1 + loc_idx' from foreign rank 'idx0 / 2'
					MPI_Recv(adj[buf_offset1+loc_idx].getAddress(), adj[buf_offset1+loc_idx].getNumBlocks(), BlockTypeMPI, idx0 / 2, 0, MPI_COMM_WORLD, &status);
			}
		}
	}
}

//Send a signal to all MPI nodes, indicating which action is requested
void sendSignal(const MPISignal signal, const int rank, const int num_mpi_threads)
{
	MPI_Request req;
	for (int i = 0; i < num_mpi_threads; i++) {
		//Don't send a signal to yourself, unless it's a lock request to the spinlock
		if (signal != REQUEST_LOCK && i == rank) continue;
		//Make sure it's an asynchronous call to avoid blocking
		MPI_Isend((void*)&signal, 1, MPI_INT, i, MPI_ANY_TAG, MPI_COMM_WORLD, &req);
	}
}
#endif
