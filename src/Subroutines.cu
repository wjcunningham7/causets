#include "hip/hip_runtime.h"
#include "Subroutines.h"

/////////////////////////////
//(C) Will Cunningham 2014 //
// Northeastern University //
// Krioukov Research Group //
/////////////////////////////

//Sort nodes temporally by tau coordinate
//O(N*log(N)) Efficiency
void quicksort(Node &nodes, int low, int high)
{
	int i, j, k;
	float key;

	if (low < high) {
		k = (low + high) / 2;
		swap(nodes, low, k);
		key = nodes.tau[low];
		i = low + 1;
		j = high;

		while (i <= j) {
			while ((i <= high) && (nodes.tau[i] <= key))
				i++;
			while ((j >= low) && (nodes.tau[j] > key))
				j--;
			if (i < j)
				swap(nodes, i, j);
		}

		swap(nodes, low, j);
		quicksort(nodes, low, j - 1);
		quicksort(nodes, j + 1, high);
	}
}

//Exchange two nodes
static void swap(Node &nodes, const int i, const int j)
{
	float4 sc = nodes.sc[i];
	float tau = nodes.tau[i];
	int k_in = nodes.k_in[i];
	int k_out = nodes.k_out[i];
	
	nodes.sc[i] = nodes.sc[j];
	nodes.tau[i] = nodes.tau[j];
	nodes.k_in[i] = nodes.k_in[j];
	nodes.k_out[i] = nodes.k_out[j];

	nodes.sc[j] = sc;
	nodes.tau[j] = tau;
	nodes.k_in[j] = k_in;
	nodes.k_out[j] = k_out;
}

//Newton-Raphson Method
//Solves Transcendental Equations
bool newton(double (*solve)(const double &x, const double * const p1, const double * const p2, const double * const p3, const float * const p4, const int * const p5, const int * p6), double *x, const int max_iter, const double tol, const double * const p1, const double * const p2, const double * const p3, const float * const p4, const int * const p5, const int * p6)
{
	if (DEBUG) assert (solve != NULL);

	double res = 1.0;
	double x1;
	int iter = 0;

	try {
		while (ABS(res, STL) > tol && iter < max_iter) {
			res = (*solve)(*x, p1, p2, p3, p4, p5, p6);
			//printf("res: %E\n", res);
			if (res != res)
				throw CausetException("NaN Error in Newton-Raphson\n");
	
			x1 = *x + res;
			//printf("x1: %E\n", x1);
	
			*x = x1;
			iter++;

			fflush(stdout);
		}
	} catch (CausetException c) {
		fprintf(stderr, "CausetException in %s: %s on line %d\n", __FILE__, c.what(), __LINE__);
		return false;
	} catch (std::exception e) {
		fprintf(stderr, "Unknown Exception in %s: %s on line %d\n", __FILE__, e.what(), __LINE__);
		return false;
	}

	//printf("Newton-Raphson Results:\n");
	//printf("Tolerance: %E\n", tol);
	//printf("%d of %d iterations performed.\n", iter, max);
	//printf("Residual: %E\n", res);
	//printf("Solution: %E\n", *x);
	//fflush(stdout);

	return true;
}
