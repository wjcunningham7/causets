#include "hip/hip_runtime.h"
#include "GPUSubroutines.h"

/////////////////////////////
//(C) Will Cunningham 2014 //
// Northeastern University //
// Krioukov Research Group //
/////////////////////////////

__global__ void Generate(Node *nodes, int N_tar, long seed)
{
	//int i = blockDim.x * blockIdx.x + threadIdx.x;
	//int j = blockDim.y * blockIdx.y + threadIdx.y;
	//if ((j * width) + i > N_tar)
	//	return;

	//Implement CURAND package here for random number generation
}

bool generateNodesGPU(Network *network)
{
	//CURAND
	hiprandGenerator_t prng;
	
	try {
		if (HIPRAND_STATUS_SUCCESS != hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_DEFAULT))
			throw CausetException("Failed to create hiprand generator.\n");
		if (HIPRAND_STATUS_SUCCESS != hiprandSetPseudoRandomGeneratorSeed(prng, (int)network->network_properties.seed))
			throw CausetException("Failed to set hiprand seed.\n");

		//Need to redesign Node for GPU so memory for points is contiguous
		//Lots of thought should go into this...
		//if (HIPRAND_STATUS_SUCCESS != hiprandGenerateUniform(prng, (float*)d_points, network->network_properties.N_tar))
		//	throw CausetException("Failed to generate hiprand uniform number distribution.\n");

		if (HIPRAND_STATUS_SUCCESS != hiprandDestroyGenerator(prng))
			throw CausetException("Failed to destroy hiprand generator.\n");
	} catch (CausetException c) {
		fprintf(stderr, "CausetException in %s: %s on line %d\n", __FILE__, c.what(), __LINE__);
		return false;
	}

	//Invoke Kernel
	Generate<<<network->network_properties.network_exec.blocks_per_grid, network->network_properties.network_exec.threads_per_block>>>((Node*)network->d_nodes, network->network_properties.N_tar, network->network_properties.seed);
	getLastCudaError("Kernel 'Generate' Failed to Execute!");
	checkCudaErrors(hipCtxSynchronize());

	//Copy Values to Host
	checkCudaErrors(hipMemcpyDtoH(network->nodes, network->d_nodes, sizeof(Node) * network->network_properties.N_tar));

	return true;
}
