#include "hip/hip_runtime.h"
#include "Validate.h"

/////////////////////////////
//(C) Will Cunningham 2014 //
// Krioukov Research Group //
// Northeastern University //
/////////////////////////////

//Debug:  Future vs Past Edges in Adjacency List
//O(1) Efficiency
void compareAdjacencyLists(const Node &nodes, const Edge &edges)
{
	if (DEBUG) {
		//No null pointers
		assert (edges.past_edges != NULL);
		assert (edges.future_edges != NULL);
		assert (edges.past_edge_row_start != NULL);
		assert (edges.future_edge_row_start != NULL);
	}

	int i, j;
	for (i = 0; i < 20; i++) {
		printf("\nNode i: %d\n", i);

		printf("Forward Connections:\n");
		if (edges.future_edge_row_start[i] == -1)
			printf("\tNo future connections.\n");
		else {
			for (j = 0; j < nodes.k_out[i] && j < 10; j++)
				printf("%d ", edges.future_edges[edges.future_edge_row_start[i]+j]);
			printf("\n");
		}

		printf("Backward Connections:\n");
		if (edges.past_edge_row_start[i] == -1)
			printf("\tNo past connections.\n");
		else {
			for (j = 0; j < nodes.k_in[i] && j < 10; j++)
				printf("%d ", edges.past_edges[edges.past_edge_row_start[i]+j]);
			printf("\n");
		}
	
		fflush(stdout);
	}
}

//Debug:  Future and Past Adjacency List Indices
//O(1) Effiency
void compareAdjacencyListIndices(const Node &nodes, const Edge &edges)
{
	if (DEBUG) {
		//No null pointers
		assert (edges.past_edges != NULL);
		assert (edges.future_edges != NULL);
		assert (edges.past_edge_row_start != NULL);
		assert (edges.future_edge_row_start != NULL);
	}

	int max1 = 20;
	int max2 = 100;
	int i, j;

	printf("\nFuture Edge Indices:\n");
	for (i = 0; i < max1; i++)
		printf("%d\n", edges.future_edge_row_start[i]);
	printf("\nPast Edge Indices:\n");
	for (i = 0; i < max1; i++)
		printf("%d\n", edges.past_edge_row_start[i]);
	fflush(stdout);

	int next_future_idx = -1;
	int next_past_idx = -1;

	for (i = 0; i < max1; i++) {
		printf("\nNode i: %d\n", i);

		printf("Out-Degrees: %d\n", nodes.k_out[i]);
		if (edges.future_edge_row_start[i] == -1) {
			printf("Pointer: 0\n");
		} else {
			for (j = 1; j < max2; j++) {
				if (edges.future_edge_row_start[i+j] != -1) {
					next_future_idx = j;
					break;
				}
			}
			printf("Pointer: %d\n", (edges.future_edge_row_start[i+next_future_idx] - edges.future_edge_row_start[i]));
		}

		printf("In-Degrees: %d\n", nodes.k_in[i]);
		if (edges.past_edge_row_start[i] == -1)
			printf("Pointer: 0\n");
		else {
			for (j = 1; j < max2; j++) {
				if (edges.past_edge_row_start[i+j] != -1) {
					next_past_idx = j;
					break;
				}
			}
			printf("Pointer: %d\n", (edges.past_edge_row_start[i+next_past_idx] - edges.past_edge_row_start[i]));
		}
		fflush(stdout);
	}
}

bool compareCoreEdgeExists(const int * const k_out, const int * const future_edges, const int * const future_edge_row_start, const bool * const core_edge_exists, const int &N_tar, const float &core_edge_fraction)
{
	if (DEBUG) {
		assert (k_out != NULL);
		assert (future_edges != NULL);
		assert (future_edge_row_start != NULL);
		assert (core_edge_exists != NULL);
		assert (N_tar > 0);
		assert (core_edge_fraction >= 0.0f && core_edge_fraction <= 1.0f);
	}
	
	int core_limit = static_cast<int>(core_edge_fraction * N_tar);
	int idx1, idx2;
	uint64_t idx12, idx21;
	int i, j;

	try {
		for (i = 0; i < core_limit; i++) {
			idx1 = i;

			if (DEBUG) {
				assert (!(future_edge_row_start[idx1] == -1 && k_out[idx1] > 0));
				assert (!(future_edge_row_start[idx1] != -1 && k_out[idx1] == 0));
			}

			for (j = 0; j < k_out[idx1]; j++) {
				idx2 = future_edges[future_edge_row_start[idx1]+j];

				if (idx2 >= core_limit)
					continue;

				idx12 = static_cast<uint64_t>(idx1) * core_limit + idx2;
				idx21 = static_cast<uint64_t>(idx2) * core_limit + idx1;

				//printf("idx12: %" PRIu64 "\tidx21: %" PRIu64 "\n", idx12, idx21);

				if (core_edge_exists[idx12] == false || core_edge_exists[idx21] == false)
					throw CausetException("Adjacency matrix does not match sparse list!\n");
			}
		}
	} catch (CausetException c) {
		fprintf(stderr, "CausetException in %s: %s on line %d\n", __FILE__, c.what(), __LINE__);
		return false;
	} catch (std::exception e) {
		fprintf(stderr, "Unknown Exception in %s: %s on line %d\n", __FILE__, e.what(), __LINE__);
		return false;
	}

	return true;
}

#ifdef CUDA_ENABLED
//Note that core_edge_exists has not been implemented in this version of the linkNodesGPU subroutine.
bool linkNodesGPU_v1(Node &nodes, const Edge &edges, bool * const &core_edge_exists, const int &N_tar, const float &k_tar, int &N_res, float &k_res, int &N_deg2, const float &core_edge_fraction, const int &edge_buffer, Stopwatch &sLinkNodesGPU, size_t &hostMemUsed, size_t &maxHostMemUsed, size_t &devMemUsed, size_t &maxDevMemUsed, const bool &compact, const bool &verbose, const bool &bench)
{
	if (DEBUG) {
		assert (nodes.crd->getDim() == 4);
		assert (!nodes.crd->isNull());
		assert (nodes.crd->w() != NULL);
		assert (nodes.crd->x() != NULL);
		assert (nodes.crd->y() != NULL);	
		assert (nodes.crd->z() != NULL);
		assert (edges.past_edges != NULL);
		assert (edges.future_edges != NULL);
		assert (edges.past_edge_row_start != NULL);
		assert (edges.future_edge_row_start != NULL);
		assert (core_edge_exists != NULL);
		assert (N_tar > 0);
		assert (k_tar > 0);
		assert (core_edge_fraction >= 0.0f && core_edge_fraction <= 1.0f);
		assert (edge_buffer >= 0);
	}

	Stopwatch sGPUOverhead = Stopwatch();
	Stopwatch sGenAdjList = Stopwatch();
	Stopwatch sBitonic0 = Stopwatch();
	Stopwatch sDecode0 = Stopwatch();
	Stopwatch sBitonic1 = Stopwatch();
	Stopwatch sDecode1 = Stopwatch();
	Stopwatch sProps = Stopwatch();

	hipDeviceptr_t d_w;
	hipDeviceptr_t d_x;
	hipDeviceptr_t d_y;
	hipDeviceptr_t d_z;
	hipDeviceptr_t d_edges;
	hipDeviceptr_t d_past_edges, d_future_edges;
	hipDeviceptr_t d_k_in, d_k_out;
	hipDeviceptr_t d_N_res, d_N_deg2;
	hipDeviceptr_t d_g_idx;

	int *g_idx;
	int j, k;

	stopwatchStart(&sLinkNodesGPU);
	stopwatchStart(&sGPUOverhead);

	//Allocate Overhead on Host
	try {
		g_idx = (int*)malloc(sizeof(int));
		if (g_idx == NULL)
			throw std::bad_alloc();
		memset(g_idx, 0, sizeof(int));
		hostMemUsed += sizeof(int);
	} catch (std::bad_alloc) {
		fprintf(stderr, "Memory allocation failure in %s on line %d!\n", __FILE__, __LINE__);
		return false;
	}

	//Allocate Global Device Memory
	checkCudaErrors(hipMalloc(&d_w, sizeof(float) * N_tar));
	checkCudaErrors(hipMalloc(&d_x, sizeof(float) * N_tar));
	checkCudaErrors(hipMalloc(&d_y, sizeof(float) * N_tar));
	checkCudaErrors(hipMalloc(&d_z, sizeof(float) * N_tar));
	devMemUsed += sizeof(float) * N_tar * 4;

	size_t d_edges_size = pow(2.0, ceil(log2(N_tar * k_tar / 2 + edge_buffer)));
	checkCudaErrors(hipMalloc(&d_edges, sizeof(uint64_t) * d_edges_size));
	devMemUsed += sizeof(uint64_t) * d_edges_size;

	checkCudaErrors(hipMalloc(&d_k_in, sizeof(int) * N_tar));
	devMemUsed += sizeof(int) * N_tar;

	checkCudaErrors(hipMalloc(&d_k_out, sizeof(int) * N_tar));
	devMemUsed += sizeof(int) * N_tar;
	
	checkCudaErrors(hipMalloc(&d_g_idx, sizeof(int)));
	devMemUsed += sizeof(int);

	memoryCheckpoint(hostMemUsed, maxHostMemUsed, devMemUsed, maxDevMemUsed);
	if (verbose)
		printMemUsed("for Parallel Node Linking", hostMemUsed, devMemUsed, 0);

	//Copy Memory from Host to Device
	checkCudaErrors(hipMemcpyHtoD(d_w, nodes.crd->w(), sizeof(float) * N_tar));
	checkCudaErrors(hipMemcpyHtoD(d_x, nodes.crd->x(), sizeof(float) * N_tar));
	checkCudaErrors(hipMemcpyHtoD(d_y, nodes.crd->y(), sizeof(float) * N_tar));
	checkCudaErrors(hipMemcpyHtoD(d_z, nodes.crd->z(), sizeof(float) * N_tar));

	//Initialize Memory on Device
	checkCudaErrors(hipMemsetD32(d_edges, 0, d_edges_size << 1));
	checkCudaErrors(hipMemsetD32(d_k_in, 0, N_tar));
	checkCudaErrors(hipMemsetD32(d_k_out, 0, N_tar));
	checkCudaErrors(hipMemsetD32(d_g_idx, 0, 1));

	//Synchronize
	checkCudaErrors(hipCtxSynchronize());

	stopwatchStop(&sGPUOverhead);

	//CUDA Grid Specifications
	unsigned int gridx_GAL = static_cast<unsigned int>(ceil(static_cast<float>(N_tar) / 2));
	unsigned int gridy_GAL = static_cast<unsigned int>(ceil((static_cast<float>(N_tar) / 2) / BLOCK_SIZE));
	dim3 blocks_per_grid_GAL(gridx_GAL, gridy_GAL, 1);
	dim3 threads_per_block_GAL(1, BLOCK_SIZE, 1);
	
	stopwatchStart(&sGenAdjList);

	//Execute Kernel
	GenerateAdjacencyLists_v1<<<blocks_per_grid_GAL, threads_per_block_GAL>>>((float*)d_w, (float*)d_x, (float*)d_y, (float*)d_z, (uint64_t*)d_edges, (int*)d_k_in, (int*)d_k_out, (int*)d_g_idx, N_tar >> 1, compact);
	getLastCudaError("Kernel 'NetworkCreator_GPU.GenerateAdjacencyLists' Failed to Execute!\n");

	//Synchronize
	checkCudaErrors(hipCtxSynchronize());

	stopwatchStop(&sGenAdjList);
	stopwatchStart(&sGPUOverhead);

	//Check Number of Connections
	checkCudaErrors(hipMemcpyDtoH(g_idx, d_g_idx, sizeof(int)));
	checkCudaErrors(hipCtxSynchronize());

	//Free Device Memory
	hipFree(d_w);
	d_w = 0;

	hipFree(d_x);
	d_x = 0;

	hipFree(d_y);
	d_y = 0;

	hipFree(d_z);
	d_z = 0;

	devMemUsed -= sizeof(float) * N_tar * 4;

	hipFree(d_g_idx);
	d_g_idx = 0;
	devMemUsed -= sizeof(int);

	try {
		if (*g_idx + 1 >= N_tar * k_tar / 2 + edge_buffer)
			throw CausetException("Not enough memory in edge adjacency list.  Increase edge buffer or decrease network size.\n");
	} catch (CausetException c) {
		fprintf(stderr, "CausetException in %s: %s on line %d\n", __FILE__, c.what(), __LINE__);
		return false;
	} catch (std::exception e) {
		fprintf(stderr, "Unknown Exception in %s: %s on line %d\n", __FILE__, e.what(), __LINE__);
		return false;
	}

	stopwatchStop(&sGPUOverhead);

	//Decode past and future edge lists using Bitonic Sort

	//CUDA Grid Specifications
	unsigned int gridx_bitonic = d_edges_size / BLOCK_SIZE;
	dim3 blocks_per_grid_bitonic(gridx_bitonic, 1, 1);
	dim3 threads_per_block(BLOCK_SIZE, 1, 1);

	stopwatchStart(&sBitonic0);

	//Execute Kernel
	for (k = 2; k <= d_edges_size; k <<= 1) {
		for (j = k >> 1; j > 0; j >>= 1) {
			BitonicSort<<<blocks_per_grid_bitonic, threads_per_block>>>((uint64_t*)d_edges, j, k);
			getLastCudaError("Kernel 'Subroutines_GPU.BitonicSort' Failed to Execute!\n");
			checkCudaErrors(hipCtxSynchronize());
		}
	}

	stopwatchStop(&sBitonic0);
	stopwatchStart(&sGPUOverhead);

	//Allocate Device Memory
	checkCudaErrors(hipMalloc(&d_future_edges, sizeof(int) * d_edges_size));
	devMemUsed += sizeof(int) * d_edges_size;

	//Initialize Memory on Device
	checkCudaErrors(hipMemsetD32(d_future_edges, 0, d_edges_size));

	stopwatchStop(&sGPUOverhead);

	//CUDA Grid Specifications
	unsigned int gridx_decode = static_cast<unsigned int>(ceil(static_cast<float>(*g_idx) / BLOCK_SIZE));
	dim3 blocks_per_grid_decode(gridx_decode, 1, 1);

	stopwatchStart(&sDecode0);

	//Execute Kernel
	DecodeFutureEdges<<<blocks_per_grid_decode, threads_per_block>>>((uint64_t*)d_edges, (int*)d_future_edges, *g_idx, d_edges_size - *g_idx);
	getLastCudaError("Kernel 'NetworkCreator_GPU.DecodeFutureEdges' Failed to Execute!\n");

	//Synchronize
	checkCudaErrors(hipCtxSynchronize());

	stopwatchStop(&sDecode0);
	stopwatchStart(&sGPUOverhead);

	//Copy Memory from Device to Host
	checkCudaErrors(hipMemcpyDtoH(edges.future_edges, d_future_edges, sizeof(int) * *g_idx));

	//Free Device Memory
	hipFree(d_future_edges);
	d_future_edges = 0;
	devMemUsed -= sizeof(int) * d_edges_size;

	stopwatchStop(&sGPUOverhead);
	stopwatchStart(&sBitonic1);

	//Resort Edges with New Encoding
	for (k = 2; k <= d_edges_size; k <<= 1) {
		for (j = k >> 1; j > 0; j >>= 1) {
			BitonicSort<<<blocks_per_grid_bitonic, threads_per_block>>>((uint64_t*)d_edges, j, k);
			getLastCudaError("Kernel 'Subroutines_GPU.BitonicSort' Failed to Execute!\n");
			checkCudaErrors(hipCtxSynchronize());
		}
	}

	stopwatchStop(&sBitonic1);
	stopwatchStart(&sGPUOverhead);

	//Allocate Device Memory
	checkCudaErrors(hipMalloc(&d_past_edges, sizeof(int) * d_edges_size));
	devMemUsed += sizeof(int) * d_edges_size;

	//Initialize Memory on Device
	checkCudaErrors(hipMemsetD32(d_past_edges, 0, d_edges_size));

	stopwatchStop(&sGPUOverhead);
	stopwatchStart(&sDecode1);

	//Execute Kernel
	DecodePastEdges<<<blocks_per_grid_decode, threads_per_block>>>((uint64_t*)d_edges, (int*)d_past_edges, *g_idx, d_edges_size - *g_idx);
	getLastCudaError("Kernel 'NetworkCreator_GPU.DecodePastEdges' Failed to Execute!\n");

	//Synchronize
	checkCudaErrors(hipCtxSynchronize());

	stopwatchStop(&sDecode1);
	stopwatchStart(&sGPUOverhead);

	//Copy Memory from Device to Host
	checkCudaErrors(hipMemcpyDtoH(edges.past_edges, d_past_edges, sizeof(int) * *g_idx));
	
	//Free Device Memory
	hipFree(d_edges);
	d_edges = 0;
	devMemUsed -= sizeof(uint64_t) * d_edges_size;

	hipFree(d_past_edges);
	d_past_edges = 0;
	devMemUsed -= sizeof(int) * d_edges_size;

	//Resulting Network Properties

	//Allocate Device Memory
	checkCudaErrors(hipMalloc(&d_N_res, sizeof(int)));
	devMemUsed += sizeof(int);

	checkCudaErrors(hipMalloc(&d_N_deg2, sizeof(int)));
	devMemUsed += sizeof(int);
	
	//Initialize Memory on Device
	checkCudaErrors(hipMemsetD32(d_N_res, 0, 1));
	checkCudaErrors(hipMemsetD32(d_N_deg2, 0, 1));

	stopwatchStop(&sGPUOverhead);

	//CUDA Grid Specifications
	unsigned int gridx_res_prop = static_cast<unsigned int>(ceil(static_cast<float>(N_tar) / BLOCK_SIZE));
	dim3 blocks_per_grid_res_prop(gridx_res_prop, 1, 1);

	stopwatchStart(&sProps);

	//Execute Kernel
	ResultingProps<<<gridx_res_prop, threads_per_block>>>((int*)d_k_in, (int*)d_k_out, (int*)d_N_res, (int*)d_N_deg2, N_tar);
	getLastCudaError("Kernel 'NetworkCreator_GPU.ResultingProps' Failed to Execute!\n");
	checkCudaErrors(hipCtxSynchronize());

	stopwatchStop(&sProps);
	stopwatchStart(&sGPUOverhead);

	//Copy Memory from Device to Host
	checkCudaErrors(hipMemcpyDtoH(nodes.k_in, d_k_in, sizeof(int) * N_tar));
	checkCudaErrors(hipMemcpyDtoH(nodes.k_out, d_k_out, sizeof(int) * N_tar));
	checkCudaErrors(hipMemcpyDtoH(&N_res, d_N_res, sizeof(int)));
	checkCudaErrors(hipMemcpyDtoH(&N_deg2, d_N_deg2, sizeof(int)));

	//Synchronize
	checkCudaErrors(hipCtxSynchronize());

	//Prefix Sum of 'k_in' and 'k_out'
	scan(nodes.k_in, nodes.k_out, edges.past_edge_row_start, edges.future_edge_row_start, N_tar);

	N_res = N_tar - N_res;
	N_deg2 = N_tar - N_deg2;
	k_res = static_cast<float>(*g_idx << 1) / N_res;

	if (DEBUG) {
		assert (N_res > 0);
		assert (N_deg2 > 0);
		assert (k_res > 0.0);
	}

	//Free Device Memory
	hipFree(d_k_in);
	d_k_in = 0;
	devMemUsed -= sizeof(int) * N_tar;

	hipFree(d_k_out);
	d_k_out = 0;
	devMemUsed -= sizeof(int) * N_tar;

	hipFree(d_N_res);
	d_N_res = 0;
	devMemUsed -= sizeof(int);

	hipFree(d_N_deg2);
	d_N_deg2 = 0;
	devMemUsed -= sizeof(int);

	stopwatchStop(&sGPUOverhead);
	stopwatchStop(&sLinkNodesGPU);

	if (!bench) {
		printf("\tCausets Successfully Connected.\n");
		printf_cyan();
		printf("\t\tUndirected Links:         %d\n", *g_idx);
		printf("\t\tResulting Network Size:   %d\n", N_res);
		printf("\t\tResulting Average Degree: %f\n", k_res);
		printf("\t\t    Incl. Isolated Nodes: %f\n", (k_res * N_res) / N_tar);
		printf_std();
		fflush(stdout);
	}
	
	if (DEBUG && !compareCoreEdgeExists(nodes.k_out, edges.future_edges, edges.future_edge_row_start, core_edge_exists, N_tar, core_edge_fraction))
		return false;

	//Print Results
	/*if (!printDegrees(nodes, N_tar, "in-degrees_GPU_v1.cset.dbg.dat", "out-degrees_GPU_v1.cset.dbg.dat")) return false;
	if (!printEdgeLists(edges, *g_idx, "past-edges_GPU_v1.cset.dbg.dat", "future-edges_GPU_v1.cset.dbg.dat")) return false;
	if (!printEdgeListPointers(edges, N_tar, "past-edge-pointers_GPU_v1.cset.dbg.dat", "future-edge-pointers_GPU_v1.cset.dbg.dat")) return false;
	printf_red();
	printf("Check files now.\n");
	printf_std();
	fflush(stdout);
	exit(0);*/

	//Free Host Memory
	free(g_idx);
	g_idx = NULL;
	hostMemUsed -= sizeof(int);

	if (verbose) {
		printf("\t\tExecution Time: %5.6f sec\n", sLinkNodesGPU.elapsedTime);
		printf("\t\t\tGPU Overhead Time: %5.6f sec\n", sGPUOverhead.elapsedTime);
		printf("\t\t\tAdjacency List Kernel Time: %5.6f sec\n", sGenAdjList.elapsedTime);
		printf("\t\t\tBitonic Sort 0 Kernel Time: %5.6f sec\n", sBitonic0.elapsedTime);
		printf("\t\t\tFuture Edge Decode Time: %5.6f sec\n", sDecode0.elapsedTime);
		printf("\t\t\tBitonic Sort 1 Kernel Time: %5.6f sec\n", sBitonic1.elapsedTime);
		printf("\t\t\tPast Edge Decode Time: %5.6f sec\n", sDecode1.elapsedTime);
		fflush(stdout);
	}

	return true;
}

bool generateLists_v1(Node &nodes, uint64_t * const &edges, bool * const core_edge_exists, int * const &g_idx, const int &N_tar, const float &core_edge_fraction, const size_t &d_edges_size, size_t &hostMemUsed, size_t &maxHostMemUsed, size_t &devMemUsed, size_t &maxDevMemUsed, const bool &compact, const bool &verbose)
{
	if (DEBUG) {
		assert (nodes.crd->getDim() == 4);
		assert (!nodes.crd->isNull());
		assert (nodes.crd->w() != NULL);
		assert (nodes.crd->x() != NULL);
		assert (nodes.crd->y() != NULL);
		assert (nodes.crd->z() != NULL);
		assert (nodes.k_in != NULL);
		assert (nodes.k_out != NULL);
		assert (edges != NULL);
		assert (core_edge_exists != NULL);
		assert (g_idx != NULL);
		assert (N_tar > 0);
		assert (core_edge_fraction >= 0.0f && core_edge_fraction <= 1.0f);
	}

	//Temporary Buffers
	hipDeviceptr_t d_w0, d_x0, d_y0, d_z0;
	hipDeviceptr_t d_w1, d_x1, d_y1, d_z1;
	hipDeviceptr_t d_k_in, d_k_out;
	hipDeviceptr_t d_edges;

	int *h_k_in;
	int *h_k_out;
	bool *h_edges;

	unsigned int core_limit = static_cast<unsigned int>(core_edge_fraction * N_tar);
	unsigned int i, j;
	bool diag;

	//Thread blocks are grouped into "mega" blocks
	size_t mblock_size = static_cast<unsigned int>(ceil(static_cast<float>(N_tar) / (2 * BLOCK_SIZE * GROUP_SIZE)));
	size_t mthread_size = mblock_size * BLOCK_SIZE;
	size_t m_edges_size = mthread_size * mthread_size;

	//DEBUG
	/*if (DEBUG) {
		printf_red();
		printf("\nTHREAD  SIZE: %d\n", THREAD_SIZE);
		printf("BLOCK   SIZE: %d\n", BLOCK_SIZE);
		printf("GROUP   SIZE: %d\n", GROUP_SIZE);
		printf("MBLOCK  SIZE: %zd\n", mblock_size);
		printf("MTHREAD SIZE: %zd\n", mthread_size);
		printf("Number of Times Kernel is Executed: %d\n\n", (GROUP_SIZE*GROUP_SIZE));
		printf_std();
		fflush(stdout);
	}*/

	//Allocate Buffers on Host
	try {
		h_k_in = (int*)malloc(sizeof(int) * mthread_size);
		if (h_k_in == NULL)
			throw std::bad_alloc();
		memset(h_k_in, 0, sizeof(int) * mthread_size);
		hostMemUsed += sizeof(int) * mthread_size;

		h_k_out = (int*)malloc(sizeof(int) * mthread_size);
		if (h_k_out == NULL)
			throw std::bad_alloc();
		memset(h_k_out, 0, sizeof(int) * mthread_size);
		hostMemUsed += sizeof(int) * mthread_size;

		h_edges = (bool*)malloc(sizeof(bool) * m_edges_size);
		if (h_edges == NULL)
			throw std::bad_alloc();
		memset(h_edges, 0, sizeof(bool) * m_edges_size);
		hostMemUsed += sizeof(bool) * m_edges_size;
	} catch (std::bad_alloc) {
		fprintf(stderr, "Memory allocation failure in %s on line %d!\n", __FILE__, __LINE__);
		return false;
	}
	
	//Allocate Node Buffers on Device
	checkCudaErrors(hipMalloc(&d_w0, sizeof(float) * mthread_size));
	checkCudaErrors(hipMalloc(&d_x0, sizeof(float) * mthread_size));
	checkCudaErrors(hipMalloc(&d_y0, sizeof(float) * mthread_size));
	checkCudaErrors(hipMalloc(&d_z0, sizeof(float) * mthread_size));
	devMemUsed += sizeof(float) * mthread_size * 4;

	checkCudaErrors(hipMalloc(&d_w1, sizeof(float) * mthread_size));
	checkCudaErrors(hipMalloc(&d_x1, sizeof(float) * mthread_size));
	checkCudaErrors(hipMalloc(&d_y1, sizeof(float) * mthread_size));
	checkCudaErrors(hipMalloc(&d_z1, sizeof(float) * mthread_size));
	devMemUsed += sizeof(float) * mthread_size * 4;

	//Allocate Degree Buffers on Device
	checkCudaErrors(hipMalloc(&d_k_in, sizeof(int) * mthread_size));
	checkCudaErrors(hipMemsetD32(d_k_in, 0, mthread_size));
	devMemUsed += sizeof(int) * mthread_size;

	checkCudaErrors(hipMalloc(&d_k_out, sizeof(int) * mthread_size));
	checkCudaErrors(hipMemsetD32(d_k_out, 0, mthread_size));
	devMemUsed += sizeof(int) * mthread_size;

	//Allocate Edge Buffer on Device
	checkCudaErrors(hipMalloc(&d_edges, sizeof(bool) * m_edges_size));
	checkCudaErrors(hipMemsetD8(d_edges, 0, m_edges_size));
	devMemUsed += sizeof(bool) * m_edges_size;

	memoryCheckpoint(hostMemUsed, maxHostMemUsed, devMemUsed, maxDevMemUsed);
	if (verbose)
		printMemUsed("for Generating Lists on GPU", hostMemUsed, devMemUsed, 0);

	//CUDA Grid Specifications
	unsigned int gridx = static_cast<unsigned int>(ceil(static_cast<float>(mthread_size) / THREAD_SIZE));
	unsigned int gridy = mblock_size;
	dim3 threads_per_block(1, BLOCK_SIZE, 1);
	dim3 blocks_per_grid(gridx, gridy, 1);

	//DEBUG
	/*printf_red();
	printf("Grid X: %u\n", gridx);
	printf("Grid Y: %u\n", gridy);
	printf_std();
	fflush(stdout);*/

	size_t final_size = N_tar - mthread_size * (2 * GROUP_SIZE - 1);
	size_t size0, size1;

	//Index 'i' marks the row and 'j' marks the column
	for (i = 0; i < 2 * GROUP_SIZE; i++) {
		for (j = 0; j < 2 * GROUP_SIZE; j++) {
			if (i > j)
				continue;

			diag = (i == j);

			size0 = (i < 2 * GROUP_SIZE - 1) ? mthread_size : final_size;
			size1 = (j < 2 * GROUP_SIZE - 1) ? mthread_size : final_size;

			//Copy node values to device buffers
			checkCudaErrors(hipMemcpyHtoD(d_w0, nodes.crd->w() + i * mthread_size, sizeof(float) * size0));
			checkCudaErrors(hipMemcpyHtoD(d_x0, nodes.crd->x() + i * mthread_size, sizeof(float) * size0));
			checkCudaErrors(hipMemcpyHtoD(d_y0, nodes.crd->y() + i * mthread_size, sizeof(float) * size0));
			checkCudaErrors(hipMemcpyHtoD(d_z0, nodes.crd->z() + i * mthread_size, sizeof(float) * size0));

			checkCudaErrors(hipMemcpyHtoD(d_w1, nodes.crd->w() + j * mthread_size, sizeof(float) * size1));
			checkCudaErrors(hipMemcpyHtoD(d_x1, nodes.crd->x() + j * mthread_size, sizeof(float) * size1));
			checkCudaErrors(hipMemcpyHtoD(d_y1, nodes.crd->y() + j * mthread_size, sizeof(float) * size1));
			checkCudaErrors(hipMemcpyHtoD(d_z1, nodes.crd->z() + j * mthread_size, sizeof(float) * size1));

			//Synchronize
			checkCudaErrors(hipCtxSynchronize());

			//Execute Kernel
			GenerateAdjacencyLists_v2<<<blocks_per_grid, threads_per_block>>>((float*)d_w0, (float*)d_x0, (float*)d_y0, (float*)d_z0, (float*)d_w1, (float*)d_x1, (float*)d_y1, (float*)d_z1, (int*)d_k_in, (int*)d_k_out, (bool*)d_edges, size0, size1, diag, compact);
			getLastCudaError("Kernel 'NetworkCreator_GPU.GenerateAdjacencyLists_v2' Failed to Execute!\n");

			//Synchronize
			checkCudaErrors(hipCtxSynchronize());

			//Copy edges to host
			checkCudaErrors(hipMemcpyDtoH(h_edges, d_edges, sizeof(bool) * m_edges_size));

			//Copy degrees to host
			checkCudaErrors(hipMemcpyDtoH(h_k_in, d_k_in, sizeof(int) * size1));
			checkCudaErrors(hipMemcpyDtoH(h_k_out, d_k_out, sizeof(int) * size0));

			//Synchronize
			checkCudaErrors(hipCtxSynchronize());

			//Transfer data from buffers
			readDegrees(nodes.k_in, h_k_in, j * mthread_size, size1);
			readDegrees(nodes.k_out, h_k_out, i * mthread_size, size0);
			readEdges(edges, h_edges, core_edge_exists, g_idx, core_limit, d_edges_size, mthread_size, size0, size1, i, j);

			//Clear Device Memory
			checkCudaErrors(hipMemsetD32(d_k_in, 0, mthread_size));
			checkCudaErrors(hipMemsetD32(d_k_out, 0, mthread_size));
			checkCudaErrors(hipMemsetD8(d_edges, 0, m_edges_size));

			//Synchronize
			checkCudaErrors(hipCtxSynchronize());			
		}
	}

	hipFree(d_w0);
	d_w0 = 0;

	hipFree(d_x0);
	d_x0 = 0;

	hipFree(d_y0);
	d_y0 = 0;

	hipFree(d_z0);
	d_z0 = 0;

	devMemUsed -= sizeof(float) * mthread_size * 4;

	hipFree(d_w1);
	d_w1 = 0;

	hipFree(d_x1);
	d_x1 = 0;

	hipFree(d_y1);
	d_y1 = 0;

	hipFree(d_z1);
	d_z1 = 0;

	devMemUsed -= sizeof(float) * mthread_size * 4;

	hipFree(d_k_in);
	d_k_in = 0;
	devMemUsed -= sizeof(int) * mthread_size;

	hipFree(d_k_out);
	d_k_out = 0;
	devMemUsed -= sizeof(int) * mthread_size;

	hipFree(d_edges);
	d_edges = 0;
	devMemUsed -= sizeof(bool) * m_edges_size;

	free(h_k_in);
	h_k_in = NULL;
	hostMemUsed -= sizeof(int) * mthread_size;

	free(h_k_out);
	h_k_out = NULL;
	hostMemUsed -= sizeof(int) * mthread_size;

	free(h_edges);
	h_edges = NULL;
	hostMemUsed -= sizeof(bool) * m_edges_size;

	return true;
}

//Decode past and future edge lists using Bitonic Sort
bool decodeLists_v1(const Edge &edges, const uint64_t * const h_edges, const int * const g_idx, const size_t &d_edges_size, size_t &hostMemUsed, size_t &maxHostMemUsed, size_t &devMemUsed, size_t &maxDevMemUsed, const bool &verbose)
{
	if (DEBUG) {
		assert (edges.past_edges != NULL);
		assert (edges.future_edges != NULL);
		assert (h_edges != NULL);
		assert (g_idx != NULL);

		assert (d_edges_size > 0);
	}

	hipDeviceptr_t d_edges;
	hipDeviceptr_t d_past_edges, d_future_edges;
	int j, k;

	//Allocate Global Device Memory
	checkCudaErrors(hipMalloc(&d_edges, sizeof(uint64_t) * d_edges_size));
	devMemUsed += sizeof(uint64_t) * d_edges_size;

	//Copy Memory from Host to Device
	checkCudaErrors(hipMemcpyHtoD(d_edges, h_edges, sizeof(uint64_t) * d_edges_size));

	//Synchronize
	checkCudaErrors(hipCtxSynchronize());

	//CUDA Grid Specifications
	unsigned int gridx_bitonic = d_edges_size / BLOCK_SIZE;
	dim3 threads_per_block(BLOCK_SIZE, 1, 1);
	dim3 blocks_per_grid_bitonic(gridx_bitonic, 1, 1);

	//Execute Kernel
	for (k = 2; k <= d_edges_size; k <<= 1) {
		for (j = k >> 1; j > 0; j >>= 1) {
			BitonicSort<<<blocks_per_grid_bitonic, threads_per_block>>>((uint64_t*)d_edges, j, k);
			getLastCudaError("Kernel 'Subroutines_GPU.BitonicSort' Failed to Execute!\n");
			checkCudaErrors(hipCtxSynchronize());
		}
	}

	//Allocate Device Memory
	checkCudaErrors(hipMalloc(&d_future_edges, sizeof(int) * d_edges_size));
	devMemUsed += sizeof(int) * d_edges_size;

	memoryCheckpoint(hostMemUsed, maxHostMemUsed, devMemUsed, maxDevMemUsed);
	if (verbose)
		printMemUsed("for Bitonic Sorting", hostMemUsed, devMemUsed, 0);

	//Initialize Memory on Device
	checkCudaErrors(hipMemsetD32(d_future_edges, 0, d_edges_size));

	//CUDA Grid Specifications
	unsigned int gridx_decode = static_cast<unsigned int>(ceil(static_cast<float>(*g_idx) / BLOCK_SIZE));
	dim3 blocks_per_grid_decode(gridx_decode, 1, 1);

	//Execute Kernel
	DecodeFutureEdges<<<blocks_per_grid_decode, threads_per_block>>>((uint64_t*)d_edges, (int*)d_future_edges, *g_idx, d_edges_size - *g_idx);
	getLastCudaError("Kernel 'NetworkCreator_GPU.DecodeFutureEdges' Failed to Execute!\n");

	//Synchronize
	checkCudaErrors(hipCtxSynchronize());

	//Copy Memory from Device to Host
	checkCudaErrors(hipMemcpyDtoH(edges.future_edges, d_future_edges, sizeof(int) * *g_idx));

	//Free Device Memory
	hipFree(d_future_edges);
	d_future_edges = 0;
	devMemUsed -= sizeof(int) * d_edges_size;

	//Resort Edges with New Encoding
	for (k = 2; k <= d_edges_size; k <<= 1) {
		for (j = k >> 1; j > 0; j >>= 1) {
			BitonicSort<<<blocks_per_grid_bitonic, threads_per_block>>>((uint64_t*)d_edges, j, k);
			getLastCudaError("Kernel 'Subroutines_GPU.BitonicSort' Failed to Execute!\n");
			checkCudaErrors(hipCtxSynchronize());
		}
	}

	//Allocate Device Memory
	checkCudaErrors(hipMalloc(&d_past_edges, sizeof(int) * d_edges_size));
	devMemUsed += sizeof(int) * d_edges_size;

	//Initialize Memory on Device
	checkCudaErrors(hipMemsetD32(d_past_edges, 0, d_edges_size));

	//Execute Kernel
	DecodePastEdges<<<blocks_per_grid_decode, threads_per_block>>>((uint64_t*)d_edges, (int*)d_past_edges, *g_idx, d_edges_size - *g_idx);
	getLastCudaError("Kernel 'NetworkCreator_GPU.DecodePastEdges' Failed to Execute!\n");

	//Synchronize
	checkCudaErrors(hipCtxSynchronize());

	//Copy Memory from Device to Host
	checkCudaErrors(hipMemcpyDtoH(edges.past_edges, d_past_edges, sizeof(int) * *g_idx));
	
	//Free Device Memory
	hipFree(d_edges);
	d_edges = 0;
	devMemUsed -= sizeof(uint64_t) * d_edges_size;

	hipFree(d_past_edges);
	d_past_edges = 0;
	devMemUsed -= sizeof(int) * d_edges_size;

	return true;
}
#endif

//Generate confusion matrix for geodesic distances in universe with matter
//Save matrix values as well as d_theta and d_eta to file
bool validateEmbedding(EVData &evd, Node &nodes, const Edge &edges, bool * const core_edge_exists, const int &N_tar, const float &k_tar, const double &N_emb, const int &N_res, const float &k_res, const int &dim, const Manifold &manifold, const double &a, const double &alpha, const float &core_edge_fraction, const int &edge_buffer, long &seed, const int &num_mpi_threads, const int &rank, Stopwatch &sValidateEmbedding, size_t &hostMemUsed, size_t &maxHostMemUsed, size_t &devMemUsed, size_t &maxDevMemUsed, const bool &universe, const bool &compact, const bool &verbose)
{
	if (DEBUG) {
		assert (nodes.crd->getDim() == 4);
		assert (!nodes.crd->isNull());
		assert (nodes.crd->w() != NULL);
		assert (nodes.crd->x() != NULL);
		assert (nodes.crd->y() != NULL);
		assert (nodes.crd->z() != NULL);
		assert (edges.future_edges != NULL);
		assert (edges.future_edge_row_start != NULL);
		assert (core_edge_exists != NULL);
		assert (N_tar > 0);
		assert (k_tar > 0.0);
		assert (dim == 3);
		assert (manifold == DE_SITTER);
		assert (a > 0.0);
		if (universe)
			assert (alpha > 0.0);
		assert (core_edge_fraction >= 0.0 && core_edge_fraction <= 1.0);
		assert (edge_buffer >= 0);
		#ifdef MPI_ENABLED
		assert (num_mpi_threads > 0);
		assert (rank >= 0);
		#endif
	}

	uint64_t stride = static_cast<uint64_t>(static_cast<double>(N_tar) * (N_tar - 1) / (N_emb * 2));
	uint64_t npairs = static_cast<uint64_t>(N_emb);
	uint64_t k;

	#ifdef MPI_ENABLED
	int edges_size = static_cast<int>(N_tar * k_tar / 2 + edge_buffer);
	int core_edges_size = static_cast<int>(POW2(core_edge_fraction * N_tar, EXACT));
	//uint64_t *rcounts = NULL;	//Receive counts used for MPI_Gatherv
	//uint64_t *displs = NULL;	//Displacements used for MPI_Gatherv
	#endif

	stopwatchStart(&sValidateEmbedding);

	//printf("Number of paths to test: %" PRIu64 "\n", static_cast<uint64_t>(N_emb));
	//printf("Stride: %" PRIu64 "\n", stride);

	try {
		evd.confusion = (uint64_t*)malloc(sizeof(uint64_t) * 4);
		if (evd.confusion == NULL)
			throw std::bad_alloc();
		memset(evd.confusion, 0, sizeof(uint64_t) * 4);
		hostMemUsed += sizeof(uint64_t) * 4;

		/*evd.fn = (float*)malloc(sizeof(float) * npairs * 2);
		if (evd.fn == NULL)
			throw std::bad_alloc();
		memset(evd.fn, 0, sizeof(float) * npairs * 2);
		hostMemUsed += sizeof(float) * npairs * 2;

		evd.fp = (float*)malloc(sizeof(float) * npairs * 2);
		if (evd.fp == NULL)
			throw std::bad_alloc();
		memset(evd.fp, 0, sizeof(float) * npairs * 2);
		hostMemUsed += sizeof(float) * npairs * 2;

		#ifdef MPI_ENABLED
		if (rank == 0) {
			rcounts = (uint64_t*)malloc(sizeof(uint64_t) * num_mpi_threads);
			if (rcounts == NULL)
				throw std::bad_alloc();
			memset(rcounts, 0, sizeof(uint64_t) * num_mpi_threads);
			hostMemUsed += sizeof(uint64_t) * num_mpi_threads;

			displs = (uint64_t*)malloc(sizeof(uint64_t) * num_mpi_threads);
			if (displs == NULL)
				throw std::bad_alloc();
			memset(displs, 0, sizeof(uint64_t) * num_mpi_threads);
			hostMemUsed += sizeof(uint64_t) * num_mpi_threads;
		}
		#endif*/

		memoryCheckpoint(hostMemUsed, maxHostMemUsed, devMemUsed, maxDevMemUsed);
		if (verbose)
			printMemUsed("for Embedding Validation", hostMemUsed, devMemUsed, rank);
	} catch (std::bad_alloc) {
		fprintf(stderr, "Memory allocation failure in %s on line %d!\n", __FILE__, __LINE__);
		return false;
	}

	#ifdef MPI_ENABLED
	// Broadcast:
	// > nodes.crd.w
	// > nodes.crd.x
	// > nodes.crd.y
	// > nodes.crd.z
	// > nodes.id.tau
	// If 'nodesAreConnected' is used
	// > nodes.k_out
	// > edges.future_edges
	// > edges.future_edge_row_start
	// > core_edge_exists
	MPI_Barrier(MPI_COMM_WORLD);
	MPI_Bcast(nodes.crd->w(), N_tar, MPI_FLOAT, 0, MPI_COMM_WORLD);
	MPI_Bcast(nodes.crd->x(), N_tar, MPI_FLOAT, 0, MPI_COMM_WORLD);
	MPI_Bcast(nodes.crd->y(), N_tar, MPI_FLOAT, 0, MPI_COMM_WORLD);
	MPI_Bcast(nodes.crd->z(), N_tar, MPI_FLOAT, 0, MPI_COMM_WORLD);
	MPI_Bcast(nodes.id.tau, N_tar, MPI_FLOAT, 0, MPI_COMM_WORLD);

	MPI_Bcast(nodes.k_out, N_tar, MPI_INT, 0, MPI_COMM_WORLD);
	MPI_Bcast(edges.future_edges, edges_size, MPI_INT, 0, MPI_COMM_WORLD);
	MPI_Bcast(edges.future_edge_row_start, N_tar, MPI_INT, 0, MPI_COMM_WORLD);
	MPI_Bcast(core_edge_exists, core_edges_size, MPI::BOOL, 0, MPI_COMM_WORLD);
	#endif

	uint64_t start = 0;
	uint64_t finish = npairs;

	#ifdef MPI_ENABLED
	uint64_t mpi_chunk = npairs / num_mpi_threads;
	start = rank * mpi_chunk;
	finish = start + mpi_chunk;
	#endif

	#ifdef _OPENMP
	#pragma omp parallel for
	#endif
	for (k = start; k < finish; k++) {
		//Choose a pair (i,j) from a single index k
		uint64_t vec_idx = k * stride + 1;
		int i = static_cast<int>(vec_idx / (N_tar - 1));
		int j = static_cast<int>(vec_idx % (N_tar - 1) + 1);
		int do_map = i >= j;

		if (j < N_tar >> 1) {
			i = i + do_map * ((((N_tar >> 1) - i) << 1) - 1);
			j = j + do_map * (((N_tar >> 1) - j) << 1);
		}

		//Embedded distance
		double distance = distanceEmb(nodes.crd->getFloat4(i), nodes.id.tau[i], nodes.crd->getFloat4(j), nodes.id.tau[j], dim, manifold, a, alpha, universe, compact);

		//Check light cone condition for 4D vs 5D
		//Null hypothesis is the nodes are not connected
		//double d_eta = ABS(static_cast<double>(nodes.crd->w(j) - nodes.crd->w(i)), STL);
		//double d_theta = ACOS(static_cast<double>(DIST_V2 ? sphProduct_v2(nodes.crd->getFloat4(i), nodes.crd->getFloat4(j)) : sphProduct_v1(nodes.crd->getFloat4(i), nodes.crd->getFloat4(j))), APPROX ? INTEGRATION : STL, VERY_HIGH_PRECISION);

		//if (d_theta < d_eta) {	//Actual Timelike (Negative)
		if (nodesAreConnected(nodes, edges.future_edges, edges.future_edge_row_start, core_edge_exists, N_tar, core_edge_fraction, i, j)) {
			if (distance > 0) {
				//True Negative (both timelike)
				#ifdef _OPENMP
				#pragma omp atomic
				#endif
				evd.confusion[1]++;
			} else {
				//False Positive
				#ifdef _OPENMP
				#pragma omp atomic
				#endif
				evd.confusion[2]++;

				/*#ifdef _OPENMP
				#pragma omp critical (tn)
				{
				#endif
				evd.fp[evd.fp_idx++] = static_cast<float>(d_eta);
				evd.fp[evd.fp_idx++] = static_cast<float>(d_theta);
				#ifdef _OPENMP
				}
				#endif*/
			}
		} else {	//Actual Spacelike (Positive)
			if (distance > 0) {
				//False Negative
				#ifdef _OPENMP
				#pragma omp atomic
				#endif
				evd.confusion[3]++;

				/*#ifdef _OPENMP
				#pragma omp critical (fp)
				{
				#endif
				evd.fn[evd.fn_idx++] = static_cast<float>(d_eta);
				evd.fn[evd.fn_idx++] = static_cast<float>(d_theta);
				#ifdef _OPENMP
				}
				#endif*/
			} else {
				//True Positive (both spacelike)
				#ifdef _OPENMP
				#pragma omp atomic
				#endif
				evd.confusion[0]++;
			}
		}		
	}

	#ifdef MPI_ENABLED
	MPI_Barrier(MPI_COMM_WORLD);

	// Reduce (In-Place):
	// > evd.confusion[0-3]
	if (rank == 0)
		MPI_Reduce(MPI_IN_PLACE, evd.confusion, 4, MPI_UINT64_T, MPI_SUM, 0, MPI_COMM_WORLD);
	else
		MPI_Reduce(evd.confusion, NULL, 4, MPI_UINT64_T, MPI_SUM, 0, MPI_COMM_WORLD);

	// Gather (In-Place):
	// > rcounts
	/*if (rank == 0) {
		rcounts[0] = evd.fn_idx;
		MPI_Gather(MPI_IN_PLACE, 0, MPI_DATATYPE_NULL, rcounts, num_mpi_threads, MPI_UINT64_T, 0, MPI_COMM_WORLD);
	} else
		MPI_Gather(rcounts, num_mpi_threads, MPI_UINT64_T, NULL, 0, MPI_DATATYPE_NULL, 0, MPI_COMM_WORLD);

	if (rank == 0) {
		int i;
		displs[0] = 0;
		for (i = 1; i < num_mpi_threads; i++)
			displs[i] = displs[i-1] + rcounts[i-1];
	}*/

	// Gatherv (In-Place):
	// > evd.fn
	// > evd.fp
	/*if (rank == 0)
		MPI_Gatherv(MPI_IN_PLACE, 0, MPI_DATATYPE_NULL, evd.fn, rcounts, displs, MPI_FLOAT, 0, MPI_COMM_WORLD);
	else
		MPI_Gatherv(evd.fn, evd.fn_idx, MPI_FLOAT, NULL, NULL, NULL, MPI_DATATYPE_NULL, 0, MPI_COMM_WORLD);*/
	
	//Free Buffers
	/*if (rank == 0) {
		free(rcounts);
		rcounts = NULL;
		hostMemUsed -= sizeof(uint64_t) * num_mpi_threads;

		free(displs);
		displs = NULL;
		hostMemUsed -= sizeof(uint64_t) * num_mpi_threads;
	}*/
	#endif

	//Number of timelike distances in 4-D native FLRW spacetime
	evd.A1T = static_cast<double>(N_res * k_res / 2);
	//Number of spacelike distances in 4-D native FLRW spacetime
	evd.A1S = static_cast<double>(N_tar) * (N_tar - 1) / 2 - evd.A1T;

	//Normalization
	//double norm = N_emb / (static_cast<uint64_t>(N_tar) * (N_tar - 1) / 2);
	//evd.A1T *= norm;
	//evd.A1S *= norm;

	stopwatchStop(&sValidateEmbedding);

	printf_mpi(rank, "\tCalculated Embedding Confusion Matrix.\n");
	if (rank == 0) printf_cyan();
	printf_mpi(rank, "\t\tTrue  Positives: %f\t(4D spacelike, 5D spacelike)\n", static_cast<double>(evd.confusion[0]) / evd.A1S);
	printf_mpi(rank, "\t\tTrue  Negatives: %f\t(4D timelike,  5D timelike)\n", static_cast<double>(evd.confusion[1]) / evd.A1T);
	if (rank == 0) printf_red();
	printf_mpi(rank, "\t\tFalse Positives: %f\t(4D timelike,  5D spacelike)\n", static_cast<double>(evd.confusion[2]) / evd.A1T);
	printf_mpi(rank, "\t\tFalse Negatives: %f\t(4D spacelike, 5D timelike)\n", static_cast<double>(evd.confusion[3]) / evd.A1S);
	if (rank == 0) printf_std();
	fflush(stdout);

	if (verbose) {
		printf_mpi(rank, "\t\tExecution Time: %5.6f sec\n", sValidateEmbedding.elapsedTime);
		fflush(stdout);
	}

	return true;
}

bool validateDistances(DVData &dvd, Node &nodes, const int &N_tar, const double &N_dst, const int &dim, const Manifold &manifold, const double &a, const double &alpha, Stopwatch &sValidateDistances, size_t &hostMemUsed, size_t &maxHostMemUsed, size_t &devMemUsed, size_t &maxDevMemUsed, const bool &universe, const bool &compact, const bool &verbose)
{
	if (DEBUG) {
		assert (nodes.crd->getDim() == 4);
		assert (!nodes.crd->isNull());
		assert (nodes.crd->w() != NULL);
		assert (nodes.crd->x() != NULL);
		assert (nodes.crd->y() != NULL);
		assert (nodes.crd->z() != NULL);
		assert (N_tar > 0);
		assert (dim == 3);
		assert (manifold == DE_SITTER);
		assert (a > 0.0);
		assert (!universe);
	}

	uint64_t stride = static_cast<uint64_t>(static_cast<double>(N_tar) * (N_tar - 1) / (N_dst * 2));
	uint64_t npairs = static_cast<uint64_t>(N_dst);
	uint64_t k;

	double tol = 0.01;

	stopwatchStart(&sValidateDistances);

	try {
		dvd.confusion = (uint64_t*)malloc(sizeof(uint64_t) * 2);
		if (dvd.confusion == NULL)
			throw std::bad_alloc();
		memset(dvd.confusion, 0, sizeof(uint64_t) * 2);
		hostMemUsed += sizeof(uint64_t) * 2;

		memoryCheckpoint(hostMemUsed, maxHostMemUsed, devMemUsed, maxDevMemUsed);
		if (verbose)
			printMemUsed("for Distance Validation", hostMemUsed, devMemUsed, 0);
	} catch (std::bad_alloc) {
		fprintf(stderr, "Memory allocation failure in %s on line %d!\n", __FILE__, __LINE__);
		return false;
	}

	uint64_t start = 0;
	uint64_t finish = npairs;

	#ifdef _OPENMP
	#pragma omp parallel for
	#endif
	for (k = start; k < finish; k++) {
		//Choose a pair (i,j) from a single index k
		uint64_t vec_idx = k * stride + 1;
		int i = static_cast<int>(vec_idx / (N_tar - 1));
		int j = static_cast<int>(vec_idx % (N_tar - 1) + 1);
		int do_map = i >= j;

		if (j < N_tar >> 1) {
			i = i + do_map * ((((N_tar >> 1) - i) << 1) - 1);
			j = j + do_map * (((N_tar >> 1) - j) << 1);
		}

		//DEBUG
		//printf("i: %d\tj: %d\n", i, j);

		//if (nodes.id.tau[i] == nodes.id.tau[j])
		//	printf("ERROR!\n");

		//Distance using embedding
		double embeddedDistance = distanceEmb(nodes.crd->getFloat4(i), nodes.id.tau[i], nodes.crd->getFloat4(j), nodes.id.tau[j], dim, manifold, a, alpha, universe, compact);

		//Distance using exact formula
		double exactDistance = distance(NULL, nodes.crd->getFloat4(i), nodes.id.tau[i], nodes.crd->getFloat4(j), nodes.id.tau[j], dim, manifold, a, alpha, 0, universe, compact);

		double dx = ACOS(sphProduct_v2(nodes.crd->getFloat4(i), nodes.crd->getFloat4(j)), STL, VERY_HIGH_PRECISION);

		if (ABS(embeddedDistance - exactDistance, STL) / embeddedDistance < tol || dx > HALF_PI) {
			#ifdef _OPENMP
			#pragma omp atomic
			#endif
			dvd.confusion[0]++;
		} else {
			#ifdef _OPENMP
			#pragma omp atomic
			#endif
			dvd.confusion[1]++;
			printf("Embedding: %f\n", embeddedDistance);
			printf("Exact:     %f\n\n", exactDistance);
		}
	}

	dvd.norm = static_cast<double>(npairs);

	stopwatchStop(&sValidateDistances);

	printf("\tCalculated Distances Confusion Matrix.\n");
	printf_cyan();
	printf("\t\tMatching    Pairs: %f\n", static_cast<double>(dvd.confusion[0]) / dvd.norm);
	printf_red();
	printf("\t\tConflicting Pairs: %f\n", static_cast<double>(dvd.confusion[1]) / dvd.norm);
	printf_std();
	fflush(stdout);

	if (verbose) {
		printf("\t\tExecution Time: %5.6f sec\n", sValidateDistances.elapsedTime);
		fflush(stdout);
	}

	return true;
}

//Write Node Coordinates to File
//O(num_vals) Efficiency
bool printValues(Node &nodes, const int num_vals, const char *filename, const char *coord)
{
	if (DEBUG) {
		//No null pointers
		assert (filename != NULL);
		assert (coord != NULL);

		//Variables in correct range
		assert (num_vals > 0);
	}

	try {
		std::ofstream outputStream;
		outputStream.open(filename);
		if (!outputStream.is_open())
			throw CausetException("Failed to open file in 'printValues' function!\n");

		int i;
		for (i = 0; i < num_vals; i++) {
			if (strcmp(coord, "tau") == 0)
				outputStream << nodes.id.tau[i] << std::endl;
			else if (strcmp(coord, "eta") == 0)
				outputStream << nodes.crd->w(i) << std::endl;	//Use for dim = 3
				//outputStream << nodes.crd->x(i) << std::endl;	//Use for dim = 1
			else if (strcmp(coord, "theta1") == 0)
				outputStream << nodes.crd->x(i) << std::endl;
			else if (strcmp(coord, "theta2") == 0)
				outputStream << nodes.crd->y(i) << std::endl;
			else if (strcmp(coord, "theta3") == 0)
				outputStream << nodes.crd->z(i) << std::endl;
			else
				throw CausetException("Unrecognized value in 'coord' parameter!\n");
		}
	
		outputStream.flush();
		outputStream.close();
	} catch (CausetException c) {
		fprintf(stderr, "CausetException in %s: %s on line %d\n", __FILE__, c.what(), __LINE__);
		return false;
	} catch (std::exception e) {
		fprintf(stderr, "Unknown Exception in %s: %s on line %d\n", __FILE__, e.what(), __LINE__);
		return false;
	}

	return true;
}

bool printDegrees(const Node &nodes, const int num_vals, const char *filename_in, const char *filename_out)
{
	if (DEBUG) {
		assert (nodes.k_in != NULL);
		assert (nodes.k_out != NULL);
		assert (filename_in != NULL);
		assert (filename_out != NULL);
		assert (num_vals > 0);
	}

	try {
		std::ofstream outputStream_in;
		outputStream_in.open(filename_in);
		if (!outputStream_in.is_open())
			throw CausetException("Failed to open in-degree file in 'printDegrees' function!\n");

		std::ofstream outputStream_out;
		outputStream_out.open(filename_out);
		if (!outputStream_out.is_open())
			throw CausetException("Failed to open out-degree file in 'printDegrees' function!\n");

		int i;
		for (i = 0; i < num_vals; i++) {
			outputStream_in << nodes.k_in[i] << std::endl;
			outputStream_out << nodes.k_out[i] << std::endl;
		}

		outputStream_in.flush();
		outputStream_in.close();

		outputStream_out.flush();
		outputStream_out.close();
	} catch (CausetException c) {
		fprintf(stderr, "CausetException in %s: %s on line %d\n", __FILE__, c.what(), __LINE__);
		return false;
	} catch (std::exception e) {
		fprintf(stderr, "Unknown Exception in %s: %s on line %d\n", __FILE__, e.what(), __LINE__);
		return false;
	}

	return true;
}

bool printEdgeLists(const Edge &edges, const int num_vals, const char *filename_past, const char *filename_future)
{
	if (DEBUG) {
		assert (edges.past_edges != NULL);
		assert (edges.future_edges != NULL);
		assert (filename_past != NULL);
		assert (filename_future != NULL);
		assert (num_vals > 0);
	}

	try {
		std::ofstream outputStream_past;
		outputStream_past.open(filename_past);
		if (!outputStream_past.is_open())
			throw CausetException("Failed to open past-edge file in 'printEdgeLists' function!\n");

		std::ofstream outputStream_future;
		outputStream_future.open(filename_future);
		if (!outputStream_future.is_open())
			throw CausetException("Failed to open future-edges file in 'printEdgeLists' function!\n");

		int i;
		for (i = 0; i < num_vals; i++) {
			outputStream_past << edges.past_edges[i] << std::endl;
			outputStream_future << edges.future_edges[i] << std::endl;
		}

		outputStream_past.flush();
		outputStream_past.close();

		outputStream_future.flush();
		outputStream_future.close();
	} catch (CausetException c) {
		fprintf(stderr, "CausetException in %s: %s on line %d\n", __FILE__, c.what(), __LINE__);
		return false;
	} catch (std::exception e) {
		fprintf(stderr, "Unknown Exception in %s: %s on line %d\n", __FILE__, e.what(), __LINE__);
		return false;
	}

	return true;
}

bool printEdgeListPointers(const Edge &edges, const int num_vals, const char *filename_past, const char *filename_future)
{
	if (DEBUG) {
		assert (edges.past_edge_row_start != NULL);
		assert (edges.future_edge_row_start != NULL);
		assert (filename_past != NULL);
		assert (filename_future != NULL);
		assert (num_vals > 0);
	}

	try {
		std::ofstream outputStream_past;
		outputStream_past.open(filename_past);
		if (!outputStream_past.is_open())
			throw CausetException("Failed to open past-edge-pointer file in 'printEdgeLists' function!\n");

		std::ofstream outputStream_future;
		outputStream_future.open(filename_future);
		if (!outputStream_future.is_open())
			throw CausetException("Failed to open future-edge-pointer file in 'printEdgeLists' function!\n");

		int i;
		for (i = 0; i < num_vals; i++) {
			outputStream_past << edges.past_edge_row_start[i] << std::endl;
			outputStream_future << edges.future_edge_row_start[i] << std::endl;
		}

		outputStream_past.flush();
		outputStream_past.close();

		outputStream_future.flush();
		outputStream_future.close();
	} catch (CausetException c) {
		fprintf(stderr, "CausetException in %s: %s on line %d\n", __FILE__, c.what(), __LINE__);
		return false;
	} catch (std::exception e) {
		fprintf(stderr, "Unknown Exception in %s: %s on line %d\n", __FILE__, e.what(), __LINE__);
		return false;
	}

	return true;
}

bool generateGeodesicLookupTable(const char *filename, const double max_tau, const double min_lambda, const double max_lambda, const double tau_step, const double lambda_step, const double &a, const bool &universe, const bool &verbose)
{
	if (DEBUG) {
		assert (filename != NULL);
		assert (max_tau > 0.0);
		assert (min_lambda < max_lambda);
		assert (tau_step > 0.0);
		assert (lambda_step > 0.0);
		assert (a > 0.0);
		assert (universe);
	}

	printf("\tGenerating FLRW geodesic lookup table...\n");
	fflush(stdout);

	Stopwatch sLookup = Stopwatch();
	IntData idata = IntData();
	idata.limit = 50;
	idata.tol = 1e-5;
	idata.workspace = gsl_integration_workspace_alloc(idata.nintervals);

	int n_tau = max_tau / tau_step;
	int n_lambda = (max_lambda - min_lambda) / lambda_step;

	double tau1, tau2;
	int i, j;

	try {
		FILE *table = fopen(filename, "wb");
		if (table == NULL)
			throw CausetException("Failed to open geodesic lookup table!\n");

		stopwatchStart(&sLookup);

		//printf("tau1\t\ttau2\t\tomega12\tlambda\n");
		for (i = 0; i < n_tau; i++) {
			tau1 = i * tau_step;
			for (j = 0; j < n_tau; j++) {
				tau2 = j * tau_step;

				//NOT yet ready for openmp

				//#ifdef _OPENMP
				//#pragma omp parallel for schedule (dynamic, 1)
				//#endif
				for (int k = 0; k < n_lambda; k++) {
					double lambda = k * lambda_step + min_lambda;
					double omega12;

					if (tau1 >= tau2 || lambda == 0.0)
						omega12 = 0.0;
					else if (lambda > 0) {
						idata.lower = tau1;
						idata.upper = tau2;
						omega12 = integrate1D(&flrwLookupKernel, (void*)&lambda, &idata, QAGS);
					} else if (lambda < 0) {
						double tau_m = geodesicMaxRescaledTime(lambda, a, universe);
						idata.lower = tau1;
						idata.upper = tau_m;
						//Integrate
						omega12 = integrate1D(&flrwLookupKernel, (void*)&lambda, &idata, QAGS);

						idata.lower = tau2;
						omega12 += integrate1D(&flrwLookupKernel, (void*)&lambda, &idata, QAGS);
					}

					//printf("%f\t%f\t%f\t%f\n", tau1, tau2, omega12, lambda);

					//Write to file
					fwrite(&tau1, sizeof(double), 1, table);
					fwrite(&tau2, sizeof(double), 1, table);
					fwrite(&omega12, sizeof(double), 1, table);
					fwrite(&lambda, sizeof(double), 1, table);
				}
			}
		}

		stopwatchStop(&sLookup);

		fclose(table);
	} catch (CausetException c) {
		fprintf(stderr, "CausetException in %s: %s on line %d\n", __FILE__, c.what(), __LINE__);
		return false;
	} catch (std::exception e) {
		fprintf(stderr, "Unknown Exception in %s: %s on line %d\n", __FILE__, e.what(), __LINE__);
		return false;
	}

	gsl_integration_workspace_free(idata.workspace);

	printf("\tCompleted!\n");
	fflush(stdout);

	if (verbose) {
		printf("\t\tExecution Time: %5.6f sec\n", sLookup.elapsedTime);
		fflush(stdout);
	}
	
	return true;
}
