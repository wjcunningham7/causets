#include "hip/hip_runtime.h"
#include "Measurements.h"

/////////////////////////////
//(C) Will Cunningham 2014 //
// Krioukov Research Group //
// Northeastern University //
/////////////////////////////

//Calculates clustering coefficient for each node in network
//O(N*k^3) Efficiency
bool measureClustering(float *& clustering, const Node &nodes, const Edge &edges, const bool * const core_edge_exists, float &average_clustering, const int &N_tar, const int &N_deg2, const float &core_edge_fraction, Stopwatch &sMeasureClustering, size_t &hostMemUsed, size_t &maxHostMemUsed, size_t &devMemUsed, size_t &maxDevMemUsed, const bool &calc_autocorr, const bool &verbose, const bool &bench)
{
	if (DEBUG) {
		assert (edges.past_edges != NULL);
		assert (edges.future_edges != NULL);
		assert (edges.past_edge_row_start != NULL);
		assert (edges.future_edge_row_start != NULL);
		assert (core_edge_exists != NULL);

		assert (N_tar > 0);
		assert (N_deg2 > 0);
		assert (core_edge_fraction >= 0.0 && core_edge_fraction <= 1.0);
	}

	float c_avg = 0.0f;

	stopwatchStart(&sMeasureClustering);

	try {
		clustering = (float*)malloc(sizeof(float) * N_tar);
		if (clustering == NULL)
			throw std::bad_alloc();
		memset(clustering, 0, sizeof(float) * N_tar);
		hostMemUsed += sizeof(float) * N_tar;
	} catch (std::bad_alloc()) {
		fprintf(stderr, "Failed to allocate memory in %s on line %d!\n", __FILE__, __LINE__);
		return false;
	}
	
	memoryCheckpoint(hostMemUsed, maxHostMemUsed, devMemUsed, maxDevMemUsed);
	if (verbose)
		printMemUsed("to Measure Clustering", hostMemUsed, devMemUsed, 0);

	//i represents the node we are calculating the clustering coefficient for (node #1 in triplet)
	//j represents the second node in the triplet
	//k represents the third node in the triplet
	//j and k are not interchanging or else the number of triangles would be doubly counted


	#ifdef _OPENMP
	#pragma omp parallel for schedule (dynamic, 1) reduction(+ : c_avg)
	#endif
	for (int i = 0; i < N_tar; i++) {
		//printf("\nNode %d:\n", i);
		//printf("\tDegrees: %d\n", (nodes.k_in[i] + nodes.k_out[i]));
		//printf("\t\tIn-Degrees: %d\n", nodes.k_in[i]);
		//printf("\t\tOut-Degrees: %d\n", nodes.k_out[i]);
		//fflush(stdout);

		//Ingore nodes of degree 0 and 1
		if (nodes.k_in[i] + nodes.k_out[i] < 2) {
			clustering[i] = 0.0f;
			continue;
		}

		float c_i = 0.0f;
		float c_k = static_cast<float>((nodes.k_in[i] + nodes.k_out[i]));
		float c_max = c_k * (c_k - 1.0f) / 2.0f;

		//(1) Consider both neighbors in the past
		if (edges.past_edge_row_start[i] != -1)
			for (int j = 0; j < nodes.k_in[i]; j++)
				//3 < 2 < 1
				for (int k = 0; k < j; k++)
					if (nodesAreConnected(nodes, edges.future_edges, edges.future_edge_row_start, core_edge_exists, N_tar, core_edge_fraction, edges.past_edges[edges.past_edge_row_start[i]+k], edges.past_edges[edges.past_edge_row_start[i]+j]))
						c_i += 1.0f;

		//(2) Consider both neighbors in the future
		if (edges.future_edge_row_start[i] != -1)
			for (int j = 0; j < nodes.k_out[i]; j++)
				//1 < 3 < 2
				for (int k = 0; k < j; k++)
					if (nodesAreConnected(nodes, edges.future_edges, edges.future_edge_row_start, core_edge_exists, N_tar, core_edge_fraction, edges.future_edges[edges.future_edge_row_start[i]+k], edges.future_edges[edges.future_edge_row_start[i]+j]))
						c_i += 1.0f;

		//(3) Consider one neighbor in the past and one in the future
		if (edges.past_edge_row_start[i] != -1 && edges.future_edge_row_start[i] != -1)
			for (int j = 0; j < nodes.k_out[i]; j++)
				for (int k = 0; k < nodes.k_in[i]; k++)
					//3 < 1 < 2
					if (nodesAreConnected(nodes, edges.future_edges, edges.future_edge_row_start, core_edge_exists, N_tar, core_edge_fraction, edges.past_edges[edges.past_edge_row_start[i]+k], edges.future_edges[edges.future_edge_row_start[i]+j]))
						c_i += 1.0f;

		if (DEBUG) assert (c_max > 0.0f);
		c_i = c_i / c_max;
		if (DEBUG) assert (c_i <= 1.0f);

		clustering[i] = c_i;
		c_avg += c_i;

		//printf("\tConnected Triplets: %f\n", (c_i * c_max));
		//printf("\tMaximum Triplets: %f\n", c_max);
		//printf("\tClustering Coefficient: %f\n\n", c_i);
		//fflush(stdout);
	}

	average_clustering = c_avg / N_deg2;
	if (DEBUG) assert (average_clustering >= 0.0f && average_clustering <= 1.0f);

	stopwatchStop(&sMeasureClustering);

	if (!bench) {
		printf("\tCalculated Clustering Coefficients.\n");
		printf_cyan();
		printf("\t\tAverage Clustering: %f\n", average_clustering);
		printf_std();
		fflush(stdout);
		if (calc_autocorr) {
			autocorr2 acClust(5);
			for (int i = 0; i < N_tar; i++)
				acClust.accum_data(clustering[i]);
			acClust.analysis();
			std::ofstream fout("clustAutoCorr.dat");
			acClust.fout_txt(fout);
			fout.close();
			printf("\t\tCalculated Autocorrelation.\n");
			fflush(stdout);
		}
	}

	if (verbose) {
		printf("\t\tExecution Time: %5.6f sec\n", sMeasureClustering.elapsedTime);
		fflush(stdout);
	}

	return true;
}

//Calculates the number of connected components in the graph
//as well as the size of the giant connected component
//Efficiency: O(xxx)
bool measureConnectedComponents(Node &nodes, const Edge &edges, const int &N_tar, const int &rank, int &N_cc, int &N_gcc, Stopwatch &sMeasureConnectedComponents, size_t &hostMemUsed, size_t &maxHostMemUsed, size_t &devMemUsed, size_t &maxDevMemUsed, const bool &verbose, const bool &bench)
{
	if (DEBUG) {
		assert (edges.past_edges != NULL);
		assert (edges.future_edges != NULL);
		assert (edges.past_edge_row_start != NULL);
		assert (edges.future_edge_row_start != NULL);
		assert (N_tar > 0);
		#ifdef MPI_ENABLED
		assert (rank >= 0);
		#endif
	}

	int elements;
	int i;

	stopwatchStart(&sMeasureConnectedComponents);

	try {
		nodes.cc_id = (int*)malloc(sizeof(int) * N_tar);
		if (nodes.cc_id == NULL)
			throw std::bad_alloc();
		memset(nodes.cc_id, 0, sizeof(int) * N_tar);
		hostMemUsed += sizeof(int) * N_tar;
	} catch (std::bad_alloc) {
		fprintf(stderr, "Memory allocation failure in %s on line %d\n", __FILE__, __LINE__);
		return false;
	}
	
	memoryCheckpoint(hostMemUsed, maxHostMemUsed, devMemUsed, maxDevMemUsed);
	if (verbose)
		printMemUsed("to Measure Components", hostMemUsed, devMemUsed, rank);

	if (rank == 0) {
		for (i = 0; i < N_tar; i++) {
			elements = 0;
			if (!nodes.cc_id[i] && (nodes.k_in[i] + nodes.k_out[i]) > 0) {
				bfsearch(nodes, edges, i, ++N_cc, elements);
			}
			if (elements > N_gcc)
				N_gcc = elements;
		}
	}

	#ifdef MPI_ENABLED
	MPI_Barrier(MPI_COMM_WORLD);
	MPI_Bcast(nodes.cc_id, N_tar, MPI_INT, 0, MPI_COMM_WORLD);
	MPI_Bcast(&N_cc, 1, MPI_INT, 0, MPI_COMM_WORLD);
	MPI_Bcast(&N_gcc, 1, MPI_INT, 0, MPI_COMM_WORLD);
	#endif

	stopwatchStop(&sMeasureConnectedComponents);

	if (DEBUG) {
		assert (N_cc > 0);
		assert (N_gcc > 1);
	}

	if (!bench) {
		printf_mpi(rank, "\tCalculated Number of Connected Components.\n");
		if (rank == 0) printf_cyan();
		printf_mpi(rank, "\t\tIdentified %d Components.\n", N_cc);
		printf_mpi(rank, "\t\tSize of Giant Component: %d\n", N_gcc);
		if (rank == 0) printf_std();
		fflush(stdout);
	}

	if (verbose) {
		printf_mpi(rank, "\t\tExecution Time: %5.6f sec\n", sMeasureConnectedComponents.elapsedTime);
		fflush(stdout);
	}

	return true;
}

//Calculates the Success Ratio using N_sr Unique Pairs of Nodes
//O(xxx) Efficiency (revise this)
bool measureSuccessRatio(const Node &nodes, const Edge &edges, bool * const core_edge_exists, float &success_ratio, const int &N_tar, const float &k_tar, const double &N_sr, const int &dim, const Manifold &manifold, const double &a, const double &zeta, const double &alpha, const float &core_edge_fraction, const int &edge_buffer, const int &num_mpi_threads, const int &rank, Stopwatch &sMeasureSuccessRatio, size_t &hostMemUsed, size_t &maxHostMemUsed, size_t &devMemUsed, size_t &maxDevMemUsed, const bool &universe, const bool &compact, const bool &verbose, const bool &bench)
{
	if (DEBUG) {
		assert (!nodes.crd->isNull());
		assert (dim == 1 || dim == 3);
		assert (manifold == DE_SITTER || manifold == HYPERBOLIC);

		if (manifold == HYPERBOLIC)
			assert (dim == 1);

		if (dim == 1) {
			assert (nodes.crd->getDim() == 2);
		} else if (dim == 3) {
			assert (nodes.crd->getDim() == 4);
			assert (nodes.crd->w() != NULL);
			assert (nodes.crd->z() != NULL);
			assert (manifold == DE_SITTER);
		}

		assert (nodes.crd->x() != NULL);
		assert (nodes.crd->y() != NULL);
		assert (edges.past_edges != NULL);
		assert (edges.future_edges != NULL);
		assert (edges.past_edge_row_start != NULL);
		assert (edges.future_edge_row_start != NULL);
		assert (core_edge_exists != NULL);

		assert (N_tar > 0);
		assert (N_sr > 0 && N_sr <= ((uint64_t)N_tar * (N_tar - 1)) >> 1);
		assert (!(dim == 1 && manifold == DE_SITTER));
		if (manifold == DE_SITTER && universe)
			assert (alpha > 0);
		assert (core_edge_fraction >= 0.0 && core_edge_fraction <= 1.0);
		assert (edge_buffer >= 0);
		#ifdef MPI_ENABLED
		assert (num_mpi_threads > 0);
		assert (rank >= 0);
		#endif
	}

	uint64_t stride = (uint64_t)N_tar * (N_tar - 1) / (static_cast<uint64_t>(N_sr) << 1);
	uint64_t npairs = static_cast<uint64_t>(N_sr);
	uint64_t n_trav = 0;
	uint64_t n_succ = 0;

	double *table;
	bool *used;
	long size = 0L;
	size_t u_size;

	#ifdef MPI_ENABLED
	int edges_size = static_cast<int>(N_tar * k_tar / 2 + edge_buffer);
	int core_edges_size = static_cast<int>(POW2(core_edge_fraction * N_tar, EXACT));
	#endif

	//Check out-degrees of earliest nodes
	//for (int t = 0; t < 100; t++)
	//	printf("%d\n", nodes.k_out[t]);
	//exit(11);

	stopwatchStart(&sMeasureSuccessRatio);

	//TEST THESE PARAMETERS
	u_size = sizeof(bool) * N_tar;
	//printf_mpi(rank, "Original u_size: %zu\n", u_size);
	#ifdef _OPENMP
	u_size *= omp_get_max_threads();
	//printf_mpi(rank, "OpenMP Threads:  %d\n", omp_get_max_threads());
	//printf_mpi(rank, "Max u_size:      %zu\n", u_size);
	#endif
	#ifdef MPI_ENABLED
	//u_size /= (num_mpi_threads / omp_get_max_threads());
	//printf_mpi(rank, "MPI Threads:     %d\n", num_mpi_threads);
	//printf_mpi(rank, "Divided u_size:  %zu\n", u_size);
	#endif

	try {
		used = (bool*)malloc(u_size);
		if (used == NULL)
			throw std::bad_alloc();
		memset(used, 0, u_size);
		hostMemUsed += u_size;
	} catch (std::bad_alloc) {
		fprintf(stderr, "Memory allocation failure in %s on line %d!\n", __FILE__, __LINE__);
		return false;
	}

	if (!getLookupTable("./etc/geodesic_table.cset.bin", &table, &size))
		return false;
	
	memoryCheckpoint(hostMemUsed, maxHostMemUsed, devMemUsed, maxDevMemUsed);
	if (verbose)
		printMemUsed("to Measure Success Ratio", hostMemUsed, devMemUsed, rank);

	#ifdef MPI_ENABLED
	//Broadcast:
	MPI_Barrier(MPI_COMM_WORLD);
	MPI_Bcast(nodes.crd->x(), N_tar, MPI_FLOAT, 0, MPI_COMM_WORLD);
	MPI_Bcast(nodes.crd->y(), N_tar, MPI_FLOAT, 0, MPI_COMM_WORLD);
	if (dim == 3) {
		MPI_Bcast(nodes.crd->w(), N_tar, MPI_FLOAT, 0, MPI_COMM_WORLD);
		MPI_Bcast(nodes.crd->z(), N_tar, MPI_FLOAT, 0, MPI_COMM_WORLD);
	}
	if (manifold == DE_SITTER)
		MPI_Bcast(nodes.id.tau, N_tar, MPI_FLOAT, 0, MPI_COMM_WORLD);
	MPI_Bcast(nodes.k_in, N_tar, MPI_INT, 0, MPI_COMM_WORLD);
	MPI_Bcast(nodes.k_out, N_tar, MPI_INT, 0, MPI_COMM_WORLD);
	MPI_Bcast(edges.past_edges, edges_size, MPI_INT, 0, MPI_COMM_WORLD);
	MPI_Bcast(edges.future_edges, edges_size, MPI_INT, 0, MPI_COMM_WORLD);
	MPI_Bcast(edges.past_edge_row_start, N_tar, MPI_INT, 0, MPI_COMM_WORLD);
	MPI_Bcast(edges.future_edge_row_start, N_tar, MPI_INT, 0, MPI_COMM_WORLD);
	MPI_Bcast(core_edge_exists, core_edges_size, MPI::BOOL, 0, MPI_COMM_WORLD);
	#endif

	uint64_t start = 0;
	uint64_t finish = npairs;

	#ifdef MPI_ENABLED
	uint64_t mpi_chunk = npairs / num_mpi_threads;
	start = rank * mpi_chunk;
	finish = start + mpi_chunk;
	#endif

	#ifdef _OPENMP
	#pragma omp parallel for schedule (dynamic, 1) reduction (+ : n_trav, n_succ)
	#endif
	for (uint64_t k = start; k < finish; k++) {
		//Pick Unique Pair
		uint64_t vec_idx = k * stride + 1;
		int i = static_cast<int>(vec_idx / (N_tar - 1));
		int j = static_cast<int>(vec_idx % (N_tar - 1) + 1);
		int do_map = i >= j;

		if (j < N_tar >> 1) {
			i = i + do_map * ((((N_tar >> 1) - i) << 1) - 1);
			j = j + do_map * (((N_tar >> 1) - j) << 1);
		}

		//If either node is isolated, continue
		if (!(nodes.k_in[i] + nodes.k_out[i]) || !(nodes.k_in[j] + nodes.k_out[j]))
			continue;

		//If the nodes are in different components, continue
		if (nodes.cc_id[i] != nodes.cc_id[j])
			continue;

		//Set all nodes to "not yet used"
		memset(used + N_tar * omp_get_thread_num(), 0, sizeof(bool) * N_tar);

		//Begin Traversal from i to j
		bool success = traversePath(nodes, edges, core_edge_exists, &used[N_tar*omp_get_thread_num()], table, N_tar, dim, manifold, a, zeta, alpha, core_edge_fraction, size, universe, compact, i, j);

		n_trav++;
		if (success)
			n_succ++;
	}

	#ifdef MPI_ENABLED
	//Reduce (In-Place):
	MPI_Barrier(MPI_COMM_WORLD);
	if (rank == 0)
		MPI_Reduce(MPI_IN_PLACE, &n_succ, 1, MPI_UINT64_T, MPI_SUM, 0, MPI_COMM_WORLD);
	else
		MPI_Reduce(&n_succ, NULL, 1, MPI_UINT64_T, MPI_SUM, 0, MPI_COMM_WORLD);

	if (rank == 0)
		MPI_Reduce(MPI_IN_PLACE, &n_trav, 1, MPI_UINT64_T, MPI_SUM, 0, MPI_COMM_WORLD);
	else
		MPI_Reduce(&n_trav, NULL, 1, MPI_UINT64_T, MPI_SUM, 0, MPI_COMM_WORLD);
	MPI_Barrier(MPI_COMM_WORLD);
	#endif

	if (rank == 0 && n_trav > 0)
		success_ratio = static_cast<float>(n_succ) / n_trav;

	free(used);
	used = NULL;
	hostMemUsed -= u_size;

	stopwatchStop(&sMeasureSuccessRatio);

	if (!bench) {
		printf_mpi(rank, "\tCalculated Success Ratio.\n");
		if (rank == 0) printf_cyan();
		printf_mpi(rank, "\t\tSuccess Ratio: %f\n", success_ratio);
		printf_mpi(rank, "\t\tTraversed Pairs: %" PRIu64 "\n", n_trav);
		if (rank == 0) printf_std();
		fflush(stdout);
	}

	if (verbose) {
		printf_mpi(rank, "\t\tExecution Time: %5.6f sec\n", sMeasureSuccessRatio.elapsedTime);
		fflush(stdout);
	}

	return true;
}

//Node Traversal Algorithm
//Returns true if the modified greedy routing algorithm successfully links 'source' and 'dest'
//O(xxx) Efficiency (revise this)
bool traversePath(const Node &nodes, const Edge &edges, const bool * const core_edge_exists, bool * const &used, const double * const table, const int &N_tar, const int &dim, const Manifold &manifold, const double &a, const double &zeta, const double &alpha, const float &core_edge_fraction, const long &size, const bool &universe, const bool &compact, int source, int dest)
{
	if (DEBUG) {
		assert (!nodes.crd->isNull());
		assert (dim == 1 || dim == 3);
		assert (manifold == DE_SITTER || manifold == HYPERBOLIC);

		if (manifold == HYPERBOLIC)
			assert (dim == 1);

		if (dim == 1) {
			assert (nodes.crd->getDim() == 2);
		} else if (dim == 3) {
			assert (nodes.crd->getDim() == 4);
			assert (nodes.crd->w() != NULL);
			assert (nodes.crd->z() != NULL);
			assert (manifold == DE_SITTER);
		}

		assert (nodes.crd->x() != NULL);
		assert (nodes.crd->y() != NULL);
		assert (nodes.k_in != NULL);
		assert (nodes.k_out != NULL);
		assert (edges.past_edges != NULL);
		assert (edges.future_edges != NULL);
		assert (edges.past_edge_row_start != NULL);
		assert (edges.future_edge_row_start != NULL);
		assert (core_edge_exists != NULL);
		assert (used != NULL);
		assert (table != NULL);
		
		assert (N_tar > 0);
		if (manifold == DE_SITTER) {
			assert (a > 0.0);
			assert (HALF_PI - zeta > 0.0);
			if (universe)
				assert (alpha > 0.0);
		}
		assert (core_edge_fraction >= 0.0 && core_edge_fraction <= 1.0);
		assert (size > 0);
		assert (source >= 0 && source < N_tar);
		assert (dest >= 0 && dest < N_tar);
	}

	float min_dist = 0.0f;
	int loc = source;
	int idx_a = source;
	int idx_b = dest;

	float dist;
	int next;

	bool success = false;

	//While the current location (loc) is not equal to the destination (dest)
	while (loc != dest) {
		next = loc;
		dist = INF;
		min_dist = INF;
		used[loc] = true;

		//These would indicate corrupted data
		if (DEBUG) {
			assert (!(edges.past_edge_row_start[loc] == -1 && nodes.k_in[loc] > 0));
			assert (!(edges.past_edge_row_start[loc] != -1 && nodes.k_in[loc] == 0));
			assert (!(edges.future_edge_row_start[loc] == -1 && nodes.k_out[loc] > 0));
			assert (!(edges.future_edge_row_start[loc] != -1 && nodes.k_out[loc] == 0));
		}

		//(1) Check past relations
		for (int m = 0; m < nodes.k_in[loc]; m++) {
			idx_a = edges.past_edges[edges.past_edge_row_start[loc]+m];

			//(A) If the current location's (loc's) past neighbor (idx_a) is the destination (idx_b) then return true
			if (idx_a == idx_b)
				return true;

			//(B) If the current location's past neighbor is directly connected to the destination then return true
			if (nodesAreConnected(nodes, edges.future_edges, edges.future_edge_row_start, core_edge_exists, N_tar, core_edge_fraction, idx_a, idx_b))
				return true;

			//(C) Otherwise find the past neighbor closest to the destination
			if (manifold == DE_SITTER) {
				if (compact)
					dist = distanceEmb(nodes.crd->getFloat4(idx_a), nodes.id.tau[idx_a], nodes.crd->getFloat4(idx_b), nodes.id.tau[idx_b], dim, manifold, a, alpha, universe, compact);
				else
					dist = distance(table, nodes.crd->getFloat4(idx_a), nodes.id.tau[idx_a], nodes.crd->getFloat4(idx_b), nodes.id.tau[idx_b], dim, manifold, a, alpha, size, universe, compact);
			} else if (manifold == HYPERBOLIC)
				dist = distanceH(nodes.crd->getFloat2(idx_a), nodes.crd->getFloat2(idx_b), dim, manifold, zeta);

			if (dist < 0)
				return false;

			//Save the minimum distance
			if (dist <= min_dist) {
				min_dist = dist;
				next = idx_a;
			}
		}

		//(2) Check future relations
		//OpenMP is implemented here for the early nodes which have lots of out-degrees
		//However, it does not appear to provide a speedup...
		#ifdef _OPENMP
		float priv_min_dist = min_dist;
		int priv_next = next;
		//bool make_parallel = nodes.k_out[loc] > 10000;
		//bool make_parallel = false;
		#pragma omp parallel shared (next, min_dist) \
				     firstprivate (idx_a, priv_min_dist, priv_next) \
				     if (false)
		{
		#pragma omp for schedule (dynamic, 1)
		#endif
		for (int m = 0; m < nodes.k_out[loc]; m++) {
			#ifdef _OPENMP
			if (priv_next == idx_b || priv_next == -1)
				continue;
			#endif

			idx_a = edges.future_edges[edges.future_edge_row_start[loc]+m];

			//(D) If the current location's future neighbor is the destination then return true
			if (idx_a == idx_b) {
				#ifdef _OPENMP
				priv_min_dist = 0.0f;
				priv_next = idx_b;
				continue;
				#else
				return true;
				#endif
			}

			//(E) If the current location's future neighbor is directly connected to the destination then return true
			if (nodesAreConnected(nodes, edges.future_edges, edges.future_edge_row_start, core_edge_exists, N_tar, core_edge_fraction, idx_a, idx_b)) {
				#ifdef _OPENMP
				priv_min_dist = 0.0f;
				priv_next = idx_b;
				continue;
				#else
				return true;
				#endif
			}

			//(F) Otherwise find the future neighbor closest to the destination
			if (manifold == DE_SITTER) {
				if (compact)
					dist = distanceEmb(nodes.crd->getFloat4(idx_a), nodes.id.tau[idx_a], nodes.crd->getFloat4(idx_b), nodes.id.tau[idx_b], dim, manifold, a, alpha, universe, compact);
				else
					dist = distance(table, nodes.crd->getFloat4(idx_a), nodes.id.tau[idx_a], nodes.crd->getFloat4(idx_b), nodes.id.tau[idx_b], dim, manifold, a, alpha, size, universe, compact);
			} else if (manifold == HYPERBOLIC)
				dist = distanceH(nodes.crd->getFloat2(idx_a), nodes.crd->getFloat2(idx_b), dim, manifold, zeta);

			if (dist < 0)
				idx_a = -1;

			#ifdef _OPENMP
			if (dist <= priv_min_dist) {
				priv_min_dist = dist;
				priv_next = idx_a;
			}
			#else
			if (dist <= min_dist) {
				min_dist = dist;
				next = idx_a;
			}
			#endif
		}

		#ifdef _OPENMP
		if (next != idx_b) {
			#pragma omp flush (min_dist)
			if (priv_min_dist <= min_dist) {
				#pragma omp critical
				{
					if (priv_min_dist <= min_dist) {
						min_dist = priv_min_dist;
						next = priv_next;
					}
				}
			}
		}
		}
		#endif

		if (next == idx_b)
			return true;
		else if (next == -1)
			return false;

		if (!used[next])
			loc = next;
		else
			break;
	}

	return success;
}

//Takes N_df measurements of in-degree and out-degree fields at time tau_m
//O(xxx) Efficiency (revise this)
bool measureDegreeField(int *& in_degree_field, int *& out_degree_field, float &avg_idf, float &avg_odf, Coordinates *& c, const int &N_tar, int &N_df, const double &tau_m, const int &dim, const Manifold &manifold, const double &a, const double &zeta, const double &alpha, const double &delta, long &seed, Stopwatch &sMeasureDegreeField, size_t &hostMemUsed, size_t &maxHostMemUsed, size_t &devMemUsed, size_t &maxDevMemUsed, const bool &universe, const bool &compact, const bool &verbose, const bool &bench)
{
	if (DEBUG) {
		//No Null Pointers
		assert (c->getDim() == 4);
		assert (!c->isNull());
		assert (c->w() != NULL);
		assert (c->x() != NULL);
		assert (c->y() != NULL);
		assert (c->z() != NULL);

		//Parameters in Correct Ranges
		assert (N_tar > 0);
		assert (N_df > 0);
		assert (tau_m > 0.0);
		assert (dim == 3);
		assert (manifold == DE_SITTER);
		assert (a > 0.0);
		assert (HALF_PI - zeta > 0.0);
		if (universe)
			assert (alpha > 0.0);
	}

	double *table;
	float4 test_node;
	double eta_m;
	double d_size/*, x, rval*/;
	float dt, dx;
	long size = 0L;
	int k_in, k_out;
	int i, j;

	//Numerical Integration Parameters
	double *params = NULL;
	double *params2 = (double*)malloc(sizeof(double));

	//Calculate theoretical values
	double k_in_theory = 0.0;
	double k_out_theory = 0.0;
	bool theoretical = universe && verbose;

	//Modify number of samples
	N_df = 1;

	IntData idata = IntData();
	//Modify these two parameters to trade off between speed and accuracy
	idata.limit = 50;
	idata.tol = 1e-5;
	if (universe && (USE_GSL || theoretical))
		idata.workspace = gsl_integration_workspace_alloc(idata.nintervals);

	stopwatchStart(&sMeasureDegreeField);

	//Allocate memory for data
	try {
		in_degree_field = (int*)malloc(sizeof(int) * N_df);
		if (in_degree_field == NULL)
			throw std::bad_alloc();
		memset(in_degree_field, 0, N_df);
		hostMemUsed += sizeof(int) * N_df;

		out_degree_field = (int*)malloc(sizeof(int) * N_df);
		if (out_degree_field == NULL)
			throw std::bad_alloc();
		memset(out_degree_field, 0, N_df);
		hostMemUsed += sizeof(int) * N_df;

		if (theoretical) {
			if (!getLookupTable("./etc/ctuc_table.cset.bin", &table, &size))
				return false;

			params = (double*)malloc(size + sizeof(double) * 4);
			if (params == NULL)
				throw std::bad_alloc();
			hostMemUsed += size + sizeof(double) * 4;

			params2 = (double*)malloc(sizeof(double));
			if (params == NULL)
				throw std::bad_alloc();
		}
	} catch (std::bad_alloc) {
		fprintf(stderr, "Memory allocation failure in %s on line %d!\n", __FILE__, __LINE__);
		return false;
	}

	memoryCheckpoint(hostMemUsed, maxHostMemUsed, devMemUsed, maxDevMemUsed);
	if (verbose)
		printMemUsed("to Measure Degree Fields", hostMemUsed, devMemUsed, 0);
	
	//Calculate eta_m
	if (universe) {
		if (USE_GSL) {
			//Numerical Integration
			idata.upper = tau_m * a;
			params2[0] = a;
			eta_m = integrate1D(&tToEtaUniverse, (void*)params2, &idata, QAGS) / alpha;
			free(params2);
		} else
			//Exact Solution
			eta_m = tauToEtaUniverseExact(tau_m, a, alpha);
	} else
		eta_m = tauToEta(tau_m);
	test_node.w = static_cast<float>(eta_m);
	
	if (theoretical) {	
		d_size = static_cast<double>(size);
		memcpy(params, &eta_m, sizeof(double));
		memcpy(params + 1, &a, sizeof(double));
		memcpy(params + 2, &alpha, sizeof(double));
		memcpy(params + 3, &d_size, sizeof(double));
		memcpy(params + 4, table, size);
	
		idata.limit = 100;
		idata.tol = 1e-4;
	
		//Theoretical Average In-Degree
		idata.lower = 0.0;
		idata.upper = eta_m;
		k_in_theory = (4.0 * M_PI * delta * POW2(POW2(alpha, EXACT), EXACT) / 3.0) * integrate1D(&degreeFieldTheory, params, &idata, QAGS);

		//Theoretical Average Out-Degree
		idata.lower = eta_m;
		idata.upper = HALF_PI - zeta;
		k_out_theory = (4.0 * M_PI * delta * POW2(POW2(alpha, EXACT), EXACT) / 3.0) * integrate1D(&degreeFieldTheory, params, &idata, QAGS);

		free(params);
		params = NULL;
		hostMemUsed -= size + sizeof(double) * 4;

		free(params2);
		params2 = NULL;

		free(table);
		table = NULL;
	}

	//Take N_df measurements of the fields
	for (i = 0; i < N_df; i++) {
		test_node.x = 1.0f;
		test_node.y = 1.0f;
		test_node.z = 1.0f;

		//Sample Theta from (0, 2pi)
		/*x = TWO_PI * ran2(&seed);
		test_node.x = static_cast<float>(x);
		if (DEBUG) assert (test_node.x > 0.0f && test_node.x < static_cast<float>(TWO_PI));

		//Sample Phi from (0, pi)
		x = HALF_PI;
		rval = ran2(&seed);
		if (!newton(&solvePhi, &x, 250, TOL, &rval, NULL, NULL, NULL, NULL, NULL)) 
			return false;
		test_node.y = static_cast<float>(x);
		if (DEBUG) assert (test_node.y > 0.0f && test_node.y < static_cast<float>(M_PI));

		//Sample Chi from (0, pi)
		test_node.z = static_cast<float>(ACOS(1.0 - 2.0 * ran2(&seed), APPROX ? INTEGRATION : STL, VERY_HIGH_PRECISION));
		if (DEBUG) assert (test_node.z > 0.0f && test_node.z < static_cast<float>(M_PI));*/

		k_in = 0;
		k_out = 0;

		//Compare test node to N_tar other nodes
		float4 new_node;
		for (j = 0; j < N_tar; j++) {
			//Calculate sign of spacetime interval
			new_node = c->getFloat4(j);
			dt = static_cast<float>(ABS(static_cast<double>(c->w(j) - test_node.w), STL));

			if (compact) {
				if (DIST_V2)
					dx = static_cast<float>(ACOS(static_cast<double>(sphProduct_v2(new_node, test_node)), APPROX ? INTEGRATION : STL, VERY_HIGH_PRECISION));
				else
					dx = static_cast<float>(ACOS(static_cast<double>(sphProduct_v1(new_node, test_node)), APPROX ? INTEGRATION : STL, VERY_HIGH_PRECISION));
			} else {
				if (DIST_V2)
					dx = static_cast<float>(SQRT(static_cast<double>(flatProduct_v2(new_node, test_node)), APPROX ? BITWISE : STL));
				else
					dx = static_cast<float>(SQRT(static_cast<double>(flatProduct_v1(new_node, test_node)), APPROX ? BITWISE : STL));
			}

			if (dx < dt) {
				//They are connected
				if (new_node.w < test_node.w)
					k_in++;
				else
					k_out++;
			}
		}

		//Save measurements
		in_degree_field[i] = k_in;
		out_degree_field[i] = k_out;

		avg_idf += k_in;
		avg_odf += k_out;
	}

	//Normalize averages
	avg_idf /= N_df;
	avg_odf /= N_df;

	stopwatchStop(&sMeasureDegreeField);

	if (universe && (USE_GSL || theoretical))
		gsl_integration_workspace_free(idata.workspace);

	if (!bench) {
		printf("\tCalculated Degree Field Values.\n");
		printf_cyan();
		printf("\t\tMeasurement Time: %f\n", tau_m);
		printf("\t\tAverage In-Degree Field: %f\n", avg_idf);
		if (theoretical) {
			printf_red();
			printf("\t\t\tTheory: %f\n", k_in_theory);
			printf_cyan();
		}
		printf("\t\tAverage Out-Degree Field: %f\n", avg_odf);
		if (theoretical) {
			printf_red();
			printf("\t\t\tTheory: %f\n", k_out_theory);
			printf_std();
		}
		printf_std();
		fflush(stdout);
	}

	if (verbose) {
		printf("\t\tExecution Time: %5.6f sec\n", sMeasureDegreeField.elapsedTime);
		fflush(stdout);
	}

	return true;
}

//Measure Causal Set Action
bool measureAction()
{
	return true;
}
