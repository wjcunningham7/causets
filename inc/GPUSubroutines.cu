#include "hip/hip_runtime.h"
#ifndef GPU_SUBROUTINES_CU_
#define GPU_SUBROUTINES_CU_

#include "Causet.h"

__global__ void Generate(Node *nodes, unsigned int N_tar, long seed);

bool generateNodesGPU(Network *network);

__global__ void Generate(Node *nodes, unsigned int N_tar, long seed)
{
	//unsigned int i = blockDim.x * blockIdx.x + threadIdx.x;
	//unsigned int j = blockDim.y * blockIdx.y + threadIdx.y;
	//if ((j * width) + i > N_tar)
	//	return;

	//Implement CURAND package here for random number generation
}

bool generateNodesGPU(Network *network)
{
	//CURAND
	hiprandGenerator_t prng;
	
	try {
		if (HIPRAND_STATUS_SUCCESS != hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_DEFAULT))
			throw CausetException("Failed to create hiprand generator.\n");
		if (HIPRAND_STATUS_SUCCESS != hiprandSetPseudoRandomGeneratorSeed(prng, (unsigned int)network->network_properties.seed))
			throw CausetException("Failed to set hiprand seed.\n");

		//Need to redesign Node for GPU so memory for points is contiguous
		//Lots of thought should go into this...
		//if (HIPRAND_STATUS_SUCCESS != hiprandGenerateUniform(prng, (float*)d_points, network->network_properties.N_tar))
		//	throw CausetException("Failed to generate hiprand uniform number distribution.\n");

		if (HIPRAND_STATUS_SUCCESS != hiprandDestroyGenerator(prng))
			throw CausetException("Failed to destroy hiprand generator.\n");
	} catch (CausetException e) {
		fprintf(stderr, e.what());
		exit(EXIT_FAILURE);
	}

	//Invoke Kernel
	Generate<<<network->network_properties.network_exec.blocks_per_grid, network->network_properties.network_exec.threads_per_block>>>((Node*)network->d_nodes, network->network_properties.N_tar, network->network_properties.seed);
	getLastCudaError("Kernel 'Generate' Failed to Execute!");
	checkCudaErrors(hipCtxSynchronize());

	//Copy Values to Host
	checkCudaErrors(hipMemcpyDtoH(network->nodes, network->d_nodes, sizeof(Node) * network->network_properties.N_tar));

	return true;
}

#endif